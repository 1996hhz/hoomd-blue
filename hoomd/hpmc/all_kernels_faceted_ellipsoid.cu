// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"

#include "ShapeFacetedEllipsoid.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeFacetedEllipsoid
template hipError_t gpu_hpmc_free_volume<ShapeFacetedEllipsoid>(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeFacetedEllipsoid::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeFacetedEllipsoid>(const hpmc_args_t& args,
                                                  const typename ShapeFacetedEllipsoid::param_type *d_params);
}; // end namespace detail

} // end namespace hpmc
