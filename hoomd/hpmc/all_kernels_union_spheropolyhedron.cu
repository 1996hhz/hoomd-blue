// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"

#include "ShapeSpheropolyhedron.h"
#include "ShapeUnion.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeUnion<ShapeSpheropolyhedron>
template hipError_t gpu_hpmc_free_volume<ShapeUnion<ShapeSpheropolyhedron> >(const hpmc_free_volume_args_t &args,
                                                  const typename ShapeUnion<ShapeSpheropolyhedron> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeUnion<ShapeSpheropolyhedron> >(const hpmc_args_t& args,
                                                  const typename ShapeUnion<ShapeSpheropolyhedron> ::param_type *d_params);
}; // end namespace detail

} // end namespace hpmc
