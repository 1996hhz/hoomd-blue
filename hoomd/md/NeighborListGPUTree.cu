#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: mphoward

#include "NeighborListGPUTree.cuh"

#include "hoomd/neighbor/LBVH.cuh"
#include "hoomd/neighbor/LBVHTraverser.cuh"

#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/remove.h>
#include "hoomd/extern/cub/hipcub/hipcub.hpp"

__global__ void gpu_nlist_mark_types_kernel(unsigned int *d_types,
                                            unsigned int *d_indexes,
                                            unsigned int *d_lbvh_errors,
                                            Scalar4 *d_last_pos,
                                            const Scalar4 *d_pos,
                                            const unsigned int N,
                                            const unsigned int nghosts,
                                            const BoxDim box,
                                            const Scalar3 ghost_width)
    {
    // compute the particle index this thread operates on
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // one thread per particle
    if (idx >= N+nghosts)
        return;

    // acquire particle data
    Scalar4 postype = d_pos[idx];
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int type = __scalar_as_int(postype.w);

    // get position in simulation box
    uchar3 periodic = box.getPeriodic();
    Scalar3 f = box.makeFraction(pos, ghost_width);

    /*
     * check if the particle is inside the unit cell + ghost layer.
     * we silently ignore ghosts outside of this width, since they could be in bonds, etc.
     */
    if ((f.x < Scalar(-0.00001) || f.x >= Scalar(1.00001)) ||
        (f.y < Scalar(-0.00001) || f.y >= Scalar(1.00001)) ||
        (f.z < Scalar(-0.00001) || f.z >= Scalar(1.00001)))
        {
        // error for owned particle
        if (idx < N)
            {
            atomicMax(d_lbvh_errors,idx+1);
            return;
            }
        else // silent for ghosts
            {
            type = NeighborListTypeSentinel;
            }
        }

    d_types[idx] = type;
    d_indexes[idx] = idx;
    // record as "last" position of owned particles
    if (idx < N)
        {
        d_last_pos[idx] = postype;
        }
    }

hipError_t gpu_nlist_mark_types(unsigned int *d_types,
                                 unsigned int *d_indexes,
                                 unsigned int *d_lbvh_errors,
                                 Scalar4 *d_last_pos,
                                 const Scalar4 *d_pos,
                                 const unsigned int N,
                                 const unsigned int nghosts,
                                 const BoxDim& box,
                                 const Scalar3 ghost_width,
                                 const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nlist_mark_types_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    const unsigned int run_block_size = min(block_size,max_block_size);
    const unsigned int num_blocks = ((N+nghosts) + run_block_size - 1)/run_block_size;
    gpu_nlist_mark_types_kernel<<<num_blocks, run_block_size>>>(d_types,
                                                                d_indexes,
                                                                d_lbvh_errors,
                                                                d_last_pos,
                                                                d_pos,
                                                                N,
                                                                nghosts,
                                                                box,
                                                                ghost_width);
    return hipSuccess;
    }

uchar2 gpu_nlist_sort_types(void *d_tmp,
                            size_t &tmp_bytes,
                            unsigned int *d_types,
                            unsigned int *d_sorted_types,
                            unsigned int *d_indexes,
                            unsigned int *d_sorted_indexes,
                            const unsigned int N,
                            const unsigned int num_bits)
    {
    hipcub::DoubleBuffer<unsigned int> d_keys(d_types, d_sorted_types);
    hipcub::DoubleBuffer<unsigned int> d_vals(d_indexes, d_sorted_indexes);

    // we counted number of bits to sort, so the range of bit indexes is [0,num_bits)
    hipcub::DeviceRadixSort::SortPairs(d_tmp, tmp_bytes, d_keys, d_vals, N, 0, num_bits);

    uchar2 swap = make_uchar2(0,0);
    if (d_tmp != NULL)
        {
        // mark that the gpu arrays should be flipped if the final result is not in the sorted array (1)
        swap.x = (d_keys.selector == 0);
        swap.y = (d_vals.selector == 0);
        }
    return swap;
    }

__global__ void gpu_nlist_count_types_kernel(unsigned int *d_first,
                                             unsigned int *d_last,
                                             const unsigned int *d_types,
                                             const unsigned int ntypes,
                                             const unsigned int N)
    {
    // compute the particle index this thread operates on
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // one thread per particle
    if (idx >= N)
        return;

    // my type
    const unsigned int type = d_types[idx];
    // look to left if not first
    const unsigned int left = (idx > 0) ? d_types[idx-1] : NeighborListTypeSentinel;
    // look to right if not last
    const unsigned int right = (idx < N-1) ? d_types[idx+1] : NeighborListTypeSentinel;

    // if left is not same as self (or idx == 0 by use of sentinel), this is the first index in the type
    if (left != type && type < ntypes)
        {
        d_first[type] = idx;
        }
    // if right is not the same as self (or idx == N-1 by use of sentinel), this is the last index in the type
    if (right != type && type < ntypes)
        {
        d_last[type] = idx + 1;
        }
    }

hipError_t gpu_nlist_count_types(unsigned int *d_first,
                                  unsigned int *d_last,
                                  const unsigned int *d_types,
                                  const unsigned int ntypes,
                                  const unsigned int N,
                                  const unsigned int block_size)

    {
    // initially, fill all cells as empty
    thrust::fill(thrust::device, d_first, d_first+ntypes, NeighborListTypeSentinel);
    hipMemset(d_last, 0, sizeof(unsigned int)*ntypes);

    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nlist_count_types_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    int run_block_size = min(block_size,max_block_size);
    gpu_nlist_count_types_kernel<<<N/run_block_size + 1, run_block_size>>>(d_first,
                                                                           d_last,
                                                                           d_types,
                                                                           ntypes,
                                                                           N);
    return hipSuccess;
    }

__global__ void gpu_nlist_copy_primitives_kernel(unsigned int *d_traverse_order,
                                                 const unsigned int *d_indexes,
                                                 const unsigned int *d_primitives,
                                                 const unsigned int N)
    {
    // one thread per particle
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    const unsigned int primitive = d_primitives[idx];
    d_traverse_order[idx] = __ldg(d_indexes + primitive);
    }

hipError_t gpu_nlist_copy_primitives(unsigned int *d_traverse_order,
                                      const unsigned int *d_indexes,
                                      const unsigned int *d_primitives,
                                      const unsigned int N,
                                      const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nlist_copy_primitives_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    int run_block_size = min(block_size,max_block_size);
    gpu_nlist_copy_primitives_kernel<<<N/run_block_size + 1, run_block_size>>>(d_traverse_order,
                                                                               d_indexes,
                                                                               d_primitives,
                                                                               N);
    return hipSuccess;
    }

// explicit templates for neighbor::LBVH with PointMapInsertOp and NullOp
template void neighbor::gpu::lbvh_gen_codes(unsigned int *, unsigned int *, const PointMapInsertOp&,
    const Scalar3, const Scalar3, const unsigned int, const unsigned int, hipStream_t);
template void neighbor::gpu::lbvh_bubble_aabbs(const neighbor::gpu::LBVHData, const PointMapInsertOp&,
    unsigned int *, const unsigned int, const unsigned int, hipStream_t);
template void neighbor::gpu::lbvh_one_primitive(const neighbor::gpu::LBVHData, const PointMapInsertOp&, hipStream_t);
template void neighbor::gpu::lbvh_traverse_ropes(NeighborListOp&, const neighbor::gpu::LBVHCompressedData&,
    const ParticleQueryOp<false,false>&, const Scalar3 *, unsigned int, unsigned int, hipStream_t);
template void neighbor::gpu::lbvh_traverse_ropes(NeighborListOp&, const neighbor::gpu::LBVHCompressedData&,
    const ParticleQueryOp<false,true>&, const Scalar3 *, unsigned int, unsigned int, hipStream_t);
template void neighbor::gpu::lbvh_traverse_ropes(NeighborListOp&, const neighbor::gpu::LBVHCompressedData&,
    const ParticleQueryOp<true,false>&, const Scalar3 *, unsigned int, unsigned int, hipStream_t);
template void neighbor::gpu::lbvh_traverse_ropes(NeighborListOp&, const neighbor::gpu::LBVHCompressedData&,
    const ParticleQueryOp<true,true>&, const Scalar3 *, unsigned int, unsigned int, hipStream_t);
