#include "hip/hip_runtime.h"
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: mphoward

/*!
 * \file mpcd/CellThermoComputeGPU.cu
 * \brief Explicitly instantiates reduction operators and declares kernel drivers
 *        for mpcd::CellThermoComputeGPU.
 */

#include "CellThermoComputeGPU.cuh"
#include "CellThermoTypes.h"

// #include "CellCommunicator.cuh"
#include "ReductionOperators.h"

#include "hoomd/extern/cub/hipcub/hipcub.hpp"

namespace mpcd
{
namespace gpu
{

//! Shuffle-based warp reduction
/*!
 * \param val Value to be reduced
 *
 * \tparam LOGICAL_WARP_SIZE Number of threads in a "logical" warp to reduce, must be a power-of-two
 *                           and less than the hardware warp size.
 * \tparam T Type of value to be reduced (inferred).
 *
 * \returns Reduced value.
 *
 * The value \a val is reduced into the 0-th lane of the "logical" warp using
 * shuffle-based intrinsics. This allows for the summation of quantities when
 * using multiple threads per object within a kernel.
 */
template<int LOGICAL_WARP_SIZE, typename T>
__device__ static T warp_reduce(T val)
    {
    static_assert(LOGICAL_WARP_SIZE <= HIPCUB_WARP_THREADS, "Logical warp size cannot exceed hardware warp size");
    static_assert(LOGICAL_WARP_SIZE && !(LOGICAL_WARP_SIZE & (LOGICAL_WARP_SIZE-1)), "Logical warp size must be a power of 2");

    #pragma unroll
    for (int dest_count = LOGICAL_WARP_SIZE/2; dest_count >= 1; dest_count /= 2)
        {
        val += hipcub::ShuffleDown(val, dest_count);
        }
    return val;
    }

namespace kernel
{
//! Begins the cell thermo compute by summing cell quantities
/*!
 * \param d_cell_vel Velocity and mass per cell (output)
 * \param d_cell_energy Energy, temperature, number of particles per cell (output)
 * \param d_cell_np Number of particles per cell
 * \param d_cell_list MPCD cell list
 * \param cli Indexer into the cell list
 * \param d_vel MPCD particle velocities
 * \param N_mpcd Number of MPCD particles
 * \param mpcd_mass Mass of MPCD particle
 * \param d_embed_vel Embedded particle velocity
 * \param d_embed_cell Embedded particle cells
 *
 * \tparam tpp Number of threads to use per cell
 *
 * \b Implementation details:
 * Using \a tpp threads per cell, the cell properties are accumulated into \a d_cell_vel
 * and \a d_cell_energy. Shuffle-based intrinsics are used to reduce the accumulated
 * properties per-cell, and the first thread for each cell writes the result into
 * global memory.
 */
template<unsigned int tpp>
__global__ void begin_cell_thermo(Scalar4 *d_cell_vel,
                                  Scalar3 *d_cell_energy,
                                  const unsigned int *d_cell_np,
                                  const unsigned int *d_cell_list,
                                  const Index2D cli,
                                  const Scalar4 *d_vel,
                                  const unsigned int N_mpcd,
                                  const Scalar mpcd_mass,
                                  const Scalar4 *d_embed_vel,
                                  const unsigned int *d_embed_cell)
    {
    // one thread per cell
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tpp * cli.getH())
        return;

    const unsigned int cell_id = idx / tpp;
    const unsigned int np = d_cell_np[cell_id];
    double4 momentum = make_double4(0.0, 0.0, 0.0, 0.0);
    double ke(0.0);

    for (unsigned int offset = (idx % tpp); offset < np; offset += tpp)
        {
        // Load particle data
        const unsigned int cur_p = d_cell_list[cli(offset, cell_id)];
        double3 vel_i;
        double mass_i;
        if (cur_p < N_mpcd)
            {
            Scalar4 vel_cell = d_vel[cur_p];
            vel_i = make_double3(vel_cell.x, vel_cell.y, vel_cell.z);
            mass_i = mpcd_mass;
            }
        else
            {
            Scalar4 vel_m = d_embed_vel[d_embed_cell[cur_p - N_mpcd]];
            vel_i = make_double3(vel_m.x, vel_m.y, vel_m.z);
            mass_i = vel_m.w;
            }

        // add momentum
        momentum.x += mass_i * vel_i.x;
        momentum.y += mass_i * vel_i.y;
        momentum.z += mass_i * vel_i.z;
        momentum.w += mass_i;

        // also compute ke of the particle
        ke += (double)(0.5) * mass_i * (vel_i.x * vel_i.x + vel_i.y * vel_i.y + vel_i.z * vel_i.z);
        }

    // reduce quantities down into the 0-th lane per logical warp
    if (tpp > 1)
        {
        momentum.x = warp_reduce<tpp>(momentum.x);
        momentum.y = warp_reduce<tpp>(momentum.y);
        momentum.z = warp_reduce<tpp>(momentum.z);
        momentum.w = warp_reduce<tpp>(momentum.w);
        ke = warp_reduce<tpp>(ke);
        }

    // 0-th lane in each warp writes the result
    if (idx % tpp == 0)
        {
        d_cell_vel[cell_id] = momentum;
        d_cell_energy[cell_id] = make_scalar3(ke, 0.0, __int_as_scalar(np));
        }
    }

//! Finalizes the cell thermo compute by properly averaging cell quantities
/*!
 * \param d_cell_vel Cell velocity and masses
 * \param d_cell_energy Cell energy and temperature
 * \param Ncell Number of cells
 * \param n_dimensions Number of dimensions in system
 *
 * \b Implementation details:
 * Using one thread per cell, the properties are averaged by mass, number of particles,
 * etc. The temperature is computed from the cell kinetic energy.
 */
__global__ void end_cell_thermo(Scalar4 *d_cell_vel,
                                Scalar3 *d_cell_energy,
                                const unsigned int Ncell,
                                const unsigned int n_dimensions)
    {
    // one thread per cell
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= Ncell)
        return;

    // average cell properties if the cell has mass
    const Scalar4 cell_vel = d_cell_vel[idx];
    Scalar3 vel_cm = make_scalar3(cell_vel.x, cell_vel.y, cell_vel.z);
    const Scalar mass = cell_vel.w;

    const Scalar3 cell_energy = d_cell_energy[idx];
    const Scalar ke = cell_energy.x;
    Scalar temp(0.0);
    const unsigned int np = __scalar_as_int(cell_energy.z);

    if (mass > 0.)
        {
        // average velocity is only defined when there is some mass in the cell
        vel_cm /= mass;

        // temperature is only defined for 2 or more particles
        if (np > 1)
            {
            const Scalar ke_cm = Scalar(0.5) * mass * dot(vel_cm, vel_cm);
            temp = Scalar(2.) * (ke - ke_cm) / Scalar(n_dimensions * (np-1));
            }
        }

    d_cell_vel[idx] = make_scalar4(vel_cm.x, vel_cm.y, vel_cm.z, mass);
    d_cell_energy[idx] = make_scalar3(ke, temp, __int_as_scalar(np));
    }

/*!
 * \param d_tmp_thermo Temporary cell packed thermo element
 * \param d_cell_vel Cell velocity to reduce
 * \param d_cell_energy Cell energy to reduce
 * \param tmp_ci Temporary cell indexer for cells undergoing reduction
 * \param ci Cell indexer Regular cell list indexer
 *
 * \b Implementation details:
 * Using one thread per \a temporary cell, the cell properties are normalized
 * in a way suitable for reduction of net properties, e.g. the cell velocities
 * are converted to momentum. The temperature is set to the cell energy, and a
 * flag is set to 1 or 0 to indicate whether this cell has an energy that should
 * be used in averaging the total temperature.
 */
__global__ void stage_net_cell_thermo(mpcd::detail::cell_thermo_element *d_tmp_thermo,
                                      const Scalar4 *d_cell_vel,
                                      const Scalar3 *d_cell_energy,
                                      const Index3D tmp_ci,
                                      const Index3D ci)
    {
    // one thread per cell
    unsigned int tmp_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tmp_idx >= tmp_ci.getNumElements())
        return;

    // use the temporary cell indexer to map to a cell, then use the real cell indexer to
    // get the read index
    uint3 cell = tmp_ci.getTriple(tmp_idx);
    const unsigned int idx = ci(cell.x, cell.y, cell.z);

    const Scalar4 vel_mass = d_cell_vel[idx];
    const double3 vel = make_double3(vel_mass.x, vel_mass.y, vel_mass.z);
    const double mass = vel_mass.w;

    mpcd::detail::cell_thermo_element thermo;
    thermo.momentum = make_double3(mass * vel.x,
                                   mass * vel.y,
                                   mass * vel.z);

    const Scalar3 cell_energy = d_cell_energy[idx];
    thermo.energy = cell_energy.x;
    if (__scalar_as_int(cell_energy.z) > 1)
        {
        thermo.temperature = cell_energy.y;
        thermo.flag = 1;
        }
    else
        {
        thermo.temperature = 0.0;
        thermo.flag = 0;
        }

    d_tmp_thermo[tmp_idx] = thermo;
    }

} // end namespace kernel

//! Templated launcher for multiple threads-per-particle kernel
/*
 * \param d_cell_vel Velocity and mass per cell (output)
 * \param d_cell_energy Energy, temperature, number of particles per cell (output)
 * \param d_cell_np Number of particles per cell
 * \param d_cell_list MPCD cell list
 * \param cli Indexer into the cell list
 * \param d_vel MPCD particle velocities
 * \param N_mpcd Number of MPCD particles
 * \param mpcd_mass Mass of MPCD particle
 * \param d_embed_vel Embedded particle velocity
 * \param d_embed_cell Embedded particle cells
 * \param block_size Number of threads per block
 * \param tpp Number of threads to use per-particle
 *
 * \tparam cur_tpp Number of threads-per-particle for this template instantiation
 *
 * Launchers are recursively instantiated at compile-time in order to match the
 * correct number of threads at runtime. If the templated number of threads matches
 * the runtime number of threads, then the kernel is launched. Otherwise, the
 * next template (with threads reduced by a factor of 2) is launched. This
 * recursion is broken by a specialized template for 0 threads, which does no
 * work.
 */
template<unsigned int cur_tpp>
inline void launch_begin_cell_thermo(Scalar4 *d_cell_vel,
                                     Scalar3 *d_cell_energy,
                                     const unsigned int *d_cell_np,
                                     const unsigned int *d_cell_list,
                                     const Index2D& cli,
                                     const Scalar4 *d_vel,
                                     const unsigned int N_mpcd,
                                     const Scalar mpcd_mass,
                                     const Scalar4 *d_embed_vel,
                                     const unsigned int *d_embed_cell,
                                     const unsigned int block_size,
                                     const unsigned int tpp)
    {
    if (cur_tpp == tpp)
        {
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))mpcd::gpu::kernel::begin_cell_thermo<cur_tpp>);
            max_block_size = attr.maxThreadsPerBlock;
            }

        unsigned int run_block_size = min(block_size, max_block_size);
        dim3 grid(cur_tpp*cli.getH() / run_block_size + 1);
        mpcd::gpu::kernel::begin_cell_thermo<cur_tpp><<<grid, run_block_size>>>(d_cell_vel,
                                                                                d_cell_energy,
                                                                                d_cell_np,
                                                                                d_cell_list,
                                                                                cli,
                                                                                d_vel,
                                                                                N_mpcd,
                                                                                mpcd_mass,
                                                                                d_embed_vel,
                                                                                d_embed_cell);
        }
    else
        {
        launch_begin_cell_thermo<cur_tpp/2>(d_cell_vel,
                                            d_cell_energy,
                                            d_cell_np,
                                            d_cell_list,
                                            cli,
                                            d_vel,
                                            N_mpcd,
                                            mpcd_mass,
                                            d_embed_vel,
                                            d_embed_cell,
                                            block_size,
                                            tpp);
        }
    }
//! Template specialization to break recursion
template<>
inline void launch_begin_cell_thermo<0>(Scalar4 *d_cell_vel,
                                        Scalar3 *d_cell_energy,
                                        const unsigned int *d_cell_np,
                                        const unsigned int *d_cell_list,
                                        const Index2D& cli,
                                        const Scalar4 *d_vel,
                                        const unsigned int N_mpcd,
                                        const Scalar mpcd_mass,
                                        const Scalar4 *d_embed_vel,
                                        const unsigned int *d_embed_cell,
                                        const unsigned int block_size,
                                        const unsigned int tpp)
    { }

/*
 * \param d_cell_vel Velocity and mass per cell (output)
 * \param d_cell_energy Energy, temperature, number of particles per cell (output)
 * \param d_cell_np Number of particles per cell
 * \param d_cell_list MPCD cell list
 * \param cli Indexer into the cell list
 * \param d_vel MPCD particle velocities
 * \param N_mpcd Number of MPCD particles
 * \param mpcd_mass Mass of MPCD particle
 * \param d_embed_vel Embedded particle velocity
 * \param d_embed_cell Embedded particle cells
 * \param block_size Number of threads per block
 *
 * \returns hipSuccess on completion
 *
 * \sa mpcd::gpu::launch_begin_cell_thermo
 * \sa mpcd::gpu::kernel::begin_cell_thermo
 */
hipError_t begin_cell_thermo(Scalar4 *d_cell_vel,
                              Scalar3 *d_cell_energy,
                              const unsigned int *d_cell_np,
                              const unsigned int *d_cell_list,
                              const Index2D& cli,
                              const Scalar4 *d_vel,
                              const unsigned int N_mpcd,
                              const Scalar mpcd_mass,
                              const Scalar4 *d_embed_vel,
                              const unsigned int *d_embed_cell,
                              const unsigned int block_size,
                              const unsigned int tpp)
    {
    launch_begin_cell_thermo<32>(d_cell_vel,
                                 d_cell_energy,
                                 d_cell_np,
                                 d_cell_list,
                                 cli,
                                 d_vel,
                                 N_mpcd,
                                 mpcd_mass,
                                 d_embed_vel,
                                 d_embed_cell,
                                 block_size,
                                 tpp);
    return hipSuccess;
    }

/*!
 * \param d_cell_vel Cell velocity and masses
 * \param d_cell_energy Cell energy and temperature
 * \param Ncell Number of cells
 * \param n_dimensions Number of dimensions in system
 *
 * \returns hipSuccess on completion
 *
 * \sa mpcd::gpu::kernel::end_cell_thermo
 */
hipError_t end_cell_thermo(Scalar4 *d_cell_vel,
                            Scalar3 *d_cell_energy,
                            const unsigned int Ncell,
                            const unsigned int n_dimensions,
                            const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))mpcd::gpu::kernel::end_cell_thermo);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);
    dim3 grid(Ncell / run_block_size + 1);
    mpcd::gpu::kernel::end_cell_thermo<<<grid, run_block_size>>>(d_cell_vel,
                                                                 d_cell_energy,
                                                                 Ncell,
                                                                 n_dimensions);

    return hipSuccess;
    }

/*!
 * \param d_tmp_thermo Temporary cell packed thermo element
 * \param d_cell_vel Cell velocity to reduce
 * \param d_cell_energy Cell energy to reduce
 * \param tmp_ci Temporary cell indexer for cells undergoing reduction
 * \param ci Cell indexer Regular cell list indexer
 * \param block_size Number of threads per block
 *
 * \returns hipSuccess on completion
 *
 * \sa mpcd::gpu::kernel::stage_net_cell_thermo
 */
hipError_t stage_net_cell_thermo(mpcd::detail::cell_thermo_element *d_tmp_thermo,
                                  const Scalar4 *d_cell_vel,
                                  const Scalar3 *d_cell_energy,
                                  const Index3D& tmp_ci,
                                  const Index3D& ci,
                                  const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))mpcd::gpu::kernel::stage_net_cell_thermo);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);
    dim3 grid(tmp_ci.getNumElements() / run_block_size + 1);
    mpcd::gpu::kernel::stage_net_cell_thermo<<<grid, run_block_size>>>(d_tmp_thermo,
                                                                       d_cell_vel,
                                                                       d_cell_energy,
                                                                       tmp_ci,
                                                                       ci);
    return hipSuccess;
    }

/*!
 * \param d_reduced Cell thermo properties reduced across all cells (output on second call)
 * \param d_tmp Temporary storage for reduction (output on first call)
 * \param tmp_bytes Number of bytes allocated for temporary storage (output on first call)
 * \param d_tmp_thermo Cell thermo properties to reduce
 * \param Ncell The number of cells to reduce across
 *
 * \returns hipSuccess on completion
 *
 * \b Implementation details:
 * CUB DeviceReduce is used to perform the reduction. Hence, this function requires
 * two calls to perform the reduction. The first call sizes the temporary storage,
 * which is returned in \a d_tmp and \a tmp_bytes. The caller must then allocate
 * the required bytes, and call the function a second time. This performs the
 * reducetion and returns the result in \a d_reduced.
 */
hipError_t reduce_net_cell_thermo(mpcd::detail::cell_thermo_element *d_reduced,
                                   void *d_tmp,
                                   size_t& tmp_bytes,
                                   const mpcd::detail::cell_thermo_element *d_tmp_thermo,
                                   const unsigned int Ncell)
    {
    hipcub::DeviceReduce::Sum(d_tmp, tmp_bytes, d_tmp_thermo, d_reduced, Ncell);
    return hipSuccess;
    }

/*
//! Explicit template instantiation of pack for cell velocity
template hipError_t pack_cell_buffer(typename mpcd::detail::CellVelocityPackOp::element *d_left_buf,
                                      typename mpcd::detail::CellVelocityPackOp::element *d_right_buf,
                                      const Index3D& left_idx,
                                      const Index3D& right_idx,
                                      const uint3& right_offset,
                                      const Scalar4 *d_props,
                                      mpcd::detail::CellVelocityPackOp pack_op,
                                      const Index3D& ci,
                                      unsigned int block_size);

//! Explicit template instantiation of pack for cell energy
template hipError_t pack_cell_buffer(typename mpcd::detail::CellEnergyPackOp::element *d_left_buf,
                                      typename mpcd::detail::CellEnergyPackOp::element *d_right_buf,
                                      const Index3D& left_idx,
                                      const Index3D& right_idx,
                                      const uint3& right_offset,
                                      const Scalar3 *d_props,
                                      mpcd::detail::CellEnergyPackOp pack_op,
                                      const Index3D& ci,
                                      unsigned int block_size);

//! Explicit template instantiation of unpack for cell velocity
template hipError_t unpack_cell_buffer(Scalar4 *d_props,
                                        mpcd::detail::CellVelocityPackOp pack_op,
                                        const Index3D& ci,
                                        const typename mpcd::detail::CellVelocityPackOp::element *d_left_buf,
                                        const typename mpcd::detail::CellVelocityPackOp::element *d_right_buf,
                                        const Index3D& left_idx,
                                        const Index3D& right_idx,
                                        const uint3& right_offset,
                                        const unsigned int block_size);

//! Explicit template instantiation of unpack for cell energy
template hipError_t unpack_cell_buffer(Scalar3 *d_props,
                                        mpcd::detail::CellEnergyPackOp pack_op,
                                        const Index3D& ci,
                                        const typename mpcd::detail::CellEnergyPackOp::element *d_left_buf,
                                        const typename mpcd::detail::CellEnergyPackOp::element *d_right_buf,
                                        const Index3D& left_idx,
                                        const Index3D& right_idx,
                                        const uint3& right_offset,
                                        const unsigned int block_size);*/

} // end namespace gpu
} // end namespace mpcd
