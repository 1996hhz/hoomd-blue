
#include <hip/hip_runtime.h>
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: mphoward

/*!
 * \file mpcd/ParticleData.cu
 * \brief Defines GPU functions and kernels used by mpcd::ParticleData
 */

#ifdef ENABLE_MPI

#include "ParticleData.cuh"

#include "hoomd/extern/cub/cub/device/device_scan.cuh"

namespace mpcd
{
namespace gpu
{
namespace kernel
{
//! Kernel to partition particle data
/*!
 * \param d_out
 * \param mask
 * \param d_pos Device array of particle positions
 * \param d_vel Device array of particle velocities
 * \param d_tag Device array of particle tags
 * \param d_pos_alt Device array of particle positions (output)
 * \param d_vel_alt Device array of particle velocities (output)
 * \param d_tag_alt Device array of particle tags (output)
 * \param d_out Output array for packed particle data
 * \param d_comm_flags Communication flags (nonzero if particle should be migrated)
 * \param d_comm_flags_out Packed communication flags
 * \param d_scan Result of exclusive prefix sum
 * \param N Number of local particles
 *
 * Particles are removed by performing a selection using the result of an
 * exclusive prefix sum, stored in \a d_scan. The scan recovers the indexes
 * of the particles. A simple example illustrating the implementation follows:
 *
 * \verbatim
 * Particles:   0 1 2 3 4
 * Flags:       0|1 1|0 0
 * d_scan       0|0 1|2 2
 *              ---------
 * scan_keep:   0|1 1|1 2
 *              ---------
 * keep:        0,3,4 -> 0,1,2
 * remove:      1,2 -> 0,1
 * \endverbatim
 */
__global__ void remove_particles(mpcd::detail::pdata_element *d_out,
                                 const unsigned int mask,
                                 const Scalar4 *d_pos,
                                 const Scalar4 *d_vel,
                                 const unsigned int *d_tag,
                                 const unsigned int *d_comm_flags,
                                 Scalar4 *d_pos_alt,
                                 Scalar4 *d_vel_alt,
                                 unsigned int *d_tag_alt,
                                 unsigned int *d_comm_flags_alt,
                                 const unsigned int *d_scan,
                                 const unsigned int N)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= N) return;

    unsigned int scan_remove = d_scan[idx];
    unsigned int scan_keep = idx - scan_remove;

    if (d_comm_flags[idx] & mask)
        {
        mpcd::detail::pdata_element p;
        p.pos = d_pos[idx];
        p.vel = d_vel[idx];
        p.tag = d_tag[idx];
        p.comm_flag = d_comm_flags[idx];

        d_out[scan_remove] = p;
        }
    else
        {
        d_pos_alt[scan_keep] = d_pos[idx];
        d_vel_alt[scan_keep] = d_vel[idx];
        d_tag_alt[scan_keep] = d_tag[idx];
        d_comm_flags_alt[scan_keep] = d_comm_flags[idx];
        }
    }

//! Kernel to transform communication flags for prefix sum
/*!
 * \param d_tmp Temporary storage to hold transformation (output)
 * \param d_comm_flags Communication flags
 * \param mask Bitwise mask for \a d_comm_flags
 * \param N Number of local particles
 *
 * Any communication flags that are bitwise AND with \a mask are transformed to
 * a 1 and stored in \a d_tmp.
 */
__global__ void mark_removed_particles(unsigned int *d_tmp,
                                       const unsigned int *d_comm_flags,
                                       const unsigned int mask,
                                       const unsigned int N)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= N) return;
    d_tmp[idx] = (d_comm_flags[idx] & mask) ? 1 : 0;
    }
} // end namespace kernel
} // end namespace gpu
} // end namespace mpcd


cudaError_t mpcd::gpu::mark_removed_particles(unsigned int *d_tmp_flag,
                                              const unsigned int *d_comm_flags,
                                              const unsigned int mask,
                                              const unsigned int N,
                                              const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        cudaFuncAttributes attr;
        cudaFuncGetAttributes(&attr, (const void*)mpcd::gpu::kernel::mark_removed_particles);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);
    dim3 grid(N / run_block_size + 1);
    mpcd::gpu::kernel::mark_removed_particles<<<grid, run_block_size>>>(d_tmp_flag,
                                                                        d_comm_flags,
                                                                        mask,
                                                                        N);
    return cudaSuccess;
    }

cudaError_t mpcd::gpu::scan_removed_particles(void *d_tmp,
                                              size_t& tmp_bytes,
                                              unsigned int *d_tmp_flag,
                                              const unsigned int N)
    {
    // in place scan is supported
    // https://groups.google.com/d/msg/cub-users/pEsYSNc2Rn4/_4ulOuwWDcoJ
    cub::DeviceScan::ExclusiveSum(d_tmp, tmp_bytes, d_tmp_flag, d_tmp_flag, N);

    return cudaSuccess;
    }

/*!
 * \param d_out Output array for packed particle data
 * \param mask Bitwise mask for \a d_comm_flags
 * \param d_pos Device array of particle positions
 * \param d_vel Device array of particle velocities
 * \param d_tag Device array of particle tags
 * \param d_comm_flags Device array of communication flags
 * \param d_pos_alt Device array of particle positions (output)
 * \param d_vel_alt Device array of particle velocities (output)
 * \param d_tag_alt Device array of particle tags (output)
 * \param d_comm_flags_alt Device array of communication flags (output)
 * \param d_scan Output from device scan of temporary flags
 * \param N Current number of particles
 *
 * \returns cudaSuccess on completion.
 */
cudaError_t mpcd::gpu::remove_particles(mpcd::detail::pdata_element *d_out,
                                        const unsigned int mask,
                                        const Scalar4 *d_pos,
                                        const Scalar4 *d_vel,
                                        const unsigned int *d_tag,
                                        const unsigned int *d_comm_flags,
                                        Scalar4 *d_pos_alt,
                                        Scalar4 *d_vel_alt,
                                        unsigned int *d_tag_alt,
                                        unsigned int *d_comm_flags_alt,
                                        unsigned int *d_scan,
                                        const unsigned int N)
    {
    // partition particle data into local and removed particles
    unsigned int block_size = 512;
    unsigned int n_blocks = N/block_size+1;

    mpcd::gpu::kernel::remove_particles<<<n_blocks, block_size>>>(d_out,
                                                                  mask,
                                                                  d_pos,
                                                                  d_vel,
                                                                  d_tag,
                                                                  d_comm_flags,
                                                                  d_pos_alt,
                                                                  d_vel_alt,
                                                                  d_tag_alt,
                                                                  d_comm_flags_alt,
                                                                  d_scan,
                                                                  N);
    return cudaSuccess;
    }


namespace mpcd
{
namespace gpu
{
namespace kernel
{
//! Kernel to partition particle data
/*!
 * \param old_nparticles old local particle count
 * \param num_add_ptls Number of particles in input array
 * \param d_pos Device array of particle positions
 * \param d_vel Device array of particle velocities
 * \param d_tag Device array of particle tags
 * \param d_comm_flags Device array of communication flags
 * \param d_in Device array of packed input particle data
 * \param mask Bitwise mask for received particles to unmask
 *
 * Particle data is appended to the end of the particle data arrays from the
 * packed buffer. Communication flags of new particles are unmasked.
 */
__global__ void add_particles(unsigned int old_nparticles,
                              unsigned int num_add_ptls,
                              Scalar4 *d_pos,
                              Scalar4 *d_vel,
                              unsigned int *d_tag,
                              unsigned int *d_comm_flags,
                              const mpcd::detail::pdata_element *d_in,
                              const unsigned int mask)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= num_add_ptls) return;

    mpcd::detail::pdata_element p = d_in[idx];

    unsigned int add_idx = old_nparticles + idx;
    d_pos[add_idx] = p.pos;
    d_vel[add_idx] = p.vel;
    d_tag[add_idx] = p.tag;
    d_comm_flags[add_idx] = p.comm_flag & ~mask;
    }
} // end namespace kernel
} // end namespace gpu
} // end namespace mpcd

/*!
 * \param old_nparticles old local particle count
 * \param num_add_ptls Number of particles in input array
 * \param d_pos Device array of particle positions
 * \param d_vel Device array of particle velocities
 * \param d_tag Device array of particle tags
 * \param d_comm_flags Device array of communication flags
 * \param d_in Device array of packed input particle data
 * \param mask Bitwise mask for received particles to unmask
 *
 * Particle data is appended to the end of the particle data arrays from the
 * packed buffer. Communication flags of new particles are unmasked.
 */
void mpcd::gpu::add_particles(unsigned int old_nparticles,
                              unsigned int num_add_ptls,
                              Scalar4 *d_pos,
                              Scalar4 *d_vel,
                              unsigned int *d_tag,
                              unsigned int *d_comm_flags,
                              const mpcd::detail::pdata_element *d_in,
                              const unsigned int mask)
    {
    unsigned int block_size = 512;
    unsigned int n_blocks = num_add_ptls/block_size + 1;

    mpcd::gpu::kernel::add_particles<<<n_blocks, block_size>>>(old_nparticles,
                                                               num_add_ptls,
                                                               d_pos,
                                                               d_vel,
                                                               d_tag,
                                                               d_comm_flags,
                                                               d_in,
                                                               mask);
    }

#endif // ENABLE_MPI
