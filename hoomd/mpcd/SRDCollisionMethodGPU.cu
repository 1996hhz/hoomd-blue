#include "hip/hip_runtime.h"
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: mphoward

/*!
 * \file mpcd/SRDCollisionMethodGPU.cu
 * \brief Defines GPU functions and kernels used by mpcd::SRDCollisionMethodGPU
 */

#include "SRDCollisionMethodGPU.cuh"
#include "hoomd/extern/saruprngCUDA.h"

#define MPCD_2PI 6.283185307179586

namespace mpcd
{
namespace gpu
{
namespace kernel
{
__global__ void srd_draw_vectors(Scalar3 *d_rotvec,
                                 const Index3D ci,
                                 const int3 origin,
                                 const uint3 global_dim,
                                 const Index3D global_ci,
                                 const unsigned int timestep,
                                 const unsigned int seed,
                                 const unsigned int Ncell)
    {
    // one thread per cell
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= Ncell)
        return;

    // get local cell triple from 1d index
    const uint3 cell = ci.getTriple(idx);
    // shift local cell by local origin, and wrap through global boundaries
    int3 global_cell = make_int3(origin.x + (int)cell.x,
                                 origin.y + (int)cell.y,
                                 origin.z + (int)cell.z);
    if (global_cell.x >= (int)global_dim.x) global_cell.x -= global_dim.x;
    else if (global_cell.x < 0) global_cell.x += global_dim.x;

    if (global_cell.y >= (int)global_dim.y) global_cell.y -= global_dim.y;
    else if (global_cell.y < 0) global_cell.y += global_dim.y;

    if (global_cell.z >= (int)global_dim.z) global_cell.z -= global_dim.z;
    else if (global_cell.z < 0) global_cell.z += global_dim.z;

    // convert global triple to 1d global index
    const unsigned int global_idx = global_ci(global_cell.x, global_cell.y, global_cell.z);

    // Initialize the PRNG using the cell index, timestep, and seed for the hash
    SaruGPU saru(global_idx, timestep, seed);

    // calculate the random rotation vector for the cell
    const double theta = saru.d(0, MPCD_2PI);
    const double u = saru.d(-1.0, 1.0);

    /*
     * Sometimes numbers get drawn really close to -1 or +1, and the machine precision difference is a really
     * small (negative) number. This causes sqrt() to fail with nan error, so we need to handle those cases by
     * forcing the sqrt() to 0.0.
     */
    double sqrtu = 0.0;
    const double one_minus_u2 = 1.0-u*u;
    if (one_minus_u2 > 0.0)
        {
        sqrtu = slow::sqrt(one_minus_u2);
        }

    d_rotvec[idx] = make_double3(sqrtu * slow::cos(theta), sqrtu*slow::sin(theta), u);
    }
__global__ void srd_rotate(Scalar4 *d_vel,
                           Scalar4 *d_vel_embed,
                           const unsigned int *d_embed_group,
                           const unsigned int *d_embed_cell_ids,
                           const Scalar4 *d_cell_vel,
                           const Scalar3 *d_rotvec,
                           const double cos_a,
                           const double one_minus_cos_a,
                           const double sin_a,
                           const unsigned int N_mpcd,
                           const unsigned int N_tot)
    {
    // one thread per particle
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N_tot)
        return;

    // load particle data
    double3 vel;
    unsigned int cell;
    // these properties are needed for the embedded particles only
    unsigned int idx(0); double mass(0);
    if (tid < N_mpcd)
        {
        const Scalar4 vel_cell = d_vel[tid];
        vel = make_double3(vel_cell.x, vel_cell.y, vel_cell.z);
        cell = __scalar_as_int(vel_cell.w);
        }
    else
        {
        idx = d_embed_group[tid - N_mpcd];

        const Scalar4 vel_mass = d_vel_embed[idx];
        vel = make_double3(vel_mass.x, vel_mass.y, vel_mass.z);
        mass = vel_mass.w;
        cell = d_embed_cell_ids[tid - N_mpcd];
        }

    // subtract average velocity
    const Scalar4 avg_vel = d_cell_vel[cell];
    vel.x -= avg_vel.x;
    vel.y -= avg_vel.y;
    vel.z -= avg_vel.z;

    // get rotation vector
    double3 rot_vec = d_rotvec[cell];

    // perform the rotation in double precision
    double3 new_vel;
    new_vel.x = (cos_a + rot_vec.x*rot_vec.x*one_minus_cos_a) * vel.x;
    new_vel.x += (rot_vec.x*rot_vec.y*one_minus_cos_a - sin_a*rot_vec.z) * vel.y;
    new_vel.x += (rot_vec.x*rot_vec.z*one_minus_cos_a + sin_a*rot_vec.y) * vel.z;

    new_vel.y = (cos_a + rot_vec.y*rot_vec.y*one_minus_cos_a) * vel.y;
    new_vel.y += (rot_vec.x*rot_vec.y*one_minus_cos_a + sin_a*rot_vec.z) * vel.x;
    new_vel.y += (rot_vec.y*rot_vec.z*one_minus_cos_a - sin_a*rot_vec.x) * vel.z;

    new_vel.z = (cos_a + rot_vec.z*rot_vec.z*one_minus_cos_a) * vel.z;
    new_vel.z += (rot_vec.x*rot_vec.z*one_minus_cos_a - sin_a*rot_vec.y) * vel.x;
    new_vel.z += (rot_vec.y*rot_vec.z*one_minus_cos_a + sin_a*rot_vec.x) * vel.y;

    new_vel.x += avg_vel.x;
    new_vel.y += avg_vel.y;
    new_vel.z += avg_vel.z;

    // set the new velocity
    if (tid < N_mpcd)
        {
        d_vel[tid] = make_scalar4(new_vel.x, new_vel.y, new_vel.z, __int_as_scalar(cell));
        }
    else
        {
        d_vel_embed[idx] = make_scalar4(new_vel.x, new_vel.y, new_vel.z, mass);
        }
    }
} // end namespace kernel

hipError_t srd_draw_vectors(Scalar3 *d_rotvec,
                             const Index3D& ci,
                             const int3 origin,
                             const uint3 global_dim,
                             const Index3D& global_ci,
                             const unsigned int timestep,
                             const unsigned int seed,
                             const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))mpcd::gpu::kernel::srd_draw_vectors);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    const unsigned int Ncell = ci.getNumElements();
    dim3 grid(Ncell / run_block_size + 1);
    mpcd::gpu::kernel::srd_draw_vectors<<<grid, run_block_size>>>(d_rotvec,
                                                                  ci,
                                                                  origin,
                                                                  global_dim,
                                                                  global_ci,
                                                                  timestep,
                                                                  seed,
                                                                  Ncell);

    return hipSuccess;
    }

hipError_t srd_rotate(Scalar4 *d_vel,
                       Scalar4 *d_vel_embed,
                       const unsigned int *d_embed_group,
                       const unsigned int *d_embed_cell_ids,
                       const Scalar4 *d_cell_vel,
                       const Scalar3 *d_rotvec,
                       const double angle,
                       const unsigned int N_mpcd,
                       const unsigned int N_tot,
                       const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))mpcd::gpu::kernel::srd_rotate);
        max_block_size = attr.maxThreadsPerBlock;
        }

    // precompute angles for rotation
    const double cos_a = slow::cos(angle);
    const double one_minus_cos_a = 1.0 - cos_a;
    const double sin_a = slow::sin(angle);

    unsigned int run_block_size = min(block_size, max_block_size);
    dim3 grid(N_tot / run_block_size + 1);
    mpcd::gpu::kernel::srd_rotate<<<grid, run_block_size>>>(d_vel,
                                                            d_vel_embed,
                                                            d_embed_group,
                                                            d_embed_cell_ids,
                                                            d_cell_vel,
                                                            d_rotvec,
                                                            cos_a,
                                                            one_minus_cos_a,
                                                            sin_a,
                                                            N_mpcd,
                                                            N_tot);

    return hipSuccess;
    }

} // end namespace gpu
} // end namespace mpcd
