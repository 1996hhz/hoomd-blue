#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2016 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <VectorMath.h>
#include "Index1D.h"

const unsigned int NO_BODY = 0xffffffff;

// Maintainer: jglaser

/*! \file ForceComposite.cu
    \brief Defines GPU kernel code for the composite particle integration on the GPU.
*/

//! Shared memory for body force and torque reduction, required allocation when the kernel is called
extern __shared__ Scalar3 sum[];

//! Calculates the body forces and torques by summing the constituent particle forces using a fixed sliding window size
/*  Compute the force and torque sum on all bodies in the system from their constituent particles. n_bodies_per_block
    bodies are handled within each block of execution on the GPU. The reason for this is to decrease
    over-parallelism and use the GPU cores more effectively when bodies are smaller than the block size. Otherwise,
    small bodies leave many threads in the block idle with nothing to do.

    On start, the properties common to each body are read in, computed, and stored in shared memory for all the threads
    working on that body to access. Then, the threads loop over all particles that are part of the body with
    a sliding window. Each loop of the window computes the force and torque for block_size/n_bodies_per_block particles
    in as many threads in parallel. These quantities are summed over enough windows to cover the whole body.

    The block_size/n_bodies_per_block partial sums are stored in shared memory. Then n_bodies_per_block partial
    reductions are performed in parallel using all threads to sum the total force and torque on each body. This looks
    just like a normal reduction, except that it terminates at a certain level in the tree. To make the math
    for the partial reduction work out, block_size must be a power of 2 as must n_bodies_per_block.

    Performance testing on GF100 with many different bodies of different sizes ranging from 4-256 particles per body
    has found that the optimum block size for most bodies is 64 threads. Performance increases for all body sizes
    as n_bodies_per_block is increased, but only up to 8. n_bodies_per_block=16 slows performance significantly.
    Based on these performance results, this kernel is hardcoded to handle only 1,2,4,8 n_bodies_per_block
    with a power of 2 block size (hardcoded to 64 in the kernel launch).

    However, there is one issue to the n_bodies_per_block parallelism reduction. If the reduction results in too few
    blocks, performance can actually be reduced. For example, if there are only 64 bodies running at the "most optimal"
    n_bodies_per_block=8 results in only 8 blocks on the GPU! That isn't even enough to heat up all 15 SMs on GF100.
    Even though n_bodies_per_block=1 is not fully optimal per block, running 64 slow blocks in parallel is faster than
    running 8 fast blocks in parallel. Testing on GF100 determines that 60 blocks is the dividing line (makes sense -
    that's 4 blocks active on each SM).
*/
__global__ void gpu_rigid_force_sliding_kernel(Scalar4* d_force,
                                                 Scalar4* d_torque,
                                                 const unsigned int *d_molecule_len,
                                                 const unsigned int *d_molecule_list,
                                                 const unsigned int *d_tag,
                                                 const unsigned int *d_rtag,
                                                 Index2D molecule_indexer,
                                                 const Scalar4 *d_postype,
                                                 const Scalar4* d_orientation,
                                                 Index2D body_indexer,
                                                 Scalar3* d_body_pos,
                                                 Scalar4* d_body_orientation,
                                                 const Scalar4* d_net_force,
                                                 const Scalar4* d_net_torque,
                                                 unsigned int n_mol,
                                                 unsigned int N,
                                                 unsigned int window_size,
                                                 unsigned int thread_mask,
                                                 unsigned int n_bodies_per_block)
    {
    // determine which body (0 ... n_bodies_per_block-1) this thread is working on
    // assign threads 0, 1, 2, ... to body 0, n, n+1, n+2, ... to body 1, and so on.
    unsigned int m = threadIdx.x / (blockDim.x / n_bodies_per_block);

    // body_force and body_torque are each shared memory arrays with 1 element per threads
    Scalar3 *body_force = sum;
    Scalar3 *body_torque = &sum[blockDim.x];

    // store body type, orientation and the index in molecule list in shared memory. Up to 16 bodies per block can
    // be handled.
    __shared__ unsigned int body_type[16];
    __shared__ Scalar4 body_orientation[16];
    __shared__ unsigned int mol_idx[16];
    __shared__ unsigned int central_idx[16];

    // each thread makes partial sums of force and torque of all the particles that this thread loops over
    Scalar3 sum_force = make_scalar3(Scalar(0.0), Scalar(0.0), Scalar(0.0));
    Scalar3 sum_torque = make_scalar3(Scalar(0.0), Scalar(0.0), Scalar(0.0));

    // thread_mask is a bitmask that masks out the high bits in threadIdx.x.
    // threadIdx.x & thread_mask is an index from 0 to block_size/n_bodies_per_block-1 and determines what offset
    // this thread is to use when accessing the particles in the body
    if ((threadIdx.x & thread_mask) == 0)
        {
        // thread 0 for this body reads in the body id and orientation and stores them in shared memory
        int group_idx = blockIdx.x*n_bodies_per_block + m;
        if (group_idx < n_mol)
            {
            mol_idx[m] = group_idx;

            // first ptl is central ptl
            central_idx[m] = d_molecule_list[molecule_indexer(group_idx, 0)];
            body_type[m] = __scalar_as_int(d_postype[central_idx[m]].w);
            body_orientation[m] = d_orientation[central_idx[m]];
            }
        else
            {
            mol_idx[m] = NO_BODY;
            }
        }

    __syncthreads();

    // compute the number of windows that we need to loop over
    unsigned int n_windows = n_mol / window_size + 1;

    // slide the window throughout the block
    for (unsigned int start = 0; start < n_windows; start++)
        {
        Scalar4 fi = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));
        Scalar4 ti = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));

        // determine the index with this body that this particle should handle
        unsigned int k = start * window_size + (threadIdx.x & thread_mask);

        if (mol_idx[m] == NO_BODY || central_idx[m] >= N)
            {
            // only local central ptls
            continue;
            }

        unsigned int mol_len = d_molecule_len[mol_idx[m]];
        unsigned int central_tag = d_tag[central_idx[m]];

        // if that index is in the body we are actually handling a real body
        if (k < mol_len)
            {
            // determine the particle idx of the particle
            unsigned int pidx = d_molecule_list[molecule_indexer(mol_idx[m],k)];

            unsigned int tag = d_tag[pidx];

            // indices are in tag order, and the first ptl is the central ptl
            unsigned int local_idx = tag-central_tag - 1;

            // if this particle is not the central particle
            if (pidx != central_idx[m])
                {
                // calculate body force and torques
                vec3<Scalar> particle_pos(d_body_pos[body_indexer(body_type[m], local_idx)]);
                fi = d_net_force[pidx];

                //will likely need to rotate these components too
                ti = d_net_torque[pidx];

                // tally the force in the per thread counter
                sum_force.x += fi.x;
                sum_force.y += fi.y;
                sum_force.z += fi.z;

                // This might require more calculations but more stable
                // particularly when rigid bodies are bigger than half the box
                vec3<Scalar> ri = rotate(quat<Scalar>(body_orientation[m]), particle_pos);

                // torque = r x f
                vec3<Scalar> del_torque(cross(ri, vec3<Scalar>(fi)));

                // tally the torque in the per thread counter
                sum_torque.x += del_torque.x;
                sum_torque.y += del_torque.y;
                sum_torque.z += del_torque.z;
                }
            }
        }

    __syncthreads();

    // put the partial sums into shared memory
    body_force[threadIdx.x] = sum_force;
    body_torque[threadIdx.x] = sum_torque;

    __syncthreads();

    // perform a set of partial reductions. Each block_size/n_bodies_per_block threads performs a sum reduction
    // just within its own group
    unsigned int offset = window_size >> 1;
    while (offset > 0)
        {
        if ((threadIdx.x & thread_mask) < offset)
            {
            body_force[threadIdx.x].x += body_force[threadIdx.x + offset].x;
            body_force[threadIdx.x].y += body_force[threadIdx.x + offset].y;
            body_force[threadIdx.x].z += body_force[threadIdx.x + offset].z;

            body_torque[threadIdx.x].x += body_torque[threadIdx.x + offset].x;
            body_torque[threadIdx.x].y += body_torque[threadIdx.x + offset].y;
            body_torque[threadIdx.x].z += body_torque[threadIdx.x + offset].z;
            }

        offset >>= 1;

        __syncthreads();
        }

    // thread 0 within this body writes out the total force and torque for the body
    if ((threadIdx.x & thread_mask) == 0 && mol_idx[m] != NO_BODY)
        {
        d_force[central_idx[m]] = make_scalar4(body_force[threadIdx.x].x, body_force[threadIdx.x].y, body_force[threadIdx.x].z, 0.0f);
        d_torque[central_idx[m]] = make_scalar4(body_torque[threadIdx.x].x, body_torque[threadIdx.x].y, body_torque[threadIdx.x].z, 0.0f);
        }
    }


/*!
*/
hipError_t gpu_rigid_force(Scalar4* d_force,
                 Scalar4* d_torque,
                 const unsigned int *d_molecule_len,
                 const unsigned int *d_molecule_list,
                 const unsigned int *d_tag,
                 const unsigned int *d_rtag,
                 Index2D molecule_indexer,
                 const Scalar4 *d_postype,
                 const Scalar4* d_orientation,
                 Index2D body_indexer,
                 Scalar3* d_body_pos,
                 Scalar4* d_body_orientation,
                 const Scalar4* d_net_force,
                 const Scalar4* d_net_torque,
                 unsigned int n_mol,
                 unsigned int N,
                 unsigned int n_bodies_per_block,
                 unsigned int block_size,
                 const hipDeviceProp_t& dev_prop)
    {
    // reset force and torque
    hipMemset(d_force, 0, sizeof(Scalar4)*N);
    hipMemset(d_torque, 0, sizeof(Scalar4)*N);

    dim3 force_grid(n_mol / n_bodies_per_block + 1, 1, 1);

    static unsigned int max_block_size = UINT_MAX;
    static hipFuncAttributes attr;
    if (max_block_size == UINT_MAX)
        {
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *)) gpu_rigid_force_sliding_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = max_block_size < block_size ? max_block_size : block_size;

    // round down to nearest power of two
    unsigned int b = 1;
    while (b * 2 < run_block_size) { b *= 2; }
    run_block_size = b;

    unsigned int window_size = run_block_size / n_bodies_per_block;
    unsigned int thread_mask = window_size - 1;

    unsigned int shared_bytes = 2 * run_block_size * sizeof(Scalar3);

    while (shared_bytes + attr.sharedSizeBytes >= dev_prop.sharedMemPerBlock)
        {
        // block size is power of two
        run_block_size /= 2;

        shared_bytes = 2 * run_block_size * sizeof(Scalar3);

        window_size = run_block_size / n_bodies_per_block;
        thread_mask = window_size - 1;
        }

    gpu_rigid_force_sliding_kernel<<< force_grid, run_block_size, shared_bytes >>>(
        d_force,
        d_torque,
        d_molecule_len,
        d_molecule_list,
        d_tag,
        d_rtag,
        molecule_indexer,
        d_postype,
        d_orientation,
        body_indexer,
        d_body_pos,
        d_body_orientation,
        d_net_force,
        d_net_torque,
        n_mol,
        N,
        window_size,
        thread_mask,
        n_bodies_per_block);

        return hipSuccess;
    }


