#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2015 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/
 
* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "ActiveForceComputeGPU.cuh"
#include "saruprngCUDA.h"
#include "EvaluatorConstraintEllipsoid.h"

#include <assert.h>

/*! \file ActiveForceComputeGPU.cu
    \brief Declares GPU kernel code for calculating active forces forces on the GPU. Used by ActiveForceComputeGPU.
*/

//! Kernel for adjusting active force vectors to align parallel to an ellipsoid surface constraint on the GPU
/*! \param N number of particles
    \param d_rtag particle tag
    \param d_pos particle positions on device
    \param d_actVec particle active force unit vector
    \param d_actMag particle active force vector magnitude
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
*/
extern "C" __global__
void gpu_compute_active_force_set_constraints_kernel(const unsigned int N,
                                                   const unsigned int *d_rtag,
                                                   const Scalar4 *d_pos,
                                                   Scalar3 *d_actVec,
                                                   const Scalar *d_actMag,
                                                   const Scalar3& P,
                                                   Scalar rx,
                                                   Scalar ry,
                                                   Scalar rz)
{
    //FILL ME IN, FINISH ACTIVE FORCE GPU CODE
    
/////////////////////////////////////
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
        return;
/////////////////////////////////////

    
    EvaluatorConstraintEllipsoid Ellipsoid(P, rx, ry, rz);
    unsigned int idx = d_rtag[i]; // recover original tag for particle indexing
    Scalar3 current_pos = make_scalar3(d_pos[idx].x, d_pos[idx].y, d_pos[idx].z);
                
    Scalar3 norm_scalar3 = Ellipsoid.evalNormal(current_pos); // the normal vector to which the particles are confined.
    vec3<Scalar> norm;
    norm = vec3<Scalar>(norm_scalar3);
    Scalar dot_prod = d_actVec[i].x * norm.x + d_actVec[i].y * norm.y + d_actVec[i].z * norm.z;

    d_actVec[i].x -= norm.x * dot_prod;
    d_actVec[i].y -= norm.y * dot_prod;
    d_actVec[i].z -= norm.z * dot_prod;

    Scalar new_norm = sqrt(d_actVec[i].x * d_actVec[i].x
                        + d_actVec[i].y * d_actVec[i].y
                        + d_actVec[i].z * d_actVec[i].z);

    d_actVec[i].x /= new_norm;
    d_actVec[i].y /= new_norm;
    d_actVec[i].z /= new_norm;
}

//! Kernel for applying rotational diffusion to active force vectors on the GPU
/*! \param N number of particles
    \param d_rtag particle tag
    \param d_pos particle positions on device
    \param d_actVec particle active force unit vector
    \param d_actMag particle active force vector magnitude
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
    \param is2D check if simulation is 2D or 3D
    \param rotationDiff particle rotational diffusion constant
    \param deltaT step size from the Integrator
    \param timestep current timestep
    \param seed seed for random number generator
*/
__global__ void gpu_compute_active_force_rotational_diffusion_kernel(const unsigned int N,
                                                   const unsigned int *d_rtag,
                                                   const Scalar4 *d_pos,
                                                   Scalar3 *d_actVec,
                                                   const Scalar *d_actMag,
                                                   const Scalar3& P,
                                                   Scalar rx,
                                                   Scalar ry,
                                                   Scalar rz,
                                                   bool is2D,
                                                   const Scalar rotationDiff,
                                                   const Scalar deltaT,
                                                   const unsigned int timestep,
                                                   const int seed)
{
    //FILL ME IN, FINISH ACTIVE FORCE GPU CODE
    
/////////////////////////////////////
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
        return;
/////////////////////////////////////
    
    
    if (is2D) // 2D
    {
        //USE VECTOR MATH TO SIMPLIFY THINGS? CHECK UNITS AND MAGNITUDES, ALL CHECK OUT?
        SaruGPU saru(i, timestep, seed);
        Scalar delta_theta; // rotational diffusion angle
        delta_theta = deltaT * rotationDiff * gaussian_rng(saru, 1.0);
        Scalar theta; // angle on plane defining orientation of active force vector
        theta = atan2(d_actVec[i].y, d_actVec[i].x);
        theta += delta_theta;
        d_actVec[i].x = cos(theta);
        d_actVec[i].y = sin(theta);

    } else // 3D: Following Stenhammar, Soft Matter, 2014
    {
        if (rx == 0) // if no constraint
        {
            //USE VECTOR MATH TO SIMPLIFY THINGS? CHECK UNITS AND MAGNITUDES OF DIFFUSION CONSTANT, ALL CHECK OUT?
            SaruGPU saru(i, timestep, seed);
            Scalar u = saru.d(0, 1.0); // generates an even distribution of random unit vectors in 3D
            Scalar v = saru.d(0, 1.0);
            Scalar theta = 2.0 * M_PI * u;
            Scalar phi = acos(2.0 * v - 1.0);
            vec3<Scalar> rand_vec;
            rand_vec.x = sin(phi) * cos(theta);
            rand_vec.y = sin(phi) * sin(theta);
            rand_vec.z = cos(phi);
            Scalar diffusion_mag = deltaT * rotationDiff * gaussian_rng(saru, 1.0);
            vec3<Scalar> delta_vec;
            delta_vec.x = d_actVec[i].y * rand_vec.z - d_actVec[i].z * rand_vec.y;
            delta_vec.y = d_actVec[i].z * rand_vec.x - d_actVec[i].x * rand_vec.z;
            delta_vec.z = d_actVec[i].x * rand_vec.y - d_actVec[i].y * rand_vec.x;
            d_actVec[i].x += delta_vec.x * diffusion_mag;
            d_actVec[i].y += delta_vec.y * diffusion_mag;
            d_actVec[i].z += delta_vec.z * diffusion_mag;
            Scalar new_mag = sqrt(d_actVec[i].x * d_actVec[i].x + d_actVec[i].y * d_actVec[i].y + d_actVec[i].z * d_actVec[i].z);
            d_actVec[i].x /= new_mag;
            d_actVec[i].y /= new_mag;
            d_actVec[i].z /= new_mag;

        } else // if constraint
        {
            EvaluatorConstraintEllipsoid Ellipsoid(P, rx, ry, rz);

            SaruGPU saru(i, timestep, seed);
            unsigned int idx = d_rtag[i]; // recover original tag for particle indexing
            Scalar3 current_pos = make_scalar3(d_pos[idx].x, d_pos[idx].y, d_pos[idx].z);
            Scalar3 norm_scalar3 = Ellipsoid.evalNormal(current_pos); // the normal vector to which the particles are confined.

            vec3<Scalar> norm;
            norm = vec3<Scalar> (norm_scalar3);

            vec3<Scalar> current_vec;
            current_vec.x = d_actVec[i].x;
            current_vec.y = d_actVec[i].y;
            current_vec.z = d_actVec[i].z;
            vec3<Scalar> aux_vec = cross(current_vec, norm); // aux vec for defining direction that active force vector rotates towards.

            Scalar delta_theta; // rotational diffusion angle
            delta_theta = deltaT * rotationDiff * gaussian_rng(saru, 1.0);

            d_actVec[i].x = cos(delta_theta) * current_vec.x + sin(delta_theta) * aux_vec.x;
            d_actVec[i].y = cos(delta_theta) * current_vec.y + sin(delta_theta) * aux_vec.y;
            d_actVec[i].z = cos(delta_theta) * current_vec.z + sin(delta_theta) * aux_vec.z;
        }
    }
}

//! Kernel for setting active force vectors on the GPU
/*! \param N number of particles
    \param d_rtag particle tag
    \param d_force particle force on device
    \param d_orientation particle orientation on device
    \param d_actVec particle active force unit vector
    \param d_actMag particle active force vector magnitude
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
    \param orientationLink check if particle orientation is linked to active force vector
*/
__global__ void gpu_compute_active_force_set_forces_kernel(const unsigned int N,
                                                   const unsigned int *d_rtag, 
                                                   Scalar4 *d_force,
                                                   const Scalar4 *d_orientation,
                                                   const Scalar3 *d_actVec,
                                                   const Scalar *d_actMag,
                                                   const Scalar3& P,
                                                   Scalar rx,
                                                   Scalar ry,
                                                   Scalar rz,
                                                   bool orientationLink)
{
    //FILL ME IN, FINISH ACTIVE FORCE GPU CODE
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

//////////////////////////////////
    if (i >= N)
        return;

    // why use groups?
    // unsigned int i = d_group_members[id];
    unsigned int idx = d_rtag[i];
//////////////////////////////////
    
    Scalar3 f;
    // unsigned int idx = h_rtag[i]; // recover original tag for particle indexing
    // rotate force according to particle orientation only if orientation is linked to active force vector and there are rigid bodies
    if (orientationLink)
    {
        vec3<Scalar> fi;
        f = make_scalar3(d_actMag[i] * d_actVec[i].x, d_actMag[i] * d_actVec[i].y, d_actMag[i] * d_actVec[i].z);
        quat<Scalar> quati(d_orientation[idx]);
        fi = rotate(quati, vec3<Scalar>(f));
        d_force[idx].x = fi.x;
        d_force[idx].y = fi.y;
        d_force[idx].z = fi.z;
    } else // no orientation link
    {
        f = make_scalar3(d_actMag[i] * d_actVec[i].x, d_actMag[i] * d_actVec[i].y, d_actMag[i] * d_actVec[i].z);
        d_force[idx].x = f.x;
        d_force[idx].y = f.y;
        d_force[idx].z = f.z;
    }
}



hipError_t gpu_compute_active_force_set_constraints(const unsigned int N,
                                                   const unsigned int *d_rtag,
                                                   const Scalar4 *d_pos,
                                                   Scalar4 *d_force,
                                                   Scalar3 *d_actVec,
                                                   const Scalar *d_actMag,
                                                   const Scalar3& P,
                                                   Scalar rx,
                                                   Scalar ry,
                                                   Scalar rz,
                                                   unsigned int block_size)
{
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipMemset(d_force, 0, sizeof(Scalar4)*N);
    gpu_compute_active_force_set_constraints_kernel<<< grid, threads>>>(N,
                                                                    d_rtag,
                                                                    d_pos,
                                                                    d_actVec,
                                                                    d_actMag,
                                                                    P,
                                                                    rx,
                                                                    ry,
                                                                    rz);

    return hipSuccess;
}

hipError_t gpu_compute_active_force_rotational_diffusion(const unsigned int N,
                                                       const unsigned int *d_rtag,
                                                       const Scalar4 *d_pos,
                                                       Scalar4 *d_force,
                                                       Scalar3 *d_actVec,
                                                       const Scalar *d_actMag,
                                                       const Scalar3& P,
                                                       Scalar rx,
                                                       Scalar ry,
                                                       Scalar rz,
                                                       bool is2D,
                                                       const Scalar rotationDiff,
                                                       const Scalar deltaT,
                                                       const unsigned int timestep,
                                                       const int seed,
                                                       unsigned int block_size)
{
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipMemset(d_force, 0, sizeof(Scalar4)*N);
    gpu_compute_active_force_rotational_diffusion_kernel<<< grid, threads>>>(N,
                                                                    d_rtag,
                                                                    d_pos,
                                                                    d_actVec,
                                                                    d_actMag,
                                                                    P,
                                                                    rx,
                                                                    ry,
                                                                    rz,
                                                                    is2D,
                                                                    rotationDiff,
                                                                    deltaT,
                                                                    timestep,
                                                                    seed);

    return hipSuccess;
}

hipError_t gpu_compute_active_force_set_forces(const unsigned int N,
                                           const unsigned int *d_rtag,
                                           Scalar4 *d_force,
                                           const Scalar4 *d_orientation,
                                           const Scalar3 *d_actVec,
                                           const Scalar *d_actMag,
                                           const Scalar3& P,
                                           Scalar rx,
                                           Scalar ry,
                                           Scalar rz,
                                           bool orientationLink,
                                           unsigned int block_size)
{
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipMemset(d_force, 0, sizeof(Scalar4)*N);
    gpu_compute_active_force_set_forces_kernel<<< grid, threads>>>( N,
                                                                    d_rtag,
                                                                    d_force,
                                                                    d_orientation,
                                                                    d_actVec,
                                                                    d_actMag,
                                                                    P,
                                                                    rx,
                                                                    ry,
                                                                    rz,
                                                                    orientationLink);

    return hipSuccess;
}










