#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008, 2009 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

Redistribution and use of HOOMD-blue, in source and binary forms, with or
without modification, are permitted, provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of HOOMD-blue's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS''
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR
ANY WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
// Maintainer: joaander / Everyone is free to add additional potentials

/*! \file AllDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairLJ.h"
#include "EvaluatorPairGauss.h"
#include "EvaluatorPairSLJ.h"
#include "EvaluatorPairYukawa.h"
#include "EvaluatorPairMorse.h"
#include "PotentialPairDPDThermoGPU.cuh"
#include "EvaluatorPairDPDThermo.h"
#include "AllDriverPotentialPairGPU.cuh"
#include "EvaluatorPairEwald.h"

hipError_t gpu_compute_ljtemp_forces(const gpu_force_data_arrays& force_data,
                                      const gpu_pdata_arrays &pdata,
                                      const gpu_boxsize &box,
                                      const unsigned int *d_n_neigh,
                                      const unsigned int *d_nlist,
                                      const Index2D& nli,
                                      const float2 *d_params,
                                      const float *d_rcutsq,
                                      const float *d_ronsq,
                                      const unsigned int ntypes,
                                      const unsigned int block_size,
                                      const unsigned int shift_mode)
    {
    return gpu_compute_pair_forces<EvaluatorPairLJ>(force_data,
                                                    pdata,
                                                    box,
                                                    d_n_neigh,
                                                    d_nlist,
                                                    nli,
                                                    d_params,
                                                    d_rcutsq,
                                                    d_ronsq,
                                                    ntypes,
                                                    block_size,
                                                    shift_mode);
    }

hipError_t gpu_compute_gauss_forces(const gpu_force_data_arrays& force_data,
                                     const gpu_pdata_arrays &pdata,
                                     const gpu_boxsize &box,
                                     const unsigned int *d_n_neigh,
                                     const unsigned int *d_nlist,
                                     const Index2D& nli,
                                     const float2 *d_params,
                                     const float *d_rcutsq,
                                     const float *d_ronsq,
                                     const unsigned int ntypes,
                                     const unsigned int block_size,
                                     const unsigned int shift_mode)
    {
    return gpu_compute_pair_forces<EvaluatorPairGauss>(force_data,
                                                       pdata,
                                                       box,
                                                       d_n_neigh,
                                                       d_nlist,
                                                       nli,
                                                       d_params,
                                                       d_rcutsq,
                                                       d_ronsq,
                                                       ntypes,
                                                       block_size,
                                                       shift_mode);
    }

hipError_t gpu_compute_slj_forces(const gpu_force_data_arrays& force_data,
                                   const gpu_pdata_arrays &pdata,
                                   const gpu_boxsize &box,
                                   const unsigned int *d_n_neigh,
                                   const unsigned int *d_nlist,
                                   const Index2D& nli,
                                   const float2 *d_params,
                                   const float *d_rcutsq,
                                   const float *d_ronsq,
                                   const unsigned int ntypes,
                                   const unsigned int block_size,
                                   const unsigned int shift_mode)
    {
    return gpu_compute_pair_forces<EvaluatorPairSLJ>(force_data,
                                                     pdata,
                                                     box,
                                                     d_n_neigh,
                                                     d_nlist,
                                                     nli,
                                                     d_params,
                                                     d_rcutsq,
                                                     d_ronsq,
                                                     ntypes,
                                                     block_size,
                                                     shift_mode);
    }

hipError_t gpu_compute_yukawa_forces(const gpu_force_data_arrays& force_data,
                                      const gpu_pdata_arrays &pdata,
                                      const gpu_boxsize &box,
                                      const unsigned int *d_n_neigh,
                                      const unsigned int *d_nlist,
                                      const Index2D& nli,
                                      const float2 *d_params,
                                      const float *d_rcutsq,
                                      const float *d_ronsq,
                                      const unsigned int ntypes,
                                      const unsigned int block_size,
                                      const unsigned int shift_mode)
    {
    return gpu_compute_pair_forces<EvaluatorPairYukawa>(force_data,
                                                        pdata,
                                                        box,
                                                        d_n_neigh,
                                                        d_nlist,
                                                        nli,
                                                        d_params,
                                                        d_rcutsq,
                                                        d_ronsq,
                                                        ntypes,
                                                        block_size,
                                                        shift_mode);
    }


hipError_t gpu_compute_morse_forces(const gpu_force_data_arrays& force_data,
                                     const gpu_pdata_arrays &pdata,
                                     const gpu_boxsize &box,
                                     const unsigned int *d_n_neigh,
                                     const unsigned int *d_nlist,
                                     const Index2D& nli,
                                     const float4 *d_params,
                                     const float *d_rcutsq,
                                     const float *d_ronsq,
                                     const unsigned int ntypes,
                                     const unsigned int block_size,
                                     const unsigned int shift_mode)
    {
    return gpu_compute_pair_forces<EvaluatorPairMorse>(force_data,
                                                       pdata,
                                                       box,
                                                       d_n_neigh,
                                                       d_nlist,
                                                       nli,
                                                       d_params,
                                                       d_rcutsq,
                                                       d_ronsq,
                                                       ntypes,
                                                       block_size,
                                                       shift_mode);
    }

hipError_t gpu_compute_dpdthermodpd_forces(const gpu_force_data_arrays& force_data,
                                            const gpu_pdata_arrays &pdata,
                                            const gpu_boxsize &box,
                                            const unsigned int *d_n_neigh,
                                            const unsigned int *d_nlist,
                                            const Index2D& nli,
                                            const float2 *d_params,
                                            const float *d_rcutsq,
                                            const int ntypes,
                                            const dpd_pair_args& args)
    {
    return gpu_compute_dpd_forces<EvaluatorPairDPDThermo>(force_data,
                                                          pdata,
                                                          box,
                                                          d_n_neigh,
                                                          d_nlist,
                                                          nli,
                                                          d_params,
                                                          d_rcutsq,
                                                          ntypes,
                                                          args);
    }                                          


hipError_t gpu_compute_dpdthermo_forces(const gpu_force_data_arrays& force_data,
                                         const gpu_pdata_arrays &pdata,
                                         const gpu_boxsize &box,
                                         const unsigned int *d_n_neigh,
                                         const unsigned int *d_nlist,
                                         const Index2D& nli,
                                         const float2 *d_params,
                                         const float *d_rcutsq,
                                         const float *d_ronsq,
                                         const unsigned int ntypes,
                                         const unsigned int block_size,
                                         const unsigned int shift_mode)
    {
    return gpu_compute_pair_forces<EvaluatorPairDPDThermo>(force_data,
                                                           pdata,
                                                           box,
                                                           d_n_neigh,
                                                           d_nlist,
                                                           nli,
                                                           d_params,
                                                           d_rcutsq,
                                                           d_ronsq,
                                                           ntypes,
                                                           block_size,
                                                           shift_mode);
    }   


hipError_t gpu_compute_ewald_forces(const gpu_force_data_arrays& force_data,
                     const gpu_pdata_arrays &pdata,
                     const gpu_boxsize &box,
                     const unsigned int *d_n_neigh,
                     const unsigned int *d_nlist,
                     const Index2D& nli,
                     const float *d_params,
                     const float *d_rcutsq,
                     const float *d_ronsq,
                     const unsigned int ntypes,
                     const unsigned int block_size,
                     const unsigned int shift_mode)
    {
    return  gpu_compute_pair_forces<EvaluatorPairEwald>(force_data,
                             pdata,
                             box,
                             d_n_neigh,
                             d_nlist,
                             nli,
                             d_params,
                             d_rcutsq,
                             d_ronsq,
                             ntypes,
                             block_size,
                             shift_mode);
    }

