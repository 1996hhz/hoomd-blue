#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "BondData.cuh"

#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file BondData.cu
    \brief Implements the helper functions (GPU version) for updating the GPU bond table
*/

//! Kernel to find the maximum number of angles per particle
__global__ void gpu_find_max_bond_number_kernel(const uint2 *bonds,
                                             const unsigned int *d_rtag,
                                             unsigned int *d_n_bonds,
                                             unsigned int num_bonds,
                                             unsigned int N,
                                             bool ghost_bonds)
    {
    int bond_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (bond_idx >= num_bonds)
        return;

    uint2 bond = bonds[bond_idx];
    unsigned int tag1 = bond.x;
    unsigned int tag2 = bond.y;
    unsigned int idx1 = d_rtag[tag1];
    unsigned int idx2 = d_rtag[tag2];

    if (idx1 < N && ((idx2 < N && !ghost_bonds) || (idx2 >= N && ghost_bonds)))
        atomicInc(&d_n_bonds[idx1], 0xffffffff);
    if (idx2 < N && ((idx1 < N && !ghost_bonds) || (idx1 >= N && ghost_bonds)))
        atomicInc(&d_n_bonds[idx2], 0xffffffff);

    }

//! Kernel to fill the GPU bond table
__global__ void gpu_fill_gpu_bond_table(const uint2 *bonds,
                                        const unsigned int *bond_type,
                                        uint2 *gpu_btable,
                                        const unsigned int pitch,
                                        const unsigned int *d_rtag,
                                        unsigned int *d_n_bonds,
                                        unsigned int num_bonds,
                                        unsigned int N,
                                        bool ghost_bonds)
    {
    int bond_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (bond_idx >= num_bonds)
        return;

    uint2 bond = bonds[bond_idx];
    unsigned int tag1 = bond.x;
    unsigned int tag2 = bond.y;
    unsigned int type = bond_type[bond_idx];
    unsigned int idx1 = d_rtag[tag1];
    unsigned int idx2 = d_rtag[tag2];

    if (idx1 < N && ((idx2 < N && !ghost_bonds) || (idx2 >= N && ghost_bonds)))
        {
        unsigned int num1 = atomicInc(&d_n_bonds[idx1],0xffffffff);
        gpu_btable[num1*pitch+idx1] = make_uint2(idx2,type);
        }
    if (idx2 < N && ((idx1 < N && !ghost_bonds) || (idx1 >= N && ghost_bonds)))
        {
        unsigned int num2 = atomicInc(&d_n_bonds[idx2],0xffffffff);
        gpu_btable[num2*pitch+idx2] = make_uint2(idx1,type);
        }
    }


//! Find the maximum number of bonds per particle
/*! \param max_bond_num Maximum number of bonds (return value)
    \param d_n_bonds Number of bonds per particle (return array)
    \param d_bonds Array of bonds
    \param num_bonds Size of bond array
    \param N Number of particles in the system
    \param d_rtag Array of reverse-lookup particle tag . particle index
    \param use_ghost_bonds True if we are only considering bonds with ghost particles
 */
hipError_t gpu_find_max_bond_number(unsigned int& max_bond_num,
                                     unsigned int *d_n_bonds,
                                     const uint2 *d_bonds,
                                     const unsigned int num_bonds,
                                     const unsigned int N,
                                     const unsigned int *d_rtag,
                                     bool use_ghost_bonds)
    {
    assert(d_bonds);
    assert(d_rtag);
    assert(d_n_bonds);

    unsigned int block_size = 512;

    // clear n_bonds array
    hipMemset(d_n_bonds, 0, sizeof(unsigned int) * N);

    gpu_find_max_bond_number_kernel<<<num_bonds/block_size + 1, block_size>>>(d_bonds,
                                                                              d_rtag,
                                                                              d_n_bonds,
                                                                              num_bonds,
                                                                              N,
                                                                              use_ghost_bonds);

    thrust::device_ptr<unsigned int> n_bonds_ptr(d_n_bonds);
    max_bond_num = *thrust::max_element(n_bonds_ptr, n_bonds_ptr + N);
    return hipSuccess;
    }

//! Construct the GPU bond table
/*! \param d_gpu_bondtable Pointer to the bond table on the GPU
    \param d_n_bonds Number of bonds per particle (return array)
    \param d_bonds Bonds array
    \param d_bond_type Array of bond types
    \param d_rtag Reverse-lookup tag->index
    \param num_bonds Number of bonds in bond list
    \param pitch Pitch of 2D bondtable array
    \param N Number of particles
    \param use_ghost_bonds True if we are only considering bonds with ghost particles
 */
hipError_t gpu_create_bondtable(uint2 *d_gpu_bondtable,
                                 unsigned int *d_n_bonds,
                                 const uint2 *d_bonds,
                                 const unsigned int *d_bond_type,
                                 const unsigned int *d_rtag,
                                 const unsigned int num_bonds,
                                 unsigned int pitch,
                                 unsigned int N,
                                 bool use_ghost_bonds)
    {
    unsigned int block_size = 512;

    // clear n_bonds array
    hipMemset(d_n_bonds, 0, sizeof(unsigned int) * N);

    gpu_fill_gpu_bond_table<<<num_bonds/block_size + 1, block_size>>>(d_bonds,
                                                                      d_bond_type,
                                                                      d_gpu_bondtable,
                                                                      pitch,
                                                                      d_rtag,
                                                                      d_n_bonds,
                                                                      num_bonds,
                                                                      N,
                                                                      use_ghost_bonds);
    return hipSuccess;
    }

