
#include <hip/hip_runtime.h>
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: jglaser

/*! \file CommunicatorGPU.cu
    \brief Implementation of communication algorithms on the GPU
*/

#ifdef ENABLE_MPI
#include "CommunicatorGPU.cuh"

#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/gather.h>
#include <thrust/scatter.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/copy.h>
#include <thrust/partition.h>
#include <thrust/count.h>

using namespace thrust;

//! Apply (global) periodic boundary conditions to a ghost particle
struct wrap_ghost_particle
    {
    const gpu_boxsize box;  //!< Dimensions of global simulation box
    const float rghost;     //!< Width of ghost layer
    const unsigned int dir; //!< Direction along which particle was received

    //! Constructor
    /*! \param _box Dimensions of global simulation box
     * \param _rghost Width of ghost layer
     * \param _dir Direction along which particle was received
     */
    wrap_ghost_particle(gpu_boxsize _box, float _rghost, unsigned int _dir)
        : box(_box), rghost(_rghost), dir(_dir)
        {
        }

    //! Apply peridoic boundary conditions
    /*! \param pos position element to apply boundary conditions to
     * \return the position element with boundary conditions applied
     */
    __host__ __device__ float4 operator()(const float4 &pos)
        {
            // wrap particles received across a global boundary back into global box
            float4 pos2 = pos;
            if (dir==0 && pos2.x >= box.xhi - rghost)
                pos2.x -= box.xhi - box.xlo;
            else if (dir==1 && pos2.x < box.xlo + rghost)
                pos2.x += box.xhi - box.xlo;
            else if (dir==2 && pos2.y >= box.yhi - rghost)
                pos2.y -= box.yhi - box.ylo;
            else if (dir==3 && pos2.y < box.ylo + rghost)
                pos2.y += box.yhi - box.ylo;
            else if (dir==4 && pos2.z >= box.zhi - rghost)
                pos2.z -= box.zhi - box.zlo;
            else if (dir==5 && pos2.z < box.zlo + rghost)
                pos2.z += box.zhi - box.zlo;
            return pos2;
        }
     };

//! Select local particles that within a boundary layer of the neighboring domain in a given direction
struct select_particle_ghost
    {
    const gpu_boxsize box;    //!< Local box dimensions
    const float r_ghost;      //!< Width of boundary layer
    const unsigned int dir;   //!< Direction of the neighboring domain

    //! Constructor
    /*! \param _box Local box dimensions
     * \param _r_ghost Width of boundary layer
     * \param _dir Direction of the neighboring domain
     */
    select_particle_ghost(const gpu_boxsize _box, float _r_ghost, unsigned int _dir)
        : box(_box), r_ghost(_r_ghost), dir(_dir)
        {
        }

    //! Apply selection criterium
    /*! \param pos the position of the particle to apply the criterium to
        \returns true if particle lies within the boundary layer
     */
    __host__ __device__ bool operator()(const float4 &pos)
        {
        return ((dir==0 && (pos.x >= box.xhi - r_ghost)) ||                  // send east
            (dir==1 && (pos.x < box.xlo + r_ghost) && (pos.x >= box.xlo)) || // send west
            (dir==2 && (pos.y >= box.yhi - r_ghost)) ||                      // send north
            (dir==3 && (pos.y < box.ylo + r_ghost) && (pos.y >= box.ylo)) || // send south
            (dir==4 && (pos.z >= box.zhi - r_ghost)) ||                      // send up
            (dir==5 && (pos.z < box.zlo + r_ghost) && (pos.z >= box.zlo)));  // send down

        }
     };

//! Structure to pack a particle data element into
struct __align__(128) pdata_element_gpu
    {
    float4 pos;               //!< Position
    float4 vel;               //!< Velocity
    float3 accel;             //!< Acceleration
    float charge;             //!< Charge
    float diameter;           //!< Diameter
    int3 image;               //!< Image
    unsigned int body;        //!< Body id
    float4 orientation;       //!< Orientation
    unsigned int global_tag;  //!< global tag
    };

//! Get the size of a \c pdata_element_gpu
/*! The CUDA compiler aligns structure members differently than the C++ compiler. This function is used
    to return the actual size as returned by the CUDA compiler.

    \returns the size of a pdata_element_gpu (in bytes)
 */
unsigned int gpu_pdata_element_size()
    {
    return sizeof(pdata_element_gpu);
    }

//! Define a thrust tuple for a particle data element
typedef thrust::tuple<float4,
                      float4,
                      float3,
                      float,
                      float,
                      int3,
                      unsigned int,
                      float4,
                      unsigned int> pdata_tuple_gpu;

//! Select particles to be sent in a specified direction
struct select_particle_migrate_gpu : public thrust::unary_function<const pdata_tuple_gpu&, bool>
    {
    const float xlo;        //!< Lower x boundary
    const float xhi;        //!< Upper x boundary
    const float ylo;        //!< Lower y boundary
    const float yhi;        //!< Upper y boundary
    const float zlo;        //!< Lower z boundary
    const float zhi;        //!< Upper z boundary
    const unsigned int dir; //!< Direction to send particles to
    const float4 *d_pos;    //!< Device array of particle positions


    //! Constructor
    /*!
     */
    select_particle_migrate_gpu(const float _xlo,
                            const float _xhi,
                            const float _ylo,
                            const float _yhi,
                            const float _zlo,
                            const float _zhi,
                            const unsigned int _dir,
                            const float4 *_d_pos)
        : xlo(_xlo), xhi(_xhi), ylo(_ylo), yhi(_yhi), zlo(_zlo), zhi(_zhi), dir(_dir), d_pos(_d_pos)
        {
        }

    //! Select a particle
    /*! t particle data to consider for sending
     * \return true if particle stays in the box
     */
    __host__ __device__ bool operator()(const unsigned int& idx)
        {
        const float4& pos = d_pos[idx];
        // we return true if the particle stays in our box,
        // false otherwise
        return !((dir == 0 && pos.x >= xhi) ||  // send east
                (dir == 1 && pos.x < xlo)  ||  // send west
                (dir == 2 && pos.y >= yhi) ||  // send north
                (dir == 3 && pos.y < ylo)  ||  // send south
                (dir == 4 && pos.z >= zhi) ||  // send up
                (dir == 5 && pos.z < zlo ));   // send down
        }

     };

//! Wrap a received particle across global box boundaries
struct wrap_received_particle
    {
    const gpu_boxsize box;   //!< Dimensions of global simulation box
    const unsigned int dir;  //!< Direction along which the particle was received

    //! Constructor
    /*! \param _box Dimensions of global simulation box
        \param _dir Direciton along whic the particle was received
     */
    wrap_received_particle(const gpu_boxsize _box, unsigned int _dir)
        : box(_box), dir(_dir)
        {
        }

   //! Wrap particle across boundaries
   /*! \param el particle data element to transform
    * \return transformed particle data element
    */
    __host__ __device__ pdata_element_gpu operator()(const pdata_element_gpu & el)
        {
        pdata_element_gpu el2 = el;
        float4& pos = el2.pos;
        int3& image = el2.image;

        if (dir == 0 && pos.x >= box.xhi)
            {
            pos.x -= box.xhi - box.xlo;
            image.x++;
            }
        else if (dir == 1 && pos.x < box.xlo)
            {
            pos.x += box.xhi - box.xlo;
            image.x--;
            }

        if (dir == 2 && pos.y >= box.yhi)
            {
            pos.y -= box.yhi - box.ylo;
            image.y++;
            }
        else if (dir == 3 && pos.y < box.ylo)
            {
            pos.y += box.yhi - box.ylo;
            image.y--;
            }

        if (dir == 4 && pos.z >= box.zhi)
            {
            pos.z -= box.zhi - box.zlo;
            image.z++;
            }
        else if (dir == 5 && pos.z < box.zlo)
            {
            pos.z += box.zhi - box.zlo;
            image.z--;
            }
        return el2;
        }

     };


//! Determine whether a received particle is to be added to the local box
struct isInBox
    {
    const gpu_boxsize box;  //!< Local box dimensions

    //! Constructor
    /* \param _box Local box dimensions
     */
    isInBox(const gpu_boxsize _box)
        : box(_box)
        {
        }

    //! Determine whether particle is in local box
    /*! \param pos Position of the particle to check
     * \return true if position is in local box
     */
    __host__ __device__ bool check_ptl(const float4& pos)
        {
        return (box.xlo <= pos.x  && pos.x < box.xhi) &&
               (box.ylo <= pos.y  && pos.y < box.yhi) &&
               (box.zlo <= pos.z  && pos.z < box.zhi);
        }

    //! Determine whether particle is in local box
    /*! \param el the particle data element to apply the criterium to
     * \return true if the particle is to be added to the local particle data
     */
    __host__ __device__ bool operator()(const pdata_element_gpu & el)
        {
        return check_ptl(el.pos);
        }

    //! Determine whether particle is in local box
    /*! \param t the particle data tuple to apply the criterium to
     * \return true if the particle is to be added to the local particle data
     */
    __host__ __device__ bool operator()(const pdata_tuple_gpu & t)
        {
        return check_ptl(thrust::get<0>(t));
        }
     };

//! Pack a particle data tuple
struct pack_pdata : public thrust::unary_function<pdata_tuple_gpu, pdata_element_gpu>
    {
    //! Transform operator
    /*! \param t Particle data tuple to pack
     * \return Packed particle data element
     */
    __host__ __device__ pdata_element_gpu operator()(const pdata_tuple_gpu& t)
        {
        pdata_element_gpu el;
        el.pos  = thrust::get<0>(t);
        el.vel  = thrust::get<1>(t);
        el.accel= thrust::get<2>(t);
        el.charge = thrust::get<3>(t);
        el.diameter = thrust::get<4>(t);
        el.image = thrust::get<5>(t);
        el.body = thrust::get<6>(t);
        el.orientation = thrust::get<7>(t);
        el.global_tag = thrust::get<8>(t);
        return el;
        }
    };

//! Unpack a particle data element
struct unpack_pdata : public thrust::unary_function<pdata_element_gpu, pdata_tuple_gpu>
    {
    //! Transform operator
    /*! \param el Particle data element to unpack
     */
    __host__ __device__ pdata_tuple_gpu operator()(const pdata_element_gpu & el)
        {
        return pdata_tuple_gpu(el.pos,
                           el.vel,
                           el.accel,
                           el.charge,
                           el.diameter,
                           el.image,
                           el.body,
                           el.orientation,
                           el.global_tag);
        }
    };


thrust::device_vector<unsigned int> *keys;       //!< Temporary vector of sort keys
thrust::device_vector<float4> *float4_tmp;       //!< Temporary vector for permutating particle data
thrust::device_vector<float3> *float3_tmp;       //!< Temporary vector for permutating particle data
thrust::device_vector<float> *float_tmp;         //!< Temporary vector for permutating particle data
thrust::device_vector<unsigned int> *uint_tmp;   //!< Temporary vector for permutating particle data
thrust::device_vector<int3> *int3_tmp;           //!< Temporary vector for permutating particle data

void gpu_allocate_tmp_storage()
    {
    keys = new thrust::device_vector<unsigned int>;
    float4_tmp = new thrust::device_vector<float4>;
    float3_tmp = new thrust::device_vector<float3>;
    float_tmp = new thrust::device_vector<float>;
    uint_tmp = new thrust::device_vector<unsigned int>;
    int3_tmp = new thrust::device_vector<int3>;
    }

void gpu_deallocate_tmp_storage()
    {
    delete keys;
    delete float4_tmp;
    delete float3_tmp;
    delete float_tmp;
    delete uint_tmp;
    delete int3_tmp;
    }

/*! Reorder the particles according to a migration criterium
 *  Particles that remain in the simulation box come first, followed by the particles that are sent in the
 *  specified direction
 *
 *  \param N Number of particles in local simulation box
 *  \param n_send_ptls Number of particles that are sent (return value)
 *  \param d_pos Array of particle positions
 *  \param d_vel Array of particle velocities
 *  \param d_accel Array of particle accelerations
 *  \param d_image Array of particle images
 *  \param d_charge Array of particle charges
 *  \param d_diameter Array of particle diameter
 *  \param d_body Array of particle body ids
 *  \param d_orientation Array of particle orientations
 *  \param d_tag Array of particle global tags
 *  \param box Dimensions of local simulation box
 *  \param dir Direction to send particles to
 */
void gpu_migrate_select_particles(unsigned int N,
                        unsigned int &n_send_ptls,
                        float4 *d_pos,
                        float4 *d_vel,
                        float3 *d_accel,
                        int3 *d_image,
                        float *d_charge,
                        float *d_diameter,
                        unsigned int *d_body,
                        float4  *d_orientation,
                        unsigned int *d_tag,
                        gpu_boxsize box,
                        unsigned int dir)
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<float4> vel_ptr(d_vel);
    thrust::device_ptr<float3> accel_ptr(d_accel);
    thrust::device_ptr<int3> image_ptr(d_image);
    thrust::device_ptr<float> charge_ptr(d_charge);
    thrust::device_ptr<float> diameter_ptr(d_diameter);
    thrust::device_ptr<unsigned int> body_ptr(d_body);
    thrust::device_ptr<float4> orientation_ptr(d_orientation);
    thrust::device_ptr<unsigned int> tag_ptr(d_tag);

    if (keys->size() < N)
        {
        unsigned int cur_size = keys->size() ? keys->size() : N;
        while (cur_size < N) cur_size *= 2;
        keys->resize(cur_size);
        float4_tmp->resize(cur_size);
        float3_tmp->resize(cur_size);
        float_tmp->resize(cur_size);
        uint_tmp->resize(cur_size);
        int3_tmp->resize(cur_size);
        }

    thrust::counting_iterator<unsigned int> count(0);
    thrust::copy(count, count + N, keys->begin());

    thrust::device_vector<unsigned int>::iterator keys_middle;

    keys_middle = thrust::stable_partition(keys->begin(),
                             keys->begin() + N,
                             select_particle_migrate_gpu(box.xlo, box.xhi, box.ylo, box.yhi, box.zlo, box.zhi, dir, d_pos));

    n_send_ptls = (keys->begin() + N) - keys_middle;

    // reorder particle data
    thrust::copy(thrust::make_permutation_iterator(pos_ptr, keys->begin()),
                 thrust::make_permutation_iterator(pos_ptr + N, keys->begin() +N),
                 float4_tmp->begin());
    thrust::copy(float4_tmp->begin(),
                 float4_tmp->begin() + N,
                 pos_ptr);

    thrust::copy(thrust::make_permutation_iterator(vel_ptr, keys->begin()),
                 thrust::make_permutation_iterator(vel_ptr + N, keys->begin() +N),
                 float4_tmp->begin());
    thrust::copy(float4_tmp->begin(),
                 float4_tmp->begin() + N,
                 vel_ptr);

    thrust::copy(thrust::make_permutation_iterator(accel_ptr, keys->begin()),
                 thrust::make_permutation_iterator(accel_ptr + N, keys->begin() +N),
                 float3_tmp->begin());
    thrust::copy(float3_tmp->begin(),
                 float3_tmp->begin() + N,
                 accel_ptr);

    thrust::copy(thrust::make_permutation_iterator(charge_ptr, keys->begin()),
                 thrust::make_permutation_iterator(charge_ptr + N, keys->begin() +N),
                 float_tmp->begin());
    thrust::copy(float_tmp->begin(),
                 float_tmp->begin() + N,
                 charge_ptr);

    thrust::copy(thrust::make_permutation_iterator(diameter_ptr, keys->begin()),
                 thrust::make_permutation_iterator(diameter_ptr + N, keys->begin() +N),
                 float_tmp->begin());
    thrust::copy(float_tmp->begin(),
                 float_tmp->begin() + N,
                 diameter_ptr);

    thrust::copy(thrust::make_permutation_iterator(image_ptr, keys->begin()),
                 thrust::make_permutation_iterator(image_ptr + N, keys->begin() +N),
                 int3_tmp->begin());
    thrust::copy(int3_tmp->begin(),
                 int3_tmp->begin() + N,
                 image_ptr);

    thrust::copy(thrust::make_permutation_iterator(body_ptr, keys->begin()),
                 thrust::make_permutation_iterator(body_ptr + N, keys->begin() +N),
                 uint_tmp->begin());
    thrust::copy(uint_tmp->begin(),
                 uint_tmp->begin() + N,
                 body_ptr);

    thrust::copy(thrust::make_permutation_iterator(orientation_ptr, keys->begin()),
                 thrust::make_permutation_iterator(orientation_ptr + N, keys->begin() +N),
                 float4_tmp->begin());
    thrust::copy(float4_tmp->begin(),
                 float4_tmp->begin() + N,
                 orientation_ptr);

    thrust::copy(thrust::make_permutation_iterator(tag_ptr, keys->begin()),
                 thrust::make_permutation_iterator(tag_ptr + N, keys->begin() +N),
                 uint_tmp->begin());
    thrust::copy(uint_tmp->begin(),
                 uint_tmp->begin() + N,
                 tag_ptr);
    }

//! Pack particle data into send buffer
/*! \param N number of particles to check for sending
   \param d_pos Array of particle positions
   \param d_vel Array of particle velocities
   \param d_accel Array of particle accelerations
   \param d_image Array of particle images
   \param d_charge Array of particle charges
   \param d_diameter Array of particle diameter
   \param d_body Array of particle body ids
   \param d_orientation Array of particle orientations
   \param d_tag Array of particle global tags
   \param d_send_buf Send buffer (has to be large enough, i.e. maxium size = number of local particles )
   \param d_send_buf_end Pointer to end of send buffer (return value)
*/
void gpu_migrate_pack_send_buffer(unsigned int N,
                           float4 *d_pos,
                           float4 *d_vel,
                           float3 *d_accel,
                           int3 *d_image,
                           float *d_charge,
                           float *d_diameter,
                           unsigned int *d_body,
                           float4  *d_orientation,
                           unsigned int *d_tag,
                           char *d_send_buf,
                           char *&d_send_buf_end)
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<float4> vel_ptr(d_vel);
    thrust::device_ptr<float3> accel_ptr(d_accel);
    thrust::device_ptr<int3> image_ptr(d_image);
    thrust::device_ptr<float> charge_ptr(d_charge);
    thrust::device_ptr<float> diameter_ptr(d_diameter);
    thrust::device_ptr<unsigned int> body_ptr(d_body);
    thrust::device_ptr<float4> orientation_ptr(d_orientation);
    thrust::device_ptr<unsigned int> tag_ptr(d_tag);
    thrust::device_ptr<pdata_element_gpu> send_buf_ptr((pdata_element_gpu *) d_send_buf);

    // we perform operations on the whole particle data
    typedef thrust::tuple<thrust::device_ptr<float4>,
                          thrust::device_ptr<float4>,
                          thrust::device_ptr<float3>,
                          thrust::device_ptr<float>,
                          thrust::device_ptr<float>,
                          thrust::device_ptr<int3>,
                          thrust::device_ptr<unsigned int>,
                          thrust::device_ptr<float4>,
                          thrust::device_ptr<unsigned int> > pdata_iterator_tuple;

    thrust::zip_iterator<pdata_iterator_tuple> pdata_first = thrust::make_tuple( pos_ptr,
                                               vel_ptr,
                                               accel_ptr,
                                               charge_ptr,
                                               diameter_ptr,
                                               image_ptr,
                                               body_ptr,
                                               orientation_ptr,
                                               tag_ptr);
    thrust::zip_iterator<pdata_iterator_tuple> pdata_end = pdata_first + N;


    // pack the particles into the send buffer
    thrust::device_ptr<pdata_element_gpu> send_buf_end_ptr =
        thrust::copy(thrust::make_transform_iterator(pdata_first, pack_pdata()),
                     thrust::make_transform_iterator(pdata_end, pack_pdata()),
                     send_buf_ptr);

    d_send_buf_end = (char *) thrust::raw_pointer_cast(send_buf_end_ptr);
    }

//! Wrap received particles across global box boundaries
/*! \param d_recv_buf Received particle data
 * \param d_recv_buf_end End of received particle data
 * \param n_recv_ptl Number of received particles (return value)
 * \param global_box Dimensions of global box
 * \param dir Direction along which particles where received
 */
void gpu_migrate_wrap_received_particles(char *d_recv_buf,
                                 char *d_recv_buf_end,
                                 unsigned int &n_recv_ptl,
                                 const gpu_boxsize& global_box,
                                 unsigned int dir)
    {
    thrust::device_ptr<pdata_element_gpu> recv_buf_ptr((pdata_element_gpu *) d_recv_buf);
    thrust::device_ptr<pdata_element_gpu> recv_buf_end_ptr((pdata_element_gpu *) d_recv_buf_end);
    thrust::transform(recv_buf_ptr, recv_buf_end_ptr, recv_buf_ptr, wrap_received_particle(global_box, dir));
    n_recv_ptl = recv_buf_end_ptr - recv_buf_ptr;
    }

//! Add received particles to local box if their positions are inside the local boundaries
/*! \param d_recv_buf Buffer of received particle data
 * \param d_recv_buf_end Pointer to end of receive buffer
 * \param d_pos Array to store particle positions
 * \param d_vel Array to store particle velocities
 * \param d_accel Array to store particle accelerations
 * \param d_image Array to store particle images
 * \param d_charge Array to store particle charges
 * \param d_diameter Array to store particle diameters
 * \param d_body Array to store particle body ids
 * \param d_orientation Array to store particle body orientations
 * \param d_tag Array to store particle global tags
 * \param box Local box dimensions
 */
void gpu_migrate_add_particles(  char *d_recv_buf,
                                 char *d_recv_buf_end,
                                 float4 *d_pos,
                                 float4 *d_vel,
                                 float3 *d_accel,
                                 int3 *d_image,
                                 float *d_charge,
                                 float *d_diameter,
                                 unsigned int *d_body,
                                 float4  *d_orientation,
                                 unsigned int *d_tag,
                                 const gpu_boxsize &box)
    {
    thrust::device_ptr<pdata_element_gpu> recv_buf_ptr((pdata_element_gpu *) d_recv_buf);
    thrust::device_ptr<pdata_element_gpu> recv_buf_end_ptr((pdata_element_gpu *) d_recv_buf_end);
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<float4> vel_ptr(d_vel);
    thrust::device_ptr<float3> accel_ptr(d_accel);
    thrust::device_ptr<int3> image_ptr(d_image);
    thrust::device_ptr<float> charge_ptr(d_charge);
    thrust::device_ptr<float> diameter_ptr(d_diameter);
    thrust::device_ptr<unsigned int> body_ptr(d_body);
    thrust::device_ptr<float4> orientation_ptr(d_orientation);
    thrust::device_ptr<unsigned int> tag_ptr(d_tag);

    thrust::copy(thrust::make_transform_iterator(recv_buf_ptr, unpack_pdata()),
                    thrust::make_transform_iterator(recv_buf_end_ptr, unpack_pdata()),
                    make_zip_iterator( thrust::make_tuple( pos_ptr,
                                               vel_ptr,
                                               accel_ptr,
                                               charge_ptr,
                                               diameter_ptr,
                                               image_ptr,
                                               body_ptr,
                                               orientation_ptr,
                                               tag_ptr) )) -
                    make_zip_iterator( thrust::make_tuple( pos_ptr,
                                               vel_ptr,
                                               accel_ptr,
                                               charge_ptr,
                                               diameter_ptr,
                                               image_ptr,
                                               body_ptr,
                                               orientation_ptr,
                                               tag_ptr) );
    }

//! Wrap received ghost particles across global box
/*! \param dir Direction along which particles were received
 * \param n Number of particles to apply periodic boundary conditions to
 * \param d_pos Array of particle positions to apply periodic boundary conditions to
 * \param global_box Dimensions of global simulation box
 * \param rghost Boundary layer width
 */
void gpu_wrap_ghost_particles(unsigned int dir,
                              unsigned int n,
                              float4 *d_pos,
                              gpu_boxsize global_box,
                              float rghost)
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::transform(pos_ptr, pos_ptr +n, pos_ptr, wrap_ghost_particle(global_box, rghost, dir));
    }

//! Construct a list of particle tags to send as ghost particles
/*! \param N number of particles to check
 * \param dir Direction in which ghost particles are sent
 * \param d_pos Array of particle positions
 * \param d_global_tag Array of particle global tags
 * \param d_copy_ghosts Array to be fillled x with global tags of particles that are to be send as ghosts
 * \param n_copy_ghosts Number of local particles that are sent in the given direction as ghosts (return value)
 * \param box Dimensions of local simulation box
 * \param r_ghost Width of boundary layer
 */
void gpu_make_exchange_ghost_list(unsigned int N,
                                  unsigned int dir,
                                  float4 *d_pos,
                                  unsigned int *d_global_tag,
                                  unsigned int* d_copy_ghosts,
                                  unsigned int &n_copy_ghosts,
                                  gpu_boxsize box,
                                  float r_ghost)
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<unsigned int> global_tag_ptr(d_global_tag);
    thrust::device_ptr<unsigned int> copy_ghosts_ptr(d_copy_ghosts);

    thrust::device_ptr<unsigned int> copy_ghosts_end_ptr;

    copy_ghosts_end_ptr = thrust::copy_if(global_tag_ptr,
                                          global_tag_ptr+N,
                                          pos_ptr,
                                          copy_ghosts_ptr,
                                          select_particle_ghost(box, r_ghost, dir));

    n_copy_ghosts =  copy_ghosts_end_ptr - copy_ghosts_ptr;
    }

//! Fill send buffers of particles we are sending as ghost particles with partial particle data
/*! \param nghost Number of ghost particles to copy into send buffers
 * \param d_copy_ghosts Array of particle tags to copy as ghost particles
 * \param d_rtag Inverse look-up array for global tags <-> local indices
 * \param d_pos Array of particle positions
 * \param d_pos_copybuf Send buffer for particle positions
 * \param d_charge Array of particle charges
 * \param d_charge_copybuf Send buffer for particle charges
 * \param d_diameter Array of particle diameters
 * \param d_diameter_copybuf Send buffer for particle diameters
 */
void gpu_exchange_ghosts(unsigned int nghost,
                         unsigned int *d_copy_ghosts,
                         unsigned int *d_rtag,
                         float4 *d_pos,
                         float4 *d_pos_copybuf,
                         float *d_charge,
                         float *d_charge_copybuf,
                         float *d_diameter,
                         float *d_diameter_copybuf)
    {
    thrust::device_ptr<unsigned int> copy_ghosts_ptr(d_copy_ghosts);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<float4> pos_copybuf_ptr(d_pos_copybuf);
    thrust::device_ptr<float> charge_ptr(d_charge);
    thrust::device_ptr<float> charge_copybuf_ptr(d_charge_copybuf);
    thrust::device_ptr<float> diameter_ptr(d_diameter);
    thrust::device_ptr<float> diameter_copybuf_ptr(d_diameter_copybuf);

    permutation_iterator<device_ptr<unsigned int>, device_ptr<unsigned int> > ghost_rtag(rtag_ptr, copy_ghosts_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, pos_ptr, pos_copybuf_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, charge_ptr, charge_copybuf_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, diameter_ptr, diameter_copybuf_ptr);
    }

//! Update global tag <-> local particle index reverse lookup array
/*! \param nptl Number of particles for which we are updating the reverse lookup tags
 * \param start_idx starting index of first particle in local particle data arrays
 * \param d_tag array of particle tags
 * \param d_rtag array of particle reverse lookup tags to store information to
 */
void gpu_update_rtag(unsigned int nptl, unsigned int start_idx, unsigned int *d_tag, unsigned int *d_rtag)
    {
    thrust::device_ptr<unsigned int> tag_ptr(d_tag);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);

    thrust::counting_iterator<unsigned int> first(start_idx);
    thrust::counting_iterator<unsigned int> last = first + nptl;
    thrust::scatter(first, last, tag_ptr, rtag_ptr);
    }

//! Copy ghost particle positions into send buffer
/*! \param nghost Number of ghost particles to copy
 * \param d_pos Array of particle positions
 * \param d_copy_ghosts Global particle tags of particles to copy
 * \param d_pos_copybuf Send buffer of ghost particle positions
 * \param d_rtag Global tag <-> local particle index reverse lookup array
 */
void gpu_copy_ghosts(unsigned int nghost,
                     float4 *d_pos,
                     unsigned int *d_copy_ghosts,
                     float4 *d_pos_copybuf,
                     unsigned int *d_rtag)
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);
    thrust::device_ptr<unsigned int> copy_ghosts_ptr(d_copy_ghosts);
    thrust::device_ptr<float4> copybuf_ptr(d_pos_copybuf);

    permutation_iterator<device_ptr<unsigned int>, device_ptr<unsigned int> > ghost_rtag(rtag_ptr, copy_ghosts_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, pos_ptr, copybuf_ptr);

    }
#endif
