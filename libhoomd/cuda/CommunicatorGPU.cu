
#include <hip/hip_runtime.h>
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: jglaser

/*! \file CommunicatorGPU.cu
    \brief Implementation of communication algorithms on the GPU
*/

#ifdef ENABLE_MPI
#include "CommunicatorGPU.cuh"
#include "ParticleData.cuh"

#include <thrust/replace.h>
#include <thrust/device_ptr.h>
#include <thrust/scatter.h>
#include <thrust/count.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>

// moderngpu
#include "moderngpu/util/mgpucontext.h"
#include "moderngpu/device/loadstore.cuh"
#include "moderngpu/device/launchbox.cuh"
#include "moderngpu/device/ctaloadbalance.cuh"
#include "moderngpu/kernels/localitysort.cuh"
#include "moderngpu/kernels/search.cuh"
#include "moderngpu/kernels/scan.cuh"
#include "moderngpu/kernels/sortedsearch.cuh"

using namespace thrust;

//! Select a particle for migration
struct select_particle_migrate_gpu : public thrust::unary_function<const Scalar4, bool>
    {
    const BoxDim box;          //!< Local simulation box dimensions
    unsigned int comm_mask;    //!< Allowed communication directions

    //! Constructor
    /*!
     */
    select_particle_migrate_gpu(const BoxDim & _box, unsigned int _comm_mask)
        : box(_box), comm_mask(_comm_mask)
        { }

    //! Select a particle
    /*! t particle data to consider for sending
     * \return true if particle stays in the box
     */
    __host__ __device__ bool operator()(const Scalar4 postype)
        {
        Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
        Scalar3 f = box.makeFraction(pos);

        unsigned int flags = 0;
        if (f.x >= Scalar(1.0)) flags |= send_east;
        if (f.x < Scalar(0.0)) flags |= send_west;
        if (f.y >= Scalar(1.0)) flags |= send_north;
        if (f.y < Scalar(0.0)) flags |= send_south;
        if (f.z >= Scalar(1.0)) flags |= send_up;
        if (f.z < Scalar(0.0)) flags |= send_down;

        // filter allowed directions
        flags &= comm_mask;

        return flags > 0;
        }

     };

//! Select a particle for migration
struct get_migrate_key_gpu : public thrust::unary_function<const pdata_element, unsigned int>
    {
    const BoxDim box;       //!< Local simulation box dimensions
    const uint3 my_pos;     //!< My domain decomposition position
    const Index3D di;             //!< Domain indexer
    const unsigned int mask; //!< Mask of allowed directions

    //! Constructor
    /*!
     */
    get_migrate_key_gpu(const BoxDim & _box, const uint3 _my_pos, const Index3D _di, const unsigned int _mask)
        : box(_box), my_pos(_my_pos), di(_di), mask(_mask)
        { }

    //! Generate key for a sent particle
    __device__ unsigned int operator()(const pdata_element p)
        {
        Scalar4 postype = p.pos;
        Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
        Scalar3 f = box.makeFraction(pos);

        int ix, iy, iz;
        ix = iy = iz = 0;

        // we allow for a tolerance, large enough so we don't loose particles
        // due to numerical precision
        const Scalar tol(1e-5);
        if (f.x >= Scalar(1.0)-tol && (mask & send_east))
            ix = 1;
        else if (f.x < tol && (mask & send_west))
            ix = -1;

        if (f.y >= Scalar(1.0)-tol && (mask & send_north))
            iy = 1;
        else if (f.y < tol && (mask & send_south))
            iy = -1;

        if (f.z >= Scalar(1.0)-tol && (mask & send_up))
            iz = 1;
        else if (f.z < tol && (mask & send_down))
            iz = -1;

        int i = my_pos.x;
        int j = my_pos.y;
        int k = my_pos.z;

        i += ix;
        if (i == (int)di.getW())
            i = 0;
        else if (i < 0)
            i += di.getW();

        j += iy;
        if (j == (int) di.getH())
            j = 0;
        else if (j < 0)
            j += di.getH();

        k += iz;
        if (k == (int) di.getD())
            k = 0;
        else if (k < 0)
            k += di.getD();

        return di(i,j,k);
        }

     };


/*! \param N Number of local particles
    \param d_pos Device array of particle positions
    \param d_tag Device array of particle tags
    \param d_rtag Device array for reverse-lookup table
    \param box Local box
    \param comm_mask Mask of allowed communication directions
    \param alloc Caching allocator
 */
void gpu_stage_particles(const unsigned int N,
                         const Scalar4 *d_pos,
                         const unsigned int *d_tag,
                         unsigned int *d_rtag,
                         const BoxDim& box,
                         const unsigned int comm_mask,
                         cached_allocator& alloc)
    {
    // Wrap particle data arrays
    thrust::device_ptr<const Scalar4> pos_ptr(d_pos);
    thrust::device_ptr<const unsigned int> tag_ptr(d_tag);

    // Wrap rtag array
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);

    // pointer from tag into rtag
    thrust::permutation_iterator<
        thrust::device_ptr<unsigned int>, thrust::device_ptr<const unsigned int> > rtag_prm(rtag_ptr, tag_ptr);

    // set flag for particles that are to be sent
    thrust::replace_if(thrust::cuda::par(alloc),
        rtag_prm, rtag_prm + N, pos_ptr,
        select_particle_migrate_gpu(box,comm_mask),
        NOT_LOCAL);
    }

/*! \param nsend Number of particles in buffer
    \param d_in Send buf (in-place sort)
    \param di Domain indexer
    \param box Local box
    \param d_keys Output array (target domains)
    \param d_begin Output array (start indices per key in send buf)
    \param d_end Output array (end indices per key in send buf)
    \param d_neighbors List of neighbor ranks
    \param mask Mask of communicating directions
    \param alloc Caching allocator
 */
void gpu_sort_migrating_particles(const unsigned int nsend,
                   pdata_element *d_in,
                   const Index3D& di,
                   const uint3 my_pos,
                   const BoxDim& box,
                   unsigned int *d_keys,
                   unsigned int *d_begin,
                   unsigned int *d_end,
                   const unsigned int *d_neighbors,
                   const unsigned int nneigh,
                   const unsigned int mask,
                   mgpu::ContextPtr mgpu_context,
                   cached_allocator& alloc)
    {
    // Wrap input & output
    thrust::device_ptr<pdata_element> in_ptr(d_in);
    thrust::device_ptr<unsigned int> keys_ptr(d_keys);
    thrust::device_ptr<const unsigned int> neighbors_ptr(d_neighbors);

    // generate keys
    thrust::transform(in_ptr, in_ptr + nsend, keys_ptr, get_migrate_key_gpu(box, my_pos, di,mask));


    // allocate temp arrays
    unsigned int *d_tmp = (unsigned int *)alloc.allocate(nsend*sizeof(unsigned int));
    thrust::device_ptr<unsigned int> tmp_ptr(d_tmp);

    pdata_element *d_in_copy = (pdata_element *)alloc.allocate(nsend*sizeof(pdata_element));
    thrust::device_ptr<pdata_element> in_copy_ptr(d_in_copy);

    // copy and fill with ascending integer sequence
    thrust::counting_iterator<unsigned int> count_it(0);
    thrust::copy(make_zip_iterator(thrust::make_tuple(count_it, in_ptr)),
        thrust::make_zip_iterator(thrust::make_tuple(count_it + nsend, in_ptr + nsend)),
        thrust::make_zip_iterator(thrust::make_tuple(tmp_ptr, in_copy_ptr)));

    // sort buffer by neighbors
    if (nsend) mgpu::LocalitySortPairs(thrust::raw_pointer_cast(keys_ptr), d_tmp, nsend, *mgpu_context);

    // reorder send buf
    thrust::gather(tmp_ptr, tmp_ptr + nsend, in_copy_ptr, in_ptr);

    mgpu::SortedSearch<mgpu::MgpuBoundsLower>(d_neighbors, nneigh,
        thrust::raw_pointer_cast(keys_ptr), nsend, d_begin, *mgpu_context);
    mgpu::SortedSearch<mgpu::MgpuBoundsUpper>(d_neighbors, nneigh,
        thrust::raw_pointer_cast(keys_ptr), nsend, d_end, *mgpu_context);

    // release temporary buffers
    alloc.deallocate((char *)d_in_copy,0);
    alloc.deallocate((char *)d_tmp,0);
    }

//! Wrap a particle in a pdata_element
struct wrap_particle_op_gpu : public thrust::unary_function<const pdata_element, pdata_element>
    {
    const BoxDim box; //!< The box for which we are applying boundary conditions

    //! Constructor
    /*!
     */
    wrap_particle_op_gpu(const BoxDim _box)
        : box(_box)
        {
        }

    //! Wrap position information inside particle data element
    /*! \param p Particle data element
     * \returns The particle data element with wrapped coordinates
     */
    __device__ pdata_element operator()(const pdata_element p)
        {
        pdata_element ret = p;
        box.wrap(ret.pos, ret.image);
        return ret;
        }
     };


/*! \param n_recv Number of particles in buffer
    \param d_in Buffer of particle data elements
    \param box Box for which to apply boundary conditions
 */
void gpu_wrap_particles(const unsigned int n_recv,
                        pdata_element *d_in,
                        const BoxDim& box)
    {
    // Wrap device ptr
    thrust::device_ptr<pdata_element> in_ptr(d_in);

    // Apply box wrap to input buffer
    thrust::transform(in_ptr, in_ptr + n_recv, in_ptr, wrap_particle_op_gpu(box));
    }

//! Reset reverse lookup tags of particles we are removing
/* \param n_delete_ptls Number of particles to delete
 * \param d_delete_tags Array of particle tags to delete
 * \param d_rtag Array for tag->idx lookup
 */
void gpu_reset_rtags(unsigned int n_delete_ptls,
                     unsigned int *d_delete_tags,
                     unsigned int *d_rtag)
    {
    thrust::device_ptr<unsigned int> delete_tags_ptr(d_delete_tags);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);

    thrust::constant_iterator<unsigned int> not_local(NOT_LOCAL);
    thrust::scatter(not_local,
                    not_local + n_delete_ptls,
                    delete_tags_ptr,
                    rtag_ptr);
    }


//! Kernel to select ghost atoms due to non-bonded interactions
struct make_ghost_exchange_plan_gpu : thrust::unary_function<const Scalar4, unsigned int>
    {
    const BoxDim box;       //!< Local box
    Scalar3 ghost_fraction; //!< Fractional width of ghost layer
    unsigned int mask;      //!< Mask of allowed communication directions

    //! Constructor
    make_ghost_exchange_plan_gpu(const BoxDim& _box, Scalar3 _ghost_fraction, unsigned int _mask)
        : box(_box), ghost_fraction(_ghost_fraction), mask(_mask)
        { }

    __device__ unsigned int operator() (const Scalar4 postype)
        {
        Scalar3 pos = make_scalar3(postype.x,postype.y,postype.z);
        Scalar3 f = box.makeFraction(pos);

        unsigned int plan = 0;

        // is particle inside ghost layer? set plan accordingly.
        if (f.x >= Scalar(1.0) - ghost_fraction.x)
            plan |= send_east;
        if (f.x < ghost_fraction.x)
            plan |= send_west;
        if (f.y >= Scalar(1.0) - ghost_fraction.y)
            plan |= send_north;
        if (f.y < ghost_fraction.y)
            plan |= send_south;
        if (f.z >= Scalar(1.0) - ghost_fraction.z)
            plan |= send_up;
        if (f.z < ghost_fraction.z)
            plan |= send_down;

        // filter out non-communiating directions
        plan &= mask;

        return plan;
        }
    };

//! Construct plans for sending non-bonded ghost particles
/*! \param d_plan Array of ghost particle plans
 * \param N number of particles to check
 * \param d_pos Array of particle positions
 * \param box Dimensions of local simulation box
 * \param r_ghost Width of boundary layer
 */
void gpu_make_ghost_exchange_plan(unsigned int *d_plan,
                                  unsigned int N,
                                  const Scalar4 *d_pos,
                                  const BoxDim &box,
                                  Scalar3 ghost_fraction,
                                  unsigned int mask,
                                  cached_allocator& alloc)
    {
    // wrap position array
    thrust::device_ptr<const Scalar4> pos_ptr(d_pos);

    // wrap plan (output) array
    thrust::device_ptr<unsigned int> plan_ptr(d_plan);

    // compute plans
    thrust::transform(thrust::cuda::par(alloc),
        pos_ptr, pos_ptr + N, plan_ptr,
        make_ghost_exchange_plan_gpu(box, ghost_fraction,mask));
    }

//! Apply adjacency masks to plan and return number of matching neighbors
struct num_neighbors_gpu
    {
    thrust::device_ptr<const unsigned int> adj_ptr;
    const unsigned int nneigh;

    num_neighbors_gpu(thrust::device_ptr<const unsigned int> _adj_ptr, unsigned int _nneigh)
        : adj_ptr(_adj_ptr), nneigh(_nneigh)
        { }

    __device__ unsigned int operator() (unsigned int plan)
        {
        unsigned int count = 0;
        for (unsigned int i = 0; i < nneigh; i++)
            {
            unsigned int adj = adj_ptr[i];
            if ((adj & plan) == adj) count++;
            }
        return count;
        }
    };

//! Apply adjacency masks to plan and integer and return nth matching neighbor rank
struct get_neighbor_rank_n : thrust::unary_function<
    thrust::tuple<unsigned int, unsigned int>, unsigned int >
    {
    thrust::device_ptr<const unsigned int> adj_ptr;
    thrust::device_ptr<const unsigned int> neighbor_ptr;
    const unsigned int nneigh;

    __host__ __device__ get_neighbor_rank_n(thrust::device_ptr<const unsigned int> _adj_ptr,
        thrust::device_ptr<const unsigned int> _neighbor_ptr,
        unsigned int _nneigh)
        : adj_ptr(_adj_ptr), neighbor_ptr(_neighbor_ptr), nneigh(_nneigh)
        { }

    __host__ __device__ get_neighbor_rank_n(const unsigned int *_d_adj,
        const unsigned int *_d_neighbor,
        unsigned int _nneigh)
        : adj_ptr(thrust::device_ptr<const unsigned int>(_d_adj)),
          neighbor_ptr(thrust::device_ptr<const unsigned int>(_d_neighbor)),
          nneigh(_nneigh)
        { }


    __device__ unsigned int operator() (thrust::tuple<unsigned int, unsigned int> t)
        {
        unsigned int plan = thrust::get<0>(t);
        unsigned int n = thrust::get<1>(t);
        unsigned int count = 0;
        unsigned int ineigh;
        for (ineigh = 0; ineigh < nneigh; ineigh++)
            {
            unsigned int adj = adj_ptr[ineigh];
            if ((adj & plan) == adj)
                {
                if (count == n) break;
                count++;
                }
            }
        return neighbor_ptr[ineigh];
        }

    __device__ unsigned int operator() (unsigned int plan, unsigned int n)
        {
        unsigned int count = 0;
        unsigned int ineigh;
        for (ineigh = 0; ineigh < nneigh; ineigh++)
            {
            unsigned int adj = adj_ptr[ineigh];
            if ((adj & plan) == adj)
                {
                if (count == n) break;
                count++;
                }
            }
        return neighbor_ptr[ineigh];
        }
    };

unsigned int gpu_exchange_ghosts_count_neighbors(
    unsigned int N,
    const unsigned int *d_ghost_plan,
    const unsigned int *d_adj,
    unsigned int *d_counts,
    unsigned int nneigh,
    mgpu::ContextPtr mgpu_context)
    {
    thrust::device_ptr<const unsigned int> ghost_plan_ptr(d_ghost_plan);
    thrust::device_ptr<const unsigned int> adj_ptr(d_adj);
    thrust::device_ptr<unsigned int> counts_ptr(d_counts);

    // compute neighbor counts
    thrust::transform(ghost_plan_ptr, ghost_plan_ptr + N, counts_ptr, num_neighbors_gpu(adj_ptr, nneigh));

    // determine output size
    unsigned int total = 0;
    if (N) mgpu::ScanExc(d_counts, N, &total, *mgpu_context);
    return total;
    }

template<typename Tuning>
__global__ void gpu_expand_neighbors_kernel(const unsigned int n_out,
    const int *d_offs,
    const unsigned int *d_tag,
    const unsigned int *d_plan,
    const unsigned int n_offs,
    const int* mp_global,
    unsigned int *d_idx_out,
    const unsigned int *d_neighbors,
    const unsigned int *d_adj,
    const unsigned int nneigh,
    unsigned int *d_neighbors_out)
    {
    typedef MGPU_LAUNCH_PARAMS Params;
    const int NT = Params::NT;
    const int VT = Params::VT;

    union Shared
        {
        int indices[NT * (VT + 1)];
        unsigned int values[NT * VT];
        };
    __shared__ Shared shared;
    int tid = threadIdx.x;
    int block = blockIdx.x;

    // Compute the input and output intervals this CTA processes.
    int4 range = mgpu::CTALoadBalance<NT, VT>(n_out, d_offs, n_offs,
        block, tid, mp_global, shared.indices, true);

    // The interval indices are in the left part of shared memory (n_out).
    // The scan of interval counts are in the right part (n_offs)
    int destCount = range.y - range.x;

    // Copy the source indices into register.
    int sources[VT];
    mgpu::DeviceSharedToReg<NT, VT>(shared.indices, tid, sources);

    __syncthreads();

    // Now use the segmented scan to fetch nth neighbor
    get_neighbor_rank_n getn(d_adj, d_neighbors, nneigh);

    // register to hold neighbors
    unsigned int neighbors[VT];

    int *intervals = shared.indices + destCount;

    #pragma unroll
    for(int i = 0; i < VT; ++i)
        {
        int index = NT * i + tid;
        int gid = range.x + index;

        if(index < destCount)
            {
            int interval = sources[i];
            int rank = gid - intervals[interval - range.z];
            int plan = d_plan[interval];
            neighbors[i] = getn(plan,rank);
            }
        }

    // write out neighbors to global mem
    mgpu::DeviceRegToGlobal<NT, VT>(destCount, neighbors, tid, d_neighbors_out + range.x);

    // store indices to global mem
    mgpu::DeviceRegToGlobal<NT, VT>(destCount, sources, tid, d_idx_out + range.x);
    }

void gpu_expand_neighbors(unsigned int n_out,
    const unsigned int *d_offs,
    const unsigned int *d_tag,
    const unsigned int *d_plan,
    unsigned int n_offs,
    unsigned int *d_idx_out,
    const unsigned int *d_neighbors,
    const unsigned int *d_adj,
    const unsigned int nneigh,
    unsigned int *d_neighbors_out,
    mgpu::CudaContext& context)
    {
    const int NT = 128;
    const int VT = 7;
    typedef mgpu::LaunchBoxVT<NT, VT> Tuning;
    int2 launch = Tuning::GetLaunchParams(context);

    int NV = launch.x * launch.y;
    int numBlocks = MGPU_DIV_UP(n_out + n_offs, NV);

    // Partition the input and output sequences so that the load-balancing
    // search results in a CTA fit in shared memory.
    MGPU_MEM(int) partitionsDevice = mgpu::MergePathPartitions<mgpu::MgpuBoundsUpper>(
        mgpu::counting_iterator<int>(0), n_out, (int *) d_offs,
        n_offs, NV, 0, mgpu::less<int>(), context);

    gpu_expand_neighbors_kernel<Tuning><<<numBlocks, launch.x, 0, context.Stream()>>>(
        n_out, (int *) d_offs, d_tag, d_plan, n_offs,
        partitionsDevice->get(), d_idx_out,
        d_neighbors, d_adj, nneigh, d_neighbors_out);
    }

void gpu_exchange_ghosts_make_indices(
    unsigned int N,
    const unsigned int *d_ghost_plan,
    const unsigned int *d_tag,
    const unsigned int *d_adj,
    const unsigned int *d_neighbors,
    const unsigned int *d_unique_neighbors,
    const unsigned int *d_counts,
    unsigned int *d_ghost_idx,
    unsigned int *d_ghost_begin,
    unsigned int *d_ghost_end,
    unsigned int nneigh,
    unsigned int n_unique_neigh,
    unsigned int n_out,
    unsigned int mask,
    mgpu::ContextPtr mgpu_context,
    cached_allocator& alloc)
    {
    // temporary array for output neighbor ranks
    unsigned int *d_out_neighbors = (unsigned int *)alloc.allocate(n_out*sizeof(unsigned int));

    /*
     * expand each tag by the number of neighbors to send the corresponding ptl to
     * and assign each copy to a different neighbor
     */

    // allocate temporary array
    gpu_expand_neighbors(n_out,
        d_counts,
        d_tag, d_ghost_plan, N, d_ghost_idx,
        d_neighbors, d_adj, nneigh,
        d_out_neighbors,
        *mgpu_context);

    // sort tags by neighbors
    if (n_out) mgpu::LocalitySortPairs(d_out_neighbors, d_ghost_idx, n_out, *mgpu_context);

    mgpu::SortedSearch<mgpu::MgpuBoundsLower>(d_unique_neighbors, n_unique_neigh,
        d_out_neighbors, n_out, d_ghost_begin, *mgpu_context);
    mgpu::SortedSearch<mgpu::MgpuBoundsUpper>(d_unique_neighbors, n_unique_neigh,
        d_out_neighbors, n_out, d_ghost_end, *mgpu_context);

    // deallocate temporary arrays
    alloc.deallocate((char *)d_out_neighbors,0);
    }

template<typename T>
__global__ void gpu_pack_kernel(
    unsigned int n_out,
    const unsigned int *d_ghost_idx,
    const T *in,
    T *out)
    {
    unsigned int buf_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (buf_idx >= n_out) return;
    unsigned int idx = d_ghost_idx[buf_idx];
    out[buf_idx] = in[idx];
    }

void gpu_exchange_ghosts_pack(
    unsigned int n_out,
    const unsigned int *d_ghost_idx,
    const unsigned int *d_tag,
    const Scalar4 *d_pos,
    const Scalar4 *d_vel,
    const Scalar *d_charge,
    const Scalar *d_diameter,
    const Scalar4 *d_orientation,
    unsigned int *d_tag_sendbuf,
    Scalar4 *d_pos_sendbuf,
    Scalar4 *d_vel_sendbuf,
    Scalar *d_charge_sendbuf,
    Scalar *d_diameter_sendbuf,
    Scalar4 *d_orientation_sendbuf,
    bool send_tag,
    bool send_pos,
    bool send_vel,
    bool send_charge,
    bool send_diameter,
    bool send_orientation)
    {
    unsigned int block_size = 256;
    unsigned int n_blocks = n_out/block_size + 1;
    if (send_tag) gpu_pack_kernel<<<n_blocks, block_size>>>(n_out, d_ghost_idx, d_tag, d_tag_sendbuf);
    if (send_pos) gpu_pack_kernel<<<n_blocks, block_size>>>(n_out, d_ghost_idx, d_pos, d_pos_sendbuf);
    if (send_vel) gpu_pack_kernel<<<n_blocks, block_size>>>(n_out, d_ghost_idx, d_vel, d_vel_sendbuf);
    if (send_charge) gpu_pack_kernel<<<n_blocks, block_size>>>(n_out, d_ghost_idx, d_charge, d_charge_sendbuf);
    if (send_diameter) gpu_pack_kernel<<<n_blocks, block_size>>>(n_out, d_ghost_idx, d_diameter, d_diameter_sendbuf);
    if (send_orientation) gpu_pack_kernel<<<n_blocks, block_size>>>(n_out, d_ghost_idx, d_orientation, d_orientation_sendbuf);
    }

void gpu_communicator_initialize_cache_config()
    {
    cudaFuncSetCacheConfig(gpu_pack_kernel<Scalar>, cudaFuncCachePreferL1);
    cudaFuncSetCacheConfig(gpu_pack_kernel<Scalar4>, cudaFuncCachePreferL1);
    cudaFuncSetCacheConfig(gpu_pack_kernel<unsigned int>, cudaFuncCachePreferL1);
    }

//! Wrap particles
struct wrap_ghost_pos_gpu : public thrust::unary_function<Scalar4, Scalar4>
    {
    const BoxDim box; //!< The box for which we are applying boundary conditions

    //! Constructor
    /*!
     */
    wrap_ghost_pos_gpu(const BoxDim _box)
        : box(_box)
        {
        }

    //! Wrap position Scalar4
    /*! \param p The position
     * \returns The wrapped position
     */
    __device__ Scalar4 operator()(Scalar4 p)
        {
        int3 image;
        box.wrap(p,image);
        return p;
        }
     };


/*! \param n_recv Number of particles in buffer
    \param d_pos The particle positions array
    \param box Box for which to apply boundary conditions
 */
void gpu_wrap_ghosts(const unsigned int n_recv,
                        Scalar4 *d_pos,
                        const BoxDim& box)
    {
    // Wrap device ptr
    thrust::device_ptr<Scalar4> pos_ptr(d_pos);

    // Apply box wrap to input buffer
    thrust::transform(pos_ptr, pos_ptr + n_recv, pos_ptr, wrap_ghost_pos_gpu(box));
    }

void gpu_exchange_ghosts_copy_buf(
    unsigned int n_recv,
    const unsigned int *d_tag_recvbuf,
    const Scalar4 *d_pos_recvbuf,
    const Scalar4 *d_vel_recvbuf,
    const Scalar *d_charge_recvbuf,
    const Scalar *d_diameter_recvbuf,
    const Scalar4 *d_orientation_recvbuf,
    unsigned int *d_tag,
    Scalar4 *d_pos,
    Scalar4 *d_vel,
    Scalar *d_charge,
    Scalar *d_diameter,
    Scalar4 *d_orientation,
    bool send_tag,
    bool send_pos,
    bool send_vel,
    bool send_charge,
    bool send_diameter,
    bool send_orientation)
    {
    if (send_tag) cudaMemcpyAsync(d_tag, d_tag_recvbuf, n_recv*sizeof(unsigned int), cudaMemcpyDeviceToDevice,0);
    if (send_pos) cudaMemcpyAsync(d_pos, d_pos_recvbuf, n_recv*sizeof(Scalar4), cudaMemcpyDeviceToDevice,0);
    if (send_vel) cudaMemcpyAsync(d_vel, d_vel_recvbuf, n_recv*sizeof(Scalar4), cudaMemcpyDeviceToDevice,0);
    if (send_charge) cudaMemcpyAsync(d_charge, d_charge_recvbuf, n_recv*sizeof(Scalar), cudaMemcpyDeviceToDevice,0);
    if (send_diameter) cudaMemcpyAsync(d_diameter, d_diameter_recvbuf, n_recv*sizeof(Scalar), cudaMemcpyDeviceToDevice,0);
    if (send_orientation) cudaMemcpyAsync(d_orientation, d_orientation_recvbuf, n_recv*sizeof(Scalar4), cudaMemcpyDeviceToDevice,0);
    }

void gpu_compute_ghost_rtags(
     unsigned int first_idx,
     unsigned int n_ghost,
     const unsigned int *d_tag,
     unsigned int *d_rtag)
    {
    thrust::device_ptr<const unsigned int> tag_ptr(d_tag);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);

    thrust::counting_iterator<unsigned int> idx(first_idx);
    thrust::scatter(idx, idx + n_ghost, tag_ptr, rtag_ptr);
    }


#endif
