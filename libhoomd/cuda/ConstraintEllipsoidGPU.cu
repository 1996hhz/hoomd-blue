#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2015 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "ConstraintEllipsoidGPU.cuh"
#include "EvaluatorConstraint.h"
#include "EvaluatorConstraintEllipsoid.h"

#include <assert.h>

/*! \file ConstraintEllipsoidGPU.cu
    \brief Defines GPU kernel code for calculating ellipsoid constraint forces. Used by ConstraintEllipsoidGPU.
*/

//! Kernel for caculating ellipsoid constraint forces on the GPU
/*! \param d_group_members List of members in the group
    \param group_size number of members in the group
    \param N number of particles in system
    \param d_pos particle positions on device
    \param P Position of the ellipsoid
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
    \param deltaT step size from the Integrator
*/
extern "C" __global__
void gpu_compute_constraint_ellipsoid_constraint_kernel(const unsigned int *d_group_members,
                                                 unsigned int group_size,
                                                 const unsigned int N,
                                                 const Scalar4 *d_pos,
                                                 Scalar3 P,
                                                 Scalar rx,
                                                 Scalar ry,
                                                 Scalar rz,
                                                 Scalar deltaT)
    {
    // start by identifying which particle we are to handle
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx >= group_size)
        return;

    unsigned int idx = d_group_members[group_idx];

    // read in position, velocity, net force, and mass
    Scalar4 pos = d_pos[idx];
    Scalar4 vel = d_vel[idx];
    Scalar4 net_force = d_net_force[idx];
    Scalar m = vel.w;

    // convert to Scalar3's for passing to the evaluators
    Scalar3 X = make_scalar3(pos.x, pos.y, pos.z);
    Scalar3 V = make_scalar3(vel.x, vel.y, vel.z);
    Scalar3 F = make_scalar3(net_force.x, net_force.y, net_force.z);

    // evaluate the constraint position
    EvaluatorConstraintEllipsoid Ellipsoid(m_P, m_rx, m_ry, m_rz);
    Scalar3 X = make_scalar3(h_pos.data[j].x, h_pos.data[j].y, h_pos.data[j].z);
    Scalar3 C = Ellipsoid.evalClosest(X);
    
    EvaluatorConstraint constraint(X, V, F, m, deltaT);
    EvaluatorConstraintSphere sphere(P, r);
    Scalar3 C = sphere.evalClosest(constraint.evalU());

    // evaluate the constraint force
    Scalar3 FC;
    Scalar virial[6];
    constraint.evalConstraintForce(FC, virial, C);

    // apply the constraint
    h_pos.data[j].x = C.x;
    h_pos.data[j].y = C.y;
    h_pos.data[j].z = C.z;


/*! \param d_group_members List of members in the group
    \param group_size number of members in the group
    \param N nunmber of particles
    \param d_pos particle positions on the device
    \param P Position of the ellipsoid
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
    \param deltaT step size from the Integrator
    \param block_size Block size to execute on the GPU

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()
*/
hipError_t gpu_compute_constraint_ellipsoid_constraint(const unsigned int *d_group_members,
                                                 unsigned int group_size,
                                                 const unsigned int N,
                                                 const Scalar4 *d_pos,
                                                 const Scalar3& P,
                                                 Scalar rx,
                                                 Scalar ry,
                                                 Scalar rz,
                                                 Scalar deltaT,
                                                 unsigned int block_size)
    {
    assert(d_group_members);

    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)group_size / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipMemset(d_force, 0, sizeof(Scalar4)*N);
    hipMemset(d_virial, 0, 6*sizeof(Scalar)*virial_pitch);
    gpu_compute_constraint_ellipsoid_constraint_kernel<<< grid, threads>>>(d_group_members,
                                                                    group_size,
                                                                    N,
                                                                    d_pos,
                                                                    P,
                                                                    rx,
                                                                    ry,
                                                                    rz,
                                                                    deltaT);

    return hipSuccess;
    }
