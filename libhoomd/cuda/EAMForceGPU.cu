#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
// Maintainer: morozov

/**
powered by:
Moscow group.
*/

#include "EAMForceGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file EAMForceGPU.cu
    \brief Defines GPU kernel code for calculating the eam forces. Used by EAMForceComputeGPU.
*/

//!< Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;
//! Texture for reading electron density
texture<float, 1, hipReadModeElementType> electronDensity_tex;
//! Texture for reading EAM pair potential
texture<float2, 1, hipReadModeElementType> pairPotential_tex;
//! Texture for reading the embedding function
texture<float, 1, hipReadModeElementType> embeddingFunction_tex;
//! Texture for reading the derivative of the electron density
texture<float, 1, hipReadModeElementType> derivativeElectronDensity_tex;
//! Texture for reading the derivative of the embedding function
texture<float, 1, hipReadModeElementType> derivativeEmbeddingFunction_tex;
//! Texture for reading the derivative of the atom embedding function
texture<float, 1, hipReadModeElementType> atomDerivativeEmbeddingFunction_tex;

//! Storage space for EAM parameters on the GPU
__constant__ EAMTexInterData eam_data_ti;

//! Kernel for computing EAM forces on the GPU
extern "C" __global__ void gpu_compute_eam_tex_inter_forces_kernel(
    float4* d_force,
    float* d_virial,
    gpu_pdata_arrays pdata,
    gpu_boxsize box,
    const unsigned int *d_n_neigh,
    const unsigned int *d_nlist,
    const Index2D nli,
    float* atomDerivativeEmbeddingFunction)
    {
    // start by identifying which particle we are to handle
    volatile int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= pdata.N)
        return;

    // load in the length of the list (MEM_TRANSFER: 4 bytes)
    int n_neigh = d_n_neigh[idx];

    // read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
    // (MEM TRANSFER: 16 bytes)
    float4 pos = tex1Dfetch(pdata_pos_tex, idx);

    // initialize the force to 0
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    // prefetch neighbor index
    int cur_neigh = 0;
    int next_neigh = d_nlist[nli(idx, 0)];
    int typei  = __float_as_int(pos.w);
    // loop over neighbors

    float atomElectronDensity  = 0.0f;
    int nr = eam_data_ti.nr;
    int ntypes = eam_data_ti.ntypes;
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
        {
        // read the current neighbor index (MEM TRANSFER: 4 bytes)
        // prefetch the next value and set the current one
        cur_neigh = next_neigh;
        next_neigh = d_nlist[nli(idx, neigh_idx+1)];

        // get the neighbor's position (MEM TRANSFER: 16 bytes)
        float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_neigh);

        // calculate dr (with periodic boundary conditions) (FLOPS: 3)
        float dx = pos.x - neigh_pos.x;
        float dy = pos.y - neigh_pos.y;
        float dz = pos.z - neigh_pos.z;
        int typej  = __float_as_int(neigh_pos.w);
        // apply periodic boundary conditions: (FLOPS 12)
        dx -= box.Lx * rintf(dx * box.Lxinv);
        dy -= box.Ly * rintf(dy * box.Lyinv);
        dz -= box.Lz * rintf(dz * box.Lzinv);

        // calculate r squard (FLOPS: 5)
        float rsq = dx*dx + dy*dy + dz*dz;
        if (rsq < eam_data_ti.r_cutsq)
            {
            float position_float = sqrtf(rsq) * eam_data_ti.rdr;
            atomElectronDensity += tex1D(electronDensity_tex, position_float + nr * (typei * ntypes + typej) + 0.5f ); //electronDensity[r_index + eam_data_ti.nr * typej] + derivativeElectronDensity[r_index + eam_data_ti.nr * typej] * position * eam_data_ti.dr;
            }
        }

    float position = atomElectronDensity * eam_data_ti.rdrho;
    /*unsigned int r_index = (unsigned int)position;
    position -= (float)r_index;*/
    atomDerivativeEmbeddingFunction[idx] = tex1D(derivativeEmbeddingFunction_tex, position + typei * eam_data_ti.nrho + 0.5f);//derivativeEmbeddingFunction[r_index + typei * eam_data_ti.nrho];

    force.w += tex1D(embeddingFunction_tex, position + typei * eam_data_ti.nrho + 0.5f);//embeddingFunction[r_index + typei * eam_data_ti.nrho] + derivativeEmbeddingFunction[r_index + typei * eam_data_ti.nrho] * position * eam_data_ti.drho;
    d_force[idx] = force;
    }

//! Second stage kernel for computing EAM forces on the GPU
extern "C" __global__ void gpu_compute_eam_tex_inter_forces_kernel_2(
    float4* d_force,
    float* d_virial,
    gpu_pdata_arrays pdata,
    gpu_boxsize box,
    const unsigned int *d_n_neigh,
    const unsigned int *d_nlist,
    const Index2D nli,
    float* atomDerivativeEmbeddingFunction)
    {
    // start by identifying which particle we are to handle
    volatile  int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= pdata.N)
        return;

    // loadj in the length of the list (MEM_TRANSFER: 4 bytes)
    int n_neigh = d_n_neigh[idx];

    // read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
    // (MEM TRANSFER: 16 bytes)
    float4 pos = tex1Dfetch(pdata_pos_tex, idx);
    int typei = __float_as_int(pos.w);
    // prefetch neighbor index
    float position;
    int cur_neigh = 0;
    int next_neigh = d_nlist[nli(idx, 0)];
    //float4 force = force_data.force[idx];
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    //force.w = force_data.force[idx].w;
    float fxi = 0.0f;
    float fyi = 0.0f;
    float fzi = 0.0f;
    float m_pe = 0.0f;
    float pairForce = 0.0f;
    float virial = 0.0f;
    force.w = force_data.force[idx].w;
    int nr = eam_data_ti.nr;
    int ntypes = eam_data_ti.ntypes;
    float adef = atomDerivativeEmbeddingFunction[idx];
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
        {
        cur_neigh = next_neigh;
        next_neigh = d_nlist[nli(idx, neigh_idx+1)];

        // get the neighbor's position (MEM TRANSFER: 16 bytes)
        float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_neigh);

        // calculate dr (with periodic boundary conditions) (FLOPS: 3)
        float dx = pos.x - neigh_pos.x;
        float dy = pos.y - neigh_pos.y;
        float dz = pos.z - neigh_pos.z;
        int typej = __float_as_int(neigh_pos.w);
        // apply periodic boundary conditions: (FLOPS 12)
        dx -= box.Lx * rintf(dx * box.Lxinv);
        dy -= box.Ly * rintf(dy * box.Lyinv);
        dz -= box.Lz * rintf(dz * box.Lzinv);

        // calculate r squard (FLOPS: 5)
        float rsq = dx*dx + dy*dy + dz*dz;

        if (rsq > eam_data_ti.r_cutsq) continue;

        float inverseR = rsqrtf(rsq);
        float r = 1.0f / inverseR;
        position = r * eam_data_ti.rdr;
        int shift = (typei>=typej)?(int)((2 * ntypes - typej -1)*typej/2 + typei) * nr:(int)((2 * ntypes - typei -1)*typei/2 + typej) * nr;
        float2 pair_potential = tex1D(pairPotential_tex, position + shift + 0.5f);
        float pair_eng =  pair_potential.x * inverseR;

        float derivativePhi = (pair_potential.y - pair_eng) * inverseR;

        float derivativeRhoI = tex1D(derivativeElectronDensity_tex, position + typei * eam_data_ti.nr + 0.5f);

        float derivativeRhoJ = tex1D(derivativeElectronDensity_tex, position + typej * eam_data_ti.nr + 0.5f);

        float fullDerivativePhi = adef * derivativeRhoJ +
                atomDerivativeEmbeddingFunction[cur_neigh] * derivativeRhoI + derivativePhi;
        pairForce = - fullDerivativePhi * inverseR;
        virial += float(1.0f/6.0f) * rsq * pairForce;

        fxi += dx * pairForce ;
        fyi += dy * pairForce ;
        fzi += dz * pairForce ;
        m_pe += pair_eng * 0.5f;
        }
        
    force.x = fxi;
    force.y = fyi;
    force.z = fzi;
    force.w += m_pe;
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    d_force[idx] = force;
    d_virial[idx] = virial;
    }

hipError_t gpu_compute_eam_tex_inter_forces(
    float4* d_force,
    float* d_virial,
    const gpu_pdata_arrays &pdata,
    const gpu_boxsize &box,
    const unsigned int *d_n_neigh,
    const unsigned int *d_nlist,
    const Index2D& nli,
    const EAMtex& eam_tex,
    const EAMTexInterArrays& eam_arrays,
    const EAMTexInterData& eam_data)
    {
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)pdata.N / (double)eam_data.block_size), 1, 1);
    dim3 threads(eam_data.block_size, 1, 1);

    // bind the texture
    pdata_pos_tex.normalized = false;
    pdata_pos_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
    if (error != hipSuccess)
        return error;

    electronDensity_tex.normalized = false;
    electronDensity_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(electronDensity_tex, eam_tex.electronDensity);
    if (error != hipSuccess)
        return error;

    pairPotential_tex.normalized = false;
    pairPotential_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(pairPotential_tex, eam_tex.pairPotential);
    if (error != hipSuccess)
        return error;

    embeddingFunction_tex.normalized = false;
    embeddingFunction_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(embeddingFunction_tex, eam_tex.embeddingFunction);
    if (error != hipSuccess)
        return error;

    derivativeElectronDensity_tex.normalized = false;
    derivativeElectronDensity_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(derivativeElectronDensity_tex, eam_tex.derivativeElectronDensity);
    if (error != hipSuccess)
        return error;

    derivativeEmbeddingFunction_tex.normalized = false;
    derivativeEmbeddingFunction_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(derivativeEmbeddingFunction_tex, eam_tex.derivativeEmbeddingFunction);
    if (error != hipSuccess)
        return error;
    // run the kernel
    hipMemcpyToSymbol(HIP_SYMBOL("eam_data_ti"), &eam_data, sizeof(EAMTexInterData));

    gpu_compute_eam_tex_inter_forces_kernel<<< grid, threads>>>(d_force,
                                                                d_virial,
                                                                pdata,
                                                                box,
                                                                d_n_neigh,
                                                                d_nlist,
                                                                nli,
                                                                eam_arrays.atomDerivativeEmbeddingFunction);

    gpu_compute_eam_tex_inter_forces_kernel_2<<< grid, threads>>>(d_force,
                                                                  d_virial,
                                                                  pdata,
                                                                  box,
                                                                  d_n_neigh,
                                                                  d_nlist,
                                                                  nli,
                                                                  eam_arrays.atomDerivativeEmbeddingFunction);

    return hipSuccess;
    }

// vim:syntax=cpp

