#include "hip/hip_runtime.h"
/**
powered by:
Moscow group.
*/

#include "gpu_settings.h"
#include "EAMTexInterForceGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file EAMTexInterForceGPU.cu
	\brief Defines GPU kernel code for calculating the eam forces. Used by EAMTexInterForceComputeGPU.
*/

//! Texture for reading particle positions
	/*
		hipArray* electronDensity;
	hipArray* pairPotential;
	hipArray* embeddingFunction;
	hipArray* derivativeElectronDensity;
	hipArray* derivativePairPotential;
	hipArray* derivativeEmbeddingFunction;
	*/
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;
texture<float, 1, hipReadModeElementType> electronDensity_tex;
texture<float2, 1, hipReadModeElementType> pairPotential_tex;
texture<float, 1, hipReadModeElementType> embeddingFunction_tex;
texture<float, 1, hipReadModeElementType> derivativeElectronDensity_tex;
//texture<float, 1, hipReadModeElementType> derivativePairPotential_tex;
texture<float, 1, hipReadModeElementType> derivativeEmbeddingFunction_tex;
texture<float, 1, hipReadModeElementType> atomDerivativeEmbeddingFunction_tex;
__constant__ EAMTexInterData eam_data;

extern "C" __global__ void gpu_compute_eam_tex_inter_forces_kernel(
	gpu_force_data_arrays force_data,
	gpu_pdata_arrays pdata,
	gpu_boxsize box,
	gpu_nlist_array nlist,
	float* atomDerivativeEmbeddingFunction)
	{
	// start by identifying which particle we are to handle
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx_local >= pdata.local_num)
		return;

	int idx_global = idx_local + pdata.local_beg;

	// load in the length of the list (MEM_TRANSFER: 4 bytes)
	int n_neigh = nlist.n_neigh[idx_global];

	// read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
	// (MEM TRANSFER: 16 bytes)
	float4 pos = tex1Dfetch(pdata_pos_tex, idx_global);

	// initialize the force to 0
	float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float virial = 0.0f;

	// prefetch neighbor index
	int cur_neigh = 0;
	int next_neigh = nlist.list[idx_global];
	int typei  = __float_as_int(pos.w);
	// loop over neighbors

	#ifdef ARCH_SM13
	// sm13 offers warp voting which makes this hardware bug workaround less of a performance penalty
	#define neigh_for for (int neigh_idx = 0; __any(neigh_idx < n_neigh); neigh_idx++)
	#else
	#define neigh_for for (int neigh_idx = 0; neigh_idx < nlist.height; neigh_idx++)
	#endif
	float atomElectronDensity  = 0.0f;
	int nr = eam_data.nr;
	int nrho = eam_data.nrho;
	int ntypes = eam_data.ntypes;
	float m_pe = 0.0f;
	for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
		{
		if (neigh_idx < n_neigh)
			{
			// read the current neighbor index (MEM TRANSFER: 4 bytes)
			// prefetch the next value and set the current one
			cur_neigh = next_neigh;
			if (neigh_idx+1 < nlist.height)
				next_neigh = nlist.list[nlist.pitch*(neigh_idx+1) + idx_global];

			// get the neighbor's position (MEM TRANSFER: 16 bytes)
			float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_neigh);

			// calculate dr (with periodic boundary conditions) (FLOPS: 3)
			float dx = pos.x - neigh_pos.x;
			float dy = pos.y - neigh_pos.y;
			float dz = pos.z - neigh_pos.z;
			int typej  = __float_as_int(neigh_pos.w);
			// apply periodic boundary conditions: (FLOPS 12)
			dx -= box.Lx * rintf(dx * box.Lxinv);
			dy -= box.Ly * rintf(dy * box.Lyinv);
			dz -= box.Lz * rintf(dz * box.Lzinv);

			// calculate r squard (FLOPS: 5)
			float rsq = dx*dx + dy*dy + dz*dz;
			if (rsq < eam_data.r_cutsq)
				{
				//���������� ������ � �������.
				 float position_float = sqrtf(rsq) * eam_data.rdr;
				 atomElectronDensity += tex1D(electronDensity_tex, position_float + nr * (typei * ntypes + typej) + 0.5f ); //electronDensity[r_index + eam_data.nr * typej] + derivativeElectronDensity[r_index + eam_data.nr * typej] * position * eam_data.dr;
				}
			}

		}


	//���������� ������ � �������.
	float position = atomElectronDensity * eam_data.rdrho;
	/*unsigned int r_index = (unsigned int)position;
	position -= (float)r_index;*/
	//��������� ��������.���������� ������������.
	atomDerivativeEmbeddingFunction[idx_global] = tex1D(derivativeEmbeddingFunction_tex, position + typei * eam_data.nrho + 0.5f);//derivativeEmbeddingFunction[r_index + typei * eam_data.nrho];

	force.w += tex1D(embeddingFunction_tex, position + typei * eam_data.nrho + 0.5f);//embeddingFunction[r_index + typei * eam_data.nrho] + derivativeEmbeddingFunction[r_index + typei * eam_data.nrho] * position * eam_data.drho;
	force_data.force[idx_local] = force;
	}
extern "C" __global__ void gpu_compute_eam_tex_inter_forces_kernel_2(
	gpu_force_data_arrays force_data,
	gpu_pdata_arrays pdata,
	gpu_boxsize box,
	gpu_nlist_array nlist,
	float* atomDerivativeEmbeddingFunction)
	{
	// start by identifying which particle we are to handle
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx_local >= pdata.local_num)
		return;

	int idx_global = idx_local + pdata.local_beg;

	// loadj in the length of the list (MEM_TRANSFER: 4 bytes)
	int n_neigh = nlist.n_neigh[idx_global];

	// read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
	// (MEM TRANSFER: 16 bytes)
	float4 pos = tex1Dfetch(pdata_pos_tex, idx_global);
	int typei = __float_as_int(pos.w);
	// prefetch neighbor index
	float position;
	int cur_neigh = 0;
	int next_neigh = nlist.list[idx_global];
	//float4 force = force_data.force[idx_local];
	float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	//force.w = force_data.force[idx_local].w;
	float fxi = 0.0f;
	float fyi = 0.0f;
	float fzi = 0.0f;
	float m_pe = 0.0f;
	float pairForce = 0.0f;
	float virial = 0.0f;
	force.w = force_data.force[idx_local].w;
	int nr = eam_data.nr;
	int nrho = eam_data.nrho;
	int ntypes = eam_data.ntypes;
	float adef = atomDerivativeEmbeddingFunction[idx_global];
	for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
		{
		if (neigh_idx < n_neigh)
			{
			cur_neigh = next_neigh;
			if (neigh_idx+1 < nlist.height)
				next_neigh = nlist.list[nlist.pitch*(neigh_idx+1) + idx_global];

			// get the neighbor's position (MEM TRANSFER: 16 bytes)
			float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_neigh);

			// calculate dr (with periodic boundary conditions) (FLOPS: 3)
			float dx = pos.x - neigh_pos.x;
			float dy = pos.y - neigh_pos.y;
			float dz = pos.z - neigh_pos.z;
			int typej = __float_as_int(neigh_pos.w);
			// apply periodic boundary conditions: (FLOPS 12)
			dx -= box.Lx * rintf(dx * box.Lxinv);
			dy -= box.Ly * rintf(dy * box.Lyinv);
			dz -= box.Lz * rintf(dz * box.Lzinv);

			// calculate r squard (FLOPS: 5)
			float rsq = dx*dx + dy*dy + dz*dz;

			if (rsq > eam_data.r_cutsq) continue;

			float inverseR = rsqrtf(rsq);
            float r = 1.0f / inverseR;
			position = r * eam_data.rdr;
			int shift = (typei>=typej)?(int)(0.5f * (2 * ntypes - typej -1)*typej + typei) * nr:(int)(0.5f * (2 * ntypes - typei -1)*typei + typej) * nr;
            float2 pair_potential = tex1D(pairPotential_tex, position + shift + 0.5f);
			float pair_eng =  pair_potential.x * inverseR;

			float derivativePhi = (pair_potential.y - pair_eng) * inverseR;

			float derivativeRhoI = tex1D(derivativeElectronDensity_tex, position + typei * eam_data.nr + 0.5f);

			float derivativeRhoJ = tex1D(derivativeElectronDensity_tex, position + typej * eam_data.nr + 0.5f);

			float fullDerivativePhi = adef * derivativeRhoJ +
				atomDerivativeEmbeddingFunction[cur_neigh] * derivativeRhoI + derivativePhi;
			 pairForce = - fullDerivativePhi * inverseR;
			virial += float(1.0f/6.0f) * rsq * pairForce;

			fxi += dx * pairForce ;
			fyi += dy * pairForce ;
			fzi += dz * pairForce ;
			m_pe += pair_eng * 0.5f;
			}
		}
		force.x = fxi;
		force.y = fyi;
		force.z = fzi;
		force.w += m_pe;
		// now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
		force_data.force[idx_local] = force;
		force_data.virial[idx_local] = virial;

	}

/*! \param force_data Force data on GPU to write forces to
	\param pdata Particle data on the GPU to perform the calculation on
	\param box Box dimensions (in GPU format) to use for periodic boundary conditions
	\param nlist Neighbor list stored on the gpu
	\param d_coeffs A \a coeff_width by \a coeff_width matrix of coefficients indexed by type
		pair i,j. The x-component is lj1 and the y-component is lj2.
	\param coeff_width Width of the \a d_coeffs matrix.
	\param eam_data.r_cutsq Precomputed r_cut*r_cut, where r_cut is the radius beyond which the
		force is set to 0
	\param block_size Block size to execute

	\returns Any error code resulting from the kernel launch

	This is just a driver for calcEAMForces_kernel, see the documentation for it for more information.
*/
hipError_t gpu_compute_eam_tex_inter_forces(
	const gpu_force_data_arrays& force_data,
	const gpu_pdata_arrays &pdata,
	const gpu_boxsize &box,
	const gpu_nlist_array &nlist,
	const EAMtex& eam_tex,
	const EAMTexInterArrays& eam_arrays,
	const EAMTexInterData& eam_data)
	{
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)pdata.local_num / (double)eam_data.block_size), 1, 1);
    dim3 threads(eam_data.block_size, 1, 1);

	// bind the texture
	pdata_pos_tex.normalized = false;
	pdata_pos_tex.filterMode = hipFilterModePoint;
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	electronDensity_tex.normalized = false;
	electronDensity_tex.filterMode = hipFilterModeLinear ;
	error = hipBindTextureToArray(electronDensity_tex, eam_tex.electronDensity);
	if (error != hipSuccess)
		return error;

	pairPotential_tex.normalized = false;
	pairPotential_tex.filterMode = hipFilterModeLinear ;
	error = hipBindTextureToArray(pairPotential_tex, eam_tex.pairPotential);
	if (error != hipSuccess)
		return error;

	embeddingFunction_tex.normalized = false;
	embeddingFunction_tex.filterMode = hipFilterModeLinear ;
	error = hipBindTextureToArray(embeddingFunction_tex, eam_tex.embeddingFunction);
	if (error != hipSuccess)
		return error;

	derivativeElectronDensity_tex.normalized = false;
	derivativeElectronDensity_tex.filterMode = hipFilterModeLinear ;
	error = hipBindTextureToArray(derivativeElectronDensity_tex, eam_tex.derivativeElectronDensity);
	if (error != hipSuccess)
		return error;
/*
	derivativePairPotential_tex.normalized = false;
	derivativePairPotential_tex.filterMode = hipFilterModeLinear ;
	error = hipBindTextureToArray(derivativePairPotential_tex, eam_tex.derivativePairPotential);
	if (error != hipSuccess)
		return error;
*/
	derivativeEmbeddingFunction_tex.normalized = false;
	derivativeEmbeddingFunction_tex.filterMode = hipFilterModeLinear ;
	error = hipBindTextureToArray(derivativeEmbeddingFunction_tex, eam_tex.derivativeEmbeddingFunction);
	if (error != hipSuccess)
		return error;
    // run the kernel
    hipMemcpyToSymbol(HIP_SYMBOL("eam_data"), &eam_data, sizeof(EAMTexInterData));

    gpu_compute_eam_tex_inter_forces_kernel<<< grid, threads>>>(force_data,
	pdata,
	box,
	nlist,
	eam_arrays.atomDerivativeEmbeddingFunction);

	gpu_compute_eam_tex_inter_forces_kernel_2<<< grid, threads>>>(force_data,
	pdata,
	box,
	nlist,
	eam_arrays.atomDerivativeEmbeddingFunction);
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

// vim:syntax=cpp
