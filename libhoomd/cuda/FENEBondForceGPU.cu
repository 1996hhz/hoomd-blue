#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008, 2009 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

Redistribution and use of HOOMD-blue, in source and binary forms, with or
without modification, are permitted, provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of HOOMD-blue's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS''
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR
ANY WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
// Maintainer: phillicl

#include "FENEBondForceGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif


/*! \file FENEBondForceGPU.cu
    \brief Defines GPU kernel code for calculating the FENE bond forces. Used by FENEBondForceComputeGPU.
*/

//! Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;

//! Texture for reading bond parameters
texture<float4, 1, hipReadModeElementType> bond_params_tex;

//! Texture for reading particle diameters
texture<float, 1, hipReadModeElementType> pdata_diam_tex;

//! Kernel for caculating FENE bond forces on the GPU
/*! \param force_data Data to write the compute forces to
    \param pdata Particle data arrays to calculate forces on
    \param box Box dimensions for periodic boundary condition handling
    \param blist Bond data to use in calculating the forces
    \param d_checkr Flag allocated on the device for use in checking for bonds that are too long
*/
extern "C" __global__ 
void gpu_compute_fene_bond_forces_kernel(gpu_force_data_arrays force_data,
                                         gpu_pdata_arrays pdata,
                                         gpu_boxsize box,
                                         gpu_bondtable_array blist,
                                         int *d_checkr)
    {
    // start by identifying which particle we are to handle
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= pdata.N)
        return;
        
    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_bonds = blist.n_bonds[idx];
    
    // read in the position of our particle. (MEM TRANSFER: 16 bytes)
    float4 pos = tex1Dfetch(pdata_pos_tex, idx);
    
    // read in the diameter of our particle.
    float diam = tex1Dfetch(pdata_diam_tex, idx);
    
    // initialize the force to 0
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    // initialize the virial to 0
    float virial = 0.0f;
    
    // loop over neighbors
    for (int bond_idx = 0; bond_idx < n_bonds; bond_idx++)
        {
        // MEM TRANSFER: 8 bytes
        // the volatile fails to compile in device emulation mode
#ifdef _DEVICEEMU
        uint2 cur_bond = blist.bonds[blist.pitch*bond_idx + idx];
#else
        // the volatile is needed to force the compiler to load the uint2 coalesced
        volatile uint2 cur_bond = blist.bonds[blist.pitch*bond_idx + idx];
#endif
        
        int cur_bond_idx = cur_bond.x;
        int cur_bond_type = cur_bond.y;
        
        // get the bonded particle's position (MEM_TRANSFER: 16 bytes)
        float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_bond_idx);
        
        // get the bonded particle's diameter
        float neigh_diam = tex1Dfetch(pdata_diam_tex, cur_bond_idx);
        
        // calculate dr (FLOPS: 3)
        float dx = pos.x - neigh_pos.x;
        float dy = pos.y - neigh_pos.y;
        float dz = pos.z - neigh_pos.z;
        
        // apply periodic boundary conditions (FLOPS: 12)
        dx -= box.Lx * rintf(dx * box.Lxinv);
        dy -= box.Ly * rintf(dy * box.Lyinv);
        dz -= box.Lz * rintf(dz * box.Lzinv);
        
        // get the bond parameters (MEM TRANSFER: 8 bytes)
        float4 params = tex1Dfetch(bond_params_tex, cur_bond_type);
        float K = params.x;
        float r_0 = params.y;
        // lj1 is defined as 4*epsilon*sigma^12
        float lj1 = 4.0f * params.w * params.z * params.z * params.z * params.z * params.z * params.z * 
                        params.z * params.z * params.z * params.z * params.z * params.z;
        // lj2 is defined as 4*epsilon*sigma^6
        float lj2 = 4.0f * params.w * params.z * params.z * params.z * params.z * params.z * params.z;
        float epsilon = params.w;
        
        
        // FLOPS: 5
        float rsq = dx*dx + dy*dy + dz*dz;
        //float r = sqrtf(rsq);
        // if particles have diameters that are not 1.0 need to correct this value by alpha
        float r = sqrtf(rsq);
        float radj =  r - (diam/2.0f + neigh_diam/2.0f - 1.0f);
        float rmdoverr = radj/r;
        rsq = radj*radj;  // This is now a diameter adjusted potential distance for diameter shifted potentials
        
        
        // calculate 1/r^2 (FLOPS: 2)
        float r2inv;
        float pastwcalimit;
        r2inv = 1.0f / rsq;
        if (rsq >= 1.2599210498f)  // comparing to the WCA limit
            pastwcalimit = 0.0f;
        else
            pastwcalimit = 1.0f;
            
        // calculate 1/r^6 (FLOPS: 2)
        float r6inv = r2inv*r2inv*r2inv;
        // calculate the force magnitude / r (FLOPS: 6)
        float wcaforcemag_divr = r2inv * r6inv * (12.0f * lj1  * r6inv - 6.0f * lj2);
        // calculate the pair energy (FLOPS: 3)
        float pair_eng = r6inv * (lj1 * r6inv - lj2) + epsilon;
        
        // FLOPS: 7
        float forcemag_divr = -K / (1.0f - rsq/(r_0*r_0))*rmdoverr + wcaforcemag_divr*rmdoverr*pastwcalimit;
        float bond_eng = -0.5f * K * r_0*r_0*logf(1.0f - rsq/(r_0*r_0));
        
        // add up the virial (FLOPS: 3)
        virial += float(1.0/6.0) * rsq * forcemag_divr;
        
        // add up the forces (FLOPS: 7)
        force.x += dx * forcemag_divr;
        force.y += dy * forcemag_divr;
        force.z += dz * forcemag_divr;
        force.w += bond_eng + pastwcalimit*pair_eng;
        
        // Checking to see if bond length restriction is violated.
        if (rsq >= r_0*r_0) *d_checkr = 1;
        
        }
        
    // energy is double counted: multiply by 0.5
    force.w *= 0.5f;
    
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes);
    force_data.force[idx] = force;
    force_data.virial[idx] = virial;
    }


/*! \param force_data Force data on GPU to write forces to
    \param pdata Particle data on the GPU to perform the calculation on
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param btable List of bonds stored on the GPU
    \param d_params K, r_0, lj1, and lj2 params packed as float4 variables
    \param d_checkr Flag allocated on the device for use in checking for bonds that are too long
    \param n_bond_types Number of bond types in d_params
    \param block_size Block size to use when performing calculations
    \param exceedsR0 output parameter set to true if any bond exceeds the length of r_0

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one float4 element per bond type. The x component contains K the spring constant
    and the y component contains r_0 the equilibrium length, z and w contain lj1 and lj2.
*/
hipError_t gpu_compute_fene_bond_forces(const gpu_force_data_arrays& force_data,
                                         const gpu_pdata_arrays &pdata,
                                         const gpu_boxsize &box,
                                         const gpu_bondtable_array &btable,
                                         float4 *d_params,
                                         int *d_checkr,
                                         unsigned int n_bond_types,
                                         int block_size,
                                         unsigned int& exceedsR0)
    {
    assert(d_params);
    // check that block_size is valid
    assert(block_size != 0);
    
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)pdata.N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);
    
    // bind the textures
    hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
    if (error != hipSuccess)
        return error;
        
    error = hipBindTexture(0, pdata_diam_tex, pdata.diameter, sizeof(float) * pdata.N);
    if (error != hipSuccess)
        return error;
        
    error = hipBindTexture(0, bond_params_tex, d_params, sizeof(float4) * n_bond_types);
    if (error != hipSuccess)
        return error;
        
    // start by zeroing check value on the device
    exceedsR0 = 0;
    error = hipMemcpy(d_checkr, &exceedsR0, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess)
        return error;
        
    // run the kernel
    gpu_compute_fene_bond_forces_kernel<<< grid, threads>>>(force_data, pdata, box, btable, d_checkr);
    
    error = hipMemcpy(&exceedsR0, d_checkr, sizeof(int), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
        return error;
        
    return hipSuccess;
    }

