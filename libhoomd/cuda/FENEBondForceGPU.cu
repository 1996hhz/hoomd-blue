#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: phillicl

#include "FENEBondForceGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif


/*! \file FENEBondForceGPU.cu
    \brief Defines GPU kernel code for calculating the FENE bond forces. Used by FENEBondForceComputeGPU.
*/

//! Texture for reading bond parameters
texture<float4, 1, hipReadModeElementType> bond_params_tex;

//! Kernel for caculating FENE bond forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param d_diameter device array of particle diameters
    \param box Box dimensions for periodic boundary condition handling
    \param blist Bond data to use in calculating the forces
    \param d_checkr Flag allocated on the device for use in checking for bonds that are too long
*/
extern "C" __global__ 
void gpu_compute_fene_bond_forces_kernel(float4* d_force,
                                         float* d_virial,
                                         const unsigned int virial_pitch,
                                         const unsigned int N,
                                         const Scalar4 *d_pos,
                                         const Scalar *d_diameter,
                                         gpu_boxsize box,
                                         gpu_bondtable_array blist,
                                         unsigned int *d_checkr)
    {
    // start by identifying which particle we are to handle
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= N)
        return;
        
    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_bonds = blist.n_bonds[idx];
    
    // read in the position of our particle. (MEM TRANSFER: 16 bytes)
    float4 pos = d_pos[idx];
    
    // read in the diameter of our particle.
    float diam = d_diameter[idx];
    
    // initialize the force to 0
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    // initialize the virial to 0
    float virialxx = 0.0f;
    float virialxy = 0.0f;
    float virialxz = 0.0f;
    float virialyy = 0.0f;
    float virialyz = 0.0f;
    float virialzz = 0.0f;


    // loop over neighbors
    for (int bond_idx = 0; bond_idx < n_bonds; bond_idx++)
        {
        // MEM TRANSFER: 8 bytes
        // the volatile fails to compile in device emulation mode
#ifdef _DEVICEEMU
        uint2 cur_bond = blist.bonds[blist.pitch*bond_idx + idx];
#else
        // the volatile is needed to force the compiler to load the uint2 coalesced
        volatile uint2 cur_bond = blist.bonds[blist.pitch*bond_idx + idx];
#endif
        
        int cur_bond_idx = cur_bond.x;
        int cur_bond_type = cur_bond.y;
        
        // get the bonded particle's position (MEM_TRANSFER: 16 bytes)
        float4 neigh_pos = d_pos[cur_bond_idx];
        
        // get the bonded particle's diameter
        float neigh_diam = d_diameter[cur_bond_idx];
        
        // calculate dr (FLOPS: 3)
        float dx = pos.x - neigh_pos.x;
        float dy = pos.y - neigh_pos.y;
        float dz = pos.z - neigh_pos.z;
        
        // apply periodic boundary conditions (FLOPS: 12)
        dx -= box.Lx * rintf(dx * box.Lxinv);
        dy -= box.Ly * rintf(dy * box.Lyinv);
        dz -= box.Lz * rintf(dz * box.Lzinv);
        
        // get the bond parameters (MEM TRANSFER: 8 bytes)
        float4 params = tex1Dfetch(bond_params_tex, cur_bond_type);
        float K = params.x;
        float r_0 = params.y;
        
        // lj1 is defined as 4*epsilon*sigma^12
        float lj1 = 4.0f * params.w * params.z * params.z * params.z * params.z * params.z * params.z * 
                        params.z * params.z * params.z * params.z * params.z * params.z;
        // lj2 is defined as 4*epsilon*sigma^6
        float lj2 = 4.0f * params.w * params.z * params.z * params.z * params.z * params.z * params.z;
        float epsilon = params.w;
       
        // FLOPS: 5
        float rsq = dx*dx + dy*dy + dz*dz;
        float rmdoverr = 1.0f;

        // if particles have diameters that are not 1.0 need to correct this value by alpha
        float rinv = rsqrtf(rsq);
        float r = 1.0f / rinv;
        float radj =  r - (diam/2.0f + neigh_diam/2.0f - 1.0f);
        rmdoverr = radj * rinv;
        rsq = radj*radj;  // This is now a diameter adjusted potential distance for diameter shifted potentials
        
        float wcaforcemag_divr = 0.0f;
        float pair_eng = 0.0f;
         
        if (rsq < 1.2599210498f && epsilon != 0.0f)  // comparing to the WCA limit
            {
            // calculate 1/r^6 (FLOPS: 2)
            float r2inv = rinv * rinv;
            float r6inv = r2inv*r2inv*r2inv;
            // calculate the force magnitude / r (FLOPS: 6)
            wcaforcemag_divr = r2inv * r6inv * (12.0f * lj1  * r6inv - 6.0f * lj2);
            // calculate the pair energy (FLOPS: 3)
            pair_eng = r6inv * (lj1 * r6inv - lj2) + epsilon;
            }
        if (!isfinite(pair_eng))
            pair_eng = 0.0f;    

        // FLOPS: 7
        float forcemag_divr = -K / (1.0f - rsq/(r_0*r_0))*rmdoverr + wcaforcemag_divr*rmdoverr;
        float bond_eng = -0.5f * K * r_0*r_0*logf(1.0f - rsq/(r_0*r_0));
        
        // detect non-finite results and zero them. This will result in the correct 0 force for r ~= 0. The energy
        // will be incorrect for r > r_0, however. Assuming that r > r_0 because K == 0, this is fine.
        if (!isfinite(forcemag_divr))
            forcemag_divr = 0.0f;
        if (!isfinite(bond_eng))
            bond_eng = 0.0f;
        
        // add up the virial (FLOPS: 3)
        float forcemag_div2r = 0.5f * forcemag_divr;
        virialxx += dx * dx * forcemag_div2r;
        virialxy += dx * dy * forcemag_div2r;
        virialxz += dx * dz * forcemag_div2r;
        virialyy += dy * dy * forcemag_div2r;
        virialyz += dy * dz * forcemag_div2r;
        virialzz += dz * dz * forcemag_div2r;

        // add up the forces (FLOPS: 7)
        force.x += dx * forcemag_divr;
        force.y += dy * forcemag_divr;
        force.z += dz * forcemag_divr;
        force.w += bond_eng + pair_eng;
        
        // Checking to see if bond length restriction is violated.
        if (rsq >= r_0*r_0) *d_checkr = 1;
        }
        
    // energy is double counted: multiply by 0.5
    force.w *= 0.5f;
    
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes);
    d_force[idx] = force;
    d_virial[0*virial_pitch+idx] = virialxx;
    d_virial[1*virial_pitch+idx] = virialxy;
    d_virial[2*virial_pitch+idx] = virialxz;
    d_virial[3*virial_pitch+idx] = virialyy;
    d_virial[4*virial_pitch+idx] = virialyz;
    d_virial[5*virial_pitch+idx] = virialzz;
    }


/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param d_diameter device array of particle diameters
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param btable List of bonds stored on the GPU
    \param d_params K, r_0, lj1, and lj2 params packed as float4 variables
    \param n_bond_types Number of bond types in d_params
    \param block_size Block size to use when performing calculations
    \param d_flags flags on the device - a 1 will be written if the r > R0

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one float4 element per bond type. The x component contains K the spring constant
    and the y component contains r_0 the equilibrium length, z and w contain lj1 and lj2.
*/
hipError_t gpu_compute_fene_bond_forces(float4* d_force,
                                         float* d_virial,
                                         const unsigned int virial_pitch,
                                         const unsigned int N,
                                         const Scalar4 *d_pos,
                                         const Scalar *d_diameter,
                                         const gpu_boxsize &box,
                                         const gpu_bondtable_array &btable,
                                         float4 *d_params,
                                         unsigned int n_bond_types,
                                         int block_size,
                                         unsigned int *d_flags)
    {
    assert(d_params);
    // check that block_size is valid
    assert(block_size != 0);
    
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);
    
    error = hipBindTexture(0, bond_params_tex, d_params, sizeof(float4) * n_bond_types);
    if (error != hipSuccess)
        return error;
        
    // run the kernel
    gpu_compute_fene_bond_forces_kernel<<< grid, threads>>>(d_force, d_virial, virial_pitch, N, d_pos, d_diameter, box, btable, d_flags);
    
    return hipSuccess;
    }

