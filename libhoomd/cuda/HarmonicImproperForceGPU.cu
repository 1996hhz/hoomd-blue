#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: dnlebard

#include "HarmonicImproperForceGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

//! SMALL a relatively small number
#define SMALL 0.001f

/*! \file HarmonicImproperForceGPU.cu
    \brief Defines GPU kernel code for calculating the harmonic improper forces. Used by HarmonicImproperForceComputeGPU.
*/

//! Texture for reading improper parameters
texture<float2, 1, hipReadModeElementType> improper_params_tex;

//! Kernel for caculating harmonic improper forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial
    \param N number of particles
    \param d_pos Device memory of particle positions
    \param box Box dimensions for periodic boundary condition handling
    \param tlist Improper data to use in calculating the forces
*/
extern "C" __global__ 
void gpu_compute_harmonic_improper_forces_kernel(float4* d_force,
                                                 float* d_virial,
                                                 const unsigned int virial_pitch,
                                                 unsigned int N,
                                                 const Scalar4 *d_pos,
                                                 gpu_boxsize box,
                                                 gpu_dihedraltable_array tlist)
    {
    // start by identifying which particle we are to handle
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
    if (idx >= N)
        return;
        
    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_impropers = tlist.n_dihedrals[idx];
    
    // read in the position of our b-particle from the a-b-c triplet. (MEM TRANSFER: 16 bytes)
    float4 idx_pos = d_pos[idx];  // we can be either a, b, or c in the a-b-c-d quartet
    float4 pos_a,pos_b,pos_c, pos_d; // allocate space for the a,b, and c atoms in the a-b-c-d quartet
    
    // initialize the force to 0
    float4 force_idx = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    
    // initialize the virial to 0
    float virial_idx[6];
    for (int i = 0; i < 6; i++)
        virial_idx[i] = 0.0f;
    
    // loop over all impropers
    for (int improper_idx = 0; improper_idx < n_impropers; improper_idx++)
        {
        // the volatile fails to compile in device emulation mode (MEM TRANSFER: 8 bytes)
#ifdef _DEVICEEMU
        uint4 cur_improper = tlist.dihedrals[tlist.pitch*improper_idx + idx];
        uint1 cur_ABCD = tlist.dihedralABCD[tlist.pitch*improper_idx + idx];
#else
        // the volatile is needed to force the compiler to load the uint2 coalesced
        volatile uint4 cur_improper = tlist.dihedrals[tlist.pitch*improper_idx + idx];
        volatile uint1 cur_ABCD = tlist.dihedralABCD[tlist.pitch*improper_idx + idx];
#endif
        
        int cur_improper_x_idx = cur_improper.x;
        int cur_improper_y_idx = cur_improper.y;
        int cur_improper_z_idx = cur_improper.z;
        int cur_improper_type = cur_improper.w;
        int cur_improper_abcd = cur_ABCD.x;
        
        // get the a-particle's position (MEM TRANSFER: 16 bytes)
        float4 x_pos = d_pos[cur_improper_x_idx];
        // get the c-particle's position (MEM TRANSFER: 16 bytes)
        float4 y_pos = d_pos[cur_improper_y_idx];
        // get the c-particle's position (MEM TRANSFER: 16 bytes)
        float4 z_pos = d_pos[cur_improper_z_idx];
        
        if (cur_improper_abcd == 0)
            {
            pos_a = idx_pos;
            pos_b = x_pos;
            pos_c = y_pos;
            pos_d = z_pos;
            }
        if (cur_improper_abcd == 1)
            {
            pos_b = idx_pos;
            pos_a = x_pos;
            pos_c = y_pos;
            pos_d = z_pos;
            }
        if (cur_improper_abcd == 2)
            {
            pos_c = idx_pos;
            pos_a = x_pos;
            pos_b = y_pos;
            pos_d = z_pos;
            }
        if (cur_improper_abcd == 3)
            {
            pos_d = idx_pos;
            pos_a = x_pos;
            pos_b = y_pos;
            pos_c = z_pos;
            }
            
        // calculate dr for a-b,c-b,and a-c(FLOPS: 9)
        float dxab = pos_a.x - pos_b.x;
        float dyab = pos_a.y - pos_b.y;
        float dzab = pos_a.z - pos_b.z;
        
        float dxcb = pos_c.x - pos_b.x;
        float dycb = pos_c.y - pos_b.y;
        float dzcb = pos_c.z - pos_b.z;
        
        float dxdc = pos_d.x - pos_c.x;
        float dydc = pos_d.y - pos_c.y;
        float dzdc = pos_d.z - pos_c.z;
        
        dxab -= box.Lx * rintf(dxab * box.Lxinv);
        dxcb -= box.Lx * rintf(dxcb * box.Lxinv);
        dxdc -= box.Lx * rintf(dxdc * box.Lxinv);
        
        dyab -= box.Ly * rintf(dyab * box.Lyinv);
        dycb -= box.Ly * rintf(dycb * box.Lyinv);
        dydc -= box.Ly * rintf(dydc * box.Lyinv);
        
        dzab -= box.Lz * rintf(dzab * box.Lzinv);
        dzcb -= box.Lz * rintf(dzcb * box.Lzinv);
        dzdc -= box.Lz * rintf(dzdc * box.Lzinv);
        
        
        // get the improper parameters (MEM TRANSFER: 12 bytes)
        float2 params = tex1Dfetch(improper_params_tex, cur_improper_type);
        float K = params.x;
        float chi = params.y;
        
        float r1 = rsqrtf(dxab*dxab + dyab*dyab + dzab*dzab);
        float r2 = rsqrtf(dxcb*dxcb + dycb*dycb + dzcb*dzcb);
        float r3 = rsqrtf(dxdc*dxdc + dydc*dydc + dzdc*dzdc);
        
        float ss1 = r1 * r1;
        float ss2 = r2 * r2;
        float ss3 = r3 * r3;
        
        // Cosine and Sin of the angle between the planes
        float c0 = (dxab*dxdc + dyab*dydc + dzab*dzdc)* r1 * r3;
        float c1 = (dxab*dxcb + dyab*dycb + dzab*dzcb)* r1 * r2;
        float c2 = -(dxdc*dxcb + dydc*dycb + dzdc*dzcb)* r3 * r2;
        
        float s1 = 1.0f - c1*c1;
        if (s1 < SMALL) s1 = SMALL;
        s1 = 1.0f / s1;
        
        float s2 = 1.0f - c2*c2;
        if (s2 < SMALL) s2 = SMALL;
        s2 = 1.0f / s2;
        
        float s12 = sqrt(s1*s2);
        float c = (c1*c2 + c0) * s12;
        
        if (c > 1.0f) c = 1.0f;
        if (c < -1.0f) c = -1.0f;
        
        float s = sqrt(1.0f - c*c);
        if (s < SMALL) s = SMALL;
        
        float domega = acosf(c) - chi;
        float a = K * domega;
        
        // calculate the energy, 1/4th for each atom
        //float improper_eng = 0.25*a*domega;
        float improper_eng = 0.125f*a*domega;  // the .125 term is 1/2 * 1/4
        
        //a = -a * 2.0/s;
        a = -a /s; // the missing 2.0 factor is to ensure K/2 is factored in for the forces
        c = c * a;
        s12 = s12 * a;
        float a11 = c*ss1*s1;
        float a22 = -ss2 * (2.0f*c0*s12 - c*(s1+s2));
        float a33 = c*ss3*s2;
        
        float a12 = -r1*r2*(c1*c*s1 + c2*s12);
        float a13 = -r1*r3*s12;
        float a23 = r2*r3*(c2*c*s2 + c1*s12);
        
        float sx2  = a22*dxcb + a23*dxdc + a12*dxab;
        float sy2  = a22*dycb + a23*dydc + a12*dyab;
        float sz2  = a22*dzcb + a23*dzdc + a12*dzab;
        
        // calculate the forces for each particle
        float ffax = a12*dxcb + a13*dxdc + a11*dxab;
        float ffay = a12*dycb + a13*dydc + a11*dyab;
        float ffaz = a12*dzcb + a13*dzdc + a11*dzab;
        
        float ffbx = -sx2 - ffax;
        float ffby = -sy2 - ffay;
        float ffbz = -sz2 - ffaz;
        
        float ffdx = a23*dxcb + a33*dxdc + a13*dxab;
        float ffdy = a23*dycb + a33*dydc + a13*dyab;
        float ffdz = a23*dzcb + a33*dzdc + a13*dzab;
        
        float ffcx = sx2 - ffdx;
        float ffcy = sy2 - ffdy;
        float ffcz = sz2 - ffdz;
        
        // and calculate the virial (symmetrized version)
        float improper_virial[6];
        improper_virial[0] = float(1./4.)*(dxab*ffax + dxcb*ffcx + (dxdc+dxcb)*ffdx);
        improper_virial[1] = float(1./8.)*((dxab*ffay + dxcb*ffcy + (dxdc+dxcb)*ffdy)
                                     +(dyab*ffax + dycb*ffcx + (dydc+dycb)*ffdx));
        improper_virial[2] = float(1./8.)*((dxab*ffaz + dxcb*ffcz + (dxdc+dxcb)*ffdz)
                                     +(dzab*ffax + dzcb*ffcx + (dzdc+dzcb)*ffdx));
        improper_virial[3] = float(1./4.)*(dyab*ffay + dycb*ffcy + (dydc+dycb)*ffdy);
        improper_virial[4] = float(1./8.)*((dyab*ffaz + dycb*ffcz + (dydc+dycb)*ffdz)
                                     +(dzab*ffay + dzcb*ffcy + (dzdc+dzcb)*ffdy));
        improper_virial[5] = float(1./4.)*(dzab*ffaz + dzcb*ffcz + (dzdc+dzcb)*ffdz);

        
        if (cur_improper_abcd == 0)
            {
            force_idx.x += ffax;
            force_idx.y += ffay;
            force_idx.z += ffaz;
            }
        if (cur_improper_abcd == 1)
            {
            force_idx.x += ffbx;
            force_idx.y += ffby;
            force_idx.z += ffbz;
            }
        if (cur_improper_abcd == 2)
            {
            force_idx.x += ffcx;
            force_idx.y += ffcy;
            force_idx.z += ffcz;
            }
        if (cur_improper_abcd == 3)
            {
            force_idx.x += ffdx;
            force_idx.y += ffdy;
            force_idx.z += ffdz;
            }
            
        force_idx.w += improper_eng;
        for (int k = 0; k < 6; k++)
            virial_idx[k] += improper_virial[k];
        }
        
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    d_force[idx] = force_idx;
    for (int k = 0; k < 6; k++)
        d_virial[k*virial_pitch+idx] = virial_idx[k];
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the device
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param ttable List of impropers stored on the GPU
    \param d_params K, sign,multiplicity params packed as padded float4 variables
    \param n_improper_types Number of improper types in d_params
    \param block_size Block size to use when performing calculations

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one float4 element per improper type. The x component contains K the spring constant
    and the y component contains sign, and the z component the multiplicity.
*/
hipError_t gpu_compute_harmonic_improper_forces(float4* d_force,
                                                 float* d_virial,
                                                 const unsigned int virial_pitch,
                                                 const unsigned int N,
                                                 const Scalar4 *d_pos,
                                                 const gpu_boxsize &box,
                                                 const gpu_dihedraltable_array &ttable,
                                                 float2 *d_params,
                                                 unsigned int n_improper_types,
                                                 int block_size)
    {
    assert(d_params);
    
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);
    
    // bind the texture
    hipError_t error = hipBindTexture(0, improper_params_tex, d_params, sizeof(float2) * n_improper_types);
    if (error != hipSuccess)
        return error;
        
    // run the kernel
    gpu_compute_harmonic_improper_forces_kernel<<< grid, threads>>>(d_force, d_virial, virial_pitch, N, d_pos, box, ttable);
    
    return hipSuccess;
    }

