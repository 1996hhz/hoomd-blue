#include "hip/hip_runtime.h"
#include "PPPM.cuh"

extern struct electrostatics_data es_data;

#define MAX(a,b) ((a) > (b) ? (a) : (b))
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MaxOrder 12
#define CONSTANT_SIZE 2048
#define SMALL 0.00001
#define LARGE 10000.0
#define EPS_HOC 1.0e-7

//! Texture for reading particle positions	
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;		

//! Texture for reading particle charges
texture<float, 1, hipReadModeElementType> pdata_charge_tex;

//! Constant memory for gridpoint weighting
__device__ __constant__ float weight_factors[CONSTANT_SIZE];

double rms(double h, double prd, double natoms)
{
    int m;
    double sum = 0.0;
    double acons[8][7]; 

    acons[1][0] = 2.0 / 3.0;
    acons[2][0] = 1.0 / 50.0;
    acons[2][1] = 5.0 / 294.0;
    acons[3][0] = 1.0 / 588.0;
    acons[3][1] = 7.0 / 1440.0;
    acons[3][2] = 21.0 / 3872.0;
    acons[4][0] = 1.0 / 4320.0;
    acons[4][1] = 3.0 / 1936.0;
    acons[4][2] = 7601.0 / 2271360.0;
    acons[4][3] = 143.0 / 28800.0;
    acons[5][0] = 1.0 / 23232.0;
    acons[5][1] = 7601.0 / 13628160.0;
    acons[5][2] = 143.0 / 69120.0;
    acons[5][3] = 517231.0 / 106536960.0;
    acons[5][4] = 106640677.0 / 11737571328.0;
    acons[6][0] = 691.0 / 68140800.0;
    acons[6][1] = 13.0 / 57600.0;
    acons[6][2] = 47021.0 / 35512320.0;
    acons[6][3] = 9694607.0 / 2095994880.0;
    acons[6][4] = 733191589.0 / 59609088000.0;
    acons[6][5] = 326190917.0 / 11700633600.0;
    acons[7][0] = 1.0 / 345600.0;
    acons[7][1] = 3617.0 / 35512320.0;
    acons[7][2] = 745739.0 / 838397952.0;
    acons[7][3] = 56399353.0 / 12773376000.0;
    acons[7][4] = 25091609.0 / 1560084480.0;
    acons[7][5] = 1755948832039.0 / 36229939200000.0;
    acons[7][6] = 4887769399.0 / 37838389248.0;

    for (m = 0; m < es_data.interpolation_order; m++) 
	sum += acons[es_data.interpolation_order][m] * pow(h*es_data.kappa,2.0*m);
    double value = es_data.q2 * pow(h*es_data.kappa,es_data.interpolation_order) *
	sqrt(es_data.kappa*prd*sqrt(2.0*M_PI)*sum/natoms) / (prd*prd);
    return value;
}

double diffpr(double hx, double hy, double hz, double Lx, double Ly, double Lz, double natoms)
{
    double lprx, lpry, lprz, kspace_prec, real_prec;
 
    lprx = rms(hx, Lx, natoms);
    lpry = rms(hy, Ly, natoms);
    lprz = rms(hz, Lz, natoms);
    kspace_prec = sqrt(lprx*lprx + lpry*lpry + lprz*lprz) / sqrt(3.0);
    real_prec = 2.0*es_data.q2 * exp(-es_data.kappa*es_data.kappa*es_data.r_cutoff*es_data.r_cutoff) / 
	sqrt(natoms*es_data.r_cutoff*Lx*Ly*Lz);
    double value = kspace_prec - real_prec;
    return value;
}

static float *gf_b;

float gf_denom(float x, float y, float z, int order)
{
    int l ;
    float sx,sy,sz;
    sz = sy = sx = 0.0;
    for (l = order-1; l >= 0; l--) {
	sx = gf_b[l] + sx*x;
	sy = gf_b[l] + sy*y;
	sz = gf_b[l] + sz*z;
    }
    float s = sx*sy*sz;
    return s*s;
}


void compute_gf_denom(int order)
{
    int k,l,m;
  
    for (l = 1; l < order; l++) gf_b[l] = 0.0;
    gf_b[0] = 1.0;
  
    for (m = 1; m < order; m++) {
	for (l = m; l > 0; l--) 
	    gf_b[l] = 4.0 * (gf_b[l]*(l-m)*(l-m-0.5)-gf_b[l-1]*(l-m-1)*(l-m-1));
	gf_b[0] = 4.0 * (gf_b[0]*(l-m)*(l-m-0.5));
    }

    int ifact = 1;
    for (k = 1; k < 2*order; k++) ifact *= k;
    float gaminv = 1.0/ifact;
    for (l = 0; l < order; l++) gf_b[l] *= gaminv;
}


void compute_rho_coeff(int assignment_order, float* rho_coeff)
{
    int j, k, l, m;
    float s;
    int order = assignment_order;
    float *a = (float*)malloc(order * (2*order+1) * sizeof(float)); 
    //    usage: a[x][y] = a[y + x*(2*order+1)]
    
    for(l=0; l<order; l++)
    {
	for(m=0; m<(2*order+1); m++)
	{
	    rho_coeff[m + l*(2*order +1)] = 0.0f;
	}
    }

    for (k = -order; k <= order; k++) 
	for (l = 0; l < order; l++) {
	    a[(k+order) + l * (2*order+1)] = 0.0f;
	}

    a[order + 0 * (2*order+1)] = 1.0f;
    for (j = 1; j < order; j++) {
	for (k = -j; k <= j; k += 2) {
	    s = 0.0;
	    for (l = 0; l < j; l++) {
		a[(k + order) + (l+1)*(2*order+1)] = (a[(k+1+order) + l * (2*order + 1)] - a[(k-1+order) + l * (2*order + 1)]) / (l+1);
		s += pow(0.5,(double) (l+1)) * (a[(k-1+order) + l * (2*order + 1)] + pow(-1.0,(double) l) * a[(k+1+order) + l * (2*order + 1)] ) / (double)(l+1);
	    }
	    a[k+order + 0 * (2*order+1)] = s;
	}
    }

    m = 0;
    for (k = -(order-1); k < order; k += 2) {
	for (l = 0; l < order; l++) {
	    rho_coeff[m + l*(2*order +1)] = a[k+order + l * (2*order + 1)];
	}
	m++;
    }
    free(a);
}


__global__ void copy_data_kernel(gpu_boxsize box_old, gpu_boxsize *box_new)
{
    box_new[0].Lx = box_old.Lx;
    box_new[0].Ly = box_old.Ly;
    box_new[0].Lz = box_old.Lz;
}

void electrostatics_allocation(const gpu_pdata_arrays &pdata, const gpu_boxsize &box, int Nx, int Ny, int Nz, int order, float kappa, float rcut_ewald)
{
    //CUDA:
    hipMalloc((void**)&(es_data.GPU_rho_real_space), sizeof(hipfftComplex)*Nx*Ny*Nz);
    hipMalloc((void**)&(es_data.GPU_green_hat), sizeof(float)*Nx*Ny*Nz);
    hipMalloc((void**)&(es_data.GPU_k_vec), sizeof(float3)*Nx*Ny*Nz);
    hipMalloc((void**)&(es_data.GPU_E_x), sizeof(hipfftComplex)*Nx*Ny*Nz);
    hipMalloc((void**)&(es_data.GPU_E_y), sizeof(hipfftComplex)*Nx*Ny*Nz);
    hipMalloc((void**)&(es_data.GPU_E_z), sizeof(hipfftComplex)*Nx*Ny*Nz);
    hipMalloc((void**)&(es_data.GPU_field), sizeof(float3)*Nx*Ny*Nz); 
    hipMalloc((void**)&(es_data.vg), sizeof(float3)*Nx*Ny*Nz);
    hipMalloc((void**)&(es_data.cuda_thermo_quantities), sizeof(float3));
    hipMalloc((void**)&(es_data.gf_b), sizeof(float)*order);
 
    es_data.CPU_rho_coeff = (float*)malloc(order * (2*order+1) * sizeof(float));
    compute_rho_coeff(order, es_data.CPU_rho_coeff);
    hipMemcpyToSymbol(HIP_SYMBOL(weight_factors), &(es_data.CPU_rho_coeff[0]), order * (2*order+1) * sizeof(float));
  
    hipfftPlan3d(&es_data.plan, Nx, Ny, Nz, HIPFFT_C2C);
  
  
    //copy information to CPU here (stupid way, but works);
    struct gpu_boxsize CPU_box;
    struct gpu_boxsize *GPU_COPY_BOX;
    hipMalloc((void**)&GPU_COPY_BOX, sizeof(struct gpu_boxsize));

    copy_data_kernel <<< 1,1 >>> (box, GPU_COPY_BOX);
      
    hipMemcpy(&CPU_box, GPU_COPY_BOX, sizeof(struct gpu_boxsize), hipMemcpyDeviceToHost);
      
    hipFree(GPU_COPY_BOX);

    /* set up for a rectangular box */
   
    float3 inverse_lattice_vector;
    float invdet = 2.0f*M_PI/(box.Lx*box.Lz*box.Lz);
    inverse_lattice_vector.x = invdet*box.Ly*box.Lz;
    inverse_lattice_vector.y = invdet*box.Lx*box.Lz;
    inverse_lattice_vector.z = invdet*box.Lx*box.Ly;
  
   
    float3* kvec_array = (float3*)malloc(Nx * Ny * Nz * sizeof(float3)); 
    int ix, iy, iz, kper, lper, mper, k, l, m;
   
    for (ix = 0; ix < Nx; ix++) {
	float3 j;
	j.x = ix > Nx/2 ? ix - Nx : ix;
	for (iy = 0; iy < Ny; iy++) {
	    j.y = iy > Ny/2 ? iy - Ny : iy;
	    for (iz = 0; iz < Nz; iz++) {
		j.z = iz > Nz/2 ? iz - Nz : iz;
		kvec_array[iz + Nz * (iy + Ny * ix)].x =  j.x*inverse_lattice_vector.x;
		kvec_array[iz + Nz * (iy + Ny * ix)].y =  j.y*inverse_lattice_vector.y;
		kvec_array[iz + Nz * (iy + Ny * ix)].z =  j.z*inverse_lattice_vector.z;
	    }
	}
    }
     
    float3* cpu_vg = (float3 *)malloc(sizeof(float3)*Nx*Ny*Nz); 

    for(int x = 0; x < Nx; x++)
    {
	for(int y = 0; y < Ny; y++)
	{
	    for(int z = 0; z < Nz; z++)
	    {
		float3 kvec = kvec_array[z + Nz * (y + Ny * x)];
		float sqk =  kvec.x*kvec.x;
		sqk += kvec.y*kvec.y;
		sqk += kvec.z*kvec.z;
	
		if (sqk == 0.0) 
		{
		    cpu_vg[z + Nz * (y + Ny * x)].x = 0.0f;
		    cpu_vg[z + Nz * (y + Ny * x)].y = 0.0f;
		    cpu_vg[z + Nz * (y + Ny * x)].z = 0.0f;
		}
		else
		{
		    float vterm = -2.0 * (1.0/sqk + 0.25/(kappa*kappa));
		    cpu_vg[z + Nz * (y + Ny * x)].x =  1.0 + vterm*kvec.x*kvec.x;
		    cpu_vg[z + Nz * (y + Ny * x)].y =  1.0 + vterm*kvec.y*kvec.y;
		    cpu_vg[z + Nz * (y + Ny * x)].z =  1.0 + vterm*kvec.z*kvec.z;
		}
	    } 
	} 
    }
    
    float* green_hat = (float*)malloc(Nx * Ny * Nz * sizeof(float)); 
   
    int assignment_order = order;
    float snx, sny, snz, snx2, sny2, snz2;
    float argx, argy, argz, wx, wy, wz, sx, sy, sz, qx, qy, qz;
    float sum1, dot1, dot2;
    float numerator, denominator, sqk;

    float unitkx = (2.0*M_PI/CPU_box.Lx);
    float unitky = (2.0*M_PI/CPU_box.Ly);
    float unitkz = (2.0*M_PI/CPU_box.Lz);
   
    
    float xprd = CPU_box.Lx; 
    float yprd = CPU_box.Ly; 
    float zprd_slab = CPU_box.Lz; 
    
    float form = 1.0;
	
    gf_b = (float *)malloc(assignment_order*sizeof(float)); 
    compute_gf_denom(assignment_order);
    hipMemcpy(es_data.gf_b, gf_b, order*sizeof(float), hipMemcpyHostToDevice);  

    float temp = floor(((kappa*xprd/(M_PI*Nx)) * 
			pow(-log(EPS_HOC),0.25)));
    int nbx = (int)temp;

    temp = floor(((kappa*yprd/(M_PI*Ny)) * 
		  pow(-log(EPS_HOC),0.25)));
    int nby = (int)temp;

    temp =  floor(((kappa*zprd_slab/(M_PI*Nz)) * 
		   pow(-log(EPS_HOC),0.25)));
    int nbz = (int)temp;

    
    for (m = 0; m < Nz; m++) {
	mper = m - Nz*(2*m/Nz);
	snz = sin(0.5*unitkz*mper*zprd_slab/Nz);
	snz2 = snz*snz;

	for (l = 0; l < Ny; l++) {
	    lper = l - Ny*(2*l/Ny);
	    sny = sin(0.5*unitky*lper*yprd/Ny);
	    sny2 = sny*sny;

	    for (k = 0; k < Nx; k++) {
		kper = k - Nx*(2*k/Nx);
		snx = sin(0.5*unitkx*kper*xprd/Nx);
		snx2 = snx*snx;
      
		sqk = pow(unitkx*kper,2.0f) + pow(unitky*lper,2.0f) + 
		    pow(unitkz*mper,2.0f);
		if (sqk != 0.0) {
		    numerator = form*12.5663706/sqk;
		    denominator = gf_denom(snx2,sny2,snz2,assignment_order);  

		    sum1 = 0.0;
		    for (ix = -nbx; ix <= nbx; ix++) {
			qx = unitkx*(kper+(float)(Nx*ix));
			sx = exp(-.25*pow(qx/kappa,2.0f));
			wx = 1.0;
			argx = 0.5*qx*xprd/(float)Nx;
			if (argx != 0.0) wx = pow(sin(argx)/argx,assignment_order);
			for (iy = -nby; iy <= nby; iy++) {
			    qy = unitky*(lper+(float)(Ny*iy));
			    sy = exp(-.25*pow(qy/kappa,2.0f));
			    wy = 1.0;
			    argy = 0.5*qy*yprd/(float)Ny;
			    if (argy != 0.0) wy = pow(sin(argy)/argy,assignment_order);
			    for (iz = -nbz; iz <= nbz; iz++) {
				qz = unitkz*(mper+(float)(Nz*iz));
				sz = exp(-.25*pow(qz/kappa,2.0f));
				wz = 1.0;
				argz = 0.5*qz*zprd_slab/(float)Nz;
				if (argz != 0.0) wz = pow(sin(argz)/argz,assignment_order);

				dot1 = unitkx*kper*qx + unitky*lper*qy + unitkz*mper*qz;
				dot2 = qx*qx+qy*qy+qz*qz;
				sum1 += (dot1/dot2) * sx*sy*sz * pow(wx*wy*wz,2.0f);
			    }
			}
		    }
		    green_hat[m + Nz * (l + Ny * k)] = numerator*sum1/denominator;
		} else green_hat[m + Nz * (l + Ny * k)] = 0.0;

	    }
	}
    }
 
    hipMemcpy(es_data.GPU_green_hat, green_hat, Nx * Ny * Nz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(es_data.GPU_k_vec, kvec_array, Nx * Ny * Nz * sizeof(float3), hipMemcpyHostToDevice);  
    hipMemcpy(es_data.vg, cpu_vg, Nx * Ny * Nz * sizeof(float3), hipMemcpyHostToDevice);  

    free(cpu_vg);
    free(gf_b);
    free(green_hat);
    free(kvec_array);
  
    float scale = 1.0f/((float)(Nx * Ny * Nz));
    es_data.CPU_energy_virial_factor = 0.5 * CPU_box.Lx * CPU_box.Ly * CPU_box.Lz * scale * scale;
}


__global__ void reset_kvec_green_hat(gpu_boxsize box, int Nx, int Ny, int Nz, int order, float kappa, float3* kvec_array, float* green_hat, float3* vg, int nbx, int nby, int nbz, float* gf_b)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if( tid < Nx*Ny*Nz) {

	int N2 = Ny*Nz;

	int xn = tid/N2;
	int yn = (tid - xn*N2)/Nz;
	int zn = (tid - xn*N2 - yn*Nz);

	float invdet = 6.28318531f/(box.Lx*box.Lz*box.Lz);
	float3 inverse_lattice_vector, j;
	float kappa2 = kappa*kappa;

	inverse_lattice_vector.x = invdet*box.Ly*box.Lz;
	inverse_lattice_vector.y = invdet*box.Lx*box.Lz;
	inverse_lattice_vector.z = invdet*box.Lx*box.Ly;

	j.x = xn > Nx/2 ? (float)(xn - Nx) : (float)xn;
	j.y = yn > Ny/2 ? (float)(yn - Ny) : (float)yn;
	j.z = zn > Nz/2 ? (float)(zn - Nz) : (float)zn;
	kvec_array[tid].x = j.x*inverse_lattice_vector.x;
	kvec_array[tid].y = j.y*inverse_lattice_vector.y;
	kvec_array[tid].z = j.z*inverse_lattice_vector.z;

	float sqk =  kvec_array[tid].x*kvec_array[tid].x + kvec_array[tid].y*kvec_array[tid].y + kvec_array[tid].z*kvec_array[tid].z;
	if(sqk == 0.0) {
	    vg[tid].x = 0.0f;
	    vg[tid].y = 0.0f;
	    vg[tid].z = 0.0f;
	}
	else {
	    float vterm = (-2.0f/sqk - 0.5f/kappa2);
	    vg[tid].x = 1.0+vterm*kvec_array[tid].x*kvec_array[tid].x;
	    vg[tid].y = 1.0+vterm*kvec_array[tid].y*kvec_array[tid].y;
	    vg[tid].z = 1.0+vterm*kvec_array[tid].z*kvec_array[tid].z;
	}

	float unitkx = (6.28318531f/box.Lx);
	float unitky = (6.28318531f/box.Ly);
	float unitkz = (6.28318531f/box.Lz);
	int ix, iy, iz, kper, lper, mper;
	float snx, sny, snz, snx2, sny2, snz2;
	float argx, argy, argz, wx, wy, wz, sx, sy, sz, qx, qy, qz;
	float sum1, dot1, dot2;
	float numerator, denominator;

	mper = zn - Nz*(2*zn/Nz);
	snz = sinf(0.5*unitkz*mper*box.Lz/Nz);
	snz2 = snz*snz;

	lper = yn - Ny*(2*yn/Ny);
	sny = sinf(0.5*unitky*lper*box.Ly/Ny);
	sny2 = sny*sny;

	kper = xn - Nx*(2*xn/Nx);
	snx = sinf(0.5*unitkx*kper*box.Lx/Nx);
	snx2 = snx*snx;
	sqk = unitkx*kper*unitkx*kper + unitky*lper*unitky*lper + unitkz*mper*unitkz*mper;


	int l;
	sz = sy = sx = 0.0;
	for (l = order-1; l >= 0; l--) {
	    sx = gf_b[l] + sx*snx2;
	    sy = gf_b[l] + sy*sny2;
	    sz = gf_b[l] + sz*snz2;
	}
	denominator = sx*sy*sz;
	denominator *= denominator;

	float W;
	if (sqk != 0.0) {
	    numerator = 12.5663706f/sqk;
	    sum1 = 0.0;
	    for (ix = -nbx; ix <= nbx; ix++) {
		qx = unitkx*(kper+(float)(Nx*ix));
		sx = expf(-.25f*qx*qx/kappa2);
		wx = 1.0f;
		argx = 0.5f*qx*box.Lx/(float)Nx;
		if (argx != 0.0) wx = powf(sinf(argx)/argx,order);
		for (iy = -nby; iy <= nby; iy++) {
		    qy = unitky*(lper+(float)(Ny*iy));
		    sy = expf(-.25f*qy*qy/kappa2);
		    wy = 1.0f;
		    argy = 0.5f*qy*box.Ly/(float)Ny;
		    if (argy != 0.0) wy = powf(sinf(argy)/argy,order);
		    for (iz = -nbz; iz <= nbz; iz++) {
			qz = unitkz*(mper+(float)(Nz*iz));
			sz = expf(-.25f*qz*qz/kappa2);
			wz = 1.0f;
			argz = 0.5f*qz*box.Lz/(float)Nz;
			if (argz != 0.0) wz = powf(sinf(argz)/argz,order);

			dot1 = unitkx*kper*qx + unitky*lper*qy + unitkz*mper*qz;
			dot2 = qx*qx+qy*qy+qz*qz;
			W = wx*wy*wz;
			sum1 += (dot1/dot2) * sx*sy*sz * W*W;
		    }
		}
	    }
	    green_hat[tid] = numerator*sum1/denominator;
	} else green_hat[tid] = 0.0;
    }
}

__device__ inline void atomicFloatAdd(float* address, float value)
{
#if (__CUDA_ARCH__ < 200)
    float old = value;
    float new_old;
    do
    {
	new_old = atomicExch(address, 0.0f);
	new_old += old;
    }
    while ((old = atomicExch(address, new_old))!=0.0f);
#else
    atomicAdd(address, value);
#endif
}


__global__ void combined_green_e_kernel(hipfftComplex* E_x, hipfftComplex* E_y, hipfftComplex* E_z, float3* k_vec, hipfftComplex* rho, int Nx, int Ny, int Nz, float* green_function)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
    if(tid < Nx * Ny * Nz)
    {
	float3 k_vec_local = k_vec[tid];
	hipfftComplex E_x_local, E_y_local, E_z_local;
	float scale_times_green = green_function[tid] / ((float)(Nx*Ny*Nz));
	hipfftComplex rho_local = rho[tid];
    
	rho[tid] = make_float2(0.0f,0.0f);
    
	rho_local.x *= scale_times_green;
	rho_local.y *= scale_times_green;
      
	E_x_local.x = k_vec_local.x * rho_local.y;
	E_x_local.y = -k_vec_local.x * rho_local.x;
    
	E_y_local.x = k_vec_local.y * rho_local.y;
	E_y_local.y = -k_vec_local.y * rho_local.x;
    
	E_z_local.x = k_vec_local.z * rho_local.y;
	E_z_local.y = -k_vec_local.z * rho_local.x;
    
    
	E_x[tid] = E_x_local;
	E_y[tid] = E_y_local;
	E_z[tid] = E_z_local;   
    }
}


__global__ void set_to_zero(hipfftComplex* array, int Nx, int Ny, int Nz)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
 
    if(tid < Nx * Ny * Nz)
    {
	hipfftComplex Zero = make_float2(0.0f,0.0f);
	array[tid] = Zero;
    }
}

__device__ inline void AddToGridpoint(int X, int Y, int Z, hipfftComplex* array, float value, int Ny, int Nz)
{
    atomicFloatAdd(&array[Z + Nz * (Y + Ny * X)].x, value);
}


__device__ float get_rho1d(float delta, int interpolation_order, int y)
{
    int l, order = interpolation_order;

    float result;
    
    result = 0.0f;
    for (l = order-1; l >= 0; l--) {
	result = weight_factors[y + l*(2*order +1)] + result * delta;
    }

    return result;
}


__global__ void assign_charges_to_grid_kernel(gpu_pdata_arrays pdata, gpu_boxsize box, hipfftComplex *rho_real_space, int Nx, int Ny, int Nz, int order)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < pdata.N)
    {
	//get particle information
	float qi = tex1Dfetch(pdata_charge_tex, idx);
	if(qi != 0.0) {
	    float4 posi = tex1Dfetch(pdata_pos_tex, idx);
	    //calculate dx, dy, dz for the charge density grid:
	    float box_dx = box.Lx / ((float)Nx);
	    float box_dy = box.Ly / ((float)Ny);
	    float box_dz = box.Lz / ((float)Nz);
    
        
	    //normalize position to gridsize:
	    posi.x += box.Lx / 2.0f;
	    posi.y += box.Ly / 2.0f;
	    posi.z += box.Lz / 2.0f;
   
	    posi.x /= box_dx;
	    posi.y /= box_dy;
	    posi.z /= box_dz;
    
    
	    float shift, shiftone, x0, y0, z0, dx, dy, dz;
	    int nlower, nupper, mx, my, mz, nxi, nyi, nzi; 
    
	    nlower = -(order-1)/2;
	    nupper = order/2;
    
	    if (order % 2) 
	    {
		shift =0.5;
		shiftone = 0.0;
	    }
	    else 
	    {
		shift = 0.0;
		shiftone = 0.5;
	    }
        
    
	    nxi = __float2int_rd(posi.x + shift);
	    nyi = __float2int_rd(posi.y + shift);
	    nzi = __float2int_rd(posi.z + shift);
    
	    dx = shiftone+(float)nxi-posi.x;
	    dy = shiftone+(float)nyi-posi.y;
	    dz = shiftone+(float)nzi-posi.z;
    
	    int n,m,l;
    
	    x0 = qi / (box_dx*box_dy*box_dz);
	    for (n = nlower; n <= nupper; n++) {
		mx = n+nxi;
		if(mx >= Nx) mx -= Nx;
		if(mx < 0)  mx += Nx;
		y0 = x0*get_rho1d(dx, order, n-nlower);
		for (m = nlower; m <= nupper; m++) {
		    my = m+nyi;
		    if(my >= Ny) my -= Ny;
		    if(my < 0)  my += Ny;
		    z0 = y0*get_rho1d(dy, order, m-nlower);
		    for (l = nlower; l <= nupper; l++) {
			mz = l+nzi;
			if(mz >= Nz) mz -= Nz;
			if(mz < 0)  mz += Nz;
			AddToGridpoint(mx, my, mz, rho_real_space, z0*get_rho1d(dz, order, l-nlower), Ny, Nz);
		    }
		}
	    }
	}
    }
}

__global__ void set_gpu_field_kernel(hipfftComplex* E_x, hipfftComplex* E_y, hipfftComplex* E_z, float3* Electric_field, int Nx, int Ny, int Nz)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < Nx * Ny * Nz)
    {
	float3 local_field;
	local_field.x = E_x[tid].x;
	local_field.y = E_y[tid].x;
	local_field.z = E_z[tid].x;
      
	Electric_field[tid] = local_field;
    }
}

__global__ void calculate_forces_kernel(gpu_force_data_arrays force_data, gpu_pdata_arrays pdata, gpu_boxsize box, float3* Electric_field, int Nx, int Ny, int Nz, int order)
{  
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < pdata.N)
    {
	//get particle information
	float qi = tex1Dfetch(pdata_charge_tex, idx);
	if(qi != 0.0) {
	    float4 posi = tex1Dfetch(pdata_pos_tex, idx);
    
	    //calculate dx, dy, dz for the charge density grid:
	    float box_dx = box.Lx / ((float)Nx);
	    float box_dy = box.Ly / ((float)Ny);
	    float box_dz = box.Lz / ((float)Nz);
    
	    //normalize position to gridsize:
	    posi.x += box.Lx / 2.0f;
	    posi.y += box.Ly / 2.0f;
	    posi.z += box.Lz / 2.0f;
   
	    posi.x /= box_dx;
	    posi.y /= box_dy;
	    posi.z /= box_dz;
    
	    float shift, shiftone, x0, y0, z0, dx, dy, dz;
	    int nlower, nupper, mx, my, mz, nxi, nyi, nzi; 
    
	    nlower = -(order-1)/2;
	    nupper = order/2;
    
	    float4 local_force = force_data.force[idx];

	    if(order % 2) 
	    {
		shift =0.5;
		shiftone = 0.0;
	    }
	    else 
	    {
		shift = 0.0;
		shiftone = 0.5;
	    }
    
    
	    nxi = __float2int_rd(posi.x + shift);
	    nyi = __float2int_rd(posi.y + shift);
	    nzi = __float2int_rd(posi.z + shift);
    
	    dx = shiftone+(float)nxi-posi.x;
	    dy = shiftone+(float)nyi-posi.y;
	    dz = shiftone+(float)nzi-posi.z;
	    int n,m,l;
    
	    for (n = nlower; n <= nupper; n++) {
		mx = n+nxi;
		if(mx >= Nx) mx -= Nx;
		if(mx < 0)  mx += Nx;
		x0 = get_rho1d(dx, order, n-nlower);
		for (m = nlower; m <= nupper; m++) {
		    my = m+nyi;
		    if(my >= Ny) my -= Ny;
		    if(my < 0)  my += Ny;
		    y0 = x0*get_rho1d(dy, order, m-nlower);
		    for (l = nlower; l <= nupper; l++) {
			mz = l+nzi;
			if(mz >= Nz) mz -= Nz;
			if(mz < 0)  mz += Nz;
			z0 = y0*get_rho1d(dz, order, l-nlower);
			float3 local_field = Electric_field[mz + Nz * (my + Ny * mx)];
			local_force.x += qi*z0*local_field.x;
			local_force.y += qi*z0*local_field.y;
			local_force.z += qi*z0*local_field.z;
		    }
		}
	    }
    
	    force_data.force[idx] = local_force;
	}
    }
} 

__global__ void calculate_thermo_quantities_kernel(hipfftComplex* rho, float* green_function, float3* GPU_virial_energy, float3* vg, int Nx, int Ny, int Nz)
{
    int threadx = blockIdx.x * blockDim.x + threadIdx.x;
    int thready = blockIdx.y * blockDim.y + threadIdx.y;

    if((threadx < Nx) && (thready < Ny))
    {
	float2 local_GPU_virial_energy = make_float2(0.0f,0.0f);
	float3 local_vg;
	float local_green, green_times_rho_square;
	hipfftComplex rho_local;
	for(int z = 0; z < Nz; z++)
	{
	    local_vg = vg[z + Nz * (thready + Ny * threadx)];
	    local_green = green_function[z + Nz * (thready + Ny * threadx)];
	    rho_local = rho[z + Nz * (thready + Ny * threadx)];
	
	    green_times_rho_square = local_green * (rho_local.x * rho_local.x + rho_local.y * rho_local.y);
	    local_GPU_virial_energy.x += green_times_rho_square * (local_vg.x + local_vg.y + local_vg.z);
	    local_GPU_virial_energy.y += green_times_rho_square ;
	}
  
	atomicFloatAdd(&GPU_virial_energy[0].x, local_GPU_virial_energy.x);
	atomicFloatAdd(&GPU_virial_energy[0].y, local_GPU_virial_energy.y);

    }
}

__global__ void get_charge(gpu_pdata_arrays pdata, float3 *GPU_virial_energy)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < pdata.N) {
	float qi = tex1Dfetch(pdata_charge_tex, idx);
	atomicFloatAdd(&GPU_virial_energy[0].x, qi);
    }
}

__global__ void get_charge_squared(gpu_pdata_arrays pdata, float3 *GPU_virial_energy)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < pdata.N) {
	float qi = tex1Dfetch(pdata_charge_tex, idx);
	atomicFloatAdd(&GPU_virial_energy[0].z, qi*qi);
    }

}

float3 calculate_thermo_quantities(const gpu_pdata_arrays &pdata, const gpu_boxsize &box)
{
    if(es_data.electrostatics_allocation_bool)
    {
	//kernel calling parameters for all grid dependent kernels
	int new_blocksize = 256;
	int new_gridsize = es_data.Nx*es_data.Ny*es_data.Nz / new_blocksize + 1;
      
	// setup the grid to run the kernel
	int blocksize = 256;
	dim3 grid( pdata.N / blocksize + 1, 1, 1);
	dim3 threads(blocksize, 1, 1);

	// bind the position texture
	pdata_pos_tex.normalized = false;
	pdata_pos_tex.filterMode = hipFilterModePoint;
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);

	// bind the charge texture
	pdata_charge_tex.normalized = false;
	pdata_charge_tex.filterMode = hipFilterModePoint;
	error = hipBindTexture(0, pdata_charge_tex, pdata.charge, sizeof(float) * pdata.N);
	  
	//assign the charge density to the gridpoints
	assign_charges_to_grid_kernel <<< grid, threads >>> (pdata, box, es_data.GPU_rho_real_space, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order);
	hipDeviceSynchronize();    
    
	//call the forward FFT for the charge density
	hipfftExecC2C(es_data.plan, es_data.GPU_rho_real_space, es_data.GPU_rho_real_space, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	  
	//calculate the virial and energy:  
	float3 CPU_virial_energy = make_float3(0.0f, 0.0f, 0.0f);
	hipMemcpy(es_data.cuda_thermo_quantities, &CPU_virial_energy, sizeof(float3), hipMemcpyHostToDevice);  
	get_charge_squared <<< grid, threads >>> (pdata, es_data.cuda_thermo_quantities);
     
	dim3 thermo_block(8,8,1);
	dim3 thermo_grid(es_data.Nx/thermo_block.x, es_data.Ny/thermo_block.y, 1);
	calculate_thermo_quantities_kernel <<< thermo_grid, thermo_block >>> (es_data.GPU_rho_real_space, es_data.GPU_green_hat, es_data.cuda_thermo_quantities, es_data.vg, es_data.Nx, es_data.Ny, es_data.Nz);
	
	//copy to CPU:
	hipMemcpy(&CPU_virial_energy, es_data.cuda_thermo_quantities, sizeof(float3), hipMemcpyDeviceToHost);

	struct gpu_boxsize CPU_box;
	struct gpu_boxsize *GPU_COPY_BOX;
	hipMalloc((void**)&GPU_COPY_BOX, sizeof(struct gpu_boxsize));

	copy_data_kernel <<< 1,1 >>> (box, GPU_COPY_BOX);
      
	hipMemcpy(&CPU_box, GPU_COPY_BOX, sizeof(struct gpu_boxsize), hipMemcpyDeviceToHost);
      
	hipFree(GPU_COPY_BOX);
      
	CPU_virial_energy.x *= es_data.CPU_energy_virial_factor / (3.0f * CPU_box.Lx * CPU_box.Ly * CPU_box.Lz);
	CPU_virial_energy.y *= es_data.CPU_energy_virial_factor;
	
	CPU_virial_energy.y -= CPU_virial_energy.z * es_data.kappa / sqrt(M_PI);

	set_to_zero <<< new_gridsize , new_blocksize >>> (es_data.GPU_rho_real_space, es_data.Nx, es_data.Ny, es_data.Nz);
	hipDeviceSynchronize();  
	return CPU_virial_energy;
	
	
    }
    else 
	return make_float3(0.0f, 0.0f, 0.0f);
}


void electrostatics_calculation(const gpu_force_data_arrays& force_data, const gpu_pdata_arrays &pdata, const gpu_boxsize &box, const float3 *d_params, const float *d_rcutsq)
{
    //first time allocation of memory-------------------------------------
    int blocksize = 256;
    dim3 grid( pdata.N / blocksize + 1, 1, 1);
    dim3 threads(blocksize, 1, 1);

	static gpu_boxsize CPU_box_old;
	struct gpu_boxsize CPU_box;
	struct gpu_boxsize *GPU_COPY_BOX;

	hipMalloc((void**)&GPU_COPY_BOX, sizeof(struct gpu_boxsize));

	copy_data_kernel <<< 1,1 >>> (box, GPU_COPY_BOX);
	hipMemcpy(&CPU_box, GPU_COPY_BOX, sizeof(struct gpu_boxsize), hipMemcpyDeviceToHost);

    if(!es_data.electrostatics_allocation_bool)
    {
	float3 *GPU_charge;


	hipMalloc((void**)&(GPU_charge), sizeof(float3));

	float3 cpu_params;
	hipMemcpy(&cpu_params, d_params, sizeof(float3), hipMemcpyDeviceToHost);
	printf("kappa = %g grid = %d order = %d\n",cpu_params.x, (int)cpu_params.y, (int)cpu_params.z);
  
	//Store values:
	es_data.Nx = cpu_params.y;
	es_data.Ny = es_data.Nx;
	es_data.Nz = es_data.Nx;

	es_data.interpolation_order = (int)cpu_params.z;

	// bind the charge texture
	pdata_charge_tex.normalized = false;
	pdata_charge_tex.filterMode = hipFilterModePoint;
	hipError_t error = hipBindTexture(0, pdata_charge_tex, pdata.charge, sizeof(float) * pdata.N);

	float3 CPU_charge = make_float3(0.0f, 0.0f, 0.0f);
	hipMemcpy(GPU_charge, &CPU_charge, sizeof(float3), hipMemcpyHostToDevice);  
	get_charge <<< grid, threads >>> (pdata, GPU_charge);
	get_charge_squared <<< grid, threads >>> (pdata, GPU_charge);
	hipMemcpy(&CPU_charge, GPU_charge, sizeof(float3), hipMemcpyDeviceToHost);
	es_data.q2 = CPU_charge.z;

	float cpu_rcutsq;
	hipMemcpy(&cpu_rcutsq, d_rcutsq, sizeof(float), hipMemcpyDeviceToHost);
      
	es_data.r_cutoff = cpu_rcutsq;

	es_data.show_virial_flag = 0;

	es_data.electrostatics_allocation_bool = 1;
	if(!(es_data.Nx == 2)&& !(es_data.Nx == 4)&& !(es_data.Nx == 8)&& !(es_data.Nx == 16)&& !(es_data.Nx == 32)&& !(es_data.Nx == 64)&& !(es_data.Nx == 128)&& !(es_data.Nx == 256)&& !(es_data.Nx == 512)&& !(es_data.Nx == 1024))
	{
	    printf("\n\n ------ ATTENTION gridsize should be a power of 2 ------ \n\n");
	}
	if (es_data.interpolation_order * (2*es_data.interpolation_order +1) > CONSTANT_SIZE)
	{
	    printf("interpolation order too high, doesn't fit into constant array\n");
	    exit(1);
	}
	if (es_data.interpolation_order > MaxOrder)
	{
	    printf("interpolation order too high\n");
	    exit(1);
	}
            
	es_data.kappa  = cpu_params.x;
      
	electrostatics_allocation(pdata, box, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order, es_data.kappa, es_data.r_cutoff);

	CPU_box_old = CPU_box; 
	float hx =  CPU_box.Lx/es_data.Nx;
	float hy =  CPU_box.Ly/es_data.Ny;
	float hz =  CPU_box.Lz/es_data.Nz;

	float lprx = rms(hx, CPU_box.Lx, pdata.N); 
	float lpry = rms(hy, CPU_box.Lz, pdata.N);
	float lprz = rms(hz, CPU_box.Lz, pdata.N);
	float lpr = sqrt(lprx*lprx + lpry*lpry + lprz*lprz) / sqrt(3.0);
	float spr = 2.0*CPU_charge.z*exp(-es_data.kappa*es_data.kappa*cpu_rcutsq) / sqrt(pdata.N*sqrt(cpu_rcutsq)*CPU_box.Lx*CPU_box.Ly*CPU_box.Lz);

     
	double RMS_error = MAX(lpr,spr);
	if(RMS_error > 0.1) {
	    printf("!!!!!!!\n!!!!!!!\n!!!!!!!\nWARNING RMS error of %g is probably too high %f %f\n!!!!!!!\n!!!!!!!\n!!!!!!!\n", RMS_error, lpr, spr);
	}
	else{
	    printf("RMS error: %g\n", RMS_error);
	}
 
	if(CPU_charge.x > 0.0001 || CPU_charge.x < -0.0001) printf("WARNING system in not neutral, the net charge is %g\n", CPU_charge.x);
     
      
	printf("allocation for electrostatics done... \n");
	int new_blocksize = 256;
	int new_gridsize = es_data.Nx*es_data.Ny*es_data.Nz / new_blocksize + 1;
	//only for the first time needed, next time it is done in function new_combined_green_e_kernel
	set_to_zero <<< new_gridsize , new_blocksize >>> (es_data.GPU_rho_real_space, es_data.Nx, es_data.Ny, es_data.Nz);
	hipDeviceSynchronize();  
     
    }

    //kernel calling parameters for all grid dependent kernels
    int new_blocksize = 256;
    int new_gridsize = es_data.Nx*es_data.Ny*es_data.Nz / new_blocksize + 1;
    
    if(fabs(CPU_box.Lx - CPU_box_old.Lx) > 0.00001 || fabs(CPU_box.Ly - CPU_box_old.Ly) > 0.00001 || fabs(CPU_box.Lz - CPU_box_old.Lz) > 0.00001 || 1) {
      	
	float temp = floor(((es_data.kappa*CPU_box.Lx/(M_PI*es_data.Nx)) *  pow(-log(EPS_HOC),0.25)));
	int nbx = (int)temp;
	temp = floor(((es_data.kappa*CPU_box.Ly/(M_PI*es_data.Ny)) * pow(-log(EPS_HOC),0.25)));
	int nby = (int)temp;
	temp =  floor(((es_data.kappa*CPU_box.Lz/(M_PI*es_data.Nz)) *  pow(-log(EPS_HOC),0.25)));
	int nbz = (int)temp;

	reset_kvec_green_hat <<< new_gridsize, new_blocksize >>>(box, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order, es_data.kappa, es_data.GPU_k_vec, es_data.GPU_green_hat, es_data.vg, nbx, nby, nbz, es_data.gf_b);
	hipDeviceSynchronize();
	CPU_box_old.Lx = CPU_box.Lx;
	CPU_box_old.Ly = CPU_box.Ly;
	CPU_box_old.Lz = CPU_box.Lz;
	float scale = 1.0f/((float)(es_data.Nx * es_data.Ny * es_data.Nz));
	es_data.CPU_energy_virial_factor = 0.5 * CPU_box.Lx * CPU_box.Ly * CPU_box.Lz * scale * scale;
    }

    // setup the grid to run the particle kernel 
    
    
    // bind the position texture
    pdata_pos_tex.normalized = false;
    pdata_pos_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);

    // bind the charge texture
    pdata_charge_tex.normalized = false;
    pdata_charge_tex.filterMode = hipFilterModePoint;
    error = hipBindTexture(0, pdata_charge_tex, pdata.charge, sizeof(float) * pdata.N);
     
    //assign the charge density to the gridpoints
    assign_charges_to_grid_kernel <<< grid, threads >>> (pdata, box, es_data.GPU_rho_real_space, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order);
    hipDeviceSynchronize();    
     
    //call the forward FFT for the charge density
    hipfftExecC2C(es_data.plan, es_data.GPU_rho_real_space, es_data.GPU_rho_real_space, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    
    combined_green_e_kernel <<< new_gridsize, new_blocksize >>> (es_data.GPU_E_x, es_data.GPU_E_y, es_data.GPU_E_z, es_data.GPU_k_vec, es_data.GPU_rho_real_space,  es_data.Nx, es_data.Ny, es_data.Nz, es_data.GPU_green_hat);
       

    //backtransform field:
    hipfftExecC2C(es_data.plan, es_data.GPU_E_x, es_data.GPU_E_x, HIPFFT_BACKWARD);
    hipfftExecC2C(es_data.plan, es_data.GPU_E_y, es_data.GPU_E_y, HIPFFT_BACKWARD);
    hipfftExecC2C(es_data.plan, es_data.GPU_E_z, es_data.GPU_E_z, HIPFFT_BACKWARD);
    
    //put field into float3 array
    set_gpu_field_kernel <<< new_gridsize, new_blocksize >>> (es_data.GPU_E_x, es_data.GPU_E_y, es_data.GPU_E_z, es_data.GPU_field, es_data.Nx, es_data.Ny, es_data.Nz);
    hipDeviceSynchronize();
    //calculate forces on particles:
    calculate_forces_kernel <<< grid, threads >>>(force_data, pdata, box, es_data.GPU_field, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order);

    hipFree(GPU_COPY_BOX);
}
