#include "hip/hip_runtime.h"
#include "PPPM.cuh"

extern struct electrostatics_data es_data;

#define MAX(a,b) ((a) > (b) ? (a) : (b))
#define MaxOrder 12
#define CONSTANT_SIZE 2048

//! Texture for reading particle positions	
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;		

//! Texture for reading particle charges
texture<float, 1, hipReadModeElementType> pdata_charge_tex;

//! Constant memory for gridpoint weighting
__device__ __constant__ float weight_factors[CONSTANT_SIZE];


typedef struct { float xx, yx, zx, xy, yy, zy, xz, yz, zz; } tensor_t;

struct coord
{
  float x;
  float y;
  float z;
};

typedef struct { float x, y, z; } cart_t;


double rms(double h, double prd, double natoms, int order, double kappa, double q2)
{
    int m;
    double sum = 0.0;
    double acons[8][7]; 

    acons[1][0] = 2.0 / 3.0;
    acons[2][0] = 1.0 / 50.0;
    acons[2][1] = 5.0 / 294.0;
    acons[3][0] = 1.0 / 588.0;
    acons[3][1] = 7.0 / 1440.0;
    acons[3][2] = 21.0 / 3872.0;
    acons[4][0] = 1.0 / 4320.0;
    acons[4][1] = 3.0 / 1936.0;
    acons[4][2] = 7601.0 / 2271360.0;
    acons[4][3] = 143.0 / 28800.0;
    acons[5][0] = 1.0 / 23232.0;
    acons[5][1] = 7601.0 / 13628160.0;
    acons[5][2] = 143.0 / 69120.0;
    acons[5][3] = 517231.0 / 106536960.0;
    acons[5][4] = 106640677.0 / 11737571328.0;
    acons[6][0] = 691.0 / 68140800.0;
    acons[6][1] = 13.0 / 57600.0;
    acons[6][2] = 47021.0 / 35512320.0;
    acons[6][3] = 9694607.0 / 2095994880.0;
    acons[6][4] = 733191589.0 / 59609088000.0;
    acons[6][5] = 326190917.0 / 11700633600.0;
    acons[7][0] = 1.0 / 345600.0;
    acons[7][1] = 3617.0 / 35512320.0;
    acons[7][2] = 745739.0 / 838397952.0;
    acons[7][3] = 56399353.0 / 12773376000.0;
    acons[7][4] = 25091609.0 / 1560084480.0;
    acons[7][5] = 1755948832039.0 / 36229939200000.0;
    acons[7][6] = 4887769399.0 / 37838389248.0;

    for (m = 0; m < order; m++) 
	sum += acons[order][m] * pow(h*kappa,2.0*m);
    double value = q2 * pow(h*kappa,order) *
	sqrt(kappa*prd*sqrt(2.0*M_PI)*sum/natoms) / (prd*prd);
    return value;
}

void scalar_multiply(tensor_t *a, float b)
{
  a->xx *= b;  a->xy *= b;  a->xz *= b;
  a->yx *= b;  a->yy *= b;  a->yz *= b;
  a->zx *= b;  a->zy *= b;  a->zz *= b;
}
  
  
void transpose(tensor_t *a, tensor_t *b)
{
  a->xx = b->xx;  a->xy = b->yx;  a->xz = b->zx;
  a->yx = b->xy;  a->yy = b->yy;  a->yz = b->zy;
  a->zx = b->xz;  a->zy = b->yz;  a->zz = b->zz;
}

float det(tensor_t *a)
{
  return -a->xz*a->yy*a->zx + a->xy*a->yz*a->zx + 
    a->xz*a->yx*a->zy - a->xx*a->yz*a->zy - 
    a->xy*a->yx*a->zz + a->xx*a->yy*a->zz;
}

void inverse(tensor_t *a, tensor_t *b)
{
  float invdet = 1.0/det(b);
  a->xx = invdet*(-b->yz*b->zy + b->yy*b->zz);
  a->xy = invdet*( b->xz*b->zy - b->xy*b->zz);
  a->xz = invdet*(-b->xz*b->yy + b->xy*b->yz);
  a->yx = invdet*( b->yz*b->zx - b->yx*b->zz);
  a->yy = invdet*(-b->xz*b->zx + b->xx*b->zz);
  a->yz = invdet*( b->xz*b->yx - b->xx*b->yz);
  a->zx = invdet*(-b->yy*b->zx + b->yx*b->zy);
  a->zy = invdet*( b->xy*b->zx - b->xx*b->zy);
  a->zz = invdet*(-b->xy*b->yx + b->xx*b->yy);
}

void multiply(float3 *a, tensor_t *t, struct coord *c)
{
  a->x = t->xx*c->x + t->xy*c->y + t->xz*c->z;
  a->y = t->yx*c->x + t->yy*c->y + t->yz*c->z;
  a->z = t->zx*c->x + t->zy*c->y + t->zz*c->z;
}

static float *gf_b;

float gf_denom(float x, float y, float z, int order)
{
  int l ;
  float sx,sy,sz;
  sz = sy = sx = 0.0;
  for (l = order-1; l >= 0; l--) {
    sx = gf_b[l] + sx*x;
    sy = gf_b[l] + sy*y;
    sz = gf_b[l] + sz*z;
  }
  float s = sx*sy*sz;
  return s*s;
}

void compute_gf_denom(int order)
{
  int k,l,m;
  
  for (l = 1; l < order; l++) gf_b[l] = 0.0;
  gf_b[0] = 1.0;
  
  for (m = 1; m < order; m++) {
    for (l = m; l > 0; l--) 
      gf_b[l] = 4.0 * (gf_b[l]*(l-m)*(l-m-0.5)-gf_b[l-1]*(l-m-1)*(l-m-1));
    gf_b[0] = 4.0 * (gf_b[0]*(l-m)*(l-m-0.5));
  }

  int ifact = 1;
  for (k = 1; k < 2*order; k++) ifact *= k;
  float gaminv = 1.0/ifact;
  for (l = 0; l < order; l++) gf_b[l] *= gaminv;
}


void compute_rho_coeff(int assignment_order, float* rho_coeff)
{
  int j, k, l, m;
  float s;
  int order = assignment_order;
  float *a = (float*)malloc(order * (2*order+1) * sizeof(float)); 
  //    usage: a[x][y] = a[y + x*(2*order+1)]
    
  for(l=0; l<order; l++)
    {
      for(m=0; m<(2*order+1); m++)
	{
	  rho_coeff[m + l*(2*order +1)] = 0.0f;
	}
    }

  for (k = -order; k <= order; k++) 
    for (l = 0; l < order; l++) {
      a[(k+order) + l * (2*order+1)] = 0.0f;
    }

  a[order + 0 * (2*order+1)] = 1.0f;
  for (j = 1; j < order; j++) {
    for (k = -j; k <= j; k += 2) {
      s = 0.0;
      for (l = 0; l < j; l++) {
	a[(k + order) + (l+1)*(2*order+1)] = (a[(k+1+order) + l * (2*order + 1)] - a[(k-1+order) + l * (2*order + 1)]) / (l+1);
	s += pow(0.5,(double) (l+1)) * (a[(k-1+order) + l * (2*order + 1)] + pow(-1.0,(double) l) * a[(k+1+order) + l * (2*order + 1)] ) / (double)(l+1);
      }
      a[k+order + 0 * (2*order+1)] = s;
    }
  }

  m = 0;
  for (k = -(order-1); k < order; k += 2) {
    for (l = 0; l < order; l++) {
      rho_coeff[m + l*(2*order +1)] = a[k+order + l * (2*order + 1)];
    }
    m++;
  }
  free(a);
}


__global__ void copy_data_kernel(gpu_boxsize box_old, gpu_boxsize *box_new)
{
  box_new[0].Lx = box_old.Lx;
  box_new[0].Ly = box_old.Ly;
  box_new[0].Lz = box_old.Lz;
}

void electrostatics_allocation(const gpu_pdata_arrays &pdata, const gpu_boxsize &box, int Nx, int Ny, int Nz, int order, float kappa, float rcut_ewald)
{
  //CUDA:
  hipMalloc((void**)&(es_data.GPU_rho_real_space), sizeof(hipfftComplex)*Nx*Ny*Nz);
  hipMalloc((void**)&(es_data.GPU_green_hat), sizeof(float)*Nx*Ny*Nz);
  hipMalloc((void**)&(es_data.GPU_k_vec), sizeof(float3)*Nx*Ny*Nz);
  hipMalloc((void**)&(es_data.GPU_E_x), sizeof(hipfftComplex)*Nx*Ny*Nz);
  hipMalloc((void**)&(es_data.GPU_E_y), sizeof(hipfftComplex)*Nx*Ny*Nz);
  hipMalloc((void**)&(es_data.GPU_E_z), sizeof(hipfftComplex)*Nx*Ny*Nz);
  hipMalloc((void**)&(es_data.GPU_field), sizeof(float3)*Nx*Ny*Nz); 
  hipMalloc((void**)&(es_data.vg), sizeof(float3)*Nx*Ny*Nz);
  hipMalloc((void**)&(es_data.cuda_thermo_quantities), sizeof(float3));
  
  
  es_data.CPU_rho_coeff = (float*)malloc(order * (2*order+1) * sizeof(float));
  compute_rho_coeff(order, es_data.CPU_rho_coeff);
  hipMemcpyToSymbol(HIP_SYMBOL(weight_factors), &(es_data.CPU_rho_coeff[0]), order * (2*order+1) * sizeof(float));
  
  hipfftPlan3d(&es_data.plan, Nx, Ny, Nz, HIPFFT_C2C);
  
  
  //copy information to CPU here (stupid way, but works);
  struct gpu_boxsize CPU_box;
  struct gpu_boxsize *GPU_COPY_BOX;
  hipMalloc((void**)&GPU_COPY_BOX, sizeof(struct gpu_boxsize));

  copy_data_kernel <<< 1,1 >>> (box, GPU_COPY_BOX);
      
  hipMemcpy(&CPU_box, GPU_COPY_BOX, sizeof(struct gpu_boxsize), hipMemcpyDeviceToHost);
      
  hipFree(GPU_COPY_BOX);
      
  /* set up for a rectangular box */
  tensor_t lattice_vectors;
  lattice_vectors.xx = CPU_box.Lx;
  lattice_vectors.yx = 0.0;
  lattice_vectors.zx = 0.0;

  lattice_vectors.xy = 0.0;
  lattice_vectors.yy = CPU_box.Ly;
  lattice_vectors.zy = 0.0;

  lattice_vectors.xz = 0.0;
  lattice_vectors.yz = 0.0;
  lattice_vectors.zz = CPU_box.Lz;
       
  tensor_t inverse_lattice_vectors;
  inverse(&inverse_lattice_vectors, &lattice_vectors);
   
  tensor_t reciprocal_lattice_vectors;
  transpose(&reciprocal_lattice_vectors, &inverse_lattice_vectors);
  scalar_multiply(&reciprocal_lattice_vectors, 2*M_PI);
  
  float3* kvec_array = (float3*)malloc(Nx * Ny * Nz * sizeof(float3)); 
  int ix, iy, iz, kper, lper, mper, k, l, m;
   
  for (ix = 0; ix < Nx; ix++) {
    struct coord j;
    j.x = ix > Nx/2 ? ix - Nx : ix;
    for (iy = 0; iy < Ny; iy++) {
      j.y = iy > Ny/2 ? iy - Ny : iy;
      for (iz = 0; iz < Nz; iz++) {
	j.z = iz > Nz/2 ? iz - Nz : iz;
	float3 kvec;
	multiply(&kvec, &reciprocal_lattice_vectors, &j);
	kvec_array[iz + Nz * (iy + Ny * ix)] = kvec;
      }
    }
  }
     
  float3* cpu_vg = (float3 *)malloc(sizeof(float3)*Nx*Ny*Nz); 

  for(int x = 0; x < Nx; x++)
    {
      for(int y = 0; y < Ny; y++)
	{
	  for(int z = 0; z < Nz; z++)
	    {
	      float3 kvec = kvec_array[z + Nz * (y + Ny * x)];
	      float sqk =  kvec.x*kvec.x;
	      sqk += kvec.y*kvec.y;
	      sqk += kvec.z*kvec.z;
	
	      if (sqk == 0.0) 
		{
		  cpu_vg[z + Nz * (y + Ny * x)].x = 0.0f;
		  cpu_vg[z + Nz * (y + Ny * x)].y = 0.0f;
		  cpu_vg[z + Nz * (y + Ny * x)].z = 0.0f;
		}
	      else
		{
		  float vterm = -2.0 * (1.0/sqk + 0.25/(kappa*kappa));
		  cpu_vg[z + Nz * (y + Ny * x)].x =  1.0 + vterm*kvec.x*kvec.x;
		  cpu_vg[z + Nz * (y + Ny * x)].y =  1.0 + vterm*kvec.y*kvec.y;
		  cpu_vg[z + Nz * (y + Ny * x)].z =  1.0 + vterm*kvec.z*kvec.z;
		}
	    } 
	} 
    }
    
  float* green_hat = (float*)malloc(Nx * Ny * Nz * sizeof(float)); 
   
  int assignment_order = order;
  float snx, sny, snz, snx2, sny2, snz2;
  float argx, argy, argz, wx, wy, wz, sx, sy, sz, qx, qy, qz;
  float sum1, dot1, dot2;
  float numerator, denominator, sqk;

  float unitkx = (2.0*M_PI/CPU_box.Lx);
  float unitky = (2.0*M_PI/CPU_box.Ly);
  float unitkz = (2.0*M_PI/CPU_box.Lz);
   
    
  float xprd = CPU_box.Lx; 
  float yprd = CPU_box.Ly; 
  float zprd_slab = CPU_box.Lz; 
    
  float form = 1.0;
	
  gf_b = (float *)malloc(assignment_order*sizeof(float)); 
  compute_gf_denom(assignment_order);
	
#define EPS_HOC 1.0e-7

  float temp = floor(((kappa*xprd/(M_PI*Nx)) * 
		      pow(-log(EPS_HOC),0.25)));
  int nbx = (int)temp;

  temp = floor(((kappa*yprd/(M_PI*Nx)) * 
		pow(-log(EPS_HOC),0.25)));
  int nby = (int)temp;

  temp =  floor(((kappa*zprd_slab/(M_PI*Nz)) * 
		 pow(-log(EPS_HOC),0.25)));
  int nbz = (int)temp;

    
  for (m = 0; m < Nz; m++) {
    mper = m - Nz*(2*m/Nz);
    snz = sin(0.5*unitkz*mper*zprd_slab/Nz);
    snz2 = snz*snz;

    for (l = 0; l < Ny; l++) {
      lper = l - Ny*(2*l/Ny);
      sny = sin(0.5*unitky*lper*yprd/Ny);
      sny2 = sny*sny;

      for (k = 0; k < Nx; k++) {
	kper = k - Nx*(2*k/Nx);
	snx = sin(0.5*unitkx*kper*xprd/Nx);
	snx2 = snx*snx;
      
	sqk = pow(unitkx*kper,2.0f) + pow(unitky*lper,2.0f) + 
	  pow(unitkz*mper,2.0f);

	if (sqk != 0.0) {
	  numerator = form*12.5663706/sqk;
	  denominator = gf_denom(snx2,sny2,snz2,assignment_order);  
	  sum1 = 0.0;
	  for (ix = -nbx; ix <= nbx; ix++) {
	    qx = unitkx*(kper+(float)(Nx*ix));
	    sx = exp(-.25*pow(qx/kappa,2.0f));
	    wx = 1.0;
	    argx = 0.5*qx*xprd/(float)Nx;
	    if (argx != 0.0) wx = pow(sin(argx)/argx,assignment_order);
	    for (iy = -nby; iy <= nby; iy++) {
	      qy = unitky*(lper+(float)(Ny*iy));
	      sy = exp(-.25*pow(qy/kappa,2.0f));
	      wy = 1.0;
	      argy = 0.5*qy*yprd/(float)Ny;
	      if (argy != 0.0) wy = pow(sin(argy)/argy,assignment_order);
	      for (iz = -nbz; iz <= nbz; iz++) {
		qz = unitkz*(mper+(float)(Nz*iz));
		sz = exp(-.25*pow(qz/kappa,2.0f));
		wz = 1.0;
		argz = 0.5*qz*zprd_slab/(float)Nz;
		if (argz != 0.0) wz = pow(sin(argz)/argz,assignment_order);

		dot1 = unitkx*kper*qx + unitky*lper*qy + unitkz*mper*qz;
		dot2 = qx*qx+qy*qy+qz*qz;
		sum1 += (dot1/dot2) * sx*sy*sz * pow(wx*wy*wz,2.0f);
	      }
	    }
	  }
	  green_hat[m + Nz * (l + Ny * k)] = numerator*sum1/denominator;
	} else green_hat[m + Nz * (l + Ny * k)] = 0.0;
      }
    }
  }
   
 
  hipMemcpy(es_data.GPU_green_hat, green_hat, Nx * Ny * Nz * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(es_data.GPU_k_vec, kvec_array, Nx * Ny * Nz * sizeof(float3), hipMemcpyHostToDevice);  

  hipMemcpy(es_data.vg, cpu_vg, Nx * Ny * Nz * sizeof(float3), hipMemcpyHostToDevice);  
  free(cpu_vg);
  free(gf_b);
  free(green_hat);
  free(kvec_array);
  
  float scale = 1.0f/((float)(Nx * Ny * Nz));
  es_data.CPU_energy_virial_factor = 0.5 * CPU_box.Lx * CPU_box.Ly * CPU_box.Lz * scale * scale;
}


__device__ inline void atomicFloatAdd(float* address, float value)
{
  float old = value;  
  float new_old;

  do
    {
      new_old = atomicExch(address, 0.0f);
      new_old += old;
    }
  while ((old = atomicExch(address, new_old))!=0.0f);
}


__global__ void combined_green_e_kernel(hipfftComplex* E_x, hipfftComplex* E_y, hipfftComplex* E_z, float3* k_vec, hipfftComplex* rho, int Nx, int Ny, int Nz, float* green_function)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(tid < Nx * Ny * Nz)
    {
      float3 k_vec_local = k_vec[tid];
      hipfftComplex E_x_local, E_y_local, E_z_local;
      float scale_times_green = green_function[tid] / ((float)(Nx*Ny*Nz));
      hipfftComplex rho_local = rho[tid];
    
      rho[tid] = make_float2(0.0f,0.0f);
    
      rho_local.x *= scale_times_green;
      rho_local.y *= scale_times_green;
      
      E_x_local.x = k_vec_local.x * rho_local.y;
      E_x_local.y = -k_vec_local.x * rho_local.x;
    
      E_y_local.x = k_vec_local.y * rho_local.y;
      E_y_local.y = -k_vec_local.y * rho_local.x;
    
      E_z_local.x = k_vec_local.z * rho_local.y;
      E_z_local.y = -k_vec_local.z * rho_local.x;
    
    
      E_x[tid] = E_x_local;
      E_y[tid] = E_y_local;
      E_z[tid] = E_z_local;   
    }
}


__global__ void set_to_zero(hipfftComplex* array, int Nx, int Ny, int Nz)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
 
  if(tid < Nx * Ny * Nz)
    {
      hipfftComplex Zero = make_float2(0.0f,0.0f);
      array[tid] = Zero;
    }
}

__device__ inline void AddToGridpoint(int X, int Y, int Z, hipfftComplex* array, float value, int Ny, int Nz)
{
  atomicFloatAdd(&array[Z + Nz * (Y + Ny * X)].x, value);
}

__device__ void compute_rho1d(float* rho1d, float dx, float dy, float dz, int interpolation_order)
{
  int k, l, k_order;
  int order = interpolation_order;

  k_order = -((1-order)/2);

    
  for (k = (1-order)/2; k <= order/2; k++) {
    rho1d[k+k_order + 0 * MaxOrder] = 0.0;
    rho1d[k+k_order + 1 * MaxOrder] = 0.0;
    rho1d[k+k_order + 2 * MaxOrder] = 0.0;
    for (l = order-1; l >= 0; l--) {
      rho1d[k+k_order + 0 * MaxOrder] = weight_factors[k+k_order + l*(2*order +1)] + rho1d[k+k_order + 0 * MaxOrder]*dx;
      rho1d[k+k_order + 1 * MaxOrder] = weight_factors[k+k_order + l*(2*order +1)] + rho1d[k+k_order + 1 * MaxOrder]*dy;
      rho1d[k+k_order + 2 * MaxOrder] = weight_factors[k+k_order + l*(2*order +1)] + rho1d[k+k_order + 2 * MaxOrder]*dz;
    }
  }
    
}

__device__ float get_rho1d(float delta, int interpolation_order, int y)
{
  int l, order = interpolation_order;

  float result;
    
  result = 0.0f;
  for (l = order-1; l >= 0; l--) {
    result = weight_factors[y + l*(2*order +1)] + result * delta;
  }

  return result;
}


__global__ void assign_charges_to_grid_kernel(gpu_pdata_arrays pdata, gpu_boxsize box, hipfftComplex *rho_real_space, int Nx, int Ny, int Nz, int order)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < pdata.N)
    {
      //get particle information
      float qi = tex1Dfetch(pdata_charge_tex, idx);
      if(qi != 0.0) {
	float4 posi = tex1Dfetch(pdata_pos_tex, idx);
	//calculate dx, dy, dz for the charge density grid:
	float box_dx = box.Lx / ((float)Nx);
	float box_dy = box.Ly / ((float)Ny);
	float box_dz = box.Lz / ((float)Nz);
    
        
	//normalize position to gridsize:
	posi.x += box.Lx / 2.0f;
	posi.y += box.Ly / 2.0f;
	posi.z += box.Lz / 2.0f;
   
	posi.x /= box_dx;
	posi.y /= box_dy;
	posi.z /= box_dz;
    
    
	float shift, shiftone, x0, y0, z0, dx, dy, dz;
	int nlower, nupper, mx, my, mz, nxi, nyi, nzi; 
    
	nlower = -(order-1)/2;
	nupper = order/2;
    
	if (order % 2) 
	  {
	    shift =0.5;
	    shiftone = 0.0;
	  }
	else 
	  {
	    shift = 0.0;
	    shiftone = 0.5;
	  }
        
    
	nxi = __float2int_rd(posi.x + shift);
	nyi = __float2int_rd(posi.y + shift);
	nzi = __float2int_rd(posi.z + shift);
    
	dx = shiftone+(float)nxi-posi.x;
	dy = shiftone+(float)nyi-posi.y;
	dz = shiftone+(float)nzi-posi.z;
    
	int n,m,l;
    
	x0 = qi / (box_dx*box_dy*box_dz);
	for (n = nlower; n <= nupper; n++) {
	  mx = n+nxi;
	  if(mx >= Nx) mx -= Nx;
	  if(mx < 0)  mx += Nx;
	  y0 = x0*get_rho1d(dx, order, n-nlower);
	  for (m = nlower; m <= nupper; m++) {
	    my = m+nyi;
	    if(my >= Ny) my -= Ny;
	    if(my < 0)  my += Ny;
	    z0 = y0*get_rho1d(dy, order, m-nlower);
	    for (l = nlower; l <= nupper; l++) {
	      mz = l+nzi;
	      if(mz >= Nz) mz -= Nz;
	      if(mz < 0)  mz += Nz;
	      AddToGridpoint(mx, my, mz, rho_real_space, z0*get_rho1d(dz, order, l-nlower), Ny, Nz);
	    }
	  }
	}
      }
    }
}

__global__ void set_gpu_field_kernel(hipfftComplex* E_x, hipfftComplex* E_y, hipfftComplex* E_z, float3* Electric_field, int Nx, int Ny, int Nz)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < Nx * Ny * Nz)
    {
      float3 local_field;
      local_field.x = E_x[tid].x;
      local_field.y = E_y[tid].x;
      local_field.z = E_z[tid].x;
      
      Electric_field[tid] = local_field;
    }
}

__global__ void calculate_forces_kernel(gpu_force_data_arrays force_data, gpu_pdata_arrays pdata, gpu_boxsize box, float3* Electric_field, int Nx, int Ny, int Nz, int order)
{  
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < pdata.N)
    {
      //get particle information
      float qi = tex1Dfetch(pdata_charge_tex, idx);
      if(qi != 0.0) {
	float4 posi = tex1Dfetch(pdata_pos_tex, idx);
    
	//calculate dx, dy, dz for the charge density grid:
	float box_dx = box.Lx / ((float)Nx);
	float box_dy = box.Ly / ((float)Ny);
	float box_dz = box.Lz / ((float)Nz);
    
	//normalize position to gridsize:
	posi.x += box.Lx / 2.0f;
	posi.y += box.Ly / 2.0f;
	posi.z += box.Lz / 2.0f;
   
	posi.x /= box_dx;
	posi.y /= box_dy;
	posi.z /= box_dz;
    
	float shift, shiftone, x0, y0, z0, dx, dy, dz;
	int nlower, nupper, mx, my, mz, nxi, nyi, nzi; 
    
	nlower = -(order-1)/2;
	nupper = order/2;
    
	float4 local_force = force_data.force[idx];

	if(order % 2) 
	  {
	    shift =0.5;
	    shiftone = 0.0;
	  }
	else 
	  {
	    shift = 0.0;
	    shiftone = 0.5;
	  }
    
    
	nxi = __float2int_rd(posi.x + shift);
	nyi = __float2int_rd(posi.y + shift);
	nzi = __float2int_rd(posi.z + shift);
    
	dx = shiftone+(float)nxi-posi.x;
	dy = shiftone+(float)nyi-posi.y;
	dz = shiftone+(float)nzi-posi.z;
 	int n,m,l;
    
  	for (n = nlower; n <= nupper; n++) {
	  mx = n+nxi;
	  if(mx >= Nx) mx -= Nx;
	  if(mx < 0)  mx += Nx;
	  x0 = get_rho1d(dx, order, n-nlower);
	  for (m = nlower; m <= nupper; m++) {
	    my = m+nyi;
	    if(my >= Ny) my -= Ny;
	    if(my < 0)  my += Ny;
	    y0 = x0*get_rho1d(dy, order, m-nlower);
	    for (l = nlower; l <= nupper; l++) {
	      mz = l+nzi;
	      if(mz >= Nz) mz -= Nz;
	      if(mz < 0)  mz += Nz;
	      z0 = y0*get_rho1d(dz, order, l-nlower);
	      float3 local_field = Electric_field[mz + Nz * (my + Ny * mx)];
	      local_force.x += qi*z0*local_field.x;
	      local_force.y += qi*z0*local_field.y;
	      local_force.z += qi*z0*local_field.z;
	    }
	  }
	}
    
	force_data.force[idx] = local_force;
      }
    }
} 

__global__ void calculate_thermo_quantities_kernel(hipfftComplex* rho, float* green_function, float3* GPU_virial_energy, float3* vg, int Nx, int Ny, int Nz)
{
  int threadx = blockIdx.x * blockDim.x + threadIdx.x;
  int thready = blockIdx.y * blockDim.y + threadIdx.y;

  if((threadx < Nx) && (thready < Ny))
    {
      float2 local_GPU_virial_energy = make_float2(0.0f,0.0f);
      float3 local_vg;
      float local_green, green_times_rho_square;
      hipfftComplex rho_local;
      for(int z = 0; z < Nz; z++)
	{
	  local_vg = vg[z + Nz * (thready + Ny * threadx)];
	  local_green = green_function[z + Nz * (thready + Ny * threadx)];
	  rho_local = rho[z + Nz * (thready + Ny * threadx)];
	
	  green_times_rho_square = local_green * (rho_local.x * rho_local.x + rho_local.y * rho_local.y);
	  local_GPU_virial_energy.x += green_times_rho_square * (local_vg.x + local_vg.y + local_vg.z);
	  local_GPU_virial_energy.y += green_times_rho_square ;
	}
  
      atomicFloatAdd(&GPU_virial_energy[0].x, local_GPU_virial_energy.x);
      atomicFloatAdd(&GPU_virial_energy[0].y, local_GPU_virial_energy.y);

    }
}

__global__ void get_charge(gpu_pdata_arrays pdata, float3 *GPU_virial_energy)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < pdata.N) {
    float qi = tex1Dfetch(pdata_charge_tex, idx);
    atomicFloatAdd(&GPU_virial_energy[0].x, qi);
  }
}

__global__ void get_charge_squared(gpu_pdata_arrays pdata, float3 *GPU_virial_energy)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < pdata.N) {
    float qi = tex1Dfetch(pdata_charge_tex, idx);
    atomicFloatAdd(&GPU_virial_energy[0].z, qi*qi);
  }

}

float3 calculate_thermo_quantities(const gpu_pdata_arrays &pdata, const gpu_boxsize &box)
{
  if(es_data.electrostatics_allocation_bool)
    {
      //kernel calling parameters for all grid dependent kernels
      int new_blockzise = 512;
      int new_gridsize = es_data.Nx*es_data.Ny*es_data.Nz / new_blockzise + 1;
      
      // setup the grid to run the kernel
      int blocksize = 512;
      dim3 grid( pdata.N / blocksize + 1, 1, 1);
      dim3 threads(blocksize, 1, 1);

      // bind the position texture
      pdata_pos_tex.normalized = false;
      pdata_pos_tex.filterMode = hipFilterModePoint;
      hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);

      // bind the charge texture
      pdata_charge_tex.normalized = false;
      pdata_charge_tex.filterMode = hipFilterModePoint;
      error = hipBindTexture(0, pdata_charge_tex, pdata.charge, sizeof(float) * pdata.N);
	  
      //assign the charge density to the gridpoints
      assign_charges_to_grid_kernel <<< grid, threads >>> (pdata, box, es_data.GPU_rho_real_space, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order);
      hipDeviceSynchronize();    
    
      //call the forward FFT for the charge density
      hipfftExecC2C(es_data.plan, es_data.GPU_rho_real_space, es_data.GPU_rho_real_space, HIPFFT_FORWARD);
      hipDeviceSynchronize();
	  
      //calculate the virial and energy:  
      float3 CPU_virial_energy = make_float3(0.0f, 0.0f, 0.0f);
      hipMemcpy(es_data.cuda_thermo_quantities, &CPU_virial_energy, sizeof(float3), hipMemcpyHostToDevice);  
      get_charge_squared <<< grid, threads >>> (pdata, es_data.cuda_thermo_quantities);
     
      dim3 thermo_block(8,8,1);
      dim3 thermo_grid(es_data.Nx/thermo_block.x, es_data.Ny/thermo_block.y, 1);
      calculate_thermo_quantities_kernel <<< thermo_grid, thermo_block >>> (es_data.GPU_rho_real_space, es_data.GPU_green_hat, es_data.cuda_thermo_quantities, es_data.vg, es_data.Nx, es_data.Ny, es_data.Nz);
	
      //copy to CPU:
      hipMemcpy(&CPU_virial_energy, es_data.cuda_thermo_quantities, sizeof(float3), hipMemcpyDeviceToHost);

      struct gpu_boxsize CPU_box;
      struct gpu_boxsize *GPU_COPY_BOX;
      hipMalloc((void**)&GPU_COPY_BOX, sizeof(struct gpu_boxsize));

      copy_data_kernel <<< 1,1 >>> (box, GPU_COPY_BOX);
      
      hipMemcpy(&CPU_box, GPU_COPY_BOX, sizeof(struct gpu_boxsize), hipMemcpyDeviceToHost);
      
      hipFree(GPU_COPY_BOX);
      
      CPU_virial_energy.x *= es_data.CPU_energy_virial_factor / (3.0f * CPU_box.Lx * CPU_box.Ly * CPU_box.Lz);
      CPU_virial_energy.y *= es_data.CPU_energy_virial_factor;
	
      CPU_virial_energy.y -= CPU_virial_energy.z * es_data.kappa / sqrt(M_PI);

      set_to_zero <<< new_gridsize , new_blockzise >>> (es_data.GPU_rho_real_space, es_data.Nx, es_data.Ny, es_data.Nz);
      hipDeviceSynchronize();  
      return CPU_virial_energy;
	
	
    }
  else 
    return make_float3(0.0f, 0.0f, 0.0f);
}


void electrostatics_calculation(const gpu_force_data_arrays& force_data, const gpu_pdata_arrays &pdata, const gpu_boxsize &box, const float3 *d_params, const float *d_rcutsq)
{
  //first time allocation of memory-------------------------------------
  int blocksize = 512;
  dim3 grid( pdata.N / blocksize + 1, 1, 1);
  dim3 threads(blocksize, 1, 1);

  if(!es_data.electrostatics_allocation_bool)
    {
      float3 cpu_params, *GPU_charge;

      hipMalloc((void**)&(GPU_charge), sizeof(float3));

      hipMemcpy(&cpu_params, d_params, sizeof(float3), hipMemcpyDeviceToHost);
      printf("kappa = %g grid = %d order = %d\n",cpu_params.x, (int)cpu_params.y, (int)cpu_params.z);

      //Store values:
      es_data.Nx = cpu_params.y;
      es_data.Ny = es_data.Nx;
      es_data.Nz = es_data.Nx;

      es_data.interpolation_order = (int)cpu_params.z;
      int interpolation_order = es_data.interpolation_order;
      es_data.show_virial_flag = 0;

      int N = es_data.Nx;

      es_data.electrostatics_allocation_bool = 1;
      if(!(N == 2)&& !(N == 4)&& !(N == 8)&& !(N == 16)&& !(N == 32)&& !(N == 64)&& !(N == 128)&& !(N == 256)&& !(N == 512)&& !(N == 1024))
	{
	  printf("\n\n ------ ATTENTION gridsize should be a power of 2 ------ \n\n");
	}
      if (interpolation_order * (2*interpolation_order +1) > CONSTANT_SIZE)
	{
	  printf("interpolation order too high, doesn't fit into constant array\n");
	  exit(1);
	}
      if (interpolation_order > MaxOrder)
	{
	  printf("interpolation order too high\n");
	  exit(1);
	}
            
      float cpu_rcutsq;
      hipMemcpy(&cpu_rcutsq, d_rcutsq, sizeof(float), hipMemcpyDeviceToHost);
      
      es_data.r_cutoff = cpu_rcutsq;
      es_data.kappa = cpu_params.x;
      
      electrostatics_allocation(pdata, box, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order, es_data.kappa, es_data.r_cutoff);

      // bind the charge texture
      pdata_charge_tex.normalized = false;
      pdata_charge_tex.filterMode = hipFilterModePoint;
      hipError_t error = hipBindTexture(0, pdata_charge_tex, pdata.charge, sizeof(float) * pdata.N);
      struct gpu_boxsize CPU_box;
      struct gpu_boxsize *GPU_COPY_BOX;
      hipMalloc((void**)&GPU_COPY_BOX, sizeof(struct gpu_boxsize));

      copy_data_kernel <<< 1,1 >>> (box, GPU_COPY_BOX);
      hipMemcpy(&CPU_box, GPU_COPY_BOX, sizeof(struct gpu_boxsize), hipMemcpyDeviceToHost);

      float3 CPU_charge = make_float3(0.0f, 0.0f, 0.0f);
      hipMemcpy(GPU_charge, &CPU_charge, sizeof(float3), hipMemcpyHostToDevice);  
      get_charge <<< grid, threads >>> (pdata, GPU_charge);
      get_charge_squared <<< grid, threads >>> (pdata, GPU_charge);
      hipMemcpy(&CPU_charge, GPU_charge, sizeof(float3), hipMemcpyDeviceToHost);

      float hx =  CPU_box.Lx/es_data.Nx;
      float hy =  CPU_box.Ly/es_data.Ny;
      float hz =  CPU_box.Lz/es_data.Nz;

      float lprx = rms(hx, CPU_box.Lx, pdata.N, interpolation_order, es_data.kappa, CPU_charge.z); 
      float lpry = rms(hy, CPU_box.Lz, pdata.N, interpolation_order, es_data.kappa, CPU_charge.z);
      float lprz = rms(hz, CPU_box.Lz, pdata.N, interpolation_order, es_data.kappa, CPU_charge.z);
      float lpr = sqrt(lprx*lprx + lpry*lpry + lprz*lprz) / sqrt(3.0);
      float spr = 2.0*CPU_charge.z*exp(-es_data.kappa*es_data.kappa*cpu_rcutsq) / sqrt(pdata.N*sqrt(cpu_rcutsq)*CPU_box.Lx*CPU_box.Ly*CPU_box.Lz);
     
      double RMS_error = MAX(lpr,spr);
      if(RMS_error > 0.1) {
	printf("!!!!!!!\n!!!!!!!\n!!!!!!!\nWARNING RMS error of %g is probably too high\n!!!!!!!\n!!!!!!!\n!!!!!!!\n", RMS_error);
      }
      else{
	printf("RMS error: %g\n", RMS_error);
      }
 
      if(CPU_charge.x > 0.0001 || CPU_charge.x < -0.0001) printf("WARNING system in not neutral, the net charge is %g\n", CPU_charge.x);
     
      
      printf("allocation for electrostatics done... \n");
      int new_blockzise = 512;
      int new_gridsize = es_data.Nx*es_data.Ny*es_data.Nz / new_blockzise + 1;
      //only for the first time needed, next time it is done in function new_combined_green_e_kernel
      set_to_zero <<< new_gridsize , new_blockzise >>> (es_data.GPU_rho_real_space, es_data.Nx, es_data.Ny, es_data.Nz);
      hipDeviceSynchronize();  
     
    }

  //kernel calling parameters for all grid dependent kernels
  int new_blockzise = 512;
  int new_gridsize = es_data.Nx*es_data.Ny*es_data.Nz / new_blockzise + 1;
    
  // setup the grid to run the particle kernel
    
    
  // bind the position texture
  pdata_pos_tex.normalized = false;
  pdata_pos_tex.filterMode = hipFilterModePoint;
  hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);

  // bind the charge texture
  pdata_charge_tex.normalized = false;
  pdata_charge_tex.filterMode = hipFilterModePoint;
  error = hipBindTexture(0, pdata_charge_tex, pdata.charge, sizeof(float) * pdata.N);
     
  //assign the charge density to the gridpoints
  assign_charges_to_grid_kernel <<< grid, threads >>> (pdata, box, es_data.GPU_rho_real_space, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order);
  hipDeviceSynchronize();    
     
  //call the forward FFT for the charge density
  hipfftExecC2C(es_data.plan, es_data.GPU_rho_real_space, es_data.GPU_rho_real_space, HIPFFT_FORWARD);
  hipDeviceSynchronize();
    
  combined_green_e_kernel <<< new_gridsize, new_blockzise >>> (es_data.GPU_E_x, es_data.GPU_E_y, es_data.GPU_E_z, es_data.GPU_k_vec, es_data.GPU_rho_real_space,  es_data.Nx, es_data.Ny, es_data.Nz, es_data.GPU_green_hat);
       
  //backtransform field:
  hipfftExecC2C(es_data.plan, es_data.GPU_E_x, es_data.GPU_E_x, HIPFFT_BACKWARD);
  hipfftExecC2C(es_data.plan, es_data.GPU_E_y, es_data.GPU_E_y, HIPFFT_BACKWARD);
  hipfftExecC2C(es_data.plan, es_data.GPU_E_z, es_data.GPU_E_z, HIPFFT_BACKWARD);
    
  //put field into float3 array
  set_gpu_field_kernel <<< new_gridsize, new_blockzise >>> (es_data.GPU_E_x, es_data.GPU_E_y, es_data.GPU_E_z, es_data.GPU_field, es_data.Nx, es_data.Ny, es_data.Nz);
  hipDeviceSynchronize();
  //calculate forces on particles:
  calculate_forces_kernel <<< grid, threads >>>(force_data, pdata, box, es_data.GPU_field, es_data.Nx, es_data.Ny, es_data.Nz, es_data.interpolation_order);
       
}
