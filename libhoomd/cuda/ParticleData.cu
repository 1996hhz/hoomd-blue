#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "ParticleData.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file ParticleData.cu
    \brief Contains GPU kernel code and data structure functions used by ParticleData
*/

//! Kernel for un-interleaving float4 input into float output
/*! \param d_out Device pointer to write un-interleaved output
    \param d_in Device pointer to read interleaved input
    \param N Number of elements in input
    \param pitch Spacing of arrays through the output

    \pre N/block_size + 1 blocks are run on the device
*/
extern "C" __global__ void uninterleave_float4_kernel(float *d_out, float4 *d_in, int N, int pitch)
    {
    int pidx  = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (pidx < N)
        {
        float4 in = d_in[pidx];
        
        d_out[pidx] = in.x;
        d_out[pidx+pitch] = in.y;
        d_out[pidx+pitch+pitch] = in.z;
        d_out[pidx+pitch+pitch+pitch] = in.w;
        }
    }


/*! The most efficient data storage on the device is to put x,y,z,type into a float4
    data structure. The most efficient storage on the CPU is x,y,z,type each as
    separate arrays. Translation between the two is best done on the device, and
    memory transfers done with one big hipMemcpy. This function, and its sister
    gpu_interleave_float4() perform the transformation between a float* with x,y,z,type
    packed non-interleaved to a float4* storing the same values interleaved.

    Performance is best when pitch is a multiple of 64.

    \param d_out Device pointer to write output to
    \param d_in Device pointer to read input from
    \param N Number of elements to interleave
    \param pitch Spacing between \c x[0] and \c y[0] in \a d_out

    \post A code snipped best describes what is done:
    \verbatim
    d_out[i] = d_in[i].x
    d_out[i+pitch] = d_in[i].y
    d_out[i+pitch*2] = d_in[i].z
    d_out[i+pitch*3] = d_in[i].w
    \endverbatim

    \returns Any error code from the kernel call retrieved via hipGetLastError()
    \note Always returns hipSuccess in release builds for performance reasons
*/
hipError_t gpu_uninterleave_float4(float *d_out, float4 *d_in, int N, int pitch)
    {
    assert(pitch >= N);
    assert(d_out);
    assert(d_in);
    assert(N > 0);
    
    const int M = 64;
    uninterleave_float4_kernel<<< N/M+1, M >>>(d_out, d_in, N, pitch);
    
    return hipSuccess;
    }

//! Kernel for interleaving float input into float4 output
/*! \param d_out Device pointer to write interleaved output
    \param d_in Device pointer to read non-interleaved input
    \param N Number of elements in output
    \param pitch Spacing of arrays through the input

    \pre N/block_size + 1 blocks are run on the device
*/
extern "C" __global__ void interleave_float4_kernel(float4 *d_out, float *d_in, int N, int pitch)
    {
    int pidx  = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (pidx < N)
        {
        float x = d_in[pidx];
        float y = d_in[pidx+pitch];
        float z = d_in[pidx+pitch+pitch];
        float w = d_in[pidx+pitch+pitch+pitch];
        
        float4 out;
        out.x = x;
        out.y = y;
        out.z = z;
        out.w = w;
        d_out[pidx] = out;
        }
    }

/*! See gpu_uninterleave_float4() for details.
    \param d_out Device pointer to write output to
    \param d_in Device pointer to read input from
    \param N Number of elements to interleave
    \param pitch Spacing between \c x[0] and \c y[0] in \a d_in

    \returns Any error code from the kernel call retrieved via hipGetLastError()
    \note Always returns hipSuccess in release builds for performance reasons
*/
hipError_t gpu_interleave_float4(float4 *d_out, float *d_in, int N, int pitch)
    {
    assert(pitch >= N);
    assert(d_out);
    assert(d_in);
    assert(N > 0);
    
    const int M = 64;
    interleave_float4_kernel<<< N/M+1, M >>>(d_out, d_in, N, pitch);
    
    return hipSuccess;
    }


