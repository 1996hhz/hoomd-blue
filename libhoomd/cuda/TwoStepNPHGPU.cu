#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: jglaser

#include "TwoStepNPHGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file TwoStepNPHGPU.cu
    \brief Defines GPU kernel code for NPT integration on the GPU. Used by TwoStepNPTGPU.
*/

/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param L_old box lengths at beginning of time step
    \param L_halfstep box lengths at t+deltaT/2
    \param L_final box lengths at t+deltaT
    \param deltaT Time to advance (for one full step)
*/
extern "C" __global__
void gpu_nph_step_one_kernel(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             float3 L_old,
                             float3 L_halfstep,
                             float3 L_final,
                             float deltaT)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // propagate velocity from t to t+1/2*deltaT and position from t to t+deltaT
    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // fetch particle position
        float4 pos = d_pos[idx];

        float px = pos.x;
        float py = pos.y;
        float pz = pos.z;
        float pw = pos.w;

        // fetch particle velocity and acceleration
        float4 vel = d_vel[idx];
        Scalar3 accel = d_accel[idx];

        float4 veltmp;

        // propagate velocity by half a time step and position by the full time step
        // according to the Nose-Hoover barostat
        veltmp.x = vel.x + (1.0f/2.0f) * deltaT*accel.x;
        px = (L_final.x/L_old.x) *(px + veltmp.x*deltaT*L_old.x*L_old.x/L_halfstep.x/L_halfstep.x);
        vel.x = L_old.x/L_final.x*veltmp.x;

        veltmp.y = vel.y + (1.0f/2.0f) * deltaT*accel.y;
        py = (L_final.y/L_old.y) *(py + veltmp.y*deltaT*L_old.y*L_old.y/L_halfstep.y/L_halfstep.y);
        vel.y = L_old.y/L_final.y*veltmp.y;

        veltmp.z = vel.z + (1.0f/2.0f) * deltaT*accel.z;
        pz = (L_final.z/L_old.z) *(pz + veltmp.z*deltaT*L_old.z*L_old.z/L_halfstep.z/L_halfstep.z);
        vel.z = L_old.z/L_final.z*veltmp.z;

        Scalar4 pos2;
        pos2.x = px;
        pos2.y = py;
        pos2.z = pz;
        pos2.w = pw;

        // write out the results
        d_pos[idx] = pos2;
        d_vel[idx] = vel;
        }
    }

/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param L_old box lengths at beginning of time step
    \param L_halfstep box lengths at t+deltaT/2
    \param L_final box box lengths at t+deltaT
    \param deltaT Time to move forward in one whole step

    This is just a kernel driver for gpu_nph_step_one_kernel(). See it for more details.
*/
hipError_t gpu_nph_step_one(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             float3 L_old,
                             float3 L_halfstep,
                             float3 L_final,
                             float deltaT)
    {
    // setup the grid to run the kernel
    unsigned int block_size = 256;
    dim3 grid( (group_size / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_nph_step_one_kernel<<< grid, threads >>>(d_pos,
                                                 d_vel,
                                                 d_accel,
                                                 d_group_members,
                                                 group_size,
                                                 L_old,
                                                 L_halfstep,
                                                 L_final,
                                                 deltaT);

    return hipSuccess;
    }

/*! \param N number of particles
    \param d_pos array of particle positions
    \param d_image array of particle images
    \param box The new box the particles where the particles now reside

    Wrap particles into new box
*/
extern "C" __global__
void gpu_nph_wrap_particles_kernel(const unsigned int N,
                             Scalar4 *d_pos,
                             int3 *d_image,
                             gpu_boxsize box)
    {
    // determine which particle this thread works on
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // scale ALL particles in the box
    if (idx < N)
        {
        // fetch particle position
        float4 pos = d_pos[idx];

        float px = pos.x;
        float py = pos.y;
        float pz = pos.z;
        float pw = pos.w;

        // read in the image flags
        int3 image = d_image[idx];

        // fix periodic boundary conditions
        float x_shift = rintf(px * box.Lxinv);
        px -= box.Lx * x_shift;
        image.x += (int)x_shift;

        float y_shift = rintf(py * box.Lyinv);
        py -= box.Ly * y_shift;
        image.y += (int)y_shift;

        float z_shift = rintf(pz * box.Lzinv);
        pz -= box.Lz * z_shift;
        image.z += (int)z_shift;

        Scalar4 pos2;
        pos2.x = px;
        pos2.y = py;
        pos2.z = pz;
        pos2.w = pw;

        // write out the results
        d_pos[idx] = pos2;
        d_image[idx] = image;
        }
    }

/*! \param N number of particles
    \param d_pos array of particle positions
    \param d_image array of particle images
    \param box The new box the particles where the particles now reside

    This is just a kernel driver for gpu_nph_wrap_particles_kernel(). See it for more details.
*/
hipError_t gpu_nph_wrap_particles(const unsigned int N,
                             Scalar4 *d_pos,
                             int3 *d_image,
                             const gpu_boxsize& box)
    {
    // setup the grid to run the kernel
    unsigned int block_size=256;
    dim3 grid( (N / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_nph_wrap_particles_kernel<<< grid, threads >>>(N, d_pos, d_image, box);

    return hipSuccess;
    }

/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param net_force array of net forces
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param deltaT Time to advance (for one full step)
*/
extern "C" __global__
void gpu_nph_step_two_kernel(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             float4 *net_force,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             float deltaT)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // fetch velocities
        float4 vel = d_vel[idx];

        // read in the net force and compute the acceleration
        float4 accel = net_force[idx];
        float mass = vel.w;
        accel.x /= mass;
        accel.y /= mass;
        accel.z /= mass;

        // propagate velocities from t+1/2*deltaT to t+deltaT
        vel.x +=  (1.0f/2.0f)*deltaT*accel.x;
        vel.y +=  (1.0f/2.0f)*deltaT*accel.y;
        vel.z +=  (1.0f/2.0f)*deltaT*accel.z;

        // write out data
        d_vel[idx] = vel;
        // since we calculate the acceleration, we need to write it for the next step
        d_accel[idx] = make_scalar3(accel.x,accel.y,accel.z);
        }
    }

/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Net force on each particle
    \param deltaT Time to move forward in one whole step

    This is just a kernel driver for gpu_nph_step_two_kernel(). See it for more details.
*/
hipError_t gpu_nph_step_two(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             float4 *d_net_force,
                             float deltaT)
    {
    // setup the grid to run the kernel
    unsigned int block_size=256;
    dim3 grid( (group_size / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_nph_step_two_kernel<<< grid, threads >>>(d_vel, d_accel, d_net_force, d_group_members, group_size, deltaT);

    return hipSuccess;
    }

// vim:syntax=cpp

