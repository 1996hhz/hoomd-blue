#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id: TwoStepNPTRigidGPU.cu 2763 2010-02-15 19:05:06Z ndtrung $
// $URL: http://codeblue.umich.edu/hoomd-blue/svn/branches/rigid-bodies/libhoomd/cuda/TwoStepNPTRigidGPU.cu $
// Maintainer: ndtrung

#include "QuaternionMath.h"
#include "TwoStepNPTRigidGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file TwoStepNPTRigidGPU.cu
    \brief Defines GPU kernel code for NPT integration on the GPU. Used by TwoStepNPTRigidGPU.
*/

//! Flag for invalid particle index, identical to the sentinel value NO_INDEX in RigidData.h
#define INVALID_INDEX 0xffffffff

/*! Maclaurine expansion
    \param x Point to take the expansion

*/
__device__ float maclaurin_series(float x)
    {
    float x2, x4;
    x2 = x * x;
    x4 = x2 * x2;
    return (1.0f + (1.0f/6.0f) * x2 + (1.0f/120.0f) * x4 + (1.0f/5040.0f) * x2 * x4 + (1.0f/362880.0f) * x4 * x4);
    }

/*! Kernel to zero virial contribution from particles from rigid bodies
    \param d_virial_rigid Virial contribution from particles in rigid bodies
    \param local_num Number of particles in this card
*/
extern "C" __global__ void gpu_npt_rigid_zero_virial_rigid_kernel(float *d_virial_rigid, 
                                                                 unsigned int local_num)
    {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // particle's index

    if (idx < local_num)
        {
        d_virial_rigid[idx] = 0.0f;
        }

    }

/*! Takes the first half-step forward for rigid bodies in the velocity-verlet NVT integration 
    \param rdata_com Body center of mass
    \param d_rigid_group Body group
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Total umber of rigid bodies
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata Thermostat/barostat data
*/

extern "C" __global__ void gpu_npt_rigid_remap_kernel(float4* rdata_com,
                                                        unsigned int *d_rigid_group,
                                                        unsigned int n_group_bodies,
                                                        unsigned int n_bodies, 
                                                        gpu_boxsize box,
                                                        gpu_npt_rigid_data npt_rdata)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= n_group_bodies)
        return;
        
    unsigned int idx_body = d_rigid_group[group_idx];
        
    float oldlo, oldhi, ctr;
    float xlo, xhi, ylo, yhi, zlo, zhi, Lx, Ly, Lz;
    float4 pos, delta;
    float dilation = npt_rdata.dilation;
    
    xlo = -box.Lx/2.0f;
    xhi = box.Lx/2.0f;
    ylo = -box.Ly/2.0f;
    yhi = box.Ly/2.0f;
    zlo = -box.Lz/2.0f;
    zhi = box.Lz/2.0f;
    Lz = zhi - zlo;
    
    float4 com = rdata_com[idx_body];
    
    delta.x = com.x - xlo;
    delta.y = com.y - ylo;
    delta.z = com.z - zlo;

    pos.x = box.Lxinv * delta.x;
    pos.y = box.Lyinv * delta.y;
    pos.z = box.Lzinv * delta.z;
    
    // reset box to new size/shape
    oldlo = xlo;
    oldhi = xhi;
    ctr = 0.5f * (oldlo + oldhi);
    xlo = (oldlo - ctr) * dilation + ctr;
    xhi = (oldhi - ctr) * dilation + ctr;
    Lx = xhi - xlo;
    
    oldlo = ylo;
    oldhi = yhi;
    ctr = 0.5f * (oldlo + oldhi);
    ylo = (oldlo - ctr) * dilation + ctr;
    yhi = (oldhi - ctr) * dilation + ctr;
    Ly = yhi - ylo;
    
    if (npt_rdata.dimension == 3)
        {
        oldlo = zlo;
        oldhi = zhi;
        ctr = 0.5f * (oldlo + oldhi);
        zlo = (oldlo - ctr) * dilation + ctr;
        zhi = (oldhi - ctr) * dilation + ctr;
        Lz = zhi - zlo;
        }
    
    // convert rigid body COMs back to box coords
    float4 newboxlo;
    newboxlo.x = -Lx/2.0f;
    newboxlo.y = -Ly/2.0f;
    newboxlo.z = -Lz/2.0f;
    
    pos.x = Lx * pos.x + newboxlo.x;
    pos.y = Ly * pos.y + newboxlo.y;
    pos.z = Lz * pos.z + newboxlo.z;
    
    // write out results
    rdata_com[idx_body].x = pos.x;
    rdata_com[idx_body].y = pos.y;
    rdata_com[idx_body].z = pos.z;
    
    if (idx_body == 0)
        {
        *(npt_rdata.new_box) = make_float4(Lx, Ly, Lz, 0.0f);
        }
    }

    
#pragma mark RIGID_STEP_ONE_KERNEL
/*! Takes the first half-step forward for rigid bodies in the velocity-verlet NVT integration 
    \param rdata_com Body center of mass
    \param rdata_vel Body velocity
    \param rdata_angmom Angular momentum
    \param rdata_angvel Angular velocity
    \param rdata_orientation Quaternion
    \param rdata_ex_space x-axis unit vector
    \param rdata_ey_space y-axis unit vector
    \param rdata_ez_space z-axis unit vector
    \param rdata_body_imagex Body image in x-direction
    \param rdata_body_imagey Body image in y-direction
    \param rdata_body_imagez Body image in z-direction
    \param rdata_conjqm Conjugate quaternion momentum
    \param d_rigid_mass Body mass
    \param d_rigid_mi Body inertia moments
    \param n_group_bodies Number of rigid bodies in my group
    \param d_rigid_force Body forces
    \param d_rigid_torque Body torques
    \param d_rigid_group Body indices
    \param n_bodies Total umber of rigid bodies
    \param npt_rdata_eta_dot_t0 Thermostat translational velocity 
    \param npt_rdata_eta_dot_r0 Thermostat rotational velocity
    \param npt_rdata_epsilon_dot Barostat velocity
    \param npt_rdata_partial_Ksum_t Body translational kinetic energy 
    \param npt_rdata_partial_Ksum_r Body rotation kinetic energy
    \param npt_rdata_nf_t Translational degrees of freedom
    \param npt_rdata_nf_r Translational degrees of freedom
    \param npt_rdata_dimension System dimesion
    \param box Box dimensions for periodic boundary condition handling
    \param deltaT Timestep 
    
*/

extern "C" __global__ void gpu_npt_rigid_step_one_body_kernel(float4* rdata_com, 
                                                            float4* rdata_vel, 
                                                            float4* rdata_angmom, 
                                                            float4* rdata_angvel,
                                                            float4* rdata_orientation, 
                                                            float4* rdata_ex_space, 
                                                            float4* rdata_ey_space, 
                                                            float4* rdata_ez_space, 
                                                            int* rdata_body_imagex, 
                                                            int* rdata_body_imagey, 
                                                            int* rdata_body_imagez,
                                                            float4* rdata_conjqm,
                                                            float *d_rigid_mass,
                                                            float4 *d_rigid_mi,
                                                            float4 *d_rigid_force,
                                                            float4 *d_rigid_torque,
                                                            unsigned int *d_rigid_group, 
                                                            unsigned int n_group_bodies,  
                                                            unsigned int n_bodies, 
                                                            float npt_rdata_eta_dot_t0, 
                                                            float npt_rdata_eta_dot_r0,
                                                            float npt_rdata_epsilon_dot, 
                                                            float* npt_rdata_partial_Ksum_t, 
                                                            float* npt_rdata_partial_Ksum_r,
                                                            unsigned int npt_rdata_nf_t,
                                                            unsigned int npt_rdata_nf_r,
                                                            unsigned int npt_rdata_dimension, 
                                                            gpu_boxsize box, 
                                                            float deltaT)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (group_idx >= n_group_bodies)
        return;
        
    // do velocity verlet update
    // v(t+deltaT/2) = v(t) + (1/2)a*deltaT
    // r(t+deltaT) = r(t) + v(t+deltaT/2)*deltaT
    float body_mass;
    float4 moment_inertia, com, vel, orientation, ex_space, ey_space, ez_space, force, torque, conjqm;
    int body_imagex, body_imagey, body_imagez;
    float4 mbody, tbody, fquat;

    float dt_half = 0.5f * deltaT;
    float onednft, onednfr, tmp, scale_t, scale_r, scale_v, akin_t, akin_r;

    onednft = 1.0f + (float) (npt_rdata_dimension) / (float) (npt_rdata_nf_t);
    onednfr = (float) (npt_rdata_dimension) / (float) (npt_rdata_nf_r);

    tmp = -1.0f * dt_half * (npt_rdata_eta_dot_t0 + onednft * npt_rdata_epsilon_dot);
    scale_t = exp(tmp);
    tmp = -1.0f * dt_half * (npt_rdata_eta_dot_r0 + onednfr * npt_rdata_epsilon_dot);
    scale_r = exp(tmp);
    tmp = dt_half * npt_rdata_epsilon_dot;
    scale_v = deltaT * __expf(tmp) * maclaurin_series(tmp);

    unsigned int idx_body = d_rigid_group[group_idx];
    body_mass = d_rigid_mass[idx_body];
    moment_inertia = d_rigid_mi[idx_body];
    com = rdata_com[idx_body];
    vel = rdata_vel[idx_body];
    orientation = rdata_orientation[idx_body];
    body_imagex = rdata_body_imagex[idx_body];
    body_imagey = rdata_body_imagey[idx_body];
    body_imagez = rdata_body_imagez[idx_body];
    force = d_rigid_force[idx_body];
    torque = d_rigid_torque[idx_body];
    conjqm = rdata_conjqm[idx_body];
    
    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);
    
    // update velocity
    float dtfm = dt_half / body_mass;
    
    float4 vel2;
    vel2.x = vel.x + dtfm * force.x;
    vel2.y = vel.y + dtfm * force.y;
    vel2.z = vel.z + dtfm * force.z;
    vel2.x *= scale_t;
    vel2.y *= scale_t;
    vel2.z *= scale_t;
    vel2.w = vel.w;
    
    tmp = vel2.x * vel2.x + vel2.y * vel2.y + vel2.z * vel2.z;
    akin_t = body_mass * tmp;
    
    // update position
    float4 pos2;
    pos2.x = com.x + vel2.x * scale_v;
    pos2.y = com.y + vel2.y * scale_v;
    pos2.z = com.z + vel2.z * scale_v;
    pos2.w = com.w;
    
    // read in body's image
    // time to fix the periodic boundary conditions
    float x_shift = rintf(pos2.x * box.Lxinv);
    pos2.x -= box.Lx * x_shift;
    body_imagex += (int)x_shift;
    
    float y_shift = rintf(pos2.y * box.Lyinv);
    pos2.y -= box.Ly * y_shift;
    body_imagey += (int)y_shift;
    
    float z_shift = rintf(pos2.z * box.Lzinv);
    pos2.z -= box.Lz * z_shift;
    body_imagez += (int)z_shift;
    
    matrix_dot(ex_space, ey_space, ez_space, torque, tbody);
    quatvec(orientation, tbody, fquat);
    
    float4 conjqm2;
    conjqm2.x = conjqm.x + deltaT * fquat.x;
    conjqm2.y = conjqm.y + deltaT * fquat.y;
    conjqm2.z = conjqm.z + deltaT * fquat.z;
    conjqm2.w = conjqm.w + deltaT * fquat.w;
    
    conjqm2.x *= scale_r;
    conjqm2.y *= scale_r;
    conjqm2.z *= scale_r;
    conjqm2.w *= scale_r;
    
    // use no_squish rotate to update p and q
    
    no_squish_rotate(3, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(2, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(1, conjqm2, orientation, moment_inertia, deltaT);
    no_squish_rotate(2, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(3, conjqm2, orientation, moment_inertia, dt_half);
    
    // update the exyz_space
    // transform p back to angmom
    // update angular velocity
    float4 angmom2;
    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);
    invquatvec(orientation, conjqm2, mbody);
    transpose_dot(ex_space, ey_space, ez_space, mbody, angmom2);
    
    angmom2.x *= 0.5f;
    angmom2.y *= 0.5f;
    angmom2.z *= 0.5f;
    
    float4 angvel2;
    computeAngularVelocity(angmom2, moment_inertia, ex_space, ey_space, ez_space, angvel2);
    
    akin_r = angmom2.x * angvel2.x + angmom2.y * angvel2.y + angmom2.z * angvel2.z;
    
    // write out the results (MEM_TRANSFER: ? bytes)

    rdata_com[idx_body] = pos2;
    rdata_vel[idx_body] = vel2;
    rdata_angmom[idx_body] = angmom2;
    rdata_angvel[idx_body] = angvel2;
    rdata_orientation[idx_body] = orientation;
    rdata_ex_space[idx_body] = ex_space;
    rdata_ey_space[idx_body] = ey_space;
    rdata_ez_space[idx_body] = ez_space;
    rdata_body_imagex[idx_body] = body_imagex;
    rdata_body_imagey[idx_body] = body_imagey;
    rdata_body_imagez[idx_body] = body_imagez;
    rdata_conjqm[idx_body] = conjqm2;
    
    npt_rdata_partial_Ksum_t[idx_body] = akin_t;
    npt_rdata_partial_Ksum_r[idx_body] = akin_r;
    }

/*! \param pdata Particle data to step forward 1/2 step
    \param rigid_data Rigid body data to step forward 1/2 step
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Particle net forces
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata Thermostat/barostat data
    \param deltaT Amount of real time to step forward in one time step
    
*/
hipError_t gpu_npt_rigid_step_one(const gpu_pdata_arrays& pdata,       
                                   const gpu_rigid_data_arrays& rigid_data,
                                   float4 *d_pdata_orientation,
                                   unsigned int *d_group_members,
                                   unsigned int group_size,
                                   float4 *d_net_force,
                                   const gpu_boxsize &box, 
                                   const gpu_npt_rigid_data& npt_rdata,
                                   float deltaT)
    {
    unsigned int n_bodies = rigid_data.n_bodies;
    unsigned int n_group_bodies = rigid_data.n_group_bodies;
    unsigned int nmax = rigid_data.nmax;
    
    // setup the grid to run the kernel for rigid bodies
    int block_size = 64;
    int n_blocks = n_group_bodies / block_size + 1;
    dim3 body_grid(n_blocks, 1, 1);
    dim3 body_threads(block_size, 1, 1);
    gpu_npt_rigid_step_one_body_kernel<<< body_grid, body_threads  >>>(rigid_data.com, 
                                                                        rigid_data.vel, 
                                                                        rigid_data.angmom, 
                                                                        rigid_data.angvel,
                                                                        rigid_data.orientation, 
                                                                        rigid_data.ex_space, 
                                                                        rigid_data.ey_space, 
                                                                        rigid_data.ez_space, 
                                                                        rigid_data.body_imagex, 
                                                                        rigid_data.body_imagey, 
                                                                        rigid_data.body_imagez,
                                                                        rigid_data.conjqm,
                                                                        rigid_data.body_mass,
                                                                        rigid_data.moment_inertia,
                                                                        rigid_data.force,
                                                                        rigid_data.torque,
                                                                        rigid_data.body_indices,
                                                                        n_group_bodies,
                                                                        n_bodies, 
                                                                        npt_rdata.eta_dot_t0, 
                                                                        npt_rdata.eta_dot_r0,
                                                                        npt_rdata.epsilon_dot, 
                                                                        npt_rdata.partial_Ksum_t,
                                                                        npt_rdata.partial_Ksum_r,
                                                                        npt_rdata.nf_t,
                                                                        npt_rdata.nf_r,
                                                                        npt_rdata.dimension,
                                                                        box, 
                                                                        deltaT);
       
    gpu_npt_rigid_remap_kernel<<< body_grid, body_threads >>>(rigid_data.com,
                                                                rigid_data.body_indices,
                                                                n_group_bodies,
                                                                n_bodies,
                                                                box, 
                                                                npt_rdata);

    block_size = 192;
    dim3 particle_grid(group_size/block_size+1, 1, 1);
    dim3 particle_threads(block_size, 1, 1);
    
    gpu_rigid_setxv_kernel<true><<< particle_grid, particle_threads >>>(pdata.pos, 
                                                                        pdata.vel,
                                                                        d_pdata_orientation,
                                                                        pdata.image,
                                                                        d_group_members,
                                                                        group_size,
                                                                        rigid_data.particle_offset,
                                                                        pdata.body,
                                                                        rigid_data.body_indices,
                                                                        rigid_data.orientation,
                                                                        rigid_data.com,
                                                                        rigid_data.vel,
                                                                        rigid_data.angvel,
                                                                        rigid_data.body_imagex,
                                                                        rigid_data.body_imagey,
                                                                        rigid_data.body_imagez,
                                                                        rigid_data.particle_indices,
                                                                        rigid_data.particle_pos,
                                                                        rigid_data.particle_orientation,
                                                                        n_group_bodies,
                                                                        pdata.N,
                                                                        nmax,
                                                                        box);
                                                                    
    return hipSuccess;
    }

#pragma mark RIGID_STEP_TWO_KERNEL
//! Takes the 2nd 1/2 step forward in the velocity-verlet NPT integration scheme
/*!  
    \param rdata_vel Body velocity
    \param rdata_angmom Angular momentum
    \param rdata_angvel Angular velocity
    \param rdata_orientation Quaternion
    \param rdata_conjqm Conjugate quaternion momentum
    \param d_rigid_mass Body mass
    \param d_rigid_mi Body inertia moments
    \param d_rigid_force Body forces
    \param d_rigid_torque Body torques
    \param d_rigid_group Body indices
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Total number of rigid bodies
    \param npt_rdata_eta_dot_t0 Thermostat translational part 
    \param npt_rdata_eta_dot_r0 Thermostat rotational part
    \param npt_rdata_epsilon_dot Barostat velocity
    \param npt_rdata_partial_Ksum_t Body translational kinetic energy 
    \param npt_rdata_partial_Ksum_r Body rotation kinetic energy
    \param npt_rdata_nf_t Translational degrees of freedom
    \param npt_rdata_nf_r Translational degrees of freedom
    \param npt_rdata_dimension System dimesion
    \param deltaT Timestep 
    \param box Box dimensions for periodic boundary condition handling
*/

extern "C" __global__ void gpu_npt_rigid_step_two_body_kernel(float4* rdata_vel, 
                                                          float4* rdata_angmom, 
                                                          float4* rdata_angvel,
                                                          float4* rdata_orientation,
                                                          float4* rdata_conjqm,
                                                          float *d_rigid_mass,
                                                          float4 *d_rigid_mi,
                                                          float4 *d_rigid_force,
                                                          float4 *d_rigid_torque,
                                                          unsigned int *d_rigid_group,
                                                          unsigned int n_group_bodies,
                                                          unsigned int n_bodies, 
                                                          float npt_rdata_eta_dot_t0, 
                                                          float npt_rdata_eta_dot_r0,
                                                          float npt_rdata_epsilon_dot, 
                                                          float* npt_rdata_partial_Ksum_t,
                                                          float* npt_rdata_partial_Ksum_r,
                                                          unsigned int npt_rdata_nf_t,
                                                          unsigned int npt_rdata_nf_r,
                                                          unsigned int npt_rdata_dimension,
                                                          gpu_boxsize box, 
                                                          float deltaT)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (group_idx >= n_group_bodies)
        return;
        
    float body_mass;
    float4 moment_inertia, vel, ex_space, ey_space, ez_space, orientation, conjqm;
    float4 force, torque;
    float4 mbody, tbody, fquat;
    
    float dt_half = 0.5f * deltaT;
    float   onednft, onednfr, tmp, scale_t, scale_r, akin_t, akin_r;
    
    onednft = 1.0f + (float) (npt_rdata_dimension) / (float) (npt_rdata_nf_t);
    onednfr = (float) (npt_rdata_dimension) / (float) (npt_rdata_nf_r);

    tmp = -1.0f * dt_half * (npt_rdata_eta_dot_t0 + onednft * npt_rdata_epsilon_dot);
    scale_t = exp(tmp);
    tmp = -1.0f * dt_half * (npt_rdata_eta_dot_r0 + onednfr * npt_rdata_epsilon_dot);
    scale_r = exp(tmp);
    
    unsigned int idx_body = d_rigid_group[group_idx];

    // Update body velocity and angmom
    body_mass = d_rigid_mass[idx_body];
    moment_inertia = d_rigid_mi[idx_body];
    vel = rdata_vel[idx_body];
    force = d_rigid_force[idx_body];
    torque = d_rigid_torque[idx_body];
    orientation = rdata_orientation[idx_body];
    conjqm = rdata_conjqm[idx_body];
    
    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);
    
    float dtfm = dt_half / body_mass;
    
    // update the velocity
    float4 vel2;
    vel2.x = scale_t * vel.x + dtfm * force.x;
    vel2.y = scale_t * vel.y + dtfm * force.y;
    vel2.z = scale_t * vel.z + dtfm * force.z;
    vel2.w = 0.0f;
    
    tmp = vel2.x * vel2.x + vel2.y * vel2.y + vel2.z * vel2.z;
    akin_t = body_mass * tmp;
    
    // update angular momentum
    matrix_dot(ex_space, ey_space, ez_space, torque, tbody);
    quatvec(orientation, tbody, fquat);
    
    float4  conjqm2, angmom2;
    conjqm2.x = scale_r * conjqm.x + deltaT * fquat.x;
    conjqm2.y = scale_r * conjqm.y + deltaT * fquat.y;
    conjqm2.z = scale_r * conjqm.z + deltaT * fquat.z;
    conjqm2.w = scale_r * conjqm.w + deltaT * fquat.w;
    
    invquatvec(orientation, conjqm2, mbody);
    transpose_dot(ex_space, ey_space, ez_space, mbody, angmom2);
    
    angmom2.x *= 0.5f;
    angmom2.y *= 0.5f;
    angmom2.z *= 0.5f;
    angmom2.w = 0.0f;
    
    // update angular velocity
    float4 angvel2;
    computeAngularVelocity(angmom2, moment_inertia, ex_space, ey_space, ez_space, angvel2);
    
    akin_r = angmom2.x * angvel2.x + angmom2.y * angvel2.y + angmom2.z * angvel2.z;
    
    rdata_vel[idx_body] = vel2;
    rdata_angmom[idx_body] = angmom2;
    rdata_angvel[idx_body] = angvel2;
    rdata_conjqm[idx_body] = conjqm2;
    
    npt_rdata_partial_Ksum_t[idx_body] = akin_t;
    npt_rdata_partial_Ksum_r[idx_body] = akin_r;
    }

/*! \param pdata Particle data to step forward 1/2 step
    \param rigid_data Rigid body data to step forward 1/2 step
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Particle net forces
    \param d_net_virial Particle net virial
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata Thermostat/barostat data
    \param deltaT Amount of real time to step forward in one time step
    
*/
hipError_t gpu_npt_rigid_step_two(const gpu_pdata_arrays &pdata, 
                                    const gpu_rigid_data_arrays& rigid_data,
                                    float4 *d_pdata_orientation,
                                    unsigned int *d_group_members,
                                    unsigned int group_size,
                                    float4 *d_net_force,
                                    float *d_net_virial,
                                    const gpu_boxsize &box, 
                                    const gpu_npt_rigid_data& npt_rdata,
                                    float deltaT)
    {
    unsigned int n_bodies = rigid_data.n_bodies;
    unsigned int n_group_bodies = rigid_data.n_group_bodies;
    unsigned int nmax = rigid_data.nmax;
                                                                                                                                                            
    unsigned int block_size = 64;
    unsigned int n_blocks = n_group_bodies / block_size + 1;                                
    dim3 body_grid(n_blocks, 1, 1);
    dim3 body_threads(block_size, 1, 1);                                                 
    gpu_npt_rigid_step_two_body_kernel<<< body_grid, body_threads >>>(rigid_data.vel, 
                                                                    rigid_data.angmom, 
                                                                    rigid_data.angvel,
                                                                    rigid_data.orientation,
                                                                    rigid_data.conjqm,
                                                                    rigid_data.body_mass,
                                                                    rigid_data.moment_inertia,
                                                                    rigid_data.force,
                                                                    rigid_data.torque,
                                                                    rigid_data.body_indices,
                                                                    n_group_bodies,
                                                                    n_bodies, 
                                                                    npt_rdata.eta_dot_t0, 
                                                                    npt_rdata.eta_dot_r0,
                                                                    npt_rdata.epsilon_dot, 
                                                                    npt_rdata.partial_Ksum_t,
                                                                    npt_rdata.partial_Ksum_r,
                                                                    npt_rdata.nf_t,
                                                                    npt_rdata.nf_r,
                                                                    npt_rdata.dimension, 
                                                                    box, 
                                                                    deltaT);

    block_size = 192;
    dim3 particle_grid(group_size/block_size+1, 1, 1);
    dim3 particle_threads(block_size, 1, 1);
    
    gpu_rigid_setxv_kernel<false><<< particle_grid, particle_threads >>>(pdata.pos, 
                                                                        pdata.vel,
                                                                        d_pdata_orientation,
                                                                        pdata.image,
                                                                        d_group_members,
                                                                        group_size,
                                                                        rigid_data.particle_offset,
                                                                        pdata.body,
                                                                        rigid_data.body_indices,
                                                                        rigid_data.orientation,
                                                                        rigid_data.com,
                                                                        rigid_data.vel,
                                                                        rigid_data.angvel,
                                                                        rigid_data.body_imagex,
                                                                        rigid_data.body_imagey,
                                                                        rigid_data.body_imagez,
                                                                        rigid_data.particle_indices,
                                                                        rigid_data.particle_pos,
                                                                        rigid_data.particle_orientation,
                                                                        n_group_bodies,
                                                                        pdata.N,
                                                                        nmax,
                                                                        box);
                                                                                                                                             
    return hipSuccess;
    }

#pragma mark RIGID_KINETIC_ENERGY_REDUCTION

//! Shared memory for kinetic energy reduction
extern __shared__ float npt_rigid_sdata[];

/*! Summing the kinetic energy of rigid bodies
    \param npt_rdata Thermostat data for rigid bodies 
    
*/
extern "C" __global__ void gpu_npt_rigid_reduce_ksum_kernel(gpu_npt_rigid_data npt_rdata)
    {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    float* body_ke_t = npt_rigid_sdata;
    float* body_ke_r = &npt_rigid_sdata[blockDim.x];
    
    float Ksum_t = 0.0f, Ksum_r=0.0f;
    
    // sum up the values in the partial sum via a sliding window
    for (int start = 0; start < npt_rdata.n_bodies; start += blockDim.x)
        {
        if (start + threadIdx.x < npt_rdata.n_bodies)
            {
            body_ke_t[threadIdx.x] = npt_rdata.partial_Ksum_t[start + threadIdx.x];
            body_ke_r[threadIdx.x] = npt_rdata.partial_Ksum_r[start + threadIdx.x];
            }
        else
            {
            body_ke_t[threadIdx.x] = 0.0f;
            body_ke_r[threadIdx.x] = 0.0f;
            }
        __syncthreads();
        
        // reduce the sum within a block
        int offset = blockDim.x >> 1;
        while (offset > 0)
            {
            if (threadIdx.x < offset)
                {
                body_ke_t[threadIdx.x] += body_ke_t[threadIdx.x + offset];
                body_ke_r[threadIdx.x] += body_ke_r[threadIdx.x + offset];
                }
            offset >>= 1;
            __syncthreads();
            }
            
        // everybody sums up Ksum
        Ksum_t += body_ke_t[0];
        Ksum_r += body_ke_r[0];
        }
        
    __syncthreads();
    
    
    if (global_idx == 0)
        {
        *npt_rdata.Ksum_t = Ksum_t;
        *npt_rdata.Ksum_r = Ksum_r;
        }
        
    }

/*! 
    \param npt_rdata Thermostat/barostat data for rigid bodies 
*/
hipError_t gpu_npt_rigid_reduce_ksum(const gpu_npt_rigid_data& npt_rdata)
    {
    // setup the grid to run the kernel
    int block_size = 128;
    dim3 grid( 1, 1, 1);
    dim3 threads(block_size, 1, 1);
    
    // run the kernel: double the block size to accomodate Ksum_t and Ksum_r
    gpu_npt_rigid_reduce_ksum_kernel<<< grid, threads, 2 * block_size * sizeof(float) >>>(npt_rdata);
    
    return hipSuccess;
    }

