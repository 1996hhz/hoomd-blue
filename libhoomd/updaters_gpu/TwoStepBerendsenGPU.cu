#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008, 2009 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

Redistribution and use of HOOMD-blue, in source and binary forms, with or
without modification, are permitted, provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of HOOMD-blue's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS''
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR
ANY WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "TwoStepBerendsenGPU.cuh"

#include <assert.h>

/*! \file TwoStepBerendsenGPU.cu
    \brief CUDA kernels for BerendsenGPU
*/

// First, the kernel code for the Berendsen thermostat
//! Kernel that applies the first step of a Berendsen integration to a group of particles
/*! \param pdata Particle data arrays to apply Berendsen thermostat to
    \param d_group_members Device array listing the indicies of the members of the group to integrate
    \param group_size Number of members in the group
    \param box Box dimensions for applying periodic boundary conditions
    \param lambda Intermediate variable computed on the host and used in integrating the velocity
    \param deltaT Length of one timestep

    This kernel executes one thread per particle and applies the theromstat to each each. It can be
    run with any 1D block size as long as block_size * num_blocks is >= the number of particles.
*/
extern "C" __global__
void gpu_berendsen_step_one_kernel(gpu_pdata_arrays pdata,
                                   unsigned int *d_group_members,
                                   unsigned int group_size,
                                   gpu_boxsize box,
                                   float lambda,
                                   float deltaT)
    {
    // determine the particle index for this thread
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // read the particle position
        float4 pos = pdata.pos[idx];

        // we need to use temporary variables to reduce global memory access
        float px = pos.x;
        float py = pos.y;
        float pz = pos.z;
        float pw = pos.w;

        // read the particle velocity and acceleration
        float4 vel = pdata.vel[idx];
        float4 accel = pdata.accel[idx];

        // integrate velocity and position forward in time
        vel.x = lambda * (vel.x + accel.x * deltaT / 2.0f);
        px += vel.x * deltaT;

        vel.y = lambda * (vel.y + accel.y * deltaT / 2.0f);
        py += vel.y * deltaT;

        vel.z = lambda * (vel.z + accel.z * deltaT / 2.0f);
        pz += vel.z * deltaT;

        // read in the image flags
        int4 image = pdata.image[idx];

        // apply the periodic boundary conditions
        float x_shift = rintf(px * box.Lxinv);
        px -= box.Lx * x_shift;
        image.x += (int)x_shift;

        float y_shift = rintf(py * box.Lyinv);
        py -= box.Ly * y_shift;
        image.y += (int) y_shift;

        float z_shift = rintf(pz * box.Lzinv);
        pz -= box.Lz * z_shift;
        image.z += (int)z_shift;

        // another temporary variable
        float4 pos2;
        pos2.x = px;
        pos2.y = py;
        pos2.z = pz;
        pos2.w = pw;

        // write the results
        pdata.pos[idx] = pos2;
        pdata.vel[idx] = vel;
        pdata.image[idx] = image;
        }
    }

//! Kernel that applies the first step of a Berendsen integration to a group of particles
/*! \param pdata Particle data arrays to apply Berendsen thermostat to
    \param d_group_members Device array listing the indicies of the members of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Current net force on the particles
    \param deltaT Length of one timestep

    This kernel executes one thread per particle and applies the theromstat to each each. It can be
    run with any 1D block size as long as block_size * num_blocks is >= the number of particles.
*/
extern "C" __global__
void gpu_berendsen_step_two_kernel(gpu_pdata_arrays pdata,
                                   unsigned int *d_group_members,
                                   unsigned int group_size,
                                   float4 *d_net_force,
                                   float deltaT)
    {
    // determine the particle index for this thread
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // read in the net force and calculate the acceleration
        float4 accel = d_net_force[idx];
        float mass = pdata.mass[idx];
        accel.x /= mass;
        accel.y /= mass;
        accel.z /= mass;

        // read in teh velocity
        float4 vel = pdata.vel[idx];

        // integrate the velocity
        vel.x = (vel.x + accel.x * deltaT / 2.0f);
        vel.y = (vel.y + accel.y * deltaT / 2.0f);
        vel.z = (vel.z + accel.z * deltaT / 2.0f);

        // write out the velocity and acceleration
        pdata.vel[idx] = vel;
        pdata.accel[idx] = accel;
        }
    }

hipError_t gpu_berendsen_step_one(const gpu_pdata_arrays &pdata,
                                   unsigned int *d_group_members,
                                   unsigned int group_size,
                                   const gpu_boxsize &box,
                                   unsigned int block_size,
                                   float lambda,
                                   float deltaT)
    {
    // setup the grid to run the kernel
    dim3 grid( (group_size / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_berendsen_step_one_kernel<<< grid, threads, block_size * sizeof(float) >>>(pdata,
                                                                                   d_group_members,
                                                                                   group_size,
                                                                                   box,
                                                                                   lambda,
                                                                                   deltaT);

    return hipSuccess;
    }

hipError_t gpu_berendsen_step_two(const gpu_pdata_arrays &pdata,
                                   unsigned int *d_group_members,
                                   unsigned int group_size,
                                   float4 *d_net_force,
                                   unsigned int block_size,
                                   float deltaT)
    {
    // setup the grid to run the kernel
    dim3 grid( (group_size / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_berendsen_step_two_kernel<<< grid, threads, block_size * sizeof(float) >>>(pdata,
                                                                                   d_group_members,
                                                                                   group_size,
                                                                                   d_net_force,
                                                                                   deltaT);

    return hipSuccess;
    }

