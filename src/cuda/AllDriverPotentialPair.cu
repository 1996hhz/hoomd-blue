#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008, 2009 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

Redistribution and use of HOOMD-blue, in source and binary forms, with or
without modification, are permitted, provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of HOOMD-blue's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS''
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR
ANY WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
// Maintainer: joaander

/*! \file PotentialPairLJGPU.cu
    \brief Defines the driver function for computing LJ pair forces on the GPU
*/

#include "EvaluatorPairLJ.h"
#include "PotentialPairLJGPU.cuh"

/*! This is just a driver function for gpu_compute_pair_forces<EvaluatorPairLJ>(). See it for details.

    \param force_data Device memory array to write calculated forces to
    \param pdata Particle data on the GPU to calculate forces on
    \param box Box dimensions used to implement periodic boundary conditions
    \param nlist Neigbhor list data on the GPU to use to calculate the forces
    \param d_params Parameters for the potential, stored per type pair
    \param d_rcutsq rcut squared, stored per type pair
    \param d_ronsq ron squared, stored per type pair
    \param ntypes Number of types in the simulation
    \param args Additional options
*/
hipError_t gpu_compute_ljtemp_forces(const gpu_force_data_arrays& force_data,
                                      const gpu_pdata_arrays &pdata,
                                      const gpu_boxsize &box,
                                      const gpu_nlist_array &nlist,
                                      float2 *d_params,
                                      float *d_rcutsq,
                                      float *d_ronsq,
                                      int ntypes,
                                      const pair_args& args)
    {
    return gpu_compute_pair_forces<EvaluatorPairLJ>(force_data,
                                                    pdata,
                                                    box,
                                                    nlist,
                                                    d_params,
                                                    d_rcutsq,
                                                    d_ronsq,
                                                    ntypes,
                                                    args);
    }