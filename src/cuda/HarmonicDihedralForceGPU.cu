#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
// Maintainer: dnlebard

#include "gpu_settings.h"
#include "HarmonicDihedralForceGPU.cuh"
#include "DihedralData.cuh" // SERIOUSLY, DO I NEED THIS HERE??

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

//! SMALL a relatively small number
#define SMALL 0.001f

/*! \file HarmonicDihedralForceGPU.cu
    \brief Defines GPU kernel code for calculating the harmonic dihedral forces. Used by HarmonicDihedralForceComputeGPU.
*/

//! Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;

//! Texture for reading dihedral parameters
texture<float4, 1, hipReadModeElementType> dihedral_params_tex;

//! Kernel for caculating harmonic dihedral forces on the GPU
/*! \param force_data Data to write the compute forces to
    \param pdata Particle data arrays to calculate forces on
    \param box Box dimensions for periodic boundary condition handling
    \param tlist Dihedral data to use in calculating the forces
*/
extern "C" __global__ 
void gpu_compute_harmonic_dihedral_forces_kernel(gpu_force_data_arrays force_data,
                                                 gpu_pdata_arrays pdata,
                                                 gpu_boxsize box,
                                                 gpu_dihedraltable_array tlist)
    {
    // start by identifying which particle we are to handle
    int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_global = idx_local + pdata.local_beg;
    
    
    if (idx_local >= pdata.local_num)
        return;
        
    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_dihedrals = tlist.n_dihedrals[idx_local];
    
    // read in the position of our b-particle from the a-b-c triplet. (MEM TRANSFER: 16 bytes)
    float4 idx_pos = tex1Dfetch(pdata_pos_tex, idx_global);  // we can be either a, b, or c in the a-b-c-d quartet
    float4 a_pos,b_pos,c_pos, d_pos; // allocate space for the a,b, and c atoms in the a-b-c-d quartet
    
    // initialize the force to 0
    float4 force_idx = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    
    // initialize the virial to 0
    float virial_idx = 0.0f;
    
    // loop over all dihedrals
    for (int dihedral_idx = 0; dihedral_idx < n_dihedrals; dihedral_idx++)
        {
        // the volatile fails to compile in device emulation mode (MEM TRANSFER: 8 bytes)
#ifdef _DEVICEEMU
        uint4 cur_dihedral = tlist.dihedrals[tlist.pitch*dihedral_idx + idx_local];
        uint1 cur_ABCD = tlist.dihedralABCD[tlist.pitch*dihedral_idx + idx_local];
#else
        // the volatile is needed to force the compiler to load the uint2 coalesced
        volatile uint4 cur_dihedral = tlist.dihedrals[tlist.pitch*dihedral_idx + idx_local];
        volatile uint1 cur_ABCD = tlist.dihedralABCD[tlist.pitch*dihedral_idx + idx_local];
#endif
        
        int cur_dihedral_x_idx = cur_dihedral.x;
        int cur_dihedral_y_idx = cur_dihedral.y;
        int cur_dihedral_z_idx = cur_dihedral.z;
        int cur_dihedral_type = cur_dihedral.w;
        int cur_dihedral_abcd = cur_ABCD.x;
        
        // get the a-particle's position (MEM TRANSFER: 16 bytes)
        float4 x_pos = tex1Dfetch(pdata_pos_tex, cur_dihedral_x_idx);
        // get the c-particle's position (MEM TRANSFER: 16 bytes)
        float4 y_pos = tex1Dfetch(pdata_pos_tex, cur_dihedral_y_idx);
        // get the c-particle's position (MEM TRANSFER: 16 bytes)
        float4 z_pos = tex1Dfetch(pdata_pos_tex, cur_dihedral_z_idx);
        
        if (cur_dihedral_abcd == 0)
            {
            a_pos = idx_pos;
            b_pos = x_pos;
            c_pos = y_pos;
            d_pos = z_pos;
            }
        if (cur_dihedral_abcd == 1)
            {
            b_pos = idx_pos;
            a_pos = x_pos;
            c_pos = y_pos;
            d_pos = z_pos;
            }
        if (cur_dihedral_abcd == 2)
            {
            c_pos = idx_pos;
            a_pos = x_pos;
            b_pos = y_pos;
            d_pos = z_pos;
            }
        if (cur_dihedral_abcd == 3)
            {
            d_pos = idx_pos;
            a_pos = x_pos;
            b_pos = y_pos;
            c_pos = z_pos;
            }
            
        // calculate dr for a-b,c-b,and a-c(FLOPS: 9)
        float dxab = a_pos.x - b_pos.x;
        float dyab = a_pos.y - b_pos.y;
        float dzab = a_pos.z - b_pos.z;
        
        float dxcb = c_pos.x - b_pos.x;
        float dycb = c_pos.y - b_pos.y;
        float dzcb = c_pos.z - b_pos.z;
        
        float dxdc = d_pos.x - c_pos.x;
        float dydc = d_pos.y - c_pos.y;
        float dzdc = d_pos.z - c_pos.z;
        
        dxab -= box.Lx * rintf(dxab * box.Lxinv);
        dxcb -= box.Lx * rintf(dxcb * box.Lxinv);
        dxdc -= box.Lx * rintf(dxdc * box.Lxinv);
        
        dyab -= box.Ly * rintf(dyab * box.Lyinv);
        dycb -= box.Ly * rintf(dycb * box.Lyinv);
        dydc -= box.Ly * rintf(dydc * box.Lyinv);
        
        dzab -= box.Lz * rintf(dzab * box.Lzinv);
        dzcb -= box.Lz * rintf(dzcb * box.Lzinv);
        dzdc -= box.Lz * rintf(dzdc * box.Lzinv);
        
        float dxcbm = -dxcb;
        float dycbm = -dycb;
        float dzcbm = -dzcb;
        
        dxcbm -= box.Lx * rintf(dxcbm * box.Lxinv);
        dycbm -= box.Ly * rintf(dycbm * box.Lyinv);
        dzcbm -= box.Lz * rintf(dzcbm * box.Lzinv);
        
        // get the dihedral parameters (MEM TRANSFER: 12 bytes)
        float4 params = tex1Dfetch(dihedral_params_tex, cur_dihedral_type);
        float K = params.x;
        float sign = params.y;
        float multi = params.z;
        
        // printf("IN CUDA CODE: k = %f sign = %f multi = %f \n",K,sign,multi);
        
        float aax = dyab*dzcbm - dzab*dycbm;
        float aay = dzab*dxcbm - dxab*dzcbm;
        float aaz = dxab*dycbm - dyab*dxcbm;
        
        float bbx = dydc*dzcbm - dzdc*dycbm;
        float bby = dzdc*dxcbm - dxdc*dzcbm;
        float bbz = dxdc*dycbm - dydc*dxcbm;
        
        float raasq = aax*aax + aay*aay + aaz*aaz;
        float rbbsq = bbx*bbx + bby*bby + bbz*bbz;
        float rgsq = dxcbm*dxcbm + dycbm*dycbm + dzcbm*dzcbm;
        float rg = sqrtf(rgsq);
        
        float rginv, raa2inv, rbb2inv;
        rginv = raa2inv = rbb2inv = 0.0f;
        if (rg > 0.0f) rginv = 1.0f/rg;
        if (raasq > 0.0f) raa2inv = 1.0f/raasq;
        if (rbbsq > 0.0f) rbb2inv = 1.0f/rbbsq;
        float rabinv = sqrtf(raa2inv*rbb2inv);
        
        float c_abcd = (aax*bbx + aay*bby + aaz*bbz)*rabinv;
        float s_abcd = rg*rabinv*(aax*dxdc + aay*dydc + aaz*dzdc);
        
        if (c_abcd > 1.0f) c_abcd = 1.0f;
        if (c_abcd < -1.0f) c_abcd = -1.0f;
        
        
        float p = 1.0f;
        float ddfab;
        float dfab = 0.0f;
        int m = __float2int_rn(multi);
        
        for (int jj = 0; jj < m; jj++)
            {
            ddfab = p*c_abcd - dfab*s_abcd;
            dfab = p*s_abcd + dfab*c_abcd;
            p = ddfab;
            }
            
/////////////////////////
// FROM LAMMPS: sin_shift is always 0... so dropping all sin_shift terms!!!!
/////////////////////////
        p *= sign;
        dfab *= sign;
        dfab *= -multi;
        p += 1.0f;
        
        if (multi < 1.0f)
            {
            p =  1.0f + sign;
            dfab = 0.0f;
            }
            
        float fg = dxab*dxcbm + dyab*dycbm + dzab*dzcbm;
        float hg = dxdc*dxcbm + dydc*dycbm + dzdc*dzcbm;
        
        float fga = fg*raa2inv*rginv;
        float hgb = hg*rbb2inv*rginv;
        float gaa = -raa2inv*rg;
        float gbb = rbb2inv*rg;
        
        float dtfx = gaa*aax;
        float dtfy = gaa*aay;
        float dtfz = gaa*aaz;
        float dtgx = fga*aax - hgb*bbx;
        float dtgy = fga*aay - hgb*bby;
        float dtgz = fga*aaz - hgb*bbz;
        float dthx = gbb*bbx;
        float dthy = gbb*bby;
        float dthz = gbb*bbz;
        
        //float df = -K * dfab;
        float df = -K * dfab * float(0.500); // the 0.5 term is for 1/2K in the forces
        
        float sx2 = df*dtgx;
        float sy2 = df*dtgy;
        float sz2 = df*dtgz;
        
        float ffax = df*dtfx;
        float ffay = df*dtfy;
        float ffaz = df*dtfz;
        
        float ffbx = sx2 - ffax;
        float ffby = sy2 - ffay;
        float ffbz = sz2 - ffaz;
        
        float ffdx = df*dthx;
        float ffdy = df*dthy;
        float ffdz = df*dthz;
        
        float ffcx = -sx2 - ffdx;
        float ffcy = -sy2 - ffdy;
        float ffcz = -sz2 - ffdz;
        
        // Now, apply the force to each individual atom a,b,c,d
        // and accumlate the energy/virial
        // compute 1/4 of the energy, 1/4 for each atom in the dihedral
        //float dihedral_eng = p*K*float(1.0/4.0);
        float dihedral_eng = p*K*float(1.0/8.0); // the 1/8th term is (1/2)K * 1/4
        
        float vx = (dxab*ffax) + (dxcb*ffcx) + (dxdc+dxcb)*ffdx;
        float vy = (dyab*ffay) + (dycb*ffcy) + (dydc+dycb)*ffdy;
        float vz = (dzab*ffaz) + (dzcb*ffcz) + (dzdc+dzcb)*ffdz;
        
        // compute 1/4 of the virial, 1/4 for each atom in the dihedral
        float dihedral_virial = float(1.0/12.0)*(vx + vy + vz);
        
        if (cur_dihedral_abcd == 0)
            {
            force_idx.x += ffax;
            force_idx.y += ffay;
            force_idx.z += ffaz;
            }
        if (cur_dihedral_abcd == 1)
            {
            force_idx.x += ffbx;
            force_idx.y += ffby;
            force_idx.z += ffbz;
            }
        if (cur_dihedral_abcd == 2)
            {
            force_idx.x += ffcx;
            force_idx.y += ffcy;
            force_idx.z += ffcz;
            }
        if (cur_dihedral_abcd == 3)
            {
            force_idx.x += ffdx;
            force_idx.y += ffdy;
            force_idx.z += ffdz;
            }
            
        force_idx.w += dihedral_eng;
        virial_idx += dihedral_virial;
        }
        
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    force_data.force[idx_local] = force_idx;
    force_data.virial[idx_local] = virial_idx;
    }

/*! \param force_data Force data on GPU to write forces to
    \param pdata Particle data on the GPU to perform the calculation on
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param ttable List of dihedrals stored on the GPU
    \param d_params K, sign,multiplicity params packed as padded float4 variables
    \param n_dihedral_types Number of dihedral types in d_params
    \param block_size Block size to use when performing calculations

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one float4 element per dihedral type. The x component contains K the spring constant
    and the y component contains sign, and the z component the multiplicity.
*/
hipError_t gpu_compute_harmonic_dihedral_forces(const gpu_force_data_arrays& force_data, const gpu_pdata_arrays &pdata, const gpu_boxsize &box, const gpu_dihedraltable_array &ttable, float4 *d_params, unsigned int n_dihedral_types, int block_size)
    {
    assert(d_params);
    
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)pdata.local_num / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);
    
    // bind the textures
    hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
    if (error != hipSuccess)
        return error;
        
    error = hipBindTexture(0, dihedral_params_tex, d_params, sizeof(float4) * n_dihedral_types);
    if (error != hipSuccess)
        return error;
        
    // run the kernel
    gpu_compute_harmonic_dihedral_forces_kernel<<< grid, threads>>>(force_data, pdata, box, ttable);
    
    if (!g_gpu_error_checking)
        {
        return hipSuccess;
        }
    else
        {
        hipDeviceSynchronize();
        return hipGetLastError();
        }
    }

