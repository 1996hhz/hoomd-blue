#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
// Maintainer: dnlebard

#include "ImproperData.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file ImproperData.cu
 	\brief Defines the data structures for storing impropers on the GPU.
*/

/*! \pre no allocations have been performed or deallocate() has been called after a previous allocate()
	\post Memory for \a n_impropers and \a impropers is allocated on the device
	\param num_local Number of particles local to the GPU on which this is being called
	\param alloc_height Number of impropers to allocate for each particle
	\note allocate() \b must be called on the GPU it is to allocate data on
*/
hipError_t gpu_impropertable_array::allocate(unsigned int num_local, unsigned int alloc_height)
	{
	// sanity checks
	assert(n_impropers == NULL);
	assert(impropers == NULL);
	assert(improperABCD == NULL);
		
	// allocate n_impropers and check for errors
	hipError_t error = hipMalloc((void**)((void*)&n_impropers), num_local*sizeof(unsigned int));
	if (error != hipSuccess)
		return error;
	
	error = hipMemset((void*)n_impropers, 0, num_local*sizeof(unsigned int));
	if (error != hipSuccess)
		return error;

        
	// hipMallocPitch fails to work for coalesced reads here (dunno why), need to calculate pitch ourselves
	// round up to the nearest multiple of 32
	height = alloc_height;
	pitch = (num_local + (32 - num_local & 31));
	error = hipMalloc((void**)((void*)&impropers), pitch * height * sizeof(uint4));
	if (error != hipSuccess)
		return error;	
	
	error = hipMemset((void*)impropers, 0, pitch * height * sizeof(uint4));
	if (error != hipSuccess)
		return error;

	error = hipMalloc((void**)((void*)&improperABCD), pitch * height * sizeof(uint1));
	if (error != hipSuccess)
		return error;	
	
	error = hipMemset((void*)improperABCD, 0, pitch * height * sizeof(uint1));
	if (error != hipSuccess)
		return error;
		
	// all done, return success
	return hipSuccess;	
	}
	
/*! \pre allocate() has been called
	\post Memory for \a n_impropers and \a impropers is freed on the device
	\note deallocate() \b must be called on the same GPU as allocate()
*/
hipError_t gpu_impropertable_array::deallocate()
	{
	// sanity checks
	assert(n_impropers != NULL);
	assert(impropers != NULL);
	assert(improperABCD != NULL);

	// free the memory
	hipError_t error = hipFree((void*)n_impropers);
	n_impropers = NULL;
	if (error != hipSuccess)
		return error;
		
	error = hipFree((void*)impropers);
	impropers = NULL;
	if (error != hipSuccess)
		return error;

	error = hipFree((void*)improperABCD);
	improperABCD = NULL;
	if (error != hipSuccess)
		return error;

	// all done, return success
	return hipSuccess;
	}
