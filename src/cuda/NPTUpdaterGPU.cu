#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/


#include "NPTUpdaterGPU.cuh"
#include "Integrator.cuh"
#include "gpu_settings.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

#include <stdio.h>

/*! \file NVTUpdaterGPU.cu
	\brief Defines GPU kernel code for NPT integration on the GPU. Used by NPTUpdaterGPU.
*/

//! Texture for reading the pdata pos array
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;
//! Texture for reading the pdata vel array
texture<float4, 1, hipReadModeElementType> pdata_vel_tex;
//! Texture for reading the pdata accel array
texture<float4, 1, hipReadModeElementType> pdata_accel_tex;

//! Shared data used by NPT kernels for sum reductions
extern __shared__ float npt_sdata[];

//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \todo document me
*/
__device__ float gpu_integrator_sum_virials_inline(unsigned int idx_local, unsigned int local_num, float **virial_data_ptrs, int num_virials)
	{
	// each block loads in the pointers
	__shared__ float *virial_ptrs[32];
	if (threadIdx.x < 32)
		virial_ptrs[threadIdx.x] = virial_data_ptrs[threadIdx.x];
	__syncthreads();

	float virial = 0.0f;
	if (idx_local < local_num)
		{
		// sum the virials
		for (int i = 0; i < num_virials; i++)
			{
			float *d_virial = virial_ptrs[i];
			float v = d_virial[idx_local];
		
			virial += v;
			}
		}
	// return the result
	return virial;
	}

//! Sums the varoius virials on the GPU
/*! \param nptdata NPT data storage structure
	\param pdata Particle data arrays
	\param virial_data_ptrs list of virial data pointers
	\param num_virials number of virial points in the list

	\a virial_data_ptrs contains up to 32 pointers. Each points to pdata.local_num float's in memory
	All virials are summed into nptdata.virial
*/
extern "C" __global__ void gpu_integrator_sum_virials_kernel(gpu_npt_data nptdata, gpu_pdata_arrays pdata, float **virial_data_ptrs, int num_virials)
	{
	// calculate the index we will be handling
	int idx_local = blockDim.x * blockIdx.x + threadIdx.x;

	float virial = gpu_integrator_sum_virials_inline(idx_local, pdata.local_num, virial_data_ptrs, num_virials);

	if (idx_local < pdata.local_num)
		{
		// write out the result
		nptdata.virial[idx_local] = virial;
		}
	}

/*! Every virial on every particle is summed up into \a nptpdata.virial

	\param nptdata NPT data storage structure
    \param pdata Particle data to write virial sum to
    \param virial_list List of pointers to virial data to sum
    \param num_virials Number of forces in \a virial_list

    \returns Any error code from the kernel call retrieved via hipGetLastError()
*/
hipError_t gpu_integrator_sum_virials(const gpu_npt_data &nptdata, const gpu_pdata_arrays &pdata, float** virial_list, int num_virials)
	{
	// sanity check
	assert(num_virials < 32);

	const int block_size = 192;

	gpu_integrator_sum_virials_kernel<<< pdata.local_num/block_size+1, block_size >>>(nptdata, pdata, virial_list, num_virials);

	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
    }


//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \param pdata Particle data arrays to integrate forward 1/2 step
	\param box Box dimensions that the particles are in
	\param d_npt_data NPT data structure for storing data specific to NPT integration
	\param exp_v_fac For inexplicable reasons, the author has decided that is is best not to document his code
	\param exp_r_fac For inexplicable reasons, the author has decided that is is best not to document his code
	\param deltaT Time to advance (for one full step)
	\param box_len_scale For inexplicable reasons, the author has decided that is is best not to document his code
	
	\todo document me
*/
extern "C" __global__ void gpu_npt_pre_step_kernel(gpu_pdata_arrays pdata, gpu_boxsize box, gpu_npt_data d_npt_data, float exp_v_fac, float exp_r_fac, float deltaT, float box_len_scale)
	{
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_global = idx_local + pdata.local_beg;
	// do Nose-Hoover integrate ??? Copied and pasted comment doesn't apply
	
	if (idx_local < pdata.local_num)
		{
		// update positions to the next timestep and update velocities to the next half step
		float4 pos = tex1Dfetch(pdata_pos_tex, idx_global);
		
		float px = pos.x;
		float py = pos.y;
		float pz = pos.z;
		float pw = pos.w;
		
		float4 vel = tex1Dfetch(pdata_vel_tex, idx_global);
		float4 accel = tex1Dfetch(pdata_accel_tex, idx_global);
		
		vel.x = vel.x*exp_v_fac*exp_v_fac + (1.0f/2.0f) * deltaT*exp_v_fac*accel.x;
		px = px*exp_r_fac*exp_r_fac + vel.x*exp_r_fac*deltaT;

		vel.y = vel.y*exp_v_fac*exp_v_fac + (1.0f/2.0f) * deltaT*exp_v_fac*accel.y;
		py = py*exp_r_fac*exp_r_fac + vel.y*exp_r_fac*deltaT;

		vel.z = vel.z*exp_v_fac*exp_v_fac + (1.0f/2.0f) * deltaT*exp_v_fac*accel.z;
		pz = pz*exp_r_fac*exp_r_fac + vel.z*exp_r_fac*deltaT;

	
		// time to fix the periodic boundary conditions	
		//	printf("Lx = %f\n", box.Lx);
		//printf("Ly = %f\n", box.Ly);
		//printf("Lz = %f\n", box.Lz);
		//printf("Lxinv = %f\n", box.Lxinv);
		  //printf("Lyinv = %f\n", box.Lyinv);
		  //printf("Lzinv = %f\n", box.Lzinv);
		//printf("box_len_scale = %f\n", box_len_scale);
		px -= box_len_scale*box.Lx * rintf(px * box.Lxinv/box_len_scale);
		py -= box_len_scale*box.Ly * rintf(py * box.Lyinv/box_len_scale);
		pz -= box_len_scale*box.Lz * rintf(pz * box.Lzinv/box_len_scale);
	
		//printf("px = %f\n", px);
		//printf("py = %f\n", py);
		//printf("pz = %f\n", pz);
		float4 pos2;
		pos2.x = px;
		pos2.y = py;
		pos2.z = pz;
		pos2.w = pw;
						
		// write out the results
		pdata.pos[idx_global] = pos2;
		pdata.vel[idx_global] = vel;
	
		// now we need to do the partial K sums  ??? copied and pasted comments do not apply
	
		// compute our contribution to the sum
		// NOTE: mass = 1.0
		}
	
	}

/*! \param pdata Particle Data to operate on
	\param box Current box dimensions the particles are in
	\param d_npt_data NPT specific data structures
	\param Xi For inexplicable reasons, the author has decided that is is best not to document his code
	\param Eta For inexplicable reasons, the author has decided that is is best not to document his code
	\param deltaT Time to move forward in one whole step

	\todo document me!
*/
hipError_t gpu_npt_pre_step(const gpu_pdata_arrays &pdata, const gpu_boxsize &box, const gpu_npt_data &d_npt_data, float Xi, float Eta, float deltaT)
	{
	// setup the grid to run the kernel
	int block_size = d_npt_data.block_size;
	dim3 grid( d_npt_data.NBlocks, 1, 1);
	dim3 threads(block_size, 1, 1);

	// bind the textures
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, pdata_accel_tex, pdata.accel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;
	
	// run the kernel
	float exp_v_fac = exp(-1.0f/4.0f*(Eta+Xi)*deltaT);
	float exp_r_fac = exp(1.0f/2.0f*Eta*deltaT);
	float box_len_scale = exp(Eta*deltaT);
	
	//printf("Eta = %f\n", Eta);
	//printf("Xi = %f\n", Xi);
	//printf("deltaT = %f\n", deltaT);

	gpu_npt_pre_step_kernel<<< grid, threads >>>(pdata, box, d_npt_data, exp_v_fac, exp_r_fac, deltaT, box_len_scale);

	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \param pdata Particle data arrays to integrate forward 1/2 step
	\param d_npt_data NPT data structure for storing data specific to NPT integration
	\param force_data_ptrs Pointers to the forces in device memory
	\param num_forces Number of forces in \a force_data_ptrs
	\param exp_v_fac For inexplicable reasons, the author has decided that is is best not to document his code
	\param deltaT Time to advance (for one full step)
	
	\todo document me
*/
extern "C" __global__ void gpu_npt_step_kernel(gpu_pdata_arrays pdata, gpu_npt_data d_npt_data, float4 **force_data_ptrs, int num_forces, float exp_v_fac, float deltaT)
	{
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_global = idx_local + pdata.local_beg;
	
	// note assumes mac is 1.0
	float4 accel = gpu_integrator_sum_forces_inline(idx_local, pdata.local_num, force_data_ptrs, num_forces);
	if (idx_local < pdata.local_num)
		{
		float4 vel = tex1Dfetch(pdata_vel_tex, idx_global);
			
		vel.x = vel.x*exp_v_fac*exp_v_fac + (1.0f/2.0f)*deltaT*exp_v_fac*accel.x;
		vel.y = vel.y*exp_v_fac*exp_v_fac + (1.0f/2.0f)*deltaT*exp_v_fac*accel.y;
		vel.z = vel.z*exp_v_fac*exp_v_fac + (1.0f/2.0f)*deltaT*exp_v_fac*accel.z;
		
		// write out data
		pdata.vel[idx_global] = vel;
		// since we calculate the acceleration, we need to write it for the next step
		pdata.accel[idx_global] = accel;
		}
	}

/*! \param pdata Particle Data to operate on
	\param d_npt_data NPT specific data structures
	\param force_data_ptrs Pointers to the forces in device memory
	\param num_forces Number of forces in \a force_data_ptrs
	\param Xi For inexplicable reasons, the author has decided that is is best not to document his code
	\param Eta For inexplicable reasons, the author has decided that is is best not to document his code
	\param deltaT Time to move forward in one whole step

	\todo document me!
*/
hipError_t gpu_npt_step(const gpu_pdata_arrays &pdata, const gpu_npt_data &d_npt_data, float4 **force_data_ptrs, int num_forces, float Xi, float Eta, float deltaT)
	{
	  // setup the grid to run the kernel
	  int block_size = d_npt_data.block_size;
	  dim3 grid( d_npt_data.NBlocks, 1, 1);
	  dim3 threads(block_size, 1, 1);
	  float exp_v_fac = exp(-1.0f/4.0f*(Eta+Xi)*deltaT);

	  // bind the texture
	  hipError_t error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	  if (error != hipSuccess)
		return error;

	  // run the kernel
	  gpu_npt_step_kernel<<< grid, threads >>>(pdata, d_npt_data, force_data_ptrs, num_forces, exp_v_fac, deltaT);
	  
	  if (!g_gpu_error_checking)
	        {
		  return hipSuccess;
		}
	  else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}
	
//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \param d_npt_data NPT specific data structures
	\todo document me!
*/
extern "C" __global__ void gpu_npt_reduce_ksum_kernel(gpu_npt_data d_npt_data)
	{
	float Ksum = 0.0f;

	// sum up the values in the partial sum via a sliding window
	for (int start = 0; start < d_npt_data.NBlocks; start += blockDim.x)
		{
		__syncthreads();
		if (start + threadIdx.x < d_npt_data.NBlocks)
			npt_sdata[threadIdx.x] = d_npt_data.partial_Ksum[start + threadIdx.x];
		else
			npt_sdata[threadIdx.x] = 0.0f;
		__syncthreads();

		// reduce the sum in parallel
		int offs = blockDim.x >> 1;
		while (offs > 0)
			{
			if (threadIdx.x < offs)
				npt_sdata[threadIdx.x] += npt_sdata[threadIdx.x + offs];
			offs >>= 1;
			__syncthreads();
			}

		// everybody sums up Ksum
		Ksum += npt_sdata[0];
		}
	
	if (threadIdx.x == 0)
	  {
		*d_npt_data.Ksum = Ksum;
		//printf("Ksum = %f\n", Ksum);
	  }
	}
	
/*! \param d_npt_data NPT specific data structures
	\todo document me!
*/
hipError_t gpu_npt_reduce_ksum(const gpu_npt_data &d_npt_data)
	{
	// setup the grid to run the kernel
	int block_size = 128;
	dim3 grid( 1, 1, 1);
	dim3 threads(block_size, 1, 1);
	
	// run the kernel
	gpu_npt_reduce_ksum_kernel<<< grid, threads, block_size*sizeof(float) >>>(d_npt_data);
	
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	\todo document me!
*/
extern "C" __global__ void gpu_npt_temperature_kernel(gpu_npt_data d_npt_data, gpu_pdata_arrays pdata)
	{
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_global = idx_local + pdata.local_beg;
	
	float vsq;
	if (idx_local < pdata.local_num)
		{
		float4 vel = tex1Dfetch(pdata_vel_tex, idx_global);
		vsq = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
		}
	else
		{
		vsq = 0.0f;
		}
		
	npt_sdata[threadIdx.x] = vsq;
	__syncthreads();

	// reduce the sum in parallel
	int offs = blockDim.x >> 1;
	while (offs > 0)
		{
		if (threadIdx.x < offs)
			npt_sdata[threadIdx.x] += npt_sdata[threadIdx.x + offs];
		offs >>= 1;
		__syncthreads();
		}

	// write out our partial sum
	if (threadIdx.x == 0)
		{
		d_npt_data.partial_Ksum[blockIdx.x] = npt_sdata[0];
		}
	}

/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	\todo document me!
*/
hipError_t gpu_npt_temperature(const gpu_npt_data &d_npt_data, const gpu_pdata_arrays &pdata)
	{
	// setup the grid to run the kernel
	int block_size = d_npt_data.block_size;
	dim3 grid( 1, 1, 1);
	dim3 threads(block_size, 1, 1);

	// bind velocity to the texture
	hipError_t error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	// run the kernel
	gpu_npt_temperature_kernel<<< grid, threads, block_size*sizeof(float) >>>(d_npt_data, pdata);
	
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}


//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute pressure of
	\todo document me!
*/
extern "C" __global__ void gpu_npt_reduce_psum_kernel(gpu_npt_data d_npt_data)
	{
	float Psum = 0.0f;

	// sum up the values in the partial sum via a sliding window
	for (int start = 0; start < d_npt_data.NBlocks; start += blockDim.x)
		{
		__syncthreads();
		if (start + threadIdx.x < d_npt_data.NBlocks)
			npt_sdata[threadIdx.x] = d_npt_data.partial_Psum[start + threadIdx.x];
		else
			npt_sdata[threadIdx.x] = 0.0f;
		__syncthreads();

		// reduce the sum in parallel
		int offs = blockDim.x >> 1;
		while (offs > 0)
			{
			if (threadIdx.x < offs)
				npt_sdata[threadIdx.x] += npt_sdata[threadIdx.x + offs];
			offs >>= 1;
			__syncthreads();
			}

		// everybody sums up Psum
		Psum += npt_sdata[0];
		}
	
	if (threadIdx.x == 0)
	  {
		*d_npt_data.Psum = Psum;
		//printf("Psum = %f\n", Psum);
	  }
	}

/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	\todo document me!
*/
hipError_t gpu_npt_reduce_psum(const gpu_npt_data &d_npt_data)
	{
	// setup the grid to run the kernel
	int block_size = 128;
	dim3 grid( 1, 1, 1);
	dim3 threads(block_size, 1, 1);
	
	// run the kernel
	gpu_npt_reduce_psum_kernel<<< grid, threads, block_size*sizeof(float) >>>(d_npt_data);
	//printf("d_npt_data.Psum = %f\n", (*d_npt_data).Psum);
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	\todo document me!
*/
extern "C" __global__ void gpu_npt_pressure_kernel(gpu_npt_data d_npt_data, gpu_pdata_arrays pdata)
	{
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	// do Nose-Hoover integrate ??? copied and pasted comment doesn't apply
	
	//printf("pdata.local_num = %d\n",  pdata.local_num);
	
	float virial = 0.0f;
	if (idx_local < pdata.local_num)
	  {
		virial = d_npt_data.virial[idx_local];
		 //printf("virial[%d] = %f\n", idx_local, virial);
	  }	

	npt_sdata[threadIdx.x] = virial;
	__syncthreads();

	// reduce the sum in parallel
	int offs = blockDim.x >> 1;
	while (offs > 0)
		{
		if (threadIdx.x < offs)
			npt_sdata[threadIdx.x] += npt_sdata[threadIdx.x + offs];
		offs >>= 1;
		__syncthreads();
		}

	// write out our partial sum
	if (threadIdx.x == 0)
		{
		d_npt_data.partial_Psum[blockIdx.x] = npt_sdata[0];
		}
	}

/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	\todo document me!
*/
hipError_t gpu_npt_pressure(const gpu_npt_data &d_npt_data, const gpu_pdata_arrays &pdata)
	{
	// setup the grid to run the kernel
	int block_size = d_npt_data.block_size;
	dim3 grid( 1, 1, 1);
	dim3 threads(block_size, 1, 1);

	// run the kernel
	gpu_npt_pressure_kernel<<< grid, threads, block_size*sizeof(float) >>>(d_npt_data, pdata);
	
	if (!g_gpu_error_checking)
		{
		 return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}	

// vim:syntax=cpp
