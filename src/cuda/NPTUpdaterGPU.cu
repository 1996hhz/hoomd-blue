#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/


#include "NPTUpdaterGPU.cuh"
#include "Integrator.cuh"
#include "gpu_settings.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

#include <stdio.h>

/*! \file NPTUpdaterGPU.cu
	\brief Defines GPU kernel code for NPT integration on the GPU. Used by NPTUpdaterGPU.
*/

//! Texture for reading the pdata pos array
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;
//! Texture for reading the pdata vel array
texture<float4, 1, hipReadModeElementType> pdata_vel_tex;
//! Texture for reading the pdata accel array
texture<float4, 1, hipReadModeElementType> pdata_accel_tex;

//! Shared data used by NPT kernels for sum reductions
extern __shared__ float npt_sdata[];

//! Sums virials from many different ForceComputes all in an inline function that can be included in any kernel.
/*! \param idx_local Local index of the running thread
	\param local_num Number of particles local to this GPU
	\param virial_data_ptrs Pointer to a list of pointers which are the arrays of virial data from the various ForceComputes
	\param num_virials Number of virials listed in \a virial_data_ptrs
	
	\note Every thread in the grid must call this function: it needs to __syncthreads()
	\note A maximum of 32 virials can be given in virial_data_ptrs
	
	gpu_integrator_sum_virials_inline() is designed to be run on one thread per particle with the
	normal thread breakdown of idx_local = threadIdx.x + blockDim.x * blockIdx.x. Full memory coalescing
	is achieved when this is the case. Each thread loops through the data pointers (which are cached
	in shared memory) and sums up the virial for particle idx_local.
	
	This inlined call is designed to be used from within other kernels.
*/
__device__ float gpu_integrator_sum_virials_inline(unsigned int idx_local, unsigned int local_num, float **virial_data_ptrs, int num_virials)
	{
	// each block loads in the pointers
	__shared__ float *virial_ptrs[32];
	if (threadIdx.x < 32)
		virial_ptrs[threadIdx.x] = virial_data_ptrs[threadIdx.x];
	__syncthreads();

	float virial = 0.0f;
	if (idx_local < local_num)
		{
		// sum the virials
		for (int i = 0; i < num_virials; i++)
			{
			float *d_virial = virial_ptrs[i];
			float v = d_virial[idx_local];
			virial += v;
			}
		}
	// return the result
	return virial;
	}

//! Sums the varoius virials on the GPU
/*! \param nptdata NPT data storage structure
	\param pdata Particle data arrays
	\param virial_data_ptrs list of virial data pointers
	\param num_virials number of virial points in the list

	\a virial_data_ptrs contains up to 32 pointers. Each points to pdata.local_num float's in memory
	All virials are summed into nptdata.virial
	
	gpu_integrator_sum_virials_kernel() is a simple driver to that uses gpu_integrator_sum_virials_inline()
	to compute the per-particle virial sums into the memory provided in nptdata.virial. One thread
	per particle is run with an arbitrary block size (a multiple of the warp size for coalescing).
*/
extern "C" __global__ void gpu_integrator_sum_virials_kernel(gpu_npt_data nptdata, gpu_pdata_arrays pdata, float **virial_data_ptrs, int num_virials)
	{
	// calculate the index we will be handling
	int idx_local = blockDim.x * blockIdx.x + threadIdx.x;

	float virial = gpu_integrator_sum_virials_inline(idx_local, pdata.local_num, virial_data_ptrs, num_virials);

	if (idx_local < pdata.local_num)
		{
		// write out the result
		nptdata.virial[idx_local] = virial;
		}
	}

/*! Every virial on every particle is summed up into \a nptpdata.virial

	\param nptdata NPT data storage structure
	\param pdata Particle data to write virial sum to
	\param virial_list List of pointers to virial data to sum
	\param num_virials Number of forces in \a virial_list

	\returns Any error code from the kernel call retrieved via hipGetLastError()
	
	This is just a kernel driver for gpu_integrator_sum_virials_kernel(). See it for more details.
*/
hipError_t gpu_integrator_sum_virials(const gpu_npt_data &nptdata, const gpu_pdata_arrays &pdata, float** virial_list, int num_virials)
	{
	// sanity check
	assert(num_virials < 32);

	const int block_size = 192;

	gpu_integrator_sum_virials_kernel<<< pdata.local_num/block_size+1, block_size >>>(nptdata, pdata, virial_list, num_virials);

	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}


//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \param pdata Particle data arrays to integrate forward 1/2 step
	\param box Box dimensions that the particles are in
	\param d_npt_data NPT data structure for storing data specific to NPT integration
	\param exp_v_fac For inexplicable reasons, the author has decided that is is best not to document his code
	\param exp_r_fac For inexplicable reasons, the author has decided that is is best not to document his code
	\param deltaT Time to advance (for one full step)
	\param box_len_scale For inexplicable reasons, the author has decided that is is best not to document his code
	
	\todo document me
*/
extern "C" __global__ void gpu_npt_pre_step_kernel(gpu_pdata_arrays pdata, gpu_boxsize box, gpu_npt_data d_npt_data, float exp_v_fac, float exp_r_fac, float deltaT, float box_len_scale)
	{
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_global = idx_local + pdata.local_beg;
	// do Nose-Hoover integrate ??? Copied and pasted comment doesn't apply
	
	if (idx_local < pdata.local_num)
		{
		// update positions to the next timestep and update velocities to the next half step
		float4 pos = tex1Dfetch(pdata_pos_tex, idx_global);
		
		float px = pos.x;
		float py = pos.y;
		float pz = pos.z;
		float pw = pos.w;
		
		float4 vel = tex1Dfetch(pdata_vel_tex, idx_global);
		float4 accel = tex1Dfetch(pdata_accel_tex, idx_global);
		
		vel.x = vel.x*exp_v_fac*exp_v_fac + (1.0f/2.0f) * deltaT*exp_v_fac*accel.x;
		px = px*exp_r_fac*exp_r_fac + vel.x*exp_r_fac*deltaT;

		vel.y = vel.y*exp_v_fac*exp_v_fac + (1.0f/2.0f) * deltaT*exp_v_fac*accel.y;
		py = py*exp_r_fac*exp_r_fac + vel.y*exp_r_fac*deltaT;

		vel.z = vel.z*exp_v_fac*exp_v_fac + (1.0f/2.0f) * deltaT*exp_v_fac*accel.z;
		pz = pz*exp_r_fac*exp_r_fac + vel.z*exp_r_fac*deltaT;

	
		// time to fix the periodic boundary conditions	
		px -= box_len_scale*box.Lx * rintf(px * box.Lxinv/box_len_scale);
		py -= box_len_scale*box.Ly * rintf(py * box.Lyinv/box_len_scale);
		pz -= box_len_scale*box.Lz * rintf(pz * box.Lzinv/box_len_scale);
	
		float4 pos2;
		pos2.x = px;
		pos2.y = py;
		pos2.z = pz;
		pos2.w = pw;
						
		// write out the results
		pdata.pos[idx_global] = pos2;
		pdata.vel[idx_global] = vel;
		}
	
	}

/*! \param pdata Particle Data to operate on
	\param box Current box dimensions the particles are in
	\param d_npt_data NPT specific data structures
	\param Xi For inexplicable reasons, the author has decided that is is best not to document his code
	\param Eta For inexplicable reasons, the author has decided that is is best not to document his code
	\param deltaT Time to move forward in one whole step

	This is just a kernel driver for gpu_integrator_pre_step_kernel(). See it for more details.
*/
hipError_t gpu_npt_pre_step(const gpu_pdata_arrays &pdata, const gpu_boxsize &box, const gpu_npt_data &d_npt_data, float Xi, float Eta, float deltaT)
	{
	// setup the grid to run the kernel
	int block_size = d_npt_data.block_size;
	dim3 grid( d_npt_data.NBlocks, 1, 1);
	dim3 threads(block_size, 1, 1);

	// bind the textures
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, pdata_accel_tex, pdata.accel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;
	
	// run the kernel
	float exp_v_fac = exp(-1.0f/4.0f*(Eta+Xi)*deltaT);
	float exp_r_fac = exp(1.0f/2.0f*Eta*deltaT);
	float box_len_scale = exp(Eta*deltaT);
	
	gpu_npt_pre_step_kernel<<< grid, threads >>>(pdata, box, d_npt_data, exp_v_fac, exp_r_fac, deltaT, box_len_scale);

	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

//! For inexplicable reasons, the author has decided that is is best not to document his code
/*! \param pdata Particle data arrays to integrate forward 1/2 step
	\param d_npt_data NPT data structure for storing data specific to NPT integration
	\param force_data_ptrs Pointers to the forces in device memory
	\param num_forces Number of forces in \a force_data_ptrs
	\param exp_v_fac For inexplicable reasons, the author has decided that is is best not to document his code
	\param deltaT Time to advance (for one full step)
	
	\todo document me
*/
extern "C" __global__ void gpu_npt_step_kernel(gpu_pdata_arrays pdata, gpu_npt_data d_npt_data, float4 **force_data_ptrs, int num_forces, float exp_v_fac, float deltaT)
	{
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_global = idx_local + pdata.local_beg;
	
	// note assumes mac is 1.0
	float4 accel = gpu_integrator_sum_forces_inline(idx_local, pdata.local_num, force_data_ptrs, num_forces);
	if (idx_local < pdata.local_num)
		{
		float4 vel = tex1Dfetch(pdata_vel_tex, idx_global);
			
		vel.x = vel.x*exp_v_fac*exp_v_fac + (1.0f/2.0f)*deltaT*exp_v_fac*accel.x;
		vel.y = vel.y*exp_v_fac*exp_v_fac + (1.0f/2.0f)*deltaT*exp_v_fac*accel.y;
		vel.z = vel.z*exp_v_fac*exp_v_fac + (1.0f/2.0f)*deltaT*exp_v_fac*accel.z;
		
		// write out data
		pdata.vel[idx_global] = vel;
		// since we calculate the acceleration, we need to write it for the next step
		pdata.accel[idx_global] = accel;
		}
	}

/*! \param pdata Particle Data to operate on
	\param d_npt_data NPT specific data structures
	\param force_data_ptrs Pointers to the forces in device memory
	\param num_forces Number of forces in \a force_data_ptrs
	\param Xi For inexplicable reasons, the author has decided that is is best not to document his code
	\param Eta For inexplicable reasons, the author has decided that is is best not to document his code
	\param deltaT Time to move forward in one whole step

	This is just a kernel driver for gpu_npt_step_kernel(). See it for more details.
*/
hipError_t gpu_npt_step(const gpu_pdata_arrays &pdata, const gpu_npt_data &d_npt_data, float4 **force_data_ptrs, int num_forces, float Xi, float Eta, float deltaT)
	{
	  // setup the grid to run the kernel
	  int block_size = d_npt_data.block_size;
	  dim3 grid( d_npt_data.NBlocks, 1, 1);
	  dim3 threads(block_size, 1, 1);
	  float exp_v_fac = exp(-1.0f/4.0f*(Eta+Xi)*deltaT);

	  // bind the texture
	  hipError_t error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	  if (error != hipSuccess)
		return error;

	  // run the kernel
	  gpu_npt_step_kernel<<< grid, threads >>>(pdata, d_npt_data, force_data_ptrs, num_forces, exp_v_fac, deltaT);
	  
	  if (!g_gpu_error_checking)
	        {
		  return hipSuccess;
		}
	  else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}
	
//! Completes the sums of m*v^2 over every particle in the simulation
/*! \param d_npt_data NPT specific data structures
	
	\pre gpu_npt_temperature_kernel() must be called first to fill out the partial sums in \a d_npt_data.
	\a d_npt_data.NBlocks partial sums are written there to be added up here.
	
	gpu_npt_reduce_ksum_kernel() is a very simple 1-block kernel run that completes the partial sums
	and writes the final m*v^2 sum for this GPU out to *d_npt_data.Ksum. It must be run with one
	block and a power of 2 for a block size with blockDim.x*sizeof(float) bytes of dynamic shared 
	memory allocated.
	
	The kernel works by going over the list of partial sums with a sliding window blockDim.x threads
	wide. Each thread participates in a fully coalesced load of the partial sums and then a parallel 
	reduction is employed to complete the sum.
*/
extern "C" __global__ void gpu_npt_reduce_ksum_kernel(gpu_npt_data d_npt_data)
	{
	float Ksum = 0.0f;

	// sum up the values in the partial sum via a sliding window
	for (int start = 0; start < d_npt_data.NBlocks; start += blockDim.x)
		{
		__syncthreads();
		if (start + threadIdx.x < d_npt_data.NBlocks)
			npt_sdata[threadIdx.x] = d_npt_data.partial_Ksum[start + threadIdx.x];
		else
			npt_sdata[threadIdx.x] = 0.0f;
		__syncthreads();

		// reduce the sum in parallel
		int offs = blockDim.x >> 1;
		while (offs > 0)
			{
			if (threadIdx.x < offs)
				npt_sdata[threadIdx.x] += npt_sdata[threadIdx.x + offs];
			offs >>= 1;
			__syncthreads();
			}

		// everybody sums up Ksum
		Ksum += npt_sdata[0];
		}
	
	if (threadIdx.x == 0)
		{
		*d_npt_data.Ksum = Ksum;
		}
	}
	
/*! \param d_npt_data NPT specific data structures
	
	This is just a driver for gpu_npt_reduce_ksum_kernel(). See it for more details.
*/
hipError_t gpu_npt_reduce_ksum(const gpu_npt_data &d_npt_data)
	{
	// setup the grid to run the kernel
	int block_size = 128;
	dim3 grid( 1, 1, 1);
	dim3 threads(block_size, 1, 1);
	
	// run the kernel
	gpu_npt_reduce_ksum_kernel<<< grid, threads, block_size*sizeof(float) >>>(d_npt_data);
	
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

//! Computes the first-pass m*v^2 sum
/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	
	\a d_npt_data.NBlocks blocks are to be run with \a d_npt_data.block_size width. Each thread
	reads in the velocity of a single particle, calculates m*v^2 and then each block makes a 
	parallel reduction pass to compute the partial m*v^2 sums. \a d_npt_data.NBlocks partial sums
	are written out to \a d_npt_data.partial_Ksum which will be later summed in gpu_npt_reduce_ksum_kernel().
*/
extern "C" __global__ void gpu_npt_temperature_kernel(gpu_npt_data d_npt_data, gpu_pdata_arrays pdata)
	{
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_global = idx_local + pdata.local_beg;
	
	float vsq;
	if (idx_local < pdata.local_num)
		{
		float4 vel = tex1Dfetch(pdata_vel_tex, idx_global);
		vsq = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
		}
	else
		{
		vsq = 0.0f;
		}
		
	npt_sdata[threadIdx.x] = vsq;
	__syncthreads();

	// reduce the sum in parallel
	int offs = blockDim.x >> 1;
	while (offs > 0)
		{
		if (threadIdx.x < offs)
			npt_sdata[threadIdx.x] += npt_sdata[threadIdx.x + offs];
		offs >>= 1;
		__syncthreads();
		}

	// write out our partial sum
	if (threadIdx.x == 0)
		{
		d_npt_data.partial_Ksum[blockIdx.x] = npt_sdata[0];
		}
	}

/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	
	This is just a driver for gpu_npt_temperature_kernel(). See it for more details.
*/
hipError_t gpu_npt_temperature(const gpu_npt_data &d_npt_data, const gpu_pdata_arrays &pdata)
	{
	// setup the grid to run the kernel
	int block_size = d_npt_data.block_size;
	dim3 grid(d_npt_data.NBlocks, 1, 1);
	dim3 threads(block_size, 1, 1);

	// bind velocity to the texture
	hipError_t error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	// run the kernel
	gpu_npt_temperature_kernel<<< grid, threads, block_size*sizeof(float) >>>(d_npt_data, pdata);
	
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}


//! Completes the virial sum over every particle in the simulation
/*! \param d_npt_data NPT specific data structures
	
	\pre gpu_npt_pressure_kernel() must be called first to fill out the partial sums in \a d_npt_data.
	\a d_npt_data.NBlocks partial sums are written there to be added up here.
	
	gpu_npt_reduce_psum_kernel() is a very simple 1-block kernel run that completes the partial sums
	and writes the final virial sum for this GPU out to *d_npt_data.Psum. It must be run with one
	block and a power of 2 for a block size with blockDim.x*sizeof(float) bytes of dynamic shared 
	memory allocated.
	
	The kernel works by going over the list of partial sums with a sliding window blockDim.x threads
	wide. Each thread participates in a fully coalesced load of the partial sums and then a parallel 
	reduction is employed to complete the sum.
*/
extern "C" __global__ void gpu_npt_reduce_psum_kernel(gpu_npt_data d_npt_data)
	{
	float Psum = 0.0f;

	// sum up the values in the partial sum via a sliding window
	for (int start = 0; start < d_npt_data.NBlocks; start += blockDim.x)
		{
		__syncthreads();
		if (start + threadIdx.x < d_npt_data.NBlocks)
			npt_sdata[threadIdx.x] = d_npt_data.partial_Psum[start + threadIdx.x];
		else
			npt_sdata[threadIdx.x] = 0.0f;
		__syncthreads();

		// reduce the sum in parallel
		int offs = blockDim.x >> 1;
		while (offs > 0)
			{
			if (threadIdx.x < offs)
				npt_sdata[threadIdx.x] += npt_sdata[threadIdx.x + offs];
			offs >>= 1;
			__syncthreads();
			}

		// everybody sums up Psum
		Psum += npt_sdata[0];
		}
	
	if (threadIdx.x == 0)
		{
		*d_npt_data.Psum = Psum;
	  	}
	}

/*! \param d_npt_data NPT specific data structures
		
	This is just a driver for gpu_npt_reduce_psum_kernel(). See it for more details.
*/
hipError_t gpu_npt_reduce_psum(const gpu_npt_data &d_npt_data)
	{
	// setup the grid to run the kernel
	int block_size = 128;
	dim3 grid( 1, 1, 1);
	dim3 threads(block_size, 1, 1);
	
	// run the kernel
	gpu_npt_reduce_psum_kernel<<< grid, threads, block_size*sizeof(float) >>>(d_npt_data);
	//printf("d_npt_data.Psum = %f\n", (*d_npt_data).Psum);
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

//! Computes the first-pass virial sum
/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	
	\a d_npt_data.NBlocks blocks are to be run with \a d_npt_data.block_size width. Each thread
	reads in the total virial on a single particle and then each block makes a 
	parallel reduction pass to compute the partial virial sums. \a d_npt_data.NBlocks partial sums
	are written out to \a d_npt_data.partial_Psum which will be later summed in gpu_npt_reduce_psum_kernel().
*/
extern "C" __global__ void gpu_npt_pressure_kernel(gpu_npt_data d_npt_data, gpu_pdata_arrays pdata)
	{
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	// do Nose-Hoover integrate ??? copied and pasted comment doesn't apply
	
	//printf("pdata.local_num = %d\n",  pdata.local_num);
	
	float virial = 0.0f;
	if (idx_local < pdata.local_num)
		{
		virial = d_npt_data.virial[idx_local];
		}

	npt_sdata[threadIdx.x] = virial;
	__syncthreads();

	// reduce the sum in parallel
	int offs = blockDim.x >> 1;
	while (offs > 0)
		{
		if (threadIdx.x < offs)
			npt_sdata[threadIdx.x] += npt_sdata[threadIdx.x + offs];
		offs >>= 1;
		__syncthreads();
		}

	// write out our partial sum
	if (threadIdx.x == 0)
		{
		d_npt_data.partial_Psum[blockIdx.x] = npt_sdata[0];
		}
	}

/*! \param d_npt_data NPT specific data structures
	\param pdata Particle data to compute temperature of
	
	This is just a driver function for gpu_npt_pressure_kernel(). See it for more details.
*/
hipError_t gpu_npt_pressure(const gpu_npt_data &d_npt_data, const gpu_pdata_arrays &pdata)
	{
	// setup the grid to run the kernel
	int block_size = d_npt_data.block_size;
	dim3 grid(d_npt_data.NBlocks, 1, 1);
	dim3 threads(block_size, 1, 1);

	// run the kernel
	gpu_npt_pressure_kernel<<< grid, threads, block_size*sizeof(float) >>>(d_npt_data, pdata);
	
	if (!g_gpu_error_checking)
		{
		 return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

// vim:syntax=cpp
