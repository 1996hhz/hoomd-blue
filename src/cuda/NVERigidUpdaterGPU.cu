#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id: NVEUpdaterGPU.cu 1676 2009-02-13 19:15:18Z joaander $
// $URL: http://svn2.assembla.com/svn/hoomd/trunk/src/cuda/NVEUpdaterGPU.cu $

#include "Integrator.cuh"
#include "NVERigidUpdaterGPU.cuh"
#include "gpu_settings.h"
#include <stdio.h>

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

#include <stdio.h>

/*! \file NVERigidUpdaterGPU.cu
	\brief Defines GPU kernel code for NVE integration on the GPU. Used by NVEUpdaterGPU.
*/

#define INVALID_INDEX 0xffffffff // identical to the sentinel value NO_INDEX in RigidData.h

//! The texture for reading the pdata pos array
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;
//! The texture for reading the pdata vel array
texture<float4, 1, hipReadModeElementType> pdata_vel_tex;
//! The texture for reading the pdata accel array
texture<float4, 1, hipReadModeElementType> pdata_accel_tex;
//! The texture for reading in the pdata image array
texture<int4, 1, hipReadModeElementType> pdata_image_tex;
//! The texture for reading in the pdata mass array
texture<float, 1, hipReadModeElementType> pdata_mass_tex;

//! The texture for reading the rigid data body mass array
texture<float, 1, hipReadModeElementType> rigid_data_body_mass_tex;
//! The texture for reading the rigid data moment of inertia array
texture<float4, 1, hipReadModeElementType> rigid_data_moment_inertia_tex;
//! The texture for reading the rigid data com array
texture<float4, 1, hipReadModeElementType> rigid_data_com_tex;
//! The texture for reading the rigid data vel array
texture<float4, 1, hipReadModeElementType> rigid_data_vel_tex;
//! The texture for reading the rigid data angualr momentum array
texture<float4, 1, hipReadModeElementType> rigid_data_angmom_tex;
//! The texture for reading the rigid data angular velocity array
texture<float4, 1, hipReadModeElementType> rigid_data_angvel_tex;
//! The texture for reading the rigid data orientation array
texture<float4, 1, hipReadModeElementType> rigid_data_orientation_tex;
//! The texture for reading the rigid data ex space array
texture<float4, 1, hipReadModeElementType> rigid_data_exspace_tex;
//! The texture for reading the rigid data ey space array
texture<float4, 1, hipReadModeElementType> rigid_data_eyspace_tex;
//! The texture for reading the rigid data ez space array
texture<float4, 1, hipReadModeElementType> rigid_data_ezspace_tex;
//! The texture for reading in the rigid data body image array
texture<int, 1, hipReadModeElementType> rigid_data_body_imagex_tex;
//! The texture for reading in the rigid data body image array
texture<int, 1, hipReadModeElementType> rigid_data_body_imagey_tex;
//! The texture for reading in the rigid data body image array
texture<int, 1, hipReadModeElementType> rigid_data_body_imagez_tex;
//! The texture for reading the rigid data particle position array
texture<float4, 1, hipReadModeElementType> rigid_data_particle_pos_tex;
//! The texture for reading the rigid data particle indices array
texture<unsigned int, 1, hipReadModeElementType> rigid_data_particle_indices_tex;
//! The texture for reading the rigid data force array
texture<float4, 1, hipReadModeElementType> rigid_data_force_tex;
//! The texture for reading the rigid data torque array
texture<float4, 1, hipReadModeElementType> rigid_data_torque_tex;

#pragma mark HELPER
//! Helper functions for rigid body quaternion update

__device__ void exyzFromQuaternion(float4& quat, float4& ex_space, float4& ey_space, float4& ez_space)
	{
	// ex_space
	ex_space.x = quat.x * quat.x + quat.y * quat.y - quat.z * quat.z - quat.w * quat.w;
	ex_space.y = 2.0 * (quat.y * quat.z + quat.x * quat.w);
	ex_space.z = 2.0 * (quat.y * quat.w - quat.x * quat.z);
	
	// ey_space
	ey_space.x = 2.0 * (quat.y * quat.z - quat.x * quat.w);
	ey_space.y = quat.x * quat.x - quat.y * quat.y + quat.z * quat.z - quat.w * quat.w;
	ey_space.z = 2.0 * (quat.z * quat.w + quat.x * quat.y);
	
	// ez_space
	ez_space.x = 2.0 * (quat.y * quat.w + quat.x * quat.z);
	ez_space.y = 2.0 * (quat.z * quat.w - quat.x * quat.y);
	ez_space.z = quat.x * quat.x - quat.y * quat.y - quat.z * quat.z + quat.w * quat.w;
	}

__device__ void computeAngularVelocity(float4& angmom, float4& moment_inertia, float4& ex_space, float4& ey_space, float4& ez_space, float4& angvel)
	{
	//! Angular velocity in the body frame
	float4 angbody;
	
	//! angbody = angmom_body / moment_inertia = transpose(rotation_matrix) * angmom / moment_inertia  
	if (moment_inertia.x == 0.0) angbody.x = 0.0;
	else angbody.x = (ex_space.x * angmom.x + ex_space.y * angmom.y 
					   + ex_space.z * angmom.z) / moment_inertia.x;
	
	if (moment_inertia.y == 0.0) angbody.y = 0.0;
	else angbody.y = (ey_space.x * angmom.x + ey_space.y * angmom.y
					   + ey_space.z * angmom.z) / moment_inertia.y;
	
	if (moment_inertia.z == 0.0) angbody.z = 0.0;
	else angbody.z = (ez_space.x * angmom.x + ez_space.y * angmom.y 
					   + ez_space.z * angmom.z) / moment_inertia.z;
	
	//! Convert to angbody to the space frame: angvel = rotation_matrix * angbody
	angvel.x = angbody.x * ex_space.x + angbody.y * ey_space.x + angbody.z * ez_space.x;
	angvel.y = angbody.x * ex_space.y + angbody.y * ey_space.y + angbody.z * ez_space.y;
	angvel.z = angbody.x * ex_space.z + angbody.y * ey_space.z + angbody.z * ez_space.z;
	}

/*! Quaternion multiply: c = a * b where a = (0, a)
 */

__device__ void multiply(float4& a, float4& b, float4& c)
	{
	c.x = -(a.x * b.y + a.y * b.z + a.z * b.w);
	c.y =   b.x * a.x + a.y * b.w - a.z * b.z;
	c.z =   b.x * a.y + a.z * b.y - a.x * b.w;
	c.w =   b.x * a.z + a.x * b.z - a.y * b.y;
	}

/*! Normalize a quaternion
 */

__device__ void normalize(float4 &q)
	{
	float norm = 1.0 / sqrt(q.x * q.x + q.y * q.y + q.z * q.z + q.w * q.w);
	q.x *= norm;
	q.y *= norm;
	q.z *= norm;
	q.w *= norm;
	}

/*! Advance the quaternion using angular momentum and angular velocity
 */
__device__ void advanceQuaternion(float4& angmom, float4& moment_inertia, float4& angvel, float4& ex_space, float4& ey_space, float4& ez_space, float4& quat, float deltaT)
	{
	float4 qhalf, qfull, omegaq;
	float dtq = 0.5 * deltaT;
	
	computeAngularVelocity(angmom, moment_inertia, ex_space, ey_space, ez_space, angvel);
	
	// Compute (w q)
	multiply(angvel, quat, omegaq);
	
	// Full update q from dq/dt = 1/2 w q
	qfull.x = quat.x + dtq * omegaq.x;
	qfull.y = quat.y + dtq * omegaq.y;
	qfull.z = quat.z + dtq * omegaq.z;
	qfull.w = quat.w + dtq * omegaq.w;
	normalize(qfull);
	
	// 1st half update from dq/dt = 1/2 w q
	qhalf.x = quat.x + 0.5 * dtq * omegaq.x;
	qhalf.y = quat.y + 0.5 * dtq * omegaq.y;
	qhalf.z = quat.z + 0.5 * dtq * omegaq.z;
	qhalf.w = quat.w + 0.5 * dtq * omegaq.w;
	normalize(qhalf);
	
	// Udpate ex, ey, ez from qhalf = update A
	exyzFromQuaternion(qhalf, ex_space, ey_space, ez_space);
	
	// Compute angular velocity from new ex_space, ey_space and ex_space
	computeAngularVelocity(angmom, moment_inertia, ex_space, ey_space, ez_space, angvel);
	
	// Compute (w qhalf)
	multiply(angvel, qhalf, omegaq);
	
	// 2nd half update from dq/dt = 1/2 w q
	qhalf.x += 0.5 * dtq * omegaq.x;
	qhalf.y += 0.5 * dtq * omegaq.y;
	qhalf.z += 0.5 * dtq * omegaq.z;
	qhalf.w += 0.5 * dtq * omegaq.w;
	normalize(qhalf);
	
	// Corrected Richardson update
	quat.x = 2.0 * qhalf.x - qfull.x;
	quat.y = 2.0 * qhalf.y - qfull.y;
	quat.z = 2.0 * qhalf.z - qfull.z;
	quat.w = 2.0 * qhalf.w - qfull.w;
	normalize(quat);
	
	exyzFromQuaternion(quat, ex_space, ey_space, ez_space);
	}

#pragma mark BODY_PRE_STEP_KERNEL
//! Takes the first half-step forward for rigid bodies in the velocity-verlet NVE integration
/*! \param rigid_data rigid data to step forward 1/2 step
	\param deltaT timestep
	\param limit If \a limit is true, then the dynamics will be limited so that particles do not move 
		a distance further than \a limit_val in one step.
	\param limit_val Length to limit particle distance movement to
	\param box Box dimensions for periodic boundary condition handling
*/
extern "C" __global__ void gpu_nve_rigid_body_pre_step_kernel(gpu_rigid_data_arrays rigid_data, gpu_boxsize box, float deltaT, bool limit, float limit_val)
	{
	unsigned int idx_local = blockIdx.x * blockDim.x + threadIdx.x;	
	unsigned int idx_body = idx_local + rigid_data.local_beg;   // since we bind ALL rigid bodies to texture
		
	// do velocity verlet update
	// v(t+deltaT/2) = v(t) + (1/2)a*deltaT
	// r(t+deltaT) = r(t) + v(t+deltaT/2)*deltaT
	 	
	if (idx_local < rigid_data.local_num) 
		{
		
		// read the body information
		float body_mass = tex1Dfetch(rigid_data_body_mass_tex, idx_body);
		float4 moment_inertia = tex1Dfetch(rigid_data_moment_inertia_tex, idx_body);
		float4 com = tex1Dfetch(rigid_data_com_tex, idx_body);
		float4 vel = tex1Dfetch(rigid_data_vel_tex, idx_body);
		float4 angmom = tex1Dfetch(rigid_data_angmom_tex, idx_body);
		float4 angvel = tex1Dfetch(rigid_data_angvel_tex, idx_body);
		float4 orientation = tex1Dfetch(rigid_data_orientation_tex, idx_body);
		float4 ex_space = tex1Dfetch(rigid_data_exspace_tex, idx_body);
		float4 ey_space = tex1Dfetch(rigid_data_eyspace_tex, idx_body);
		float4 ez_space = tex1Dfetch(rigid_data_ezspace_tex, idx_body);
		int body_imagex = tex1Dfetch(rigid_data_body_imagex_tex, idx_body);
		int body_imagey = tex1Dfetch(rigid_data_body_imagey_tex, idx_body);
		int body_imagez = tex1Dfetch(rigid_data_body_imagez_tex, idx_body);			
		float4 force = tex1Dfetch(rigid_data_force_tex, idx_body);
		float4 torque = tex1Dfetch(rigid_data_torque_tex, idx_body);

		// update the velocity
		float dtfm = (1.0f/2.0f) * deltaT / body_mass;
		float4 vel2;
		vel2.x = vel.x + dtfm * force.x;
		vel2.y = vel.y + dtfm * force.y;
		vel2.z = vel.z + dtfm * force.z;
		vel2.w = vel.w;
			
		// update the position
		float4 pos2;
		pos2.x = com.x + vel2.x * deltaT;
		pos2.y = com.y + vel2.y * deltaT;
		pos2.z = com.z + vel2.z * deltaT;
		pos2.w = com.w;
						
		// read in the body's image
		// read the body's velocity and acceleration (MEM TRANSFER: 16 bytes)
					
		// time to fix the periodic boundary conditions (FLOPS: 15)
		float x_shift = rintf(pos2.x * box.Lxinv);
		pos2.x -= box.Lx * x_shift;
		body_imagex += (int)x_shift;
		
		float y_shift = rintf(pos2.y * box.Lyinv);
		pos2.y -= box.Ly * y_shift;
		body_imagey += (int)y_shift;
		
		float z_shift = rintf(pos2.z * box.Lzinv);
		pos2.z -= box.Lz * z_shift;
		body_imagez += (int)z_shift;
	
		// update the angular momentum
		float4 angmom2;
		angmom2.x = angmom.x + (1.0f/2.0f) * deltaT * torque.x;
		angmom2.y = angmom.y + (1.0f/2.0f) * deltaT * torque.y;
		angmom2.z = angmom.z + (1.0f/2.0f) * deltaT * torque.z;
			
		advanceQuaternion(angmom2, moment_inertia, angvel, ex_space, ey_space, ez_space, orientation, deltaT); 
			
		// write out the results (MEM_TRANSFER: ? bytes)
		rigid_data.com[idx_body] = pos2;
		rigid_data.vel[idx_body] = vel2;
		rigid_data.angmom[idx_body] = angmom2;
		rigid_data.angvel[idx_body] = angvel;
		rigid_data.orientation[idx_body] = orientation;
		rigid_data.ex_space[idx_body] = ex_space;
		rigid_data.ey_space[idx_body] = ey_space;
		rigid_data.ez_space[idx_body] = ez_space;
		rigid_data.body_imagex[idx_body] = body_imagex;
		rigid_data.body_imagey[idx_body] = body_imagey;
		rigid_data.body_imagez[idx_body] = body_imagez;
		}
	}

#pragma mark PARTICLE_PRE_STEP_KERNEL

extern "C" __global__ void gpu_nve_rigid_particle_pre_step_kernel(gpu_pdata_arrays pdata, gpu_rigid_data_arrays rigid_data, gpu_boxsize box, float deltaT, bool limit, float limit_val)
	{
	unsigned int idx_particle = blockIdx.x * blockDim.x + threadIdx.x;	
	unsigned int idx_body = blockIdx.x + rigid_data.local_beg; 	

	unsigned int idx_particle_index = tex1Dfetch(rigid_data_particle_indices_tex, idx_particle);
	// Since we use nmax for all rigid bodies, there might be some empty slot for particles in a rigid body
	// the particle index of these empty slots is set to be INVALID_INDEX.
	if (idx_body < rigid_data.n_bodies && idx_particle_index != INVALID_INDEX) 
		{
		float4 com = tex1Dfetch(rigid_data_com_tex, idx_body);
		float4 vel = tex1Dfetch(rigid_data_vel_tex, idx_body);
		float4 angvel = tex1Dfetch(rigid_data_angvel_tex, idx_body);
		float4 ex_space = tex1Dfetch(rigid_data_exspace_tex, idx_body);
		float4 ey_space = tex1Dfetch(rigid_data_eyspace_tex, idx_body);
		float4 ez_space = tex1Dfetch(rigid_data_ezspace_tex, idx_body);
		float4 particle_pos = tex1Dfetch(rigid_data_particle_pos_tex, idx_particle);
		
		// project the position in the body frame to the space frame: ri = rotation_matrix * particle_pos
		float4 ri;
		ri.x = ex_space.x * particle_pos.x + ey_space.x * particle_pos.y + ez_space.x * particle_pos.z;
		ri.y = ex_space.y * particle_pos.x + ey_space.y * particle_pos.y + ez_space.y * particle_pos.z;
		ri.z = ex_space.z * particle_pos.x + ey_space.z * particle_pos.y + ez_space.z * particle_pos.z;
		
		// time to fix the periodic boundary conditions (FLOPS: 15)
		int4 image = tex1Dfetch(pdata_image_tex, idx_particle);
		float4 pos = tex1Dfetch(pdata_pos_tex, idx_particle);
		
		// x_particle = com + ri
		float4 pos2;
		pos2.x = com.x + ri.x;
		pos2.y = com.y + ri.y;
		pos2.z = com.z + ri.z;
		pos2.w = pos.w;
		
		float x_shift = rintf(pos2.x * box.Lxinv);
		pos2.x -= box.Lx * x_shift;
		image.x += (int)x_shift;
		
		float y_shift = rintf(pos2.y * box.Lyinv);
		pos2.y -= box.Ly * y_shift;
		image.y += (int)y_shift;
		
		float z_shift = rintf(pos2.z * box.Lzinv);
		pos2.z -= box.Lz * z_shift;
		image.z += (int)z_shift;
		
		// v_particle = vel + angvel x ri
		float4 vel2;
		vel2.x = vel.x + angvel.y * ri.z - angvel.z * ri.y;
		vel2.y = vel.y + angvel.z * ri.x - angvel.x * ri.z;
		vel2.z = vel.z + angvel.x * ri.y - angvel.y * ri.x;
		
		// write out the results (MEM_TRANSFER: ? bytes)
		pdata.pos[idx_particle_index] = pos2;
		pdata.vel[idx_particle_index] = vel2;
		pdata.image[idx_particle_index] = image;
		}

	}	


/*! \param pdata Particle data to step forward 1/2 step
	\param box Box dimensions for periodic boundary condition handling
	\param deltaT Amount of real time to step forward in one time step
	\param limit If \a limit is true, then the dynamics will be limited so that particles do not move 
		a distance further than \a limit_val in one step.
	\param limit_val Length to limit particle distance movement to
*/
hipError_t gpu_nve_rigid_body_pre_step(const gpu_pdata_arrays& pdata, const gpu_rigid_data_arrays& rigid_data, const gpu_boxsize &box, float deltaT, bool limit, float limit_val)
{
	unsigned int n_bodies = rigid_data.n_bodies;
	unsigned int nmax = rigid_data.nmax;
		
	// setup the grid to run the rigid body kernel 
	int body_block_size = 1; // for the initial step of rigid bodies, no need of particle data; each thread takes care of a rigid body
	dim3 body_grid(n_bodies / body_block_size, 1, 1);	
	dim3 body_threads(body_block_size, 1, 1);
		
	// setup the grid to run the particle kernel 
	int particle_block_size = nmax; // maximum number of particles in a rigid body: each thread in a block takes care of a particle in a rigid body
	dim3 particle_grid(n_bodies, 1, 1);	
	dim3 particle_threads(particle_block_size, 1, 1);

	// bind the textures for rigid bodies: body mass, com, vel, images, angmom, angvel, force and torque
	hipError_t error = hipBindTexture(0, rigid_data_body_mass_tex, rigid_data.body_mass, sizeof(float) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_moment_inertia_tex, rigid_data.moment_inertia, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_com_tex, rigid_data.com, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, rigid_data_vel_tex, rigid_data.vel, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, rigid_data_angvel_tex, rigid_data.angvel, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
		
	error = hipBindTexture(0, rigid_data_angmom_tex, rigid_data.angmom, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_orientation_tex, rigid_data.orientation, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_exspace_tex, rigid_data.ex_space, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
		
	error = hipBindTexture(0, rigid_data_eyspace_tex, rigid_data.ey_space, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_ezspace_tex, rigid_data.ez_space, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
		
	error = hipBindTexture(0, rigid_data_body_imagex_tex, rigid_data.body_imagex, sizeof(int) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_body_imagey_tex, rigid_data.body_imagey, sizeof(int) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_body_imagez_tex, rigid_data.body_imagez, sizeof(int) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_particle_pos_tex, rigid_data.particle_pos, sizeof(float4) * n_bodies * nmax);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_particle_indices_tex, rigid_data.particle_indices, sizeof(unsigned int) * n_bodies * nmax);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_force_tex, rigid_data.force, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, rigid_data_torque_tex, rigid_data.torque, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
		
	// bind the textures for particles: pos, vel, accel and image of ALL particles
	error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, pdata_accel_tex, pdata.accel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, pdata_image_tex, pdata.image, sizeof(int4) * pdata.N);
	if (error != hipSuccess)
		return error;

    // run the kernel for bodies
    gpu_nve_rigid_body_pre_step_kernel<<< body_grid, body_threads >>>(rigid_data, box, deltaT, limit, limit_val);
	
	// run the kernel for particles
	gpu_nve_rigid_particle_pre_step_kernel<<< particle_grid, particle_threads >>>(pdata, rigid_data, box, deltaT, limit, limit_val);
		
		
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
			
	}

#pragma mark BODY_STEP_KERNEL

//! Takes the 2nd 1/2 step forward in the velocity-verlet NVE integration scheme
/*! \param pdata Particle data to step forward in time
	\param force_data_ptrs List of pointers to forces on each particle
	\param num_forces Number of forces listed in \a force_data_ptrs
	\param deltaT Amount of real time to step forward in one time step
	\param limit If \a limit is true, then the dynamics will be limited so that particles do not move 
		a distance further than \a limit_val in one step.
	\param limit_val Length to limit particle distance movement to
*/

extern __shared__ float4 sum[];

extern "C" __global__ void gpu_nve_rigid_body_step_kernel(gpu_pdata_arrays pdata, gpu_rigid_data_arrays rigid_data, float4 **force_data_ptrs, int num_forces, float deltaT, bool limit, float limit_val)
	{
	int idx_particle = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_body = blockIdx.x + rigid_data.local_beg; 

	float4 *body_force = sum;
	float4 *body_torque = &sum[blockDim.x];
	
	// do velocity verlet update
	// v(t+deltaT/2) = v(t) + (1/2)a*deltaT
	
	body_force[threadIdx.x] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	body_torque[threadIdx.x] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	
	__syncthreads();
	
	unsigned int idx_particle_index = tex1Dfetch(rigid_data_particle_indices_tex, idx_particle);
	// Since we use nmax for all rigid bodies, there might be some empty slot for particles in a rigid body
	// the particle index of these empty slots is set to be 0xffffffff.
	float4 particle_accel = make_float4(0.0f, 0.0f, 0.0f, 0.0f);	
	particle_accel = gpu_integrator_sum_forces_inline(idx_particle_index, pdata.local_num, force_data_ptrs, num_forces);		
	if (idx_body < rigid_data.local_num && idx_particle_index != INVALID_INDEX)
		{
		// read the body information
		float4 ex_space = tex1Dfetch(rigid_data_exspace_tex, idx_body);
		float4 ey_space = tex1Dfetch(rigid_data_eyspace_tex, idx_body);
		float4 ez_space = tex1Dfetch(rigid_data_ezspace_tex, idx_body);
		float4 particle_pos = tex1Dfetch(rigid_data_particle_pos_tex, idx_particle);
			
		// calculate body force and torques
		float particle_mass = tex1Dfetch(pdata_mass_tex, idx_particle_index);
		
		// project the position in the body frame to the space frame: ri = rotation_matrix * particle_pos
		float4 ri, fi, torquei;
		ri.x = ex_space.x * particle_pos.x + ey_space.x * particle_pos.y + ez_space.x * particle_pos.z;
		ri.y = ex_space.y * particle_pos.x + ey_space.y * particle_pos.y + ez_space.y * particle_pos.z;
		ri.z = ex_space.z * particle_pos.x + ey_space.z * particle_pos.y + ez_space.z * particle_pos.z;
		ri.z = 0.0;
		
		fi.x = particle_mass * particle_accel.x;
		fi.y = particle_mass * particle_accel.y;
		fi.z = particle_mass * particle_accel.z;
		fi.w = 0.0;
	
		body_force[threadIdx.x].x = fi.x;
		body_force[threadIdx.x].y = fi.y;
		body_force[threadIdx.x].z = fi.z;
		body_force[threadIdx.x].w = 0.0;
		
		torquei.x = ri.y * fi.z - ri.z * fi.y;
		torquei.y = ri.z * fi.x - ri.x * fi.z;
		torquei.z = ri.x * fi.y - ri.y * fi.x;
		torquei.w = 0.0;
		
		body_torque[threadIdx.x].x = torquei.x;
		body_torque[threadIdx.x].y = torquei.y;
		body_torque[threadIdx.x].z = torquei.z;
		body_torque[threadIdx.x].w = torquei.w;
	
		
	//	printf("force on %d (%d) = %f\t%f\t%f; %f\n", idx_body, threadIdx.x, body_force[threadIdx.x].x, body_force[threadIdx.x].y, body_force[threadIdx.x].z, particle_mass);
	//	printf("torque on %d (%d) = %f\t%f\t%f; %f\n", idx_body, threadIdx.x, body_torque[threadIdx.x].x, body_torque[threadIdx.x].y, body_torque[threadIdx.x].z);
	//	printf("accel on %d (%d) = %f\t%f\t%f\n", idx_body, threadIdx.x, particle_accel.x, particle_accel.y, particle_accel.z);
	
		}
	
	__syncthreads();
			
	unsigned int offset = blockDim.x >> 1;
	
	while (offset > 0)
		{
		if (threadIdx.x < offset)
			{
			body_force[threadIdx.x].x += body_force[threadIdx.x + offset].x;
			body_force[threadIdx.x].y += body_force[threadIdx.x + offset].y;
			body_force[threadIdx.x].z += body_force[threadIdx.x + offset].z;
			body_force[threadIdx.x].w += body_force[threadIdx.x + offset].w;
								
			body_torque[threadIdx.x].x += body_torque[threadIdx.x + offset].x;
			body_torque[threadIdx.x].y += body_torque[threadIdx.x + offset].y;
			body_torque[threadIdx.x].z += body_torque[threadIdx.x + offset].z;
			body_torque[threadIdx.x].w += body_torque[threadIdx.x + offset].w;
			
			}
				
		offset >>= 1;
				
		__syncthreads();
		}	
	
	if (idx_body < rigid_data.local_num)
		{
		// Every thread now has its own copy of body force and torque
		float4 force2 = body_force[0];
		float4 torque2 = body_torque[0];
		
		float body_mass = tex1Dfetch(rigid_data_body_mass_tex, idx_body);
		float4 vel = tex1Dfetch(rigid_data_vel_tex, idx_body);
		float4 angmom = tex1Dfetch(rigid_data_angmom_tex, idx_body);
		
		// update the velocity
		float dtfm = (1.0f/2.0f) * deltaT / body_mass;
		float4 vel2;
		vel2.x = vel.x + dtfm * force2.x;
		vel2.y = vel.y + dtfm * force2.y;
		vel2.z = vel.z + dtfm * force2.z;
		
		// update the angular momentum
		float4 angmom2;
		angmom2.x = angmom.x + (1.0f/2.0f) * deltaT * torque2.x;
		angmom2.y = angmom.y + (1.0f/2.0f) * deltaT * torque2.y;
		angmom2.z = angmom.z + (1.0f/2.0f) * deltaT * torque2.z;
		
		// write out the results
		rigid_data.force[idx_body] = force2;
		rigid_data.torque[idx_body] = torque2;
		rigid_data.vel[idx_body] = vel2;
		rigid_data.angmom[idx_body] = angmom2;
		
		}
	}

#pragma mark PARTICLE_STEP_KERNEL

extern "C" __global__ void gpu_nve_rigid_particle_step_kernel(gpu_pdata_arrays pdata, gpu_rigid_data_arrays rigid_data, bool limit, float limit_val)
	{
	int idx_particle = blockIdx.x * blockDim.x + threadIdx.x; // each thread for a particle in a rigid body
	int idx_body = blockIdx.x + rigid_data.local_beg; 
	
	unsigned int idx_particle_index = tex1Dfetch(rigid_data_particle_indices_tex, idx_particle);
	// Since we use nmax for all rigid bodies, there might be some empty slot for particles in a rigid body
	// the particle index of these empty slots is set to be 0xffffffff.
	if (idx_body < rigid_data.local_num && idx_particle_index != INVALID_INDEX)
		{
		// get the rigid body information
		float4 vel = tex1Dfetch(rigid_data_vel_tex, idx_body);
		float4 angvel = tex1Dfetch(rigid_data_angvel_tex, idx_body);
		float4 ex_space = tex1Dfetch(rigid_data_exspace_tex, idx_body);
		float4 ey_space = tex1Dfetch(rigid_data_eyspace_tex, idx_body);
		float4 ez_space = tex1Dfetch(rigid_data_ezspace_tex, idx_body);
		float4 particle_pos = tex1Dfetch(rigid_data_particle_pos_tex, idx_particle);
		
		// project the position in the body frame to the space frame: ri = rotation_matrix * particle_pos
		float4 ri;
		ri.x = ex_space.x * particle_pos.x + ey_space.x * particle_pos.y + ez_space.x * particle_pos.z;
		ri.y = ex_space.y * particle_pos.x + ey_space.y * particle_pos.y + ez_space.y * particle_pos.z;
		ri.z = ex_space.z * particle_pos.x + ey_space.z * particle_pos.y + ez_space.z * particle_pos.z;		
		
		// v_particle = v_com + angvel x xr
		float4 vel2;
		vel2.x = vel.x + angvel.y * ri.z - angvel.z * ri.y;
		vel2.y = vel.y + angvel.z * ri.x - angvel.x * ri.z;
		vel2.z = vel.z + angvel.x * ri.y - angvel.y * ri.x;
		
		// write out the results 
		pdata.vel[idx_particle_index] = vel2;

		}
	}

/*! \param pdata Particle data to step forward in time
	\param force_data_ptrs List of pointers to forces on each particle
	\param num_forces Number of forces listed in \a force_data_ptrs
	\param deltaT Amount of real time to step forward in one time step
	\param limit If \a limit is true, then the dynamics will be limited so that particles do not move 
		a distance further than \a limit_val in one step.
	\param limit_val Length to limit particle distance movement to
*/
hipError_t gpu_nve_rigid_body_step(const gpu_pdata_arrays &pdata, const gpu_rigid_data_arrays& rigid_data, float4 **force_data_ptrs, int num_forces, float deltaT, bool limit, float limit_val)
	{	
	unsigned int n_bodies = rigid_data.n_bodies;
	unsigned int nmax = rigid_data.nmax;
	
	// setup the grid to run the particle kernel 
	int block_size = nmax; // each thread in a block takes care of a particle in a rigid body
	dim3 grid(n_bodies, 1, 1);	
	dim3 threads(block_size, 1, 1);
	
	// bind the textures for ALL rigid bodies
	hipError_t error = hipBindTexture(0, rigid_data_body_mass_tex, rigid_data.body_mass, sizeof(float) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_vel_tex, rigid_data.vel, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_angvel_tex, rigid_data.angvel, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
		
	error = hipBindTexture(0, rigid_data_angmom_tex, rigid_data.angmom, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_orientation_tex, rigid_data.orientation, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_exspace_tex, rigid_data.ex_space, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_eyspace_tex, rigid_data.ey_space, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_ezspace_tex, rigid_data.ez_space, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
		
	error = hipBindTexture(0, rigid_data_particle_pos_tex, rigid_data.particle_pos, sizeof(float4) * n_bodies * nmax);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_particle_indices_tex, rigid_data.particle_indices, sizeof(unsigned int) * n_bodies * nmax);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_force_tex, rigid_data.force, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	error = hipBindTexture(0, rigid_data_torque_tex, rigid_data.torque, sizeof(float4) * n_bodies);
	if (error != hipSuccess)
		return error;
	
	// bind the textures for particles
	
	error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, pdata_mass_tex, pdata.mass, sizeof(float) * pdata.N);
	if (error != hipSuccess)
		return error;
	
	
	// run the kernel for bodies
	
    gpu_nve_rigid_body_step_kernel<<< grid, threads, nmax * sizeof(float4) >>>(pdata, rigid_data, force_data_ptrs, num_forces, deltaT, limit, limit_val);
	
	
	
	// run the kernel for particles
	gpu_nve_rigid_particle_step_kernel<<< grid, threads >>>(pdata, rigid_data, limit, limit_val);
	
	
	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

