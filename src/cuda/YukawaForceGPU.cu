#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$

#include "YukawaForceGPU.cuh"
#include "gpu_settings.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file YukawaForceGPU.cu
    \brief Defines GPU kernel code for calculating Yukawa pair forces. Used by YukawaForceComputeGPU.
*/

//! Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;

//! Kernel for calculating yukawa forces
/*! This kerenel is called to calculate the lennard-jones forces on all N particles

    \param force_data Device memory array to write calculated forces to
    \param pdata Particle data on the GPU to calculate forces on
    \param box Box dimensions used to implement periodic boundary conditions
    \param nlist Neigbhor list data on the GPU to use to calculate the forces
    \param d_coeffs Coefficients to the lennard jones force.
    \param coeff_width Width of the coefficient matrix
    \param r_cutsq Precalculated r_cut*r_cut, where r_cut is the radius beyond which forces are
        set to 0
    \param kappa Screening Length

    \a coeffs is a pointer to a matrix in memory. \c coeffs[i*coeff_width+j] is epsilon for the type pair \a i, \a j.
    The values in d_coeffs are read into shared memory, so
    \c coeff_width*coeff_width*sizeof(float) bytes of extern shared memory must be allocated for the kernel call.

    Developer information:
    Each block will calculate the forces on a block of particles.
    Each thread will calculate the total force on one particle.
    The neighborlist is arranged in columns so that reads are fully coalesced when doing this.
*/
extern "C" __global__ void gpu_compute_yukawa_forces_kernel(gpu_force_data_arrays force_data, gpu_pdata_arrays pdata, gpu_boxsize box, gpu_nlist_array nlist, float *d_coeffs, int coeff_width, float r_cutsq, float kappa)
    {
    // read in the coefficients
    extern __shared__ float s_coeffs[];
    for (int cur_offset = 0; cur_offset < coeff_width*coeff_width; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < coeff_width*coeff_width)
            s_coeffs[cur_offset + threadIdx.x] = d_coeffs[cur_offset + threadIdx.x];
        }
    __syncthreads();
    
    // start by identifying which particle we are to handle
    int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx_local >= pdata.local_num)
        return;
        
    int idx_global = idx_local + pdata.local_beg;
    
    // load in the length of the list (MEM_TRANSFER: 4 bytes)
    int n_neigh = nlist.n_neigh[idx_global];
    
    // read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
    // (MEM TRANSFER: 16 bytes)
    float4 pos = tex1Dfetch(pdata_pos_tex, idx_global);
    
    // initialize the force to 0
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float virial = 0.0f;
    
    // loop over neighbors
#ifdef ARCH_SM13
    // sm13 offers warp voting which makes this hardware bug workaround less of a performance penalty
    for (int neigh_idx = 0; __any(neigh_idx < n_neigh); neigh_idx++)
#else
    for (int neigh_idx = 0; neigh_idx < nlist.height; neigh_idx++)
#endif
        {
        if (neigh_idx < n_neigh)
            {
            // read the current neighbor index (MEM TRANSFER: 4 bytes)
            int cur_neigh = nlist.list[nlist.pitch*neigh_idx + idx_global];
            
            // get the neighbor's position (MEM TRANSFER: 16 bytes)
            float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_neigh);
            
            // calculate dr (with periodic boundary conditions) (FLOPS: 3)
            float dx = pos.x - neigh_pos.x;
            float dy = pos.y - neigh_pos.y;
            float dz = pos.z - neigh_pos.z;
            
            // apply periodic boundary conditions: (FLOPS 12)
            dx -= box.Lx * rintf(dx * box.Lxinv);
            dy -= box.Ly * rintf(dy * box.Lyinv);
            dz -= box.Lz * rintf(dz * box.Lzinv);
            
            // calculate r squard (FLOPS: 5)
            float rsq = dx*dx + dy*dy + dz*dz;
            
            // calculate r and rinv (FLOPS: 2)
            float r = sqrtf(rsq);
            
            float rinv;
            if (rsq >= r_cutsq)
                rinv = 0.0f;
            else
                rinv = 1.0f / r;
                
            // calculate 1/r^2 (FLOPS: 1)
            float r2inv = rinv*rinv;
            
            // lookup the coefficients between this combination of particle types
            int typ_pair = __float_as_int(neigh_pos.w) * coeff_width + __float_as_int(pos.w);
            float epsilon = s_coeffs[typ_pair];
            
            // calculate the force magnitude / r (FLOPS: 6)
            float forcemag_divr = epsilon*expf(-kappa*r)*r2inv*(kappa + rinv);
            
            // calculate the virial (FLOPS: 3)
            virial += float(1.0/6.0) * rsq * forcemag_divr;
            // calculate the pair energy (FLOPS: 3)
            float pair_eng = epsilon*expf(-kappa*r)*rinv;
            
            // add up the force vector components (FLOPS: 7)
            force.x += dx * forcemag_divr;
            force.y += dy * forcemag_divr;
            force.z += dz * forcemag_divr;
            force.w += pair_eng;
            }
        }
        
    // potential energy per particle must be halved
    force.w *= 0.5f;
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    force_data.force[idx_local] = force;
    force_data.virial[idx_local] = virial;
    }


/*! \param force_data Force data on GPU to write forces to
    \param pdata Particle data on the GPU to perform the calculation on
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param nlist Neighbor list stored on the gpu
    \param d_coeffs Coefficients to the lennard jones force.
    \param coeff_width Width of the coefficient matrix
    \param r_cutsq Precomputed r_cut*r_cut, where r_cut is the radius beyond which the
        force is set to 0
    \param kappa Screening Length
    \param block_size Block size to execute

    This is just a driver for calcYukawaForces_kernel, see it for more details.

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()
*/
hipError_t gpu_compute_yukawa_forces(const gpu_force_data_arrays& force_data, const gpu_pdata_arrays &pdata, const gpu_boxsize &box, const gpu_nlist_array &nlist, float *d_coeffs, int coeff_width, float r_cutsq, float kappa, int block_size)
    {
    assert(d_coeffs);
    assert(coeff_width > 0);
    
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)pdata.local_num / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);
    
    // bind the texture
    pdata_pos_tex.normalized = false;
    pdata_pos_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata.pos, sizeof(float4) * pdata.N);
    if (error != hipSuccess)
        return error;
        
    // run the kernel
    gpu_compute_yukawa_forces_kernel<<< grid, threads, sizeof(float)*coeff_width*coeff_width >>>(force_data, pdata, box, nlist, d_coeffs, coeff_width, r_cutsq, kappa);
    
    if (!g_gpu_error_checking)
        {
        return hipSuccess;
        }
    else
        {
        hipDeviceSynchronize();
        return hipGetLastError();
        }
    }

// vim:syntax=cpp
