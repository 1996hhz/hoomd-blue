#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id: fenebondforce_kernel.cu 1158 2008-09-01 15:41:21Z phillicl $
// $URL: https://svn2.assembla.com/svn/hoomd/tags/hoomd-0.7.0/src/cuda/fenebondforce_kernel.cu $

#include "gpu_forces.h"
#include "gpu_pdata.h"
#include "gpu_settings.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif


/*! \file fenebondforce_kernel.cu
	\brief Contains code that implements the fene bond force sum on the GPU.
*/

//! Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;

//! Texture for reading bond parameters
texture<float4, 1, hipReadModeElementType> bond_params_tex;

extern "C" __global__ void calcFENEBondForces_kernel(float4 *d_forces, gpu_pdata_arrays pdata, gpu_bondtable_array blist, gpu_boxsize box)
	{
	// start by identifying which particle we are to handle
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (pidx >= pdata.N)
		return;
	
	// load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
	int n_bonds = blist.n_bonds[pidx];

	// read in the position of our particle. (MEM TRANSFER: 16 bytes)
	float4 pos = tex1Dfetch(pdata_pos_tex, pidx);

	// initialize the force to 0
	float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	
	// loop over neighbors
	for (int bond_idx = 0; bond_idx < n_bonds; bond_idx++)
		{
		// MEM TRANSFER: 8 bytes
		// the volatile fails to compile in device emulation mode
		#ifdef _DEVICEEMU
		uint2 cur_bond = blist.bonds[blist.pitch*bond_idx + pidx];
		#else
		// the volatile is needed to force the compiler to load the uint2 coalesced
		volatile uint2 cur_bond = blist.bonds[blist.pitch*bond_idx + pidx];
		#endif
		
		int cur_bond_idx = cur_bond.x;
		int cur_bond_type = cur_bond.y;
		
		// get the bonded particle's position (MEM_TRANSFER: 16 bytes)
		float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_bond_idx);
	
		// calculate dr (FLOPS: 3)
		float dx = pos.x - neigh_pos.x;
		float dy = pos.y - neigh_pos.y;
		float dz = pos.z - neigh_pos.z;
		
		// apply periodic boundary conditions (FLOPS: 12)
		dx -= box.Lx * rintf(dx * box.Lxinv);
		dy -= box.Ly * rintf(dy * box.Lyinv);
		dz -= box.Lz * rintf(dz * box.Lzinv);
		
		// get the bond parameters (MEM TRANSFER: 8 bytes)
		float4 params = tex1Dfetch(bond_params_tex, cur_bond_type);
		float K = params.x;
		float r_0 = params.y;
		float lj1 = params.z;
		float lj2 = params.w;

						
		// FLOPS: 5
		float rsq = dx*dx + dy*dy + dz*dz;
		//float r = sqrtf(rsq);
		
		// calculate 1/r^2 (FLOPS: 2)
		float r2inv;
		if (rsq >= 1.01944064370214f)  // comparing to the WCA limit
			r2inv = 0.0f;
		else
			r2inv = 1.0f / rsq;
	
		// calculate 1/r^6 (FLOPS: 2)
		float r6inv = r2inv*r2inv*r2inv;
		// calculate the force magnitude / r (FLOPS: 6)
		float wcaforcemag_divr = r2inv * r6inv * (12.0f * lj1  * r6inv - 6.0f * lj2);
		// calculate the pair energy (FLOPS: 3)
		// For WCA interaction, this energy is low by epsilon.  This is corrected in the logger.
		float pair_eng = r6inv * (lj1 * r6inv - lj2);
		
		// FLOPS: 7
		float forcemag_divr = -K / (1.0f - rsq/(r_0*r_0)) + wcaforcemag_divr;
		float bond_eng = -0.5f * K * r_0*r_0*logf(1.0f - rsq/(r_0*r_0));
				
		// add up the forces (FLOPS: 7)
		force.x += dx * forcemag_divr;
		force.y += dy * forcemag_divr;
		force.z += dz * forcemag_divr;
		force.w += bond_eng + pair_eng;
		
		// Checking to see if bond length restriction is violated.
		if (rsq >= r_0*r_0) *blist.checkr = 1;
		
		}
		
	// energy is double counted: multiply by 0.5
	force.w *= 0.5f;
	
	// now that the force calculation is complete, write out the result (MEM TRANSFER: 16 bytes);
	d_forces[pidx] = force;
	}


/*! \param d_forces Device memory to write forces to
	\param pdata Particle data on the GPU to perform the calculation on
	\param box Box dimensions (in GPU format) to use for periodic boundary conditions
	\param btable List of bonds stored on the GPU
	\param d_params K, r_0, lj1, and lj2 params packed as float4 variables
	\param n_bond_types Number of bond types in d_params
	\param block_size Block size to use when performing calculations
	\param exceedsR0 output parameter set to true if any bond exceeds the length of r_0
	
	\returns Any error code resulting from the kernel launch
	\note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()
	
	\a d_params should include one float4 element per bond type. The x component contains K the spring constant
	and the y component contains r_0 the equilibrium length, z and w contain lj1 and lj2.
*/
hipError_t gpu_fenebondforce_sum(float4 *d_forces, gpu_pdata_arrays *pdata, gpu_boxsize *box, gpu_bondtable_array *btable, float4 *d_params, unsigned int n_bond_types, int block_size, unsigned int& exceedsR0)
	{
	assert(pdata);
	assert(btable);
	assert(d_params);
	// check that block_size is valid
	assert(block_size != 0);

	// setup the grid to run the kernel
	dim3 grid( (int)ceil((double)pdata->N/ (double)block_size), 1, 1);
	dim3 threads(block_size, 1, 1);

	// bind the textures
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata->pos, sizeof(float4) * pdata->N);
	if (error != hipSuccess)
		return error;
		
	error = hipBindTexture(0, bond_params_tex, d_params, sizeof(float4) * n_bond_types);
	if (error != hipSuccess)
		return error;
		
	// start by zeroing check value on the device
	error = hipMemcpy(btable->checkr, &exceedsR0,
			sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		return error;

			
	// run the kernel
	calcFENEBondForces_kernel<<< grid, threads>>>(d_forces, *pdata, *btable, *box);
	

	error = hipMemcpy(&exceedsR0, btable->checkr,
			sizeof(int), hipMemcpyDeviceToHost);	
	if (error != hipSuccess)
		return error;

	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}
