#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$

#include "gpu_pdata.h"
#include "gpu_updaters.h"
#include "gpu_integrator.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

#include <stdio.h>

/*! \file gpu_nve_kernel.cu
	\brief Contains kernel code for the NVE integrator on the GPU
*/

//! The texture for reading the pdata pos array
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;
texture<float4, 1, hipReadModeElementType> pdata_vel_tex;
texture<float4, 1, hipReadModeElementType> pdata_accel_tex;

extern "C" __global__ void nve_pre_step_kernel(gpu_pdata_arrays pdata, float deltaT, bool limit, float limit_val, gpu_boxsize box)
	{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int pidx = idx + pdata.local_beg;
	// do velocity verlet update
	// r(t+deltaT) = r(t) + v(t)*deltaT + (1/2)a(t)*deltaT^2
	// v(t+deltaT/2) = v(t) + (1/2)a*deltaT
	
	if (idx < pdata.local_num)
		{
		// read the particle's posision (MEM TRANSFER: 16 bytes)
		float4 pos = tex1Dfetch(pdata_pos_tex, pidx);
		
		float px = pos.x;
		float py = pos.y;
		float pz = pos.z;
		float pw = pos.w;
		
		// read the particle's velocity and acceleration (MEM TRANSFER: 32 bytes)
		float4 vel = tex1Dfetch(pdata_vel_tex, pidx);
		float4 accel = tex1Dfetch(pdata_accel_tex, pidx);
		
		// update the position (FLOPS: 15)
		float dx = vel.x * deltaT + (1.0f/2.0f) * accel.x * deltaT * deltaT;
		float dy = vel.y * deltaT + (1.0f/2.0f) * accel.y * deltaT * deltaT;
		float dz = vel.z * deltaT + (1.0f/2.0f) * accel.z * deltaT * deltaT;
		
		// limit the movement of the particles
		if (limit)
			{
			float len = sqrtf(dx*dx + dy*dy + dz*dz);
			if (len > limit_val)
				{
				dx = dx / len * limit_val;
				dy = dy / len * limit_val;
				dz = dz / len * limit_val;
				}
			}
		
		// FLOPS: 3
		px += dx;
		py += dy;
		pz += dz;
		
		// update the velocity (FLOPS: 9)
		vel.x += (1.0f/2.0f) * accel.x * deltaT;
		vel.y += (1.0f/2.0f) * accel.y * deltaT;
		vel.z += (1.0f/2.0f) * accel.z * deltaT;
		
		// time to fix the periodic boundary conditions (FLOPS: 12)
		px -= box.Lx * rintf(px * box.Lxinv);
		py -= box.Ly * rintf(py * box.Lyinv);
		pz -= box.Lz * rintf(pz * box.Lzinv);
	
		float4 pos2;
		pos2.x = px;
		pos2.y = py;
		pos2.z = pz;
		pos2.w = pw;
						
		// write out the results (MEM_TRANSFER: 32 bytes)
		pdata.pos[pidx] = pos2;
		pdata.vel[pidx] = vel;
		}	
	}

hipError_t nve_pre_step(gpu_pdata_arrays *pdata, gpu_boxsize *box, float deltaT, bool limit, float limit_val)
	{
    assert(pdata);

    // setup the grid to run the kernel
    int M = 256;
    dim3 grid( (pdata->local_num/M) + 1, 1, 1);
    dim3 threads(M, 1, 1);

	// bind the textures
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata->pos, sizeof(float4) * pdata->N);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, pdata_vel_tex, pdata->vel, sizeof(float4) * pdata->N);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, pdata_accel_tex, pdata->accel, sizeof(float4) * pdata->N);
	if (error != hipSuccess)
		return error;

    // run the kernel
    nve_pre_step_kernel<<< grid, threads >>>(*pdata, deltaT, limit, limit_val, *box);
	
	#ifdef NDEBUG
	return hipSuccess;
	#else
	hipDeviceSynchronize();
	return hipGetLastError();
	#endif
	}


extern "C" __global__ void nve_step_kernel(gpu_pdata_arrays pdata, float4 **force_data_ptrs, int num_forces, float deltaT, bool limit, float limit_val)
	{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int pidx = idx + pdata.local_beg;
	// v(t+deltaT) = v(t+deltaT/2) + 1/2 * a(t+deltaT)*deltaT

	// sum the acceleration on this particle: (MEM TRANSFER: 16 bytes * number of forces FLOPS: 3 * number of forces)
	float4 accel = integrator_sum_forces_inline(idx, pidx, pdata.local_num, force_data_ptrs, num_forces);
	if (idx < pdata.local_num)
		{
		// read the current particle velocity (MEM TRANSFER: 16 bytes)
		float4 vel = tex1Dfetch(pdata_vel_tex, pidx);
		
		// update the velocity (FLOPS: 6)
		vel.x += (1.0f/2.0f) * accel.x * deltaT;
		vel.y += (1.0f/2.0f) * accel.y * deltaT;
		vel.z += (1.0f/2.0f) * accel.z * deltaT;
		
		if (limit)
			{
			float vel_len = sqrtf(vel.x*vel.x + vel.y*vel.y + vel.z*vel.z);
			if ( (vel_len*deltaT) > limit_val)
				{
				vel.x = vel.x / vel_len * limit_val / deltaT;
				vel.y = vel.y / vel_len * limit_val / deltaT;
				vel.z = vel.z / vel_len * limit_val / deltaT;
				}
			}
		
		// write out data (MEM TRANSFER: 32 bytes)
		pdata.vel[pidx] = vel;
		// since we calculate the acceleration, we need to write it for the next step
		pdata.accel[pidx] = accel;
		}
	}
	
hipError_t nve_step(gpu_pdata_arrays *pdata, float4 **force_data_ptrs, int num_forces, float deltaT, bool limit, float limit_val)
	{
    assert(pdata);

    // setup the grid to run the kernel
    int M = 192;
    dim3 grid( (pdata->local_num/M) + 1, 1, 1);
    dim3 threads(M, 1, 1);

	// bind the texture
	hipError_t error = hipBindTexture(0, pdata_vel_tex, pdata->vel, sizeof(float4) * pdata->N);
	if (error != hipSuccess)
		return error;

    // run the kernel
    nve_step_kernel<<< grid, threads >>>(*pdata, force_data_ptrs, num_forces, deltaT, limit, limit_val);

	#ifdef NDEBUG
	return hipSuccess;
	#else
	hipDeviceSynchronize();
	return hipGetLastError();
	#endif
	}
