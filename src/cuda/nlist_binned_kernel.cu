#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
#include "gpu_nlist.h"
#include "gpu_pdata.h"
#include <stdio.h>

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

// textures used in this kernel: commented out because textures are managed globally currently
texture<unsigned int, 2, hipReadModeElementType> nlist_idxlist_tex;
texture<uint4, 1, hipReadModeElementType> nlist_bincoord_tex;
texture<uint4, 1, hipReadModeElementType> nlist_exclude_tex;

//! Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;

/*! \file nlist_binned_kernel.cu
	\brief Contains code for the kernel that implements the binned O(N) neighbor list on the GPU
*/

// upating the list from the bins will involve looping over all the particles in the bin and 
// comparing them to all particles in this and neighboring bins
// each block will process one bin. Since each particle is only placed in one bin, each block thus processes
// a block of particles (though which particles it processes is random)
// Empty bin entries will be set to 0xffffffff to allow for efficient handling

#define EMPTY_BIN 0xffffffff

extern "C" __global__ void updateFromBins_new(gpu_pdata_arrays pdata, gpu_bin_array bins, gpu_nlist_array nlist, float r_maxsq, unsigned int actual_Nmax, gpu_boxsize box, float scalex, float scaley, float scalez)
	{
	// each thread is going to compute the neighbor list for a single particle
	int my_pidx = blockDim.x * blockIdx.x + threadIdx.x;
	
	// quit early if we are past the end of the array
	if (my_pidx >= pdata.N)
		return;
	
	// first, determine which bin this particle belongs to
	float4 my_pos = tex1Dfetch(pdata_pos_tex, my_pidx);
	uint4 exclude = tex1Dfetch(nlist_exclude_tex, my_pidx);
	
	unsigned int ib = (unsigned int)((my_pos.x+box.Lx/2.0f)*scalex);
	unsigned int jb = (unsigned int)((my_pos.y+box.Ly/2.0f)*scaley);
	unsigned int kb = (unsigned int)((my_pos.z+box.Lz/2.0f)*scalez);

	// need to handle the case where the particle is exactly at the box hi
	if (ib == bins.Mx)
		ib = 0;
	if (jb == bins.My)
		jb = 0;
	if (kb == bins.Mz)
		kb = 0;
			
	int my_bin = ib*(bins.Mz*bins.My) + jb * bins.Mz + kb;	

	// each thread will determine the neighborlist of a single particle
	int n_neigh = 0;	// count number of neighbors found so far
	
	// we will need to loop over all neighboring bins. In order to do that, we need to know what bin we are actually in!
	// this could be a messy bunch of modulus operations, so we just read it out of an array that has been pre-computed for us :)
	uint4 coords = tex1Dfetch(nlist_bincoord_tex, my_bin);
	int my_i = coords.x;
	int my_j = coords.y;
	int my_k = coords.z;
	
	// loop through the 27 neighboring bins
	for (int cur_i = int(my_i) - 1; cur_i <= int(my_i) + 1; cur_i++)
		{
		for (int cur_j = int(my_j) - 1; cur_j <= int(my_j) + 1; cur_j++)
			{
			for (int cur_k = int(my_k) - 1; cur_k <= int(my_k) + 1; cur_k++)
				{
				// apply boundary conditions to the current bin
				int a = cur_i;
				if (a < 0) 
					a += bins.Mx;
				if (a >= bins.Mx)
					a -= bins.Mx;

				int b = cur_j;
				if (b < 0) 
					b += bins.My;
				if (b >= bins.My)
					b -= bins.My;

				int c = cur_k;
				if (c < 0) 
					c += bins.Mz;
				if (c >= bins.Mz)
					c -= bins.Mz;
					
				// now: we finally know the current bin to compare to
				int neigh_bin = a*bins.Mz*bins.My + b*bins.Mz + c;
				
				// now, we are set to loop through the array
				for (int cur_offset = 0; cur_offset < actual_Nmax; cur_offset++)
					{
					unsigned int cur_neigh = tex2D(nlist_idxlist_tex, cur_offset, neigh_bin);
					
					if (cur_neigh != EMPTY_BIN)
						{
						float4 neigh_pos = tex1Dfetch(pdata_pos_tex, cur_neigh);
					
						float dx = my_pos.x - neigh_pos.x;
						dx = dx - box.Lx * rintf(dx * box.Lxinv);
	
						float dy = my_pos.y - neigh_pos.y;
						dy = dy - box.Ly * rintf(dy * box.Lyinv);
	
						float dz = my_pos.z - neigh_pos.z;
						dz = dz - box.Lz * rintf(dz * box.Lzinv);
	
						float dr = dx*dx + dy*dy + dz*dz;
						int not_excluded = (exclude.x != cur_neigh) & (exclude.y != cur_neigh) & (exclude.z != cur_neigh) & (exclude.w != cur_neigh);
						
						if (dr < r_maxsq && (my_pidx != cur_neigh) && not_excluded)
							{
							// check for overflow
							if (n_neigh < nlist.height)
								{
								nlist.list[my_pidx + n_neigh*nlist.pitch] = cur_neigh;
								n_neigh++;
								}
							else
								*nlist.overflow = 1;
							}
						}
					}
				}
			}
		}
		
	nlist.n_neigh[my_pidx] = n_neigh;
	nlist.last_updated_pos[my_pidx] = my_pos;
	}
	
hipError_t gpu_nlist_binned(gpu_pdata_arrays *pdata, gpu_boxsize *box, gpu_bin_array *bins, gpu_nlist_array *nlist, float r_maxsq, int curNmax, int block_size)
	{
	assert(bins);
	assert(pdata);
	assert(nlist);
	assert(block_size > 0);

	// setup the grid to run the kernel
	int nblocks = (int)ceil((double)pdata->N/ (double)block_size);
	
	dim3 grid(nblocks, 1, 1);
	dim3 threads(block_size, 1, 1);

	// bind the textures
	hipError_t error = hipBindTexture(0, pdata_pos_tex, pdata->pos, sizeof(float4) * pdata->N);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, nlist_bincoord_tex, bins->bin_coord, sizeof(uint4)*bins->Mx*bins->My*bins->Mz);
	if (error != hipSuccess)
		return error;

	error = hipBindTexture(0, nlist_exclude_tex, nlist->exclusions, sizeof(uint4) * pdata->N);
	if (error != hipSuccess)
		return error;
	
	nlist_idxlist_tex.normalized = false;
	nlist_idxlist_tex.filterMode = hipFilterModePoint;
	error = hipBindTextureToArray(nlist_idxlist_tex, bins->idxlist_array);
	if (error != hipSuccess)
		return error;
	
	// zero the overflow check
	error = hipMemset(nlist->overflow, 0, sizeof(int));
	if (error != hipSuccess)
		return error;
	
	// make even bin dimensions
	float binx = (box->Lx) / float(bins->Mx);
	float biny = (box->Ly) / float(bins->My);
	float binz = (box->Lz) / float(bins->Mz);

	// precompute scale factors to eliminate division in inner loop
	float scalex = 1.0f / binx;
	float scaley = 1.0f / biny;
	float scalez = 1.0f / binz;

	// run the kernel
	updateFromBins_new<<< grid, threads>>>(*pdata, *bins, *nlist, r_maxsq, curNmax, *box, scalex, scaley, scalez);
	
	#ifdef NDEBUG
	return hipSuccess;
	#else
	hipDeviceSynchronize();
	return hipGetLastError();
	#endif
	}

// vim:syntax=cpp
