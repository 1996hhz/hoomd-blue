#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
#include "gpu_nlist.h"
#include "gpu_pdata.h"
#include <stdio.h>

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file nlist_nsq_kernel.cu
	\brief Contains kernel code that implements the O(N^2) kernel on the GPU
*/

const int NLIST_BLOCK_SIZE = 128;
// generate the neighbor list
extern "C" __global__ void generateNlistNSQ(gpu_pdata_arrays pdata, gpu_nlist_array nlist, float r_maxsq, gpu_boxsize box) 
	{
	// each thread is to compute the neighborlist for a single particle i
	// each block will load a bunch of particles into shared mem and then each thread will compare it's particle
	// to each particle in shmem to see if they are a neighbor. Since all threads in the block access the same 
	// shmem element at the same time, the value is broadcast and there are no bank conflicts

	// the way this funciton loads data, all data arrays need to be padded so they have a multiple of 
	// blockDim.x elements. 

	// shared data to store all of the particles we compare against
	__shared__ float sdata[NLIST_BLOCK_SIZE*4];
	
	// load in the particle
	int pidx = blockIdx.x * NLIST_BLOCK_SIZE + threadIdx.x;

	float4 pos = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	if (pidx < pdata.N)
		pos = pdata.pos[pidx];
		
	float px = pos.x;
	float py = pos.y;
	float pz = pos.z;

	// track the number of neighbors added so far
	int n_neigh = 0;
	
	uint4 exclude = make_uint4(0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff);
	if (pidx < pdata.N)
		exclude = nlist.exclusions[pidx];
	
	// each block is going to loop over all N particles (this assumes memory is padded to a multiple of blockDim.x)
	// in blocks of blockDim.x
	for (int start = 0; start < pdata.N; start += NLIST_BLOCK_SIZE)
		{
		// load data
		float4 neigh_pos = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		if (start + threadIdx.x < pdata.N)
			neigh_pos = pdata.pos[start + threadIdx.x];
		
		// make sure everybody is caught up before we stomp on the memory
		__syncthreads();
		sdata[threadIdx.x] = neigh_pos.x;
		sdata[threadIdx.x + NLIST_BLOCK_SIZE] = neigh_pos.y;
		sdata[threadIdx.x + 2*NLIST_BLOCK_SIZE] = neigh_pos.z;
		sdata[threadIdx.x + 3*NLIST_BLOCK_SIZE] = neigh_pos.w; //< unused, but try to get compiler to fully coalesce reads

		// ensure all data is loaded
		__syncthreads();

		// now each thread loops over every particle in shmem, but doesn't loop past the end of the particle list (since
		// the block might extend that far)
		int end_offset= NLIST_BLOCK_SIZE;
		end_offset = min(end_offset, pdata.N - start);

		if (pidx < pdata.N)
			{ 

			for (int cur_offset = 0; cur_offset < end_offset; cur_offset++)
				{
				// calculate dr
				float dx = px - sdata[cur_offset];
				dx = dx - box.Lx * rintf(dx * box.Lxinv);
				
				if (dx*dx < r_maxsq)
					{
					float dy = py - sdata[cur_offset + NLIST_BLOCK_SIZE];
					dy = dy - box.Ly * rintf(dy * box.Lyinv);
				
					if (dy*dy < r_maxsq)
						{
						float dz = pz - sdata[cur_offset + 2*NLIST_BLOCK_SIZE];
						dz = dz - box.Lz * rintf(dz * box.Lzinv);
				
						float drsq = dx*dx + dy*dy + dz*dz;
	
						// we don't add if we are comparing to ourselves, and we don't add if we are above the cut
						if ((drsq < r_maxsq) && ((start + cur_offset) != pidx) && exclude.x != (start + cur_offset) && exclude.y != (start + cur_offset) && exclude.z != (start + cur_offset) && exclude.w != (start + cur_offset))
							{
							nlist.list[pidx + (1 + n_neigh)*nlist.pitch] = start+cur_offset;
							n_neigh++;
							}
					
						}
					}
				}
			}
		}

	// now that we are done: update the first row that lists the number of neighbors
	if (pidx < pdata.N)
		{
		nlist.list[pidx] = n_neigh;
		nlist.last_updated_pos[pidx] = pdata.pos[pidx];
		}
	}

// driver function for the kernel
hipError_t gpu_nlist_nsq(gpu_pdata_arrays *pdata, gpu_boxsize *box, gpu_nlist_array *nlist, float r_maxsq)
	{
	assert(pdata);
	assert(nlist);
	
	// setup the grid to run the kernel
	int M = NLIST_BLOCK_SIZE;
	dim3 grid( (pdata->N/M) + 1, 1, 1);
	dim3 threads(M, 1, 1);
	
	// run the kernel
	generateNlistNSQ<<< grid, threads >>>(*pdata, *nlist, r_maxsq, *box);
	#ifdef NDEBUG
	return hipSuccess;
	#else
	hipDeviceSynchronize();
	return hipGetLastError();
	#endif	
	}

// vim:syntax=cpp
