/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id$
// $URL$
// Maintainer: joaander


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

// safe call macros
#define CUDA_SAFE_CALL( call) do {                                         \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
    exit(EXIT_FAILURE);                                                      \
    } } while (0)

#define CUT_CHECK_ERROR(errorMessage) do {                                 \
    hipDeviceSynchronize();                                                \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    } } while (0)


//*************** parameters of the benchmark
unsigned int g_N;
float g_Lx;
float g_Ly;
float g_Lz;
float g_rcut;
const unsigned int g_Nmax = 128;	// Maximum number of particles each cell can hold
const float tweak_dist = 0.1f;

//*************** data structures
float4 *gh_pos, *gd_pos;			// particle positions
unsigned int g_Mx;	// X-dimension of the cell grid
unsigned int g_My;	// Y-dimension of the cell grid
unsigned int g_Mz;	// Z-dimension of the cell grid
unsigned int *gd_idxlist;	// \a Mx x \a My x \a Mz x \a Nmax 4D array holding the indices of the particles in each cell
unsigned int *gh_idxlist;	// \a Mx x \a My x \a Mz x \a Nmax 4D array holding the indices of the particles in each cell
unsigned int *gd_bin_size;	// number of particles in each bin
unsigned int *gh_bin_size;	// number of particles in each bin

unsigned int *gd_old_idxlist;	// old data for the above array (swapped on each update call)
unsigned int *gd_old_bin_size;	// old data for the above array (swapped on each update call)

uint4 *gd_bin_coords;	// pre-calculated bin coordinates for each bin
uint4 *gh_bin_coords;	// pre-calculated bin coordinates for each bin

unsigned int *g_ref_idxlist;	// reference idxlist for correctness comparison
unsigned int *g_ref_bin_size;	// reference bin_size for correctness comparison

//*************** functions for allocating and freeing the data structures
void allocate_data()
	{
	// allocate particle positions
	gh_pos = (float4 *)malloc(sizeof(float4) * g_N);
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_pos, sizeof(float4) * g_N));
	
	// determine grid dimensions
	g_Mx = int((g_Lx) / (g_rcut));
	g_My = int((g_Ly) / (g_rcut));
	g_Mz = int((g_Lz) / (g_rcut));
	
	// allocate bins
	unsigned int Nbins = g_Mx * g_My * g_Mz;
	gh_idxlist = (unsigned int *)malloc(Nbins * g_Nmax * sizeof(unsigned int));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_idxlist, Nbins * g_Nmax * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_old_idxlist, Nbins * g_Nmax * sizeof(unsigned int)));
	gh_bin_size = (unsigned int *)malloc(Nbins * sizeof(unsigned int));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_bin_size, Nbins * g_Nmax * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_old_bin_size, Nbins * g_Nmax * sizeof(unsigned int)));
	
	g_ref_idxlist = (unsigned int *)malloc(Nbins * g_Nmax * sizeof(unsigned int));
	g_ref_bin_size = (unsigned int *)malloc(Nbins * sizeof(unsigned int));
	
	gh_bin_coords = (uint4*)malloc(Nbins*sizeof(uint4));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_bin_coords, Nbins*sizeof(uint4)));
	}
	
void free_data()
	{
	// free host memory
	free(gh_pos);
	free(gh_idxlist);
	free(gh_bin_size);
	free(g_ref_idxlist);
	free(g_ref_bin_size);
	free(gh_bin_coords);
	
	// free GPU memory
	CUDA_SAFE_CALL(hipFree(gd_pos));
	CUDA_SAFE_CALL(hipFree(gd_idxlist));
	CUDA_SAFE_CALL(hipFree(gd_old_idxlist));
	CUDA_SAFE_CALL(hipFree(gd_bin_size));
	CUDA_SAFE_CALL(hipFree(gd_old_bin_size));
	CUDA_SAFE_CALL(hipFree(gd_bin_coords));
	}
	
void initialize_data()
	{
	// initialize particles randomly
	for (unsigned int i = 0; i < g_N; i++)
		{
		gh_pos[i].x = float((rand())/float(RAND_MAX) - 0.5)*g_Lx;
		gh_pos[i].y = float((rand())/float(RAND_MAX) - 0.5)*g_Ly;
		gh_pos[i].z = float((rand())/float(RAND_MAX) - 0.5)*g_Lz;
		gh_pos[i].w = 0.0f;
		}
	
	// copy particles to the device
	CUDA_SAFE_CALL(hipMemcpy(gd_pos, gh_pos, sizeof(float4)*g_N, hipMemcpyHostToDevice));
	
	// zero all other memory
	memset(gh_idxlist, 0, sizeof(unsigned int)*g_Mx*g_My*g_Mz*g_Nmax);
	memset(g_ref_idxlist, 0, sizeof(unsigned int)*g_Mx*g_My*g_Mz*g_Nmax);
	memset(gh_bin_size, 0, sizeof(unsigned int)*g_Mx*g_My*g_Mz);
	memset(g_ref_bin_size, 0, sizeof(unsigned int)*g_Mx*g_My*g_Mz);
	
	CUDA_SAFE_CALL(hipMemset(gd_idxlist, 0, sizeof(unsigned int)*g_Mx*g_My*g_Mz*g_Nmax));
	CUDA_SAFE_CALL(hipMemset(gd_old_idxlist, 0, sizeof(unsigned int)*g_Mx*g_My*g_Mz*g_Nmax));
	CUDA_SAFE_CALL(hipMemset(gd_bin_size, 0, sizeof(unsigned int)*g_Mx*g_My*g_Mz));
	CUDA_SAFE_CALL(hipMemset(gd_old_bin_size, 0, sizeof(unsigned int)*g_Mx*g_My*g_Mz));
	
	// initialize the bin coords
	for (unsigned int i = 0; i < g_Mx; i++)
		for (unsigned int j = 0; j < g_My; j++)
			for (unsigned int k = 0; k < g_Mz; k++)
				gh_bin_coords[i*(g_Mz*g_My) + j * g_Mz + k] = make_uint4(i,j,k,0);
	
	CUDA_SAFE_CALL(hipMemcpy(gd_bin_coords, gh_bin_coords, g_Mx*g_My*g_Mz*sizeof(uint4), hipMemcpyHostToDevice));
	}
	
// moves the particles a "little bit" randomly and copies the new positions to the device
void tweak_data()
	{
	float xhi = g_Lx / 2.0f;
	float xlo = -xhi;
	float yhi = g_Ly / 2.0f;
	float ylo = -yhi;
	float zhi = g_Lz / 2.0f;
	float zlo = -zhi;	
	
	for (unsigned int i = 0; i < g_N; i++)
		{
		// OK, so it is a poorly distributed tweak. So what, it serves it's purpose.
		float x = float((rand())/float(RAND_MAX) - 0.5);
		float y = float((rand())/float(RAND_MAX) - 0.5);
		float z = float((rand())/float(RAND_MAX) - 0.5);
		float len = sqrt(x*x + y*y + z*z);
		x = x / len * tweak_dist;
		y = y / len * tweak_dist;
		z = z / len * tweak_dist;
		
		gh_pos[i].x += x;
		gh_pos[i].y += y;
		gh_pos[i].z += z;
		
		// fix up boundary conditions
		if (gh_pos[i].x >= xhi)
			gh_pos[i].x -= g_Lx;
		if (gh_pos[i].x <= xlo)
			gh_pos[i].x += g_Lx;
		if (gh_pos[i].y >= yhi)
			gh_pos[i].y -= g_Ly;
		if (gh_pos[i].y <= ylo)
			gh_pos[i].y += g_Ly;
		if (gh_pos[i].z >= zhi)
			gh_pos[i].z -= g_Lz;
		if (gh_pos[i].z <= zlo)
			gh_pos[i].z += g_Lz;
		}
		
	// update the data on the device
	hipMemcpy(gd_pos, gh_pos, sizeof(float4)*g_N, hipMemcpyHostToDevice);
	}

// sorts the data to mimic HOOMD's standard data pattern (sort of)
void sort_data()
	{
	printf("sorting....\n");
	unsigned int * bin_list = (unsigned int*)malloc(sizeof(unsigned int) *g_N);
	// make even bin dimensions
	float binx = g_Lx / float(g_Mx);
	float biny = g_Ly / float(g_My);
	float binz = g_Lz / float(g_Mz);
	
	float xlo = -g_Lx/2.0f;
	float ylo = -g_Lx/2.0f;
	float zlo = -g_Lx/2.0f;

	// precompute scale factors to eliminate division in inner loop
	float scalex = 1.0f / binx;
	float scaley = 1.0f / biny;
	float scalez = 1.0f / binz;
	
	for (unsigned int i = 0; i < g_N; i++)
		{
		// find the bin each particle belongs in
		unsigned int ib = (unsigned int)((gh_pos[i].x-xlo)*scalex);
		unsigned int jb = (unsigned int)((gh_pos[i].y-ylo)*scaley);
		unsigned int kb = (unsigned int)((gh_pos[i].z-zlo)*scalez);
		
		// need to handle the case where the particle is exactly at the box hi
		if (ib == g_Mx)
			ib = 0;
		if (jb == g_My)
			jb = 0;
		if (kb == g_Mz)
			kb = 0;
			
		// update the bin
		unsigned int bin = ib*(g_Mz*g_My) + jb * g_Mz + kb;
		bin_list[i] = bin;
		}

	bool swapped = false;
	do
		{
		swapped = false;
		for (unsigned int i = 0; i < g_N-1; i++)
			{
			if (bin_list[i] > bin_list[i+1])
				{
				unsigned int tmp = bin_list[i+1];
				bin_list[i+1] = bin_list[i];
				bin_list[i] = tmp;
				
				float4 tmpf = gh_pos[i+1];
				gh_pos[i+1] = gh_pos[i];
				gh_pos[i] = tmpf;
				swapped = true;
				}
			}
		} while (swapped);
	
		
	free(bin_list);
	// update the data on the device
	hipMemcpy(gd_pos, gh_pos, sizeof(float4)*g_N, hipMemcpyHostToDevice);
	printf("	done.\n");
	}
	
__global__ void fast_memclear_kernal(unsigned int *d_data, unsigned int N)
	{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	d_data[idx] = 0;
	}


void rebin_particles_host(unsigned int *idxlist, unsigned int *bin_size, float4 *pos, unsigned int N, float Lx, float Ly, float Lz, unsigned int Mx, unsigned int My, unsigned int Mz, unsigned int Nmax);
	

//****************** verify gh_idxlist and gh_bin_size vs the reference ones
bool verify()
	{
	// generate the reference data
	rebin_particles_host(g_ref_idxlist, g_ref_bin_size, gh_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax);	
	
	for (unsigned int bin = 0; bin < g_Mx*g_My*g_Mz; bin++)
		{
		// check bin sizes first
		if (gh_bin_size[bin] != g_ref_bin_size[bin])
			{
			printf("bin sizes differ for bin %d : %d != %d\n", bin, gh_bin_size[bin], g_ref_bin_size[bin]);
			return false;
			}
			
		// now check every single particle in the bins
		unsigned int size = gh_bin_size[bin];
		for (unsigned int ref_i = 0; ref_i < size; ref_i++)
			{
			unsigned int particle_i = g_ref_idxlist[bin*g_Nmax + ref_i];
			
			bool found = false;
			for (unsigned int j = 0; j < size; j++)
				{
				if (particle_i == gh_idxlist[bin*g_Nmax + j])
					{
					found = true;
					break;
					}
				}
			if (!found)
				{
				printf("particle %d not present in bin %d\n", particle_i, bin);
				return false;
				}
			}
		}
	return true;
	}

//****************** bins the data on the host from scratch
void rebin_particles_host(unsigned int *idxlist, unsigned int *bin_size, float4 *pos, unsigned int N, float Lx, float Ly, float Lz, unsigned int Mx, unsigned int My, unsigned int Mz, unsigned int Nmax)
	{
	// clear the bin sizes
	for (unsigned int i = 0; i < Mx*My*Mz; i++)
		bin_size[i] = 0;
		
	// make even bin dimensions
	float binx = Lx / float(Mx);
	float biny = Ly / float(My);
	float binz = Lz / float(Mz);
	
	float xlo = -Lx/2.0f;
	float ylo = -Lx/2.0f;
	float zlo = -Lx/2.0f;

	// precompute scale factors to eliminate division in inner loop
	float scalex = 1.0f / binx;
	float scaley = 1.0f / biny;
	float scalez = 1.0f / binz;
	
	// bin each particle
	for (unsigned int i = 0; i < N; i++)
		{
		// find the bin each particle belongs in
		unsigned int ib = (unsigned int)((pos[i].x-xlo)*scalex);
		unsigned int jb = (unsigned int)((pos[i].y-ylo)*scaley);
		unsigned int kb = (unsigned int)((pos[i].z-zlo)*scalez);
		
		// need to handle the case where the particle is exactly at the box hi
		if (ib == Mx)
			ib = 0;
		if (jb == My)
			jb = 0;
		if (kb == Mz)
			kb = 0;
			
		// update the bin
		unsigned int bin = ib*(Mz*My) + jb * Mz + kb;
		unsigned int size = bin_size[bin];
		if (size < Nmax)
			idxlist[bin*Nmax + size] = i;
		else
			{
			printf("Error, bins overflowed!\n");
			exit(1);
			}
		bin_size[bin]++;
		}
	}
	
// benchmark the host rebinning
void bmark_host_rebinning(bool include_memcpy)
	{
	// warm up
	rebin_particles_host(gh_idxlist, gh_bin_size, gh_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax);
	
	// verify results
	if (!verify())
		{
		printf("Invalid results in host bmark!\n");
		return;
		}
	
	// benchmarks
	timeval start;
	gettimeofday(&start, NULL);
	
	unsigned int iters = 1000;
	for (unsigned int i = 0; i < iters; i++)
		{
		if (include_memcpy)
			hipMemcpy(gh_pos, gd_pos, g_N*sizeof(unsigned int), hipMemcpyDeviceToHost);
					
		rebin_particles_host(gh_idxlist, gh_bin_size, gh_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax);
		
		if (include_memcpy)
			{
			hipMemcpy(gd_idxlist, gh_idxlist, g_Mx*g_My*g_Mz*g_Nmax*sizeof(unsigned int), hipMemcpyHostToDevice);
			hipMemcpy(gd_bin_size, gh_bin_size, g_Mx*g_My*g_Mz*sizeof(unsigned int), hipMemcpyHostToDevice);
			}
		}
	
	timeval end;
	gettimeofday(&end, NULL);
	float t = (end.tv_sec - start.tv_sec)*1000.0f + (end.tv_usec - start.tv_usec)/1000.0f;
	float avg_t = t/float(iters);
	
	if (include_memcpy)
		printf("Host w/device memcpy: ");
	else
		printf("Host                : ");
	printf("%f ms\n", avg_t);
	}

#if CUDA_ARCH >= 11

//*************************** simple method of binning on the GPU
// Run one thread per particle
// determine the bin that particle belongs in
// atomicInc the bin size
// write the particle into the bin
// done.
__global__ void rebin_simple_kernel(unsigned int *d_idxlist, unsigned int *d_bin_size, float4 *d_pos, unsigned int N, float xlo, float ylo, float zlo, unsigned int Mx, unsigned int My, unsigned int Mz, unsigned int Nmax, float scalex, float scaley, float scalez)
	{
	// read in the particle that belongs to this thread
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;
		
	float4 pos = d_pos[idx];
	
	// determine which bin it belongs in
	unsigned int ib = (unsigned int)((pos.x-xlo)*scalex);
	unsigned int jb = (unsigned int)((pos.y-ylo)*scaley);
	unsigned int kb = (unsigned int)((pos.z-zlo)*scalez);
	
	// need to handle the case where the particle is exactly at the box hi
	if (ib == Mx)
		ib = 0;
	if (jb == My)
		jb = 0;
	if (kb == Mz)
		kb = 0;
		
	unsigned int bin = ib*(Mz*My) + jb * Mz + kb;
	unsigned int size = atomicInc(&d_bin_size[bin], 0xffffffff);
	if (size < Nmax)
		d_idxlist[bin*Nmax + size] = idx;
	}
	
void rebin_particles_simple(unsigned int *idxlist, unsigned int *bin_size, float4 *pos, unsigned int N, float Lx, float Ly, float Lz, unsigned int Mx, unsigned int My, unsigned int Mz, unsigned int Nmax)
	{
	// run one particle per thread
	int block_size = 128;
	int n_blocks = (int)ceil(float(N)/(float)block_size);

	// make even bin dimensions
	float binx = Lx / float(Mx);
	float biny = Ly / float(My);
	float binz = Lz / float(Mz);
	
	float xlo = -Lx/2.0f;
	float ylo = -Lx/2.0f;
	float zlo = -Lx/2.0f;

	// precompute scale factors to eliminate division in inner loop
	float scalex = 1.0f / binx;
	float scaley = 1.0f / biny;
	float scalez = 1.0f / binz;
	
	// call the kernel
	//cudaMemset(gd_bin_size, 0, sizeof(unsigned int)*Mx*My*Mz);
	fast_memclear_kernal<<<(int)ceil(float(Mx*My*Mz)/(float)block_size), block_size>>>(gd_bin_size, Mx*My*Mz);
	rebin_simple_kernel<<<n_blocks, block_size>>>(idxlist, bin_size, pos, N, xlo, ylo, zlo, Mx, My, Mz, Nmax, scalex, scaley, scalez);
	}
	
// benchmark the device rebinning
void bmark_simple_rebinning()
	{
	// warm up
	rebin_particles_simple(gd_idxlist, gd_bin_size, gd_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax);
	CUT_CHECK_ERROR("kernel failed");
	// copy back from device
	CUDA_SAFE_CALL(cudaMemcpy(gh_idxlist, gd_idxlist, g_Mx*g_My*g_Mz*g_Nmax*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	CUDA_SAFE_CALL(cudaMemcpy(gh_bin_size, gd_bin_size, g_Mx*g_My*g_Mz*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	
	// verify results
	if (!verify())
		{
		printf("Invalid results in GPU/simple bmark!\n");
		return;
		}
	
	// benchmarks
	float total_time = 0.0f;
	cudaEvent_t start, end;
    cudaEventCreate(&start);
    cudaEventCreate(&end);
	
	
	unsigned int iters = 1000;
	for (unsigned int i = 0; i < iters; i++)
		{
		cudaEventRecord(start, 0);
		rebin_particles_simple(gd_idxlist, gd_bin_size, gd_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax);
		cudaEventRecord(end, 0);
		
		float tmp;
		cudaEventSynchronize(end);
		cudaEventElapsedTime(&tmp, start, end);
		total_time += tmp;
		}
	
	float avg_t = total_time/float(iters);
	
	// copy back from device
	CUDA_SAFE_CALL(cudaMemcpy(gh_idxlist, gd_idxlist, g_Mx*g_My*g_Mz*g_Nmax*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	CUDA_SAFE_CALL(cudaMemcpy(gh_bin_size, gd_bin_size, g_Mx*g_My*g_Mz*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	
	// verify results again to be sure
	if (!verify())
		{
		printf("Invalid results at end of GPU/simple bmark!\n");
		return;
		}	
	
	printf("GPU/simple          : ");
	printf("%f ms\n", avg_t);
	}


//*************************** simple method of binning on the GPU - with sorting
// Run one thread per particle
// determine the bin that particle belongs in
// sort the particles based on the bin
// calculate the number of particles added to each bin in the sorted array
// atomicInc the bin size in global memory
// write the particle into the bin
// done.

// bitonic sort from CUDA SDK
template<class T> __device__ inline void swap(T & a, T & b)
	{
	T tmp = a;
	a = b;
	b = tmp;
	}

template<class T, unsigned int block_size> __device__ inline void bitonic_sort(T *shared)
	{
	unsigned int tid = threadIdx.x;
	
	// Parallel bitonic sort.
	#pragma unroll
	for (int k = 2; k <= block_size; k *= 2)
		{
		// Bitonic merge:
		#pragma unroll
		for (int j = k / 2; j>0; j /= 2)
			{
			int ixj = tid ^ j;
			
			if (ixj > tid)
				{
				if ((tid & k) == 0)
					{
					if (shared[tid] > shared[ixj])
						{
						swap(shared[tid], shared[ixj]);
						}
					}
				else
					{
					if (shared[tid] < shared[ixj])
						{
						swap(shared[tid], shared[ixj]);
						}
					}
				}
				
			__syncthreads();
			}
		}
	}
	
struct bin_id_pair
	{
	unsigned int bin;
	unsigned int id;
	unsigned int start_offset;	// pad to minimize bank conflicts
	};
	
__device__ inline bin_id_pair make_bin_id_pair(unsigned int bin, unsigned int id)
	{
	bin_id_pair res;
	res.bin = bin;
	res.id = id;
	res.start_offset = 0;
	return res;
	}
	
__device__ inline bool operator< (const bin_id_pair& a, const bin_id_pair& b)
	{
	if (a.bin == b.bin)
		return (a.id < b.id);
	else
		return (a.bin < b.bin);
	}

__device__ inline bool operator> (const bin_id_pair& a, const bin_id_pair& b)
	{
	if (a.bin == b.bin)
		return (a.id > b.id);
	else
		return (a.bin > b.bin);	
	}

template<class T, unsigned int block_size> __device__ inline void scan_naive(T *temp)
	{
	int thid = threadIdx.x;
	
	int pout = 0;
	int pin = 1;
	
	#pragma unroll
	for (int offset = 1; offset < block_size; offset *= 2)
		{
		pout = 1 - pout;
		pin  = 1 - pout;
		__syncthreads();
		
		temp[pout*block_size+thid] = temp[pin*block_size+thid];
		
		if (thid >= offset)
			temp[pout*block_size+thid] += temp[pin*block_size+thid - offset];
		}
		
	__syncthreads();
	// bring the data back to the initial array
	if (pout == 1)
		{
		pout = 1 - pout;
		pin  = 1 - pout;
		temp[pout*block_size+thid] = temp[pin*block_size+thid];
		__syncthreads();
		}
	}

template<unsigned int block_size> __global__ void rebin_simple_sort_kernel(unsigned int *d_idxlist, unsigned int *d_bin_size, float4 *d_pos, unsigned int N, float xlo, float ylo, float zlo, unsigned int Mx, unsigned int My, unsigned int Mz, unsigned int Nmax, float scalex, float scaley, float scalez)
	{
	// sentinel to label a bin as invalid
	const unsigned int INVALID_BIN = 0xffffffff;
	
	// read in the particle that belongs to this thread
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	float4 pos = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	if (idx < N)
		pos = d_pos[idx];
	
	// determine which bin it belongs in
	unsigned int ib = (unsigned int)((pos.x-xlo)*scalex);
	unsigned int jb = (unsigned int)((pos.y-ylo)*scaley);
	unsigned int kb = (unsigned int)((pos.z-zlo)*scalez);
	
	// need to handle the case where the particle is exactly at the box hi
	if (ib == Mx)
		ib = 0;
	if (jb == My)
		jb = 0;
	if (kb == Mz)
		kb = 0;
		
	unsigned int bin = ib*(Mz*My) + jb * Mz + kb;
	
	// if we are past the end of the array, mark the bin as invalid
	if (idx >= N)
		bin = INVALID_BIN;
	
	// load up shared memory
	__shared__ bin_id_pair bin_pairs[block_size];
	bin_pairs[threadIdx.x] = make_bin_id_pair(bin, idx);
	__syncthreads();
	
	// sort it 
	bitonic_sort<bin_id_pair, block_size>(bin_pairs);
	
	// identify the breaking points
	__shared__ unsigned int unique[block_size*2+1];
	
	bool is_unique = false;
	if (threadIdx.x > 0 && bin_pairs[threadIdx.x].bin != bin_pairs[threadIdx.x-1].bin)
		is_unique = true;
	
	unique[threadIdx.x] = 0;
	if (is_unique)
		unique[threadIdx.x] = 1;
	
	// threadIdx.x = 0 is unique: but we don't want to count it in the scan
	if (threadIdx.x == 0)
		is_unique = true;
	
	__syncthreads();
	
	// scan to find addresses to write to
	scan_naive<unsigned int, block_size>(unique);
	
	// determine start location of each unique value in the array
	// save shared memory by reusing the temp data in the unique[] array
	unsigned int *start = &unique[block_size];
	
	if (is_unique)
		start[unique[threadIdx.x]] = threadIdx.x;
				
	// boundary condition: need one past the end
	if (threadIdx.x == 0)
		start[unique[block_size-1]+1] = block_size;
	
	__syncthreads();
	
	bool is_valid = (bin_pairs[threadIdx.x].bin < Mx*My*Mz);
	
	// now: each unique start point does it's own atomicAdd to find the starting offset
	// the is_valid check is to prevent writing to out of bounds memory at the tail end of the array
	if (is_unique && is_valid)
		bin_pairs[unique[threadIdx.x]].start_offset = atomicAdd(&d_bin_size[bin_pairs[threadIdx.x].bin], start[unique[threadIdx.x]+1] - start[unique[threadIdx.x]]);
	
	__syncthreads();
	
	// finally! we can write out all the particles
	// the is_valid check is to prevent writing to out of bounds memory at the tail end of the array
	unsigned int offset = bin_pairs[unique[threadIdx.x]].start_offset;
	if (offset + threadIdx.x - start[unique[threadIdx.x]] < Nmax && is_valid)
		d_idxlist[bin_pairs[threadIdx.x].bin*Nmax + offset + threadIdx.x - start[unique[threadIdx.x]]] = bin_pairs[threadIdx.x].id;
	}
	
void rebin_particles_simple_sort(unsigned int *idxlist, unsigned int *bin_size, float4 *pos, unsigned int N, float Lx, float Ly, float Lz, unsigned int Mx, unsigned int My, unsigned int Mz, unsigned int Nmax, unsigned int block_size)
	{
	// run one particle per thread
	int n_blocks = (int)ceil(float(N)/(float)block_size);

	// make even bin dimensions
	float binx = Lx / float(Mx);
	float biny = Ly / float(My);
	float binz = Lz / float(Mz);
	
	float xlo = -Lx/2.0f;
	float ylo = -Lx/2.0f;
	float zlo = -Lx/2.0f;

	// precompute scale factors to eliminate division in inner loop
	float scalex = 1.0f / binx;
	float scaley = 1.0f / biny;
	float scalez = 1.0f / binz;
	
	// call the kernel
	//cudaMemset(gd_bin_size, 0, sizeof(unsigned int)*Mx*My*Mz);
	fast_memclear_kernal<<<(int)ceil(float(Mx*My*Mz)/(float)block_size), block_size>>>(gd_bin_size, Mx*My*Mz);
	
	if (block_size == 32)
		rebin_simple_sort_kernel<32><<<n_blocks, block_size>>>(idxlist, bin_size, pos, N, xlo, ylo, zlo, Mx, My, Mz, Nmax, scalex, scaley, scalez);
	else if (block_size == 64)
		rebin_simple_sort_kernel<64><<<n_blocks, block_size>>>(idxlist, bin_size, pos, N, xlo, ylo, zlo, Mx, My, Mz, Nmax, scalex, scaley, scalez);
	else if (block_size == 128)
		rebin_simple_sort_kernel<128><<<n_blocks, block_size>>>(idxlist, bin_size, pos, N, xlo, ylo, zlo, Mx, My, Mz, Nmax, scalex, scaley, scalez);
	else if (block_size == 256)
		rebin_simple_sort_kernel<256><<<n_blocks, block_size>>>(idxlist, bin_size, pos, N, xlo, ylo, zlo, Mx, My, Mz, Nmax, scalex, scaley, scalez);
	else if (block_size == 512)
		rebin_simple_sort_kernel<512><<<n_blocks, block_size>>>(idxlist, bin_size, pos, N, xlo, ylo, zlo, Mx, My, Mz, Nmax, scalex, scaley, scalez);
	else
		{
		printf("invalid block size!\n");
		exit(1);
		}
	}
	
// benchmark the device rebinning
void bmark_simple_sort_rebinning(unsigned int block_size)
	{
	// warm up
	rebin_particles_simple_sort(gd_idxlist, gd_bin_size, gd_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax, block_size);
	CUT_CHECK_ERROR("kernel failed");
	// copy back from device
	CUDA_SAFE_CALL(cudaMemcpy(gh_idxlist, gd_idxlist, g_Mx*g_My*g_Mz*g_Nmax*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	CUDA_SAFE_CALL(cudaMemcpy(gh_bin_size, gd_bin_size, g_Mx*g_My*g_Mz*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	
	// verify results
	if (!verify())
		{
		printf("Invalid results in GPU/simple/sort bmark!\n");
		return;
		}
	
	// benchmarks
	float total_time = 0.0f;
	cudaEvent_t start, end;
    cudaEventCreate(&start);
    cudaEventCreate(&end);
	
	
	unsigned int iters = 1000;
	for (unsigned int i = 0; i < iters; i++)
		{
		cudaEventRecord(start, 0);
		rebin_particles_simple_sort(gd_idxlist, gd_bin_size, gd_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax, block_size);
		cudaEventRecord(end, 0);
		
		float tmp;
		cudaEventSynchronize(end);
		cudaEventElapsedTime(&tmp, start, end);
		total_time += tmp;
		}
	
	float avg_t = total_time/float(iters);
	
	// copy back from device
	CUDA_SAFE_CALL(cudaMemcpy(gh_idxlist, gd_idxlist, g_Mx*g_My*g_Mz*g_Nmax*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	CUDA_SAFE_CALL(cudaMemcpy(gh_bin_size, gd_bin_size, g_Mx*g_My*g_Mz*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	
	// verify results again to be sure
	if (!verify())
		{
		printf("Invalid results at end of GPU/simple/sort bmark!\n");
		return;
		}	
	
	printf("GPU/simple/sort/%3d : ", block_size);
	printf("%f ms\n", avg_t);
	}

	
//*************************** simple update method of binning on the GPU
// Run one thread per bin
// loop through all particles in neighboring bins
// determine which of those particles belong in this bin, and write them there
// done.
texture<float4, 1, cudaReadModeElementType> pos_tex;
texture<unsigned int, 1, cudaReadModeElementType> in_bin_size_tex;
texture<unsigned int, 1, cudaReadModeElementType> in_idxlist_tex;

__global__ void update_simple_kernel(unsigned int *d_out_idxlist, unsigned int *d_out_bin_size, uint4* d_bin_coords, float xlo, float ylo, float zlo, unsigned int Mx, unsigned int My, unsigned int Mz, unsigned int Nmax, float scalex, float scaley, float scalez)
	{
	// find the coordinates of our bin
	unsigned int bin = blockIdx.x * blockDim.x + threadIdx.x;
	if (bin >= Mx*My*Mz)
		return;
	uint4 coords = d_bin_coords[bin];
	int bin_i = coords.x;
	int bin_j = coords.y;
	int bin_k = coords.z;
	
	// intialize the new particles in this bin to 0
	unsigned int bin_size = 0;
	
	// loop through all the neighboring bins and find particles that now belong in this bin
	for (int cur_i = bin_i - 1; cur_i <= bin_i+1; cur_i++)
		{
		int neigh_i = cur_i;
		if (neigh_i == -1)
			neigh_i = Mx-1;
		if (neigh_i == Mx)
			neigh_i = 0;
			
		for (int cur_j = bin_j - 1; cur_j <= bin_j+1; cur_j++)
			{
			int neigh_j = cur_j;
			if (neigh_j == -1)
				neigh_j = My-1;
			if (neigh_j == My)
				neigh_j = 0;
				
			for (int cur_k = bin_k - 1; cur_k <= bin_k+1; cur_k++)
				{
				int neigh_k = cur_k;
				if (neigh_k == -1)
					neigh_k = Mz-1;
				if (neigh_k == Mz)
					neigh_k = 0;
					
				// determine the index of the neighboring bin
				unsigned int neigh_bin = neigh_i*(Mz*My) + neigh_j * Mz + neigh_k;
				
				// loop through all particles in that neighboring bin
				unsigned int neigh_bin_size = tex1Dfetch(in_bin_size_tex, neigh_bin);
				
				for (unsigned int cur_particle = 0; cur_particle < neigh_bin_size; cur_particle++)
					{
					// read in the current particle in the neighboring bin
					unsigned int pidx = tex1Dfetch(in_idxlist_tex, neigh_bin*Nmax + cur_particle);
					float4 pos = tex1Dfetch(pos_tex, pidx);
					
					// determine which bin the particle should be in
					unsigned int ib = (unsigned int)((pos.x-xlo)*scalex);
					unsigned int jb = (unsigned int)((pos.y-ylo)*scaley);
					unsigned int kb = (unsigned int)((pos.z-zlo)*scalez);
					
					// need to handle the case where the particle is exactly at the box hi
					if (ib == Mx)
						ib = 0;
					if (jb == My)
						jb = 0;
					if (kb == Mz)
						kb = 0;
						
					unsigned int cur_bin = ib*(Mz*My) + jb * Mz + kb;
					
					// if that is this bin, add it to the bin
					if (bin == cur_bin && bin_size < Nmax)
						{
						d_out_idxlist[bin*Nmax + bin_size] = pidx;
						bin_size++;
						}
					}
				}
			}
		}
		
	d_out_bin_size[bin] = bin_size;
	}


void update_particles_simple(float4 *pos, unsigned int N, float Lx, float Ly, float Lz, unsigned int Mx, unsigned int My, unsigned int Mz, unsigned int Nmax)
	{
	// run one bin per thread
	int block_size = 256;
	int n_blocks = (int)ceil(float(Mx*My*Mz)/(float)block_size);

	// make even bin dimensions
	float binx = Lx / float(Mx);
	float biny = Ly / float(My);
	float binz = Lz / float(Mz);
	
	float xlo = -Lx/2.0f;
	float ylo = -Lx/2.0f;
	float zlo = -Lx/2.0f;

	// precompute scale factors to eliminate division in inner loop
	float scalex = 1.0f / binx;
	float scaley = 1.0f / biny;
	float scalez = 1.0f / binz;
	
	// swap the pointers
	unsigned int * tmp;
	tmp = gd_idxlist;
	gd_idxlist = gd_old_idxlist;
	gd_old_idxlist = tmp;
	
	tmp = gd_bin_size;
	gd_bin_size = gd_old_bin_size;
	gd_old_bin_size = tmp;
	
	// bind the textures
	cudaBindTexture(0, pos_tex, pos, sizeof(float4) * N);
	cudaBindTexture(0, in_bin_size_tex, gd_old_bin_size, sizeof(unsigned int) * Mx*My*Mz);
	cudaBindTexture(0, in_idxlist_tex, gd_old_idxlist, sizeof(unsigned int) * Mx*My*Mz*Nmax);
	
	// call the kernel
	update_simple_kernel<<<n_blocks, block_size>>>(gd_idxlist, gd_bin_size, gd_bin_coords, xlo, ylo, zlo, Mx, My, Mz, Nmax, scalex, scaley, scalez);
	}

// benchmark the device rebinning
void bmark_simple_updating()
	{
	// warm up
	rebin_particles_simple(gd_idxlist, gd_bin_size, gd_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax);
	CUT_CHECK_ERROR("kernel failed");
	update_particles_simple(gd_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax);
		
	// copy back from device
	CUDA_SAFE_CALL(cudaMemcpy(gh_idxlist, gd_idxlist, g_Mx*g_My*g_Mz*g_Nmax*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	CUDA_SAFE_CALL(cudaMemcpy(gh_bin_size, gd_bin_size, g_Mx*g_My*g_Mz*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	
	// verify results
	if (!verify())
		{
		printf("Invalid results in GPU/update bmark!\n");
		return;
		}
	
	// benchmarks
	float total_time = 0.0f;
	cudaEvent_t start, end;
    cudaEventCreate(&start);
    cudaEventCreate(&end);
	
	unsigned int iters = 1000;
	for (unsigned int i = 0; i < iters; i++)
		{
		tweak_data();
		
		cudaThreadSynchronize();
		
		cudaEventRecord(start, 0);
		update_particles_simple(gd_pos, g_N, g_Lx, g_Ly, g_Lz, g_Mx, g_My, g_Mz, g_Nmax);
		cudaEventRecord(end, 0);
		
		float tmp;
		cudaEventSynchronize(end);
		cudaEventElapsedTime(&tmp, start, end);
		total_time += tmp;
		}
	
	float avg_t = total_time/float(iters);
	
	// copy back from device
	CUDA_SAFE_CALL(cudaMemcpy(gh_idxlist, gd_idxlist, g_Mx*g_My*g_Mz*g_Nmax*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	CUDA_SAFE_CALL(cudaMemcpy(gh_bin_size, gd_bin_size, g_Mx*g_My*g_Mz*sizeof(unsigned int), cudaMemcpyDeviceToHost));
	
	// verify results again to be sure
	if (!verify())
		{
		printf("Invalid results at end of GPU/update bmark!\n");
		return;
		}	
	
	printf("GPU/update          : ");
	printf("%f ms\n", avg_t);
	}

#endif

int main(int argc, char **argv)
	{
	#ifdef ENABLE_CAC_GPU_ID
	if (!getenv("CAC_GPU_ID"))
		printf("Error! Compiled with CAC_GPU_ID support, but no $CAC_GPU_ID specified\n");
	else
		cudaSetDevice(atoi(getenv("CAC_GPU_ID")));
	#endif
	hipSetDevice(1);
	
	// choose defaults if no args specified
	if (argc == 1)
		{
		g_N = 64000;
		g_rcut = 3.8f;
		}
	if (argc == 2)
		{
		g_N = atoi(argv[1]);
		g_rcut = 3.8f;
		}
	if (argc == 3)
		{
		g_N = atoi(argv[1]);
		g_rcut = atof(argv[2]);
		}
		
	float L = pow(float(M_PI/6.0)*float(g_N) / 0.20f, 1.0f/3.0f);
	g_Lx = g_Ly = g_Lz = L;
	
	// setup
	printf("Running gpu_binning microbenchmark: %d %f\n", g_N, g_rcut);
	allocate_data();
	initialize_data();
	sort_data();
	
	// normally, data in HOOMD is not perfectly sorted:
	for (unsigned int i = 0; i < 100; i++)
		tweak_data();
	
	// run the various benchmarks
	bmark_host_rebinning(false);
	bmark_host_rebinning(true);
	#if CUDA_ARCH >= 11
	bmark_simple_rebinning();
	bmark_simple_sort_rebinning(32);
	bmark_simple_sort_rebinning(64);
	bmark_simple_sort_rebinning(128);
	bmark_simple_sort_rebinning(256);
	bmark_simple_sort_rebinning(512);
	bmark_simple_updating();
	#endif
	
	free_data();
	
	return 0;
	}
