#include "hip/hip_runtime.h"
#include "ExampleUpdater.cuh"

// include gpu_settings.h for g_gpu_error_checking
#include <hoomd/gpu_settings.h>

// First, the kernel code for zeroing the velocities on the GPU
//! Kernel that zeroes velocities on the GPU
/*! \param pdata Particle data arrays to zero the velocities of
    
    This kernel executes one thread per particle and zeros the velocity of each. It can be run with any 1D block size
    as long as block_size * num_blocks is >= the number of particles.
*/
extern "C" __global__ 
void gpu_zero_velocities_kernel(gpu_pdata_arrays pdata)
    {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < pdata.local_num)
        {
        pdata.vel[idx] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        }
    }

/*! \param pdata Particle data arrays to zero the velocities of
    This is just a driver for gpu_zero_velocities_kernel(), see it for the details
*/
hipError_t gpu_zero_velocities(const gpu_pdata_arrays &pdata)
    {
    // setup the grid to run the kernel
    int block_size = 256;
    dim3 grid( (int)ceil((double)pdata.local_num / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);
    
    // run the kernel
   gpu_zero_velocities_kernel<<< grid, threads >>>(pdata);
    if (!g_gpu_error_checking)
        {
        return hipSuccess;
        }
    else
        {
        hipDeviceSynchronize();
        return hipGetLastError();
        }
    }

